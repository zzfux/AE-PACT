#include "hip/hip_runtime.h"
#include <hipDNN.h>
#include <cassert>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include <set>

const char *cudnnAlgName[] = {
    "HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM",
    "HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM",
    "HIPDNN_CONVOLUTION_FWD_ALGO_GEMM",
    "HIPDNN_CONVOLUTION_FWD_ALGO_DIRECT",
    "HIPDNN_CONVOLUTION_FWD_ALGO_FFT",
    "HIPDNN_CONVOLUTION_FWD_ALGO_FFT_TILING",
    "HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD",
    "HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED",
    "HIPDNN_CONVOLUTION_FWD_ALGO_COUNT"};

void conv_kernel_wrapper(int N_B, int N_C, int N_H, int N_W, int N_F, int N_R, int N_S, int PaddingH, int PaddingW,
                        int StrideH, int StrideW, int N_X, int N_Y, const float *Input, const float *Kernel, float *Output, int itr, int algo) {
    hipSetDevice(0);
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);
    float *dev_Input;
    float *dev_Kernel;
    float *dev_Output;

    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
            /*format=*/HIPDNN_TENSOR_NCHW,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/N_B,
            /*channels=*/N_C,
            /*image_height=*/N_H,
            /*image_width=*/N_W));

    hipdnnFilterDescriptor_t kernel_descriptor;
    checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*format=*/HIPDNN_TENSOR_NCHW,
            /*out_channels=*/N_F,
            /*in_channels=*/N_C,
            /*kernel_height=*/N_R,
            /*kernel_width=*/N_S));

    hipdnnConvolutionDescriptor_t convolution_descriptor;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
            /*pad_height=*/PaddingH,
            /*pad_width=*/PaddingW,
            /*vertical_stride=*/StrideH,
            /*horizontal_stride=*/StrideW,
            /*dilation_height=*/DilationH,
            /*dilation_width=*/DilationW,
            /*mode=*/HIPDNN_CROSS_CORRELATION,
            /*computeType=*/HIPDNN_DATA_FLOAT));

    // see out put dim
    int channels{0}, height{0}, width{0}, batch_size{0};
    checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convolution_descriptor,
                                                     input_descriptor,
                                                     kernel_descriptor,
                                                     &batch_size,
                                                     &channels,
                                                     &height,
                                                     &width));

    //printf( "Output Image: %d x %d x %d x %d \n", height, width, channels, batch_size);

    assert(batch_size == N_B);
    assert(channels == N_F);
    assert(height == N_Y);
    assert(width == N_X);

    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
            /*format=*/HIPDNN_TENSOR_NCHW,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/N_B,
            /*output_channels=*/N_F,
            /*output_height=*/N_Y,
            /*output_width=*/N_X));
    
    size_t workspace_bytes{0};
    std::set<hipdnnConvolutionFwdAlgo_t> algs;


    size_t t_bytes {0};
    auto err = hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                    input_descriptor,
                                                    kernel_descriptor,
                                                    convolution_descriptor,
                                                    output_descriptor,
                                                    hipdnnConvolutionFwdAlgo_t(algo),
                                                    &t_bytes);
    
    

    checkCUDNN(err);
    workspace_bytes = max(t_bytes, workspace_bytes);






    //printf("Workspace size: %.5f MB", (workspace_bytes / 1048576.0) );
    assert(workspace_bytes >= 0);
    void *d_workspace{nullptr};
    CHECK(hipMalloc(&d_workspace, workspace_bytes));
    const float alpha = 1.0f, beta = 0.0f;

    //printf("\n===========================================================\n");
    printf("Input size: %d X %d X %d X %d\n", N_B, N_C, N_H, N_W);
    printf("Kernel size: %d X %d X %d X %d\n", N_F, N_C, N_R, N_S);
    printf("Output size: %d X %d X %d X %d\n", N_B, N_F, N_Y, N_X);
    printf("Algorithm: %s\n", cudnnAlgName[algo]);
    //printf("===========================================================\n");


   
        
    CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
    CHECK(hipMemcpy(dev_Kernel, Kernel, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));

    CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
    CHECK(hipMemcpy(dev_Input, Input, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));

    CHECK(hipMalloc(&dev_Output, sizeof(float) * N_B * N_F * N_Y * N_X));
    CHECK(hipMemset(dev_Output, 0, sizeof(float) * N_B * N_F * N_Y * N_X));
    hipdnnConvolutionFwdAlgo_t convolution_algorithm = hipdnnConvolutionFwdAlgo_t(algo);
    checkCUDNN(hipdnnConvolutionForward(cudnn,
                                        &alpha,
                                        input_descriptor,
                                        dev_Input,
                                        kernel_descriptor,
                                        dev_Kernel,
                                        convolution_descriptor,
                                        convolution_algorithm,
                                        d_workspace,
                                        workspace_bytes,
                                        &beta,
                                        output_descriptor,
                                        dev_Output));


    hipEvent_t start;
    hipEventCreate(&start);
    hipEvent_t stop;
    hipEventCreate(&stop);

        
    CHECK(hipFree(d_workspace));

    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyFilterDescriptor(kernel_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor);

    hipdnnDestroy(cudnn);

}



