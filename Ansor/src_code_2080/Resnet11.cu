#include "hip/hip_runtime.h"

            #include <cassert>
            #include <stdlib.h>
            #include <hip/hip_runtime.h>
            #include "common.h"
            

            
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[4];
  __shared__ float pad_temp_shared[208];
  __shared__ float kernel_shared[512];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((((int)threadIdx.x) * 2))] = data[(((((rc_outer_outer * 3136) + (((((int)threadIdx.x) * 2) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) * 2) % 13)))];
    pad_temp_shared[(((((int)threadIdx.x) * 2) + 1))] = data[(((((rc_outer_outer * 3136) + ((((((int)threadIdx.x) * 2) + 1) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((((int)threadIdx.x) * 2) + 1) % 13)))];
    if (((int)threadIdx.x) < 48) {
      pad_temp_shared[(((((int)threadIdx.x) * 2) + 112))] = data[(((((rc_outer_outer * 3136) + ((((((int)threadIdx.x) * 2) + 112) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((((int)threadIdx.x) * 2) + 8) % 13)))];
    }
    if (((int)threadIdx.x) < 48) {
      pad_temp_shared[(((((int)threadIdx.x) * 2) + 113))] = data[(((((rc_outer_outer * 3136) + ((((((int)threadIdx.x) * 2) + 113) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((((int)threadIdx.x) * 2) + 9) % 13)))];
    }
    kernel_shared[(((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 7) * 8192) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    kernel_shared[((((int)threadIdx.x) + 56))] = kernel[((((((((int)blockIdx.x) / 7) * 8192) + (((((int)threadIdx.x) + 56) >> 4) * 256)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 8) & 15)))];
    kernel_shared[((((int)threadIdx.x) + 112))] = kernel[(((((((((int)blockIdx.x) / 7) * 8192) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 1792))];
    kernel_shared[((((int)threadIdx.x) + 168))] = kernel[((((((((int)blockIdx.x) / 7) * 8192) + (((((int)threadIdx.x) + 168) >> 4) * 256)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 8) & 15)))];
    kernel_shared[((((int)threadIdx.x) + 224))] = kernel[(((((((((int)blockIdx.x) / 7) * 8192) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 3584))];
    kernel_shared[((((int)threadIdx.x) + 280))] = kernel[((((((((int)blockIdx.x) / 7) * 8192) + (((((int)threadIdx.x) + 280) >> 4) * 256)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 8) & 15)))];
    kernel_shared[((((int)threadIdx.x) + 336))] = kernel[(((((((((int)blockIdx.x) / 7) * 8192) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 5376))];
    kernel_shared[((((int)threadIdx.x) + 392))] = kernel[((((((((int)blockIdx.x) / 7) * 8192) + (((((int)threadIdx.x) + 392) >> 4) * 256)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 8) & 15)))];
    kernel_shared[((((int)threadIdx.x) + 448))] = kernel[(((((((((int)blockIdx.x) / 7) * 8192) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 7168))];
    if (((int)threadIdx.x) < 8) {
      kernel_shared[((((int)threadIdx.x) + 504))] = kernel[((((((((int)blockIdx.x) / 7) * 8192) + (((((int)threadIdx.x) + 504) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 8)))];
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((rc_inner * 13) + ((((int)threadIdx.x) % 7) * 2)))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + rc_inner))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((rc_inner * 13) + ((((int)threadIdx.x) % 7) * 2)))] * kernel_shared[(((((((int)threadIdx.x) / 7) * 16) + rc_inner) + 128))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((rc_inner * 13) + ((((int)threadIdx.x) % 7) * 2)))] * kernel_shared[(((((((int)threadIdx.x) / 7) * 16) + rc_inner) + 256))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((rc_inner * 13) + ((((int)threadIdx.x) % 7) * 2)))] * kernel_shared[(((((((int)threadIdx.x) / 7) * 16) + rc_inner) + 384))]));
    }
  }
  compute[((((((((int)blockIdx.x) / 7) * 1568) + ((((int)threadIdx.x) / 7) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + (((int)threadIdx.x) % 7)))] = compute_local[(0)];
  compute[(((((((((int)blockIdx.x) / 7) * 1568) + ((((int)threadIdx.x) / 7) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + (((int)threadIdx.x) % 7)) + 392))] = compute_local[(1)];
  compute[(((((((((int)blockIdx.x) / 7) * 1568) + ((((int)threadIdx.x) / 7) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + (((int)threadIdx.x) % 7)) + 784))] = compute_local[(2)];
  compute[(((((((((int)blockIdx.x) / 7) * 1568) + ((((int)threadIdx.x) / 7) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + (((int)threadIdx.x) % 7)) + 1176))] = compute_local[(3)];
}



void conv_kernel_wrapper(int N_B, int N_C, int N_H, int N_W, int N_F, int N_R, int N_S, int PaddingH, int PaddingW,
                        int StrideH, int StrideW, int N_X, int N_Y, const float *Input, 
                        const float *Kernel, float *Output, int itr) {
                        hipEvent_t start;
                        CHECK(hipEventCreate(&start));
                        hipEvent_t stop;
                        CHECK(hipEventCreate(&stop));
                        float *dev_Input;
                        float *dev_Kernel;
                        float *dev_Output;
                    
                        CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                        CHECK(hipMemcpy(dev_Kernel, Kernel, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                        CHECK(hipMemcpy(dev_Input, Input, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Output, sizeof(float) * N_B * N_F * N_Y * N_X));
                        CHECK(hipMemset(dev_Output, 0, sizeof(float) * N_B * N_F * N_Y * N_X));
                    
dim3 size_grid(112,1,1);

dim3 size_block(56,1,1);
 
                       printf("warmup....\n");
                        default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                    
                        double *time_arr = (double *)malloc(itr * sizeof(double));
                        for (int j = 1; j < itr; j++) {
                            CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                            CHECK(hipMemcpy(dev_Kernel, Kernel+j*N_F * N_C * N_R * N_S, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                            CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                            CHECK(hipMemcpy(dev_Input, Input+j*N_B * N_C * N_H * N_W, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                            /* Record the start event*/
                            CHECK(hipEventRecord(start, NULL));
                            default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                            /* Record the end event */
                            CHECK(hipEventRecord(stop, NULL));
                            /* Wait for the stop event to complete*/
                            CHECK(hipEventSynchronize(stop));
                            float msec = 0.0f;
                            hipEventElapsedTime(&msec, start, stop);
                            time_arr[j] = msec;
                        }
                    
                       
                    
                        //hipError_t err = hipGetLastError();
                        //if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
                        double msecTotal = 0;
                        for (int j = 1; j < itr; j++) {   
                            printf("%0.5lf\n", time_arr[j]);
                            msecTotal += time_arr[j];
                        }
                    
                        double msecPerMatrixMul = (msecTotal) / itr;
                        double flopsPerMatrixMul = 2.0 * N_B * N_C * N_X * N_Y * N_F * N_R * N_S;
                    
                        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
                    
                        printf("\n===========================================================\n");
                        printf("Input size: %d X %d X %d X %d\n", N_B, N_C, N_H, N_W);
                        printf("Kernel size: %d X %d X %d X %d\n", N_F, N_C, N_R, N_S);
                        printf("Output size: %d X %d X %d X %d\n", N_B, N_F, N_Y, N_X);
                        printf("flops: %0.5lf\n", (flopsPerMatrixMul * 1.0e-9f));
                        printf("Time is %0.5lf \n", (msecPerMatrixMul / 1000.0f));
                        printf("Gflops %0.5lf \n", gigaFlops);
                        printf("===========================================================\n");
                    
                        /*write back to Host Output -- batch size is 1*/
                        CHECK(hipMemcpy(Output, dev_Output, sizeof(float) * N_B * N_F * N_X * N_Y, hipMemcpyDeviceToHost));
                    
                        CHECK(hipFree(dev_Input));
                        CHECK(hipFree(dev_Kernel));
                        CHECK(hipFree(dev_Output));
                    
                    }
            