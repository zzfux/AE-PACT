#include "hip/hip_runtime.h"

            #include <cassert>
            #include <stdlib.h>
            #include <hip/hip_runtime.h>
            #include "common.h"
            

            
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[32];
  __shared__ float pad_temp_shared[420];
  __shared__ float kernel_shared[144];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  compute_local[(7)] = 0.000000e+00f;
  compute_local[(8)] = 0.000000e+00f;
  compute_local[(9)] = 0.000000e+00f;
  compute_local[(10)] = 0.000000e+00f;
  compute_local[(11)] = 0.000000e+00f;
  compute_local[(12)] = 0.000000e+00f;
  compute_local[(13)] = 0.000000e+00f;
  compute_local[(14)] = 0.000000e+00f;
  compute_local[(15)] = 0.000000e+00f;
  compute_local[(16)] = 0.000000e+00f;
  compute_local[(17)] = 0.000000e+00f;
  compute_local[(18)] = 0.000000e+00f;
  compute_local[(19)] = 0.000000e+00f;
  compute_local[(20)] = 0.000000e+00f;
  compute_local[(21)] = 0.000000e+00f;
  compute_local[(22)] = 0.000000e+00f;
  compute_local[(23)] = 0.000000e+00f;
  compute_local[(24)] = 0.000000e+00f;
  compute_local[(25)] = 0.000000e+00f;
  compute_local[(26)] = 0.000000e+00f;
  compute_local[(27)] = 0.000000e+00f;
  compute_local[(28)] = 0.000000e+00f;
  compute_local[(29)] = 0.000000e+00f;
  compute_local[(30)] = 0.000000e+00f;
  compute_local[(31)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((((int)threadIdx.x) * 2))] = data[((((((rc_outer_outer * 75076) + (((((int)blockIdx.x) % 272) >> 2) * 1096)) + ((((int)threadIdx.x) / 35) * 274)) + ((((int)blockIdx.x) & 3) * 68)) + ((((int)threadIdx.x) % 35) * 2)))];
    pad_temp_shared[(((((int)threadIdx.x) * 2) + 1))] = data[((((((rc_outer_outer * 75076) + (((((int)blockIdx.x) % 272) >> 2) * 1096)) + ((((((int)threadIdx.x) * 2) + 1) / 70) * 274)) + ((((int)blockIdx.x) & 3) * 68)) + (((((int)threadIdx.x) * 2) + 1) % 70)))];
    if (((int)threadIdx.x) < 74) {
      pad_temp_shared[(((((int)threadIdx.x) * 2) + 272))] = data[((((((rc_outer_outer * 75076) + (((((int)blockIdx.x) % 272) >> 2) * 1096)) + ((((((int)threadIdx.x) * 2) + 272) / 70) * 274)) + ((((int)blockIdx.x) & 3) * 68)) + (((((int)threadIdx.x) * 2) + 62) % 70)))];
    }
    if (((int)threadIdx.x) < 74) {
      pad_temp_shared[(((((int)threadIdx.x) * 2) + 273))] = data[((((((rc_outer_outer * 75076) + (((((int)blockIdx.x) % 272) >> 2) * 1096)) + ((((((int)threadIdx.x) * 2) + 273) / 70) * 274)) + ((((int)blockIdx.x) & 3) * 68)) + (((((int)threadIdx.x) * 2) + 63) % 70)))];
    }
    kernel_shared[(((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 272) * 4608) + ((((int)threadIdx.x) / 9) * 288)) + (rc_outer_outer * 9)) + (((int)threadIdx.x) % 9)))];
    if (((int)threadIdx.x) < 8) {
      kernel_shared[((((int)threadIdx.x) + 136))] = kernel[((((((((int)blockIdx.x) / 272) * 4608) + (((((int)threadIdx.x) + 136) / 9) * 288)) + (rc_outer_outer * 9)) + (((int)threadIdx.x) + 1)))];
    }
    __syncthreads();
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 34) * 2))] * kernel_shared[(((((int)threadIdx.x) / 34) * 36))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 1))] * kernel_shared[(((((int)threadIdx.x) / 34) * 36))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 70))] * kernel_shared[(((((int)threadIdx.x) / 34) * 36))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 71))] * kernel_shared[(((((int)threadIdx.x) / 34) * 36))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 140))] * kernel_shared[(((((int)threadIdx.x) / 34) * 36))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 141))] * kernel_shared[(((((int)threadIdx.x) / 34) * 36))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 210))] * kernel_shared[(((((int)threadIdx.x) / 34) * 36))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 211))] * kernel_shared[(((((int)threadIdx.x) / 34) * 36))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((((int)threadIdx.x) % 34) * 2))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 9))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 1))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 9))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 70))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 9))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 71))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 9))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 140))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 9))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 141))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 9))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 210))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 9))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 211))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 9))]));
    compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(((((int)threadIdx.x) % 34) * 2))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 18))]));
    compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 1))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 18))]));
    compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 70))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 18))]));
    compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 71))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 18))]));
    compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 140))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 18))]));
    compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 141))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 18))]));
    compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 210))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 18))]));
    compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 211))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 18))]));
    compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[(((((int)threadIdx.x) % 34) * 2))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 27))]));
    compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 1))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 27))]));
    compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 70))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 27))]));
    compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 71))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 27))]));
    compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 140))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 27))]));
    compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 141))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 27))]));
    compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 210))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 27))]));
    compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 211))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 27))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 70))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 3))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 71))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 3))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 140))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 3))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 141))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 3))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 210))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 3))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 211))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 3))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 280))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 3))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 281))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 3))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 70))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 12))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 71))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 12))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 140))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 12))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 141))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 12))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 210))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 12))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 211))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 12))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 280))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 12))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 281))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 12))]));
    compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 70))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 21))]));
    compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 71))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 21))]));
    compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 140))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 21))]));
    compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 141))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 21))]));
    compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 210))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 21))]));
    compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 211))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 21))]));
    compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 280))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 21))]));
    compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 281))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 21))]));
    compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 70))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 30))]));
    compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 71))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 30))]));
    compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 140))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 30))]));
    compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 141))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 30))]));
    compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 210))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 30))]));
    compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 211))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 30))]));
    compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 280))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 30))]));
    compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 281))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 30))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 140))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 6))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 141))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 6))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 210))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 6))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 211))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 6))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 280))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 6))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 281))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 6))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 350))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 6))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 351))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 6))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 140))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 15))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 141))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 15))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 210))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 15))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 211))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 15))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 280))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 15))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 281))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 15))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 350))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 15))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 351))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 15))]));
    compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 140))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 24))]));
    compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 141))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 24))]));
    compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 210))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 24))]));
    compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 211))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 24))]));
    compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 280))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 24))]));
    compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 281))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 24))]));
    compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 350))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 24))]));
    compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 351))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 24))]));
    compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 140))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 33))]));
    compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 141))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 33))]));
    compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 210))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 33))]));
    compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 211))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 33))]));
    compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 280))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 33))]));
    compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 281))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 33))]));
    compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 350))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 33))]));
    compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 351))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 33))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 1))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 1))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 2))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 1))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 71))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 1))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 72))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 1))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 141))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 1))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 142))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 1))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 211))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 1))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 212))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 1))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 1))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 10))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 2))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 10))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 71))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 10))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 72))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 10))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 141))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 10))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 142))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 10))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 211))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 10))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 212))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 10))]));
    compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 1))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 19))]));
    compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 2))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 19))]));
    compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 71))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 19))]));
    compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 72))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 19))]));
    compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 141))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 19))]));
    compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 142))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 19))]));
    compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 211))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 19))]));
    compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 212))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 19))]));
    compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 1))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 28))]));
    compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 2))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 28))]));
    compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 71))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 28))]));
    compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 72))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 28))]));
    compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 141))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 28))]));
    compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 142))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 28))]));
    compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 211))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 28))]));
    compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 212))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 28))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 71))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 4))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 72))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 4))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 141))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 4))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 142))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 4))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 211))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 4))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 212))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 4))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 281))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 4))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 282))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 4))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 71))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 13))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 72))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 13))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 141))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 13))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 142))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 13))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 211))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 13))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 212))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 13))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 281))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 13))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 282))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 13))]));
    compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 71))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 22))]));
    compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 72))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 22))]));
    compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 141))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 22))]));
    compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 142))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 22))]));
    compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 211))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 22))]));
    compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 212))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 22))]));
    compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 281))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 22))]));
    compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 282))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 22))]));
    compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 71))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 31))]));
    compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 72))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 31))]));
    compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 141))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 31))]));
    compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 142))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 31))]));
    compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 211))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 31))]));
    compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 212))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 31))]));
    compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 281))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 31))]));
    compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 282))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 31))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 141))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 7))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 142))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 7))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 211))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 7))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 212))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 7))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 281))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 7))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 282))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 7))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 351))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 7))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 352))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 7))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 141))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 16))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 142))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 16))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 211))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 16))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 212))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 16))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 281))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 16))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 282))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 16))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 351))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 16))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 352))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 16))]));
    compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 141))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 25))]));
    compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 142))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 25))]));
    compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 211))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 25))]));
    compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 212))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 25))]));
    compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 281))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 25))]));
    compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 282))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 25))]));
    compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 351))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 25))]));
    compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 352))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 25))]));
    compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 141))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 34))]));
    compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 142))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 34))]));
    compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 211))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 34))]));
    compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 212))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 34))]));
    compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 281))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 34))]));
    compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 282))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 34))]));
    compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 351))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 34))]));
    compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 352))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 34))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 2))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 2))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 3))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 2))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 72))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 2))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 73))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 2))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 142))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 2))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 143))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 2))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 212))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 2))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 213))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 2))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 2))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 11))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 3))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 11))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 72))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 11))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 73))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 11))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 142))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 11))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 143))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 11))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 212))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 11))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 213))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 11))]));
    compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 2))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 20))]));
    compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 3))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 20))]));
    compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 72))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 20))]));
    compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 73))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 20))]));
    compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 142))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 20))]));
    compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 143))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 20))]));
    compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 212))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 20))]));
    compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 213))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 20))]));
    compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 2))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 29))]));
    compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 3))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 29))]));
    compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 72))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 29))]));
    compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 73))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 29))]));
    compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 142))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 29))]));
    compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 143))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 29))]));
    compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 212))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 29))]));
    compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 213))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 29))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 72))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 5))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 73))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 5))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 142))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 5))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 143))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 5))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 212))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 5))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 213))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 5))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 282))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 5))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 283))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 5))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 72))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 14))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 73))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 14))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 142))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 14))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 143))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 14))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 212))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 14))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 213))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 14))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 282))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 14))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 283))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 14))]));
    compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 72))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 23))]));
    compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 73))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 23))]));
    compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 142))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 23))]));
    compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 143))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 23))]));
    compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 212))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 23))]));
    compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 213))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 23))]));
    compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 282))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 23))]));
    compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 283))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 23))]));
    compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 72))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 32))]));
    compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 73))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 32))]));
    compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 142))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 32))]));
    compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 143))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 32))]));
    compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 212))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 32))]));
    compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 213))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 32))]));
    compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 282))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 32))]));
    compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 283))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 32))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 142))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 8))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 143))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 8))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 212))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 8))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 213))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 8))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 282))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 8))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 283))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 8))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 352))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 8))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 353))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 8))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 142))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 17))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 143))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 17))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 212))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 17))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 213))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 17))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 282))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 17))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 283))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 17))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 352))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 17))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 353))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 17))]));
    compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 142))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 26))]));
    compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 143))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 26))]));
    compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 212))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 26))]));
    compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 213))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 26))]));
    compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 282))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 26))]));
    compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 283))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 26))]));
    compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 352))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 26))]));
    compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 353))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 26))]));
    compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 142))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 35))]));
    compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 143))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 35))]));
    compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 212))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 35))]));
    compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 213))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 35))]));
    compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 282))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 35))]));
    compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 283))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 35))]));
    compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 352))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 35))]));
    compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[((((((int)threadIdx.x) % 34) * 2) + 353))] * kernel_shared[((((((int)threadIdx.x) / 34) * 36) + 35))]));
  }
  for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 4; ++yy_inner) {
      for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
        compute[((((((((((((int)blockIdx.x) / 272) * 1183744) + ((((int)threadIdx.x) / 34) * 295936)) + (ff_inner * 73984)) + (((((int)blockIdx.x) % 272) >> 2) * 1088)) + (yy_inner * 272)) + ((((int)blockIdx.x) & 3) * 68)) + ((((int)threadIdx.x) % 34) * 2)) + xx_inner))] = compute_local[((((ff_inner * 8) + (yy_inner * 2)) + xx_inner))];
      }
    }
  }
}



void conv_kernel_wrapper(int N_B, int N_C, int N_H, int N_W, int N_F, int N_R, int N_S, int PaddingH, int PaddingW,
                        int StrideH, int StrideW, int N_X, int N_Y, const float *Input, 
                        const float *Kernel, float *Output, int itr) {
                        hipEvent_t start;
                        CHECK(hipEventCreate(&start));
                        hipEvent_t stop;
                        CHECK(hipEventCreate(&stop));
                        float *dev_Input;
                        float *dev_Kernel;
                        float *dev_Output;
                    
                        CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                        CHECK(hipMemcpy(dev_Kernel, Kernel, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                        CHECK(hipMemcpy(dev_Input, Input, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Output, sizeof(float) * N_B * N_F * N_Y * N_X));
                        CHECK(hipMemset(dev_Output, 0, sizeof(float) * N_B * N_F * N_Y * N_X));
                    
dim3 size_grid(1088,1,1);

dim3 size_block(136,1,1);
 
                       printf("warmup....\n");
                        default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                    
                        double *time_arr = (double *)malloc(itr * sizeof(double));
                        for (int j = 1; j < itr; j++) {
                            CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                            CHECK(hipMemcpy(dev_Kernel, Kernel+j*N_F * N_C * N_R * N_S, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                            CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                            CHECK(hipMemcpy(dev_Input, Input+j*N_B * N_C * N_H * N_W, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                            /* Record the start event*/
                            CHECK(hipEventRecord(start, NULL));
                            default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                            /* Record the end event */
                            CHECK(hipEventRecord(stop, NULL));
                            /* Wait for the stop event to complete*/
                            CHECK(hipEventSynchronize(stop));
                            float msec = 0.0f;
                            hipEventElapsedTime(&msec, start, stop);
                            time_arr[j] = msec;
                        }
                    
                       
                    
                        //hipError_t err = hipGetLastError();
                        //if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
                        double msecTotal = 0;
                        for (int j = 1; j < itr; j++) {   
                            printf("%0.5lf\n", time_arr[j]);
                            msecTotal += time_arr[j];
                        }
                    
                        double msecPerMatrixMul = (msecTotal) / itr;
                        double flopsPerMatrixMul = 2.0 * N_B * N_C * N_X * N_Y * N_F * N_R * N_S;
                    
                        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
                    
                        printf("\n===========================================================\n");
                        printf("Input size: %d X %d X %d X %d\n", N_B, N_C, N_H, N_W);
                        printf("Kernel size: %d X %d X %d X %d\n", N_F, N_C, N_R, N_S);
                        printf("Output size: %d X %d X %d X %d\n", N_B, N_F, N_Y, N_X);
                        printf("flops: %0.5lf\n", (flopsPerMatrixMul * 1.0e-9f));
                        printf("Time is %0.5lf \n", (msecPerMatrixMul / 1000.0f));
                        printf("Gflops %0.5lf \n", gigaFlops);
                        printf("===========================================================\n");
                    
                        /*write back to Host Output -- batch size is 1*/
                        CHECK(hipMemcpy(Output, dev_Output, sizeof(float) * N_B * N_F * N_X * N_Y, hipMemcpyDeviceToHost));
                    
                        CHECK(hipFree(dev_Input));
                        CHECK(hipFree(dev_Kernel));
                        CHECK(hipFree(dev_Output));
                    
                    }
            