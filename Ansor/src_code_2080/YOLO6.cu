#include "hip/hip_runtime.h"

            #include <cassert>
            #include <stdlib.h>
            #include <hip/hip_runtime.h>
            #include "common.h"
            

            
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[68];
  __shared__ float pad_temp_shared[288];
  __shared__ float kernel_shared[576];
  for (int ff_c_outer_inner_init = 0; ff_c_outer_inner_init < 2; ++ff_c_outer_inner_init) {
    compute_local[((ff_c_outer_inner_init * 34))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 1))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 2))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 3))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 4))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 5))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 6))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 7))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 8))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 9))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 10))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 11))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 12))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 13))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 14))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 15))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 16))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 17))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 18))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 19))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 20))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 21))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 22))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 23))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 24))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 25))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 26))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 27))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 28))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 29))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 30))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 31))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 32))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 34) + 33))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 128; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((((int)threadIdx.x) * 2))] = data[((((rc_outer_outer * 2592) + ((((int)blockIdx.x) % 17) * 72)) + (((int)threadIdx.x) * 2)))];
    pad_temp_shared[(((((int)threadIdx.x) * 2) + 1))] = data[(((((rc_outer_outer * 2592) + ((((int)blockIdx.x) % 17) * 72)) + (((int)threadIdx.x) * 2)) + 1))];
    pad_temp_shared[(((((int)threadIdx.x) * 2) + 64))] = data[(((((rc_outer_outer * 2592) + ((((int)blockIdx.x) % 17) * 72)) + (((int)threadIdx.x) * 2)) + 64))];
    pad_temp_shared[(((((int)threadIdx.x) * 2) + 65))] = data[(((((rc_outer_outer * 2592) + ((((int)blockIdx.x) % 17) * 72)) + (((int)threadIdx.x) * 2)) + 65))];
    pad_temp_shared[(((((int)threadIdx.x) * 2) + 128))] = data[(((((rc_outer_outer * 2592) + ((((((int)threadIdx.x) * 2) + 128) / 144) * 1296)) + ((((int)blockIdx.x) % 17) * 72)) + (((((int)threadIdx.x) * 2) + 128) % 144)))];
    pad_temp_shared[(((((int)threadIdx.x) * 2) + 129))] = data[(((((rc_outer_outer * 2592) + ((((((int)threadIdx.x) * 2) + 129) / 144) * 1296)) + ((((int)blockIdx.x) % 17) * 72)) + (((((int)threadIdx.x) * 2) + 129) % 144)))];
    pad_temp_shared[(((((int)threadIdx.x) * 2) + 192))] = data[(((((rc_outer_outer * 2592) + ((((((int)threadIdx.x) * 2) + 192) / 144) * 1296)) + ((((int)blockIdx.x) % 17) * 72)) + ((((int)threadIdx.x) * 2) + 48)))];
    pad_temp_shared[(((((int)threadIdx.x) * 2) + 193))] = data[(((((rc_outer_outer * 2592) + ((((((int)threadIdx.x) * 2) + 193) / 144) * 1296)) + ((((int)blockIdx.x) % 17) * 72)) + ((((int)threadIdx.x) * 2) + 49)))];
    if (((int)threadIdx.x) < 16) {
      pad_temp_shared[(((((int)threadIdx.x) * 2) + 256))] = data[(((((rc_outer_outer * 2592) + ((((((int)threadIdx.x) * 2) + 256) / 144) * 1296)) + ((((int)blockIdx.x) % 17) * 72)) + ((((int)threadIdx.x) * 2) + 112)))];
    }
    if (((int)threadIdx.x) < 16) {
      pad_temp_shared[(((((int)threadIdx.x) * 2) + 257))] = data[(((((rc_outer_outer * 2592) + ((((((int)threadIdx.x) * 2) + 257) / 144) * 1296)) + ((((int)blockIdx.x) % 17) * 72)) + ((((int)threadIdx.x) * 2) + 113)))];
    }
      ((float2*)(kernel_shared + ((((int)threadIdx.x) * 2))))[0] = ((float2*)(kernel + ((((((((int)blockIdx.x) / 17) * 73728) + ((((int)threadIdx.x) / 9) * 2304)) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) % 9) * 2)))))[0];
      ((float2*)(kernel_shared + (((((int)threadIdx.x) * 2) + 64))))[0] = ((float2*)(kernel + ((((((((int)blockIdx.x) / 17) * 73728) + ((((((int)threadIdx.x) * 2) + 64) / 18) * 2304)) + (rc_outer_outer * 18)) + (((((int)threadIdx.x) * 2) + 10) % 18)))))[0];
      ((float2*)(kernel_shared + (((((int)threadIdx.x) * 2) + 128))))[0] = ((float2*)(kernel + ((((((((int)blockIdx.x) / 17) * 73728) + ((((((int)threadIdx.x) * 2) + 128) / 18) * 2304)) + (rc_outer_outer * 18)) + (((((int)threadIdx.x) * 2) + 2) % 18)))))[0];
      ((float2*)(kernel_shared + (((((int)threadIdx.x) * 2) + 192))))[0] = ((float2*)(kernel + ((((((((int)blockIdx.x) / 17) * 73728) + ((((((int)threadIdx.x) * 2) + 192) / 18) * 2304)) + (rc_outer_outer * 18)) + (((((int)threadIdx.x) * 2) + 12) % 18)))))[0];
      ((float2*)(kernel_shared + (((((int)threadIdx.x) * 2) + 256))))[0] = ((float2*)(kernel + ((((((((int)blockIdx.x) / 17) * 73728) + ((((((int)threadIdx.x) * 2) + 256) / 18) * 2304)) + (rc_outer_outer * 18)) + (((((int)threadIdx.x) * 2) + 4) % 18)))))[0];
      ((float2*)(kernel_shared + (((((int)threadIdx.x) * 2) + 320))))[0] = ((float2*)(kernel + ((((((((int)blockIdx.x) / 17) * 73728) + ((((((int)threadIdx.x) * 2) + 320) / 18) * 2304)) + (rc_outer_outer * 18)) + (((((int)threadIdx.x) * 2) + 14) % 18)))))[0];
      ((float2*)(kernel_shared + (((((int)threadIdx.x) * 2) + 384))))[0] = ((float2*)(kernel + ((((((((int)blockIdx.x) / 17) * 73728) + ((((((int)threadIdx.x) * 2) + 384) / 18) * 2304)) + (rc_outer_outer * 18)) + (((((int)threadIdx.x) * 2) + 6) % 18)))))[0];
      ((float2*)(kernel_shared + (((((int)threadIdx.x) * 2) + 448))))[0] = ((float2*)(kernel + ((((((((int)blockIdx.x) / 17) * 73728) + ((((((int)threadIdx.x) * 2) + 448) / 18) * 2304)) + (rc_outer_outer * 18)) + (((((int)threadIdx.x) * 2) + 16) % 18)))))[0];
      ((float2*)(kernel_shared + (((((int)threadIdx.x) * 2) + 512))))[0] = ((float2*)(kernel + ((((((((int)blockIdx.x) / 17) * 73728) + ((((((int)threadIdx.x) * 2) + 512) / 18) * 2304)) + (rc_outer_outer * 18)) + (((((int)threadIdx.x) * 2) + 8) % 18)))))[0];
    __syncthreads();
    for (int ry_outer_inner = 0; ry_outer_inner < 3; ++ry_outer_inner) {
      for (int ff_c_outer_inner = 0; ff_c_outer_inner < 2; ++ff_c_outer_inner) {
        for (int xx_c_outer_inner = 0; xx_c_outer_inner < 17; ++xx_c_outer_inner) {
          compute_local[(((ff_c_outer_inner * 34) + (xx_c_outer_inner * 2)))] = (compute_local[(((ff_c_outer_inner * 34) + (xx_c_outer_inner * 2)))] + (pad_temp_shared[((((ry_outer_inner * 36) + ((((int)threadIdx.x) & 1) * 36)) + (xx_c_outer_inner * 2)))] * kernel_shared[(((((((int)threadIdx.x) >> 1) * 36) + (ff_c_outer_inner * 18)) + (ry_outer_inner * 3)))]));
          compute_local[((((ff_c_outer_inner * 34) + (xx_c_outer_inner * 2)) + 1))] = (compute_local[((((ff_c_outer_inner * 34) + (xx_c_outer_inner * 2)) + 1))] + (pad_temp_shared[(((((ry_outer_inner * 36) + ((((int)threadIdx.x) & 1) * 36)) + (xx_c_outer_inner * 2)) + 1))] * kernel_shared[(((((((int)threadIdx.x) >> 1) * 36) + (ff_c_outer_inner * 18)) + (ry_outer_inner * 3)))]));
          compute_local[(((ff_c_outer_inner * 34) + (xx_c_outer_inner * 2)))] = (compute_local[(((ff_c_outer_inner * 34) + (xx_c_outer_inner * 2)))] + (pad_temp_shared[(((((ry_outer_inner * 36) + ((((int)threadIdx.x) & 1) * 36)) + (xx_c_outer_inner * 2)) + 1))] * kernel_shared[((((((((int)threadIdx.x) >> 1) * 36) + (ff_c_outer_inner * 18)) + (ry_outer_inner * 3)) + 1))]));
          compute_local[((((ff_c_outer_inner * 34) + (xx_c_outer_inner * 2)) + 1))] = (compute_local[((((ff_c_outer_inner * 34) + (xx_c_outer_inner * 2)) + 1))] + (pad_temp_shared[(((((ry_outer_inner * 36) + ((((int)threadIdx.x) & 1) * 36)) + (xx_c_outer_inner * 2)) + 2))] * kernel_shared[((((((((int)threadIdx.x) >> 1) * 36) + (ff_c_outer_inner * 18)) + (ry_outer_inner * 3)) + 1))]));
          compute_local[(((ff_c_outer_inner * 34) + (xx_c_outer_inner * 2)))] = (compute_local[(((ff_c_outer_inner * 34) + (xx_c_outer_inner * 2)))] + (pad_temp_shared[(((((ry_outer_inner * 36) + ((((int)threadIdx.x) & 1) * 36)) + (xx_c_outer_inner * 2)) + 2))] * kernel_shared[((((((((int)threadIdx.x) >> 1) * 36) + (ff_c_outer_inner * 18)) + (ry_outer_inner * 3)) + 2))]));
          compute_local[((((ff_c_outer_inner * 34) + (xx_c_outer_inner * 2)) + 1))] = (compute_local[((((ff_c_outer_inner * 34) + (xx_c_outer_inner * 2)) + 1))] + (pad_temp_shared[(((((ry_outer_inner * 36) + ((((int)threadIdx.x) & 1) * 36)) + (xx_c_outer_inner * 2)) + 3))] * kernel_shared[((((((((int)threadIdx.x) >> 1) * 36) + (ff_c_outer_inner * 18)) + (ry_outer_inner * 3)) + 2))]));
          compute_local[(((ff_c_outer_inner * 34) + (xx_c_outer_inner * 2)))] = (compute_local[(((ff_c_outer_inner * 34) + (xx_c_outer_inner * 2)))] + (pad_temp_shared[(((((ry_outer_inner * 36) + ((((int)threadIdx.x) & 1) * 36)) + (xx_c_outer_inner * 2)) + 144))] * kernel_shared[((((((((int)threadIdx.x) >> 1) * 36) + (ff_c_outer_inner * 18)) + (ry_outer_inner * 3)) + 9))]));
          compute_local[((((ff_c_outer_inner * 34) + (xx_c_outer_inner * 2)) + 1))] = (compute_local[((((ff_c_outer_inner * 34) + (xx_c_outer_inner * 2)) + 1))] + (pad_temp_shared[(((((ry_outer_inner * 36) + ((((int)threadIdx.x) & 1) * 36)) + (xx_c_outer_inner * 2)) + 145))] * kernel_shared[((((((((int)threadIdx.x) >> 1) * 36) + (ff_c_outer_inner * 18)) + (ry_outer_inner * 3)) + 9))]));
          compute_local[(((ff_c_outer_inner * 34) + (xx_c_outer_inner * 2)))] = (compute_local[(((ff_c_outer_inner * 34) + (xx_c_outer_inner * 2)))] + (pad_temp_shared[(((((ry_outer_inner * 36) + ((((int)threadIdx.x) & 1) * 36)) + (xx_c_outer_inner * 2)) + 145))] * kernel_shared[((((((((int)threadIdx.x) >> 1) * 36) + (ff_c_outer_inner * 18)) + (ry_outer_inner * 3)) + 10))]));
          compute_local[((((ff_c_outer_inner * 34) + (xx_c_outer_inner * 2)) + 1))] = (compute_local[((((ff_c_outer_inner * 34) + (xx_c_outer_inner * 2)) + 1))] + (pad_temp_shared[(((((ry_outer_inner * 36) + ((((int)threadIdx.x) & 1) * 36)) + (xx_c_outer_inner * 2)) + 146))] * kernel_shared[((((((((int)threadIdx.x) >> 1) * 36) + (ff_c_outer_inner * 18)) + (ry_outer_inner * 3)) + 10))]));
          compute_local[(((ff_c_outer_inner * 34) + (xx_c_outer_inner * 2)))] = (compute_local[(((ff_c_outer_inner * 34) + (xx_c_outer_inner * 2)))] + (pad_temp_shared[(((((ry_outer_inner * 36) + ((((int)threadIdx.x) & 1) * 36)) + (xx_c_outer_inner * 2)) + 146))] * kernel_shared[((((((((int)threadIdx.x) >> 1) * 36) + (ff_c_outer_inner * 18)) + (ry_outer_inner * 3)) + 11))]));
          compute_local[((((ff_c_outer_inner * 34) + (xx_c_outer_inner * 2)) + 1))] = (compute_local[((((ff_c_outer_inner * 34) + (xx_c_outer_inner * 2)) + 1))] + (pad_temp_shared[(((((ry_outer_inner * 36) + ((((int)threadIdx.x) & 1) * 36)) + (xx_c_outer_inner * 2)) + 147))] * kernel_shared[((((((((int)threadIdx.x) >> 1) * 36) + (ff_c_outer_inner * 18)) + (ry_outer_inner * 3)) + 11))]));
        }
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
    for (int xx_inner = 0; xx_inner < 34; ++xx_inner) {
      compute[((((((((((int)blockIdx.x) / 17) * 36992) + ((((int)threadIdx.x) >> 1) * 2312)) + (ff_inner * 1156)) + ((((int)blockIdx.x) % 17) * 68)) + ((((int)threadIdx.x) & 1) * 34)) + xx_inner))] = compute_local[(((ff_inner * 34) + xx_inner))];
    }
  }
}



void conv_kernel_wrapper(int N_B, int N_C, int N_H, int N_W, int N_F, int N_R, int N_S, int PaddingH, int PaddingW,
                        int StrideH, int StrideW, int N_X, int N_Y, const float *Input, 
                        const float *Kernel, float *Output, int itr) {
                        hipEvent_t start;
                        CHECK(hipEventCreate(&start));
                        hipEvent_t stop;
                        CHECK(hipEventCreate(&stop));
                        float *dev_Input;
                        float *dev_Kernel;
                        float *dev_Output;
                    
                        CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                        CHECK(hipMemcpy(dev_Kernel, Kernel, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                        CHECK(hipMemcpy(dev_Input, Input, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Output, sizeof(float) * N_B * N_F * N_Y * N_X));
                        CHECK(hipMemset(dev_Output, 0, sizeof(float) * N_B * N_F * N_Y * N_X));
                    
dim3 size_grid(272,1,1);

dim3 size_block(32,1,1);
 
                       printf("warmup....\n");
                        default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                    
                        double *time_arr = (double *)malloc(itr * sizeof(double));
                        for (int j = 1; j < itr; j++) {
                            CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                            CHECK(hipMemcpy(dev_Kernel, Kernel+j*N_F * N_C * N_R * N_S, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                            CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                            CHECK(hipMemcpy(dev_Input, Input+j*N_B * N_C * N_H * N_W, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                            /* Record the start event*/
                            CHECK(hipEventRecord(start, NULL));
                            default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                            /* Record the end event */
                            CHECK(hipEventRecord(stop, NULL));
                            /* Wait for the stop event to complete*/
                            CHECK(hipEventSynchronize(stop));
                            float msec = 0.0f;
                            hipEventElapsedTime(&msec, start, stop);
                            time_arr[j] = msec;
                        }
                    
                       
                    
                        //hipError_t err = hipGetLastError();
                        //if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
                        double msecTotal = 0;
                        for (int j = 1; j < itr; j++) {   
                            printf("%0.5lf\n", time_arr[j]);
                            msecTotal += time_arr[j];
                        }
                    
                        double msecPerMatrixMul = (msecTotal) / itr;
                        double flopsPerMatrixMul = 2.0 * N_B * N_C * N_X * N_Y * N_F * N_R * N_S;
                    
                        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
                    
                        printf("\n===========================================================\n");
                        printf("Input size: %d X %d X %d X %d\n", N_B, N_C, N_H, N_W);
                        printf("Kernel size: %d X %d X %d X %d\n", N_F, N_C, N_R, N_S);
                        printf("Output size: %d X %d X %d X %d\n", N_B, N_F, N_Y, N_X);
                        printf("flops: %0.5lf\n", (flopsPerMatrixMul * 1.0e-9f));
                        printf("Time is %0.5lf \n", (msecPerMatrixMul / 1000.0f));
                        printf("Gflops %0.5lf \n", gigaFlops);
                        printf("===========================================================\n");
                    
                        /*write back to Host Output -- batch size is 1*/
                        CHECK(hipMemcpy(Output, dev_Output, sizeof(float) * N_B * N_F * N_X * N_Y, hipMemcpyDeviceToHost));
                    
                        CHECK(hipFree(dev_Input));
                        CHECK(hipFree(dev_Kernel));
                        CHECK(hipFree(dev_Output));
                    
                    }
            