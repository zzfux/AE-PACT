#include "hip/hip_runtime.h"

            #include <cassert>
            #include <stdlib.h>
            #include <hip/hip_runtime.h>
            #include "common.h"
            

            
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[32];
  __shared__ float pad_temp_shared[4352];
  __shared__ float kernel_shared[1024];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  compute_local[(7)] = 0.000000e+00f;
  compute_local[(8)] = 0.000000e+00f;
  compute_local[(9)] = 0.000000e+00f;
  compute_local[(10)] = 0.000000e+00f;
  compute_local[(11)] = 0.000000e+00f;
  compute_local[(12)] = 0.000000e+00f;
  compute_local[(13)] = 0.000000e+00f;
  compute_local[(14)] = 0.000000e+00f;
  compute_local[(15)] = 0.000000e+00f;
  compute_local[(16)] = 0.000000e+00f;
  compute_local[(17)] = 0.000000e+00f;
  compute_local[(18)] = 0.000000e+00f;
  compute_local[(19)] = 0.000000e+00f;
  compute_local[(20)] = 0.000000e+00f;
  compute_local[(21)] = 0.000000e+00f;
  compute_local[(22)] = 0.000000e+00f;
  compute_local[(23)] = 0.000000e+00f;
  compute_local[(24)] = 0.000000e+00f;
  compute_local[(25)] = 0.000000e+00f;
  compute_local[(26)] = 0.000000e+00f;
  compute_local[(27)] = 0.000000e+00f;
  compute_local[(28)] = 0.000000e+00f;
  compute_local[(29)] = 0.000000e+00f;
  compute_local[(30)] = 0.000000e+00f;
  compute_local[(31)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
      ((float4*)(pad_temp_shared + ((((int)threadIdx.x) * 4))))[0] = ((float4*)(data + (((((rc_outer_outer * 147968) + ((((int)threadIdx.x) / 34) * 4624)) + ((((int)blockIdx.x) % 34) * 136)) + ((((int)threadIdx.x) % 34) * 4)))))[0];
      ((float4*)(pad_temp_shared + (((((int)threadIdx.x) * 4) + 544))))[0] = ((float4*)(data + ((((((rc_outer_outer * 147968) + ((((int)threadIdx.x) / 34) * 4624)) + ((((int)blockIdx.x) % 34) * 136)) + ((((int)threadIdx.x) % 34) * 4)) + 18496))))[0];
      ((float4*)(pad_temp_shared + (((((int)threadIdx.x) * 4) + 1088))))[0] = ((float4*)(data + ((((((rc_outer_outer * 147968) + ((((int)threadIdx.x) / 34) * 4624)) + ((((int)blockIdx.x) % 34) * 136)) + ((((int)threadIdx.x) % 34) * 4)) + 36992))))[0];
      ((float4*)(pad_temp_shared + (((((int)threadIdx.x) * 4) + 1632))))[0] = ((float4*)(data + ((((((rc_outer_outer * 147968) + ((((int)threadIdx.x) / 34) * 4624)) + ((((int)blockIdx.x) % 34) * 136)) + ((((int)threadIdx.x) % 34) * 4)) + 55488))))[0];
      ((float4*)(pad_temp_shared + (((((int)threadIdx.x) * 4) + 2176))))[0] = ((float4*)(data + ((((((rc_outer_outer * 147968) + ((((int)threadIdx.x) / 34) * 4624)) + ((((int)blockIdx.x) % 34) * 136)) + ((((int)threadIdx.x) % 34) * 4)) + 73984))))[0];
      ((float4*)(pad_temp_shared + (((((int)threadIdx.x) * 4) + 2720))))[0] = ((float4*)(data + ((((((rc_outer_outer * 147968) + ((((int)threadIdx.x) / 34) * 4624)) + ((((int)blockIdx.x) % 34) * 136)) + ((((int)threadIdx.x) % 34) * 4)) + 92480))))[0];
      ((float4*)(pad_temp_shared + (((((int)threadIdx.x) * 4) + 3264))))[0] = ((float4*)(data + ((((((rc_outer_outer * 147968) + ((((int)threadIdx.x) / 34) * 4624)) + ((((int)blockIdx.x) % 34) * 136)) + ((((int)threadIdx.x) % 34) * 4)) + 110976))))[0];
      ((float4*)(pad_temp_shared + (((((int)threadIdx.x) * 4) + 3808))))[0] = ((float4*)(data + ((((((rc_outer_outer * 147968) + ((((int)threadIdx.x) / 34) * 4624)) + ((((int)blockIdx.x) % 34) * 136)) + ((((int)threadIdx.x) % 34) * 4)) + 129472))))[0];
    kernel_shared[(((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 34) * 8192) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)))];
    kernel_shared[((((int)threadIdx.x) + 136))] = kernel[((((((((int)blockIdx.x) / 34) * 8192) + (((((int)threadIdx.x) + 136) >> 5) * 256)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) + 8) & 31)))];
    kernel_shared[((((int)threadIdx.x) + 272))] = kernel[((((((((int)blockIdx.x) / 34) * 8192) + (((((int)threadIdx.x) + 272) >> 5) * 256)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) + 16) & 31)))];
    kernel_shared[((((int)threadIdx.x) + 408))] = kernel[((((((((int)blockIdx.x) / 34) * 8192) + (((((int)threadIdx.x) + 408) >> 5) * 256)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) + 24) & 31)))];
    kernel_shared[((((int)threadIdx.x) + 544))] = kernel[(((((((((int)blockIdx.x) / 34) * 8192) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 4352))];
    kernel_shared[((((int)threadIdx.x) + 680))] = kernel[((((((((int)blockIdx.x) / 34) * 8192) + (((((int)threadIdx.x) + 680) >> 5) * 256)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) + 8) & 31)))];
    kernel_shared[((((int)threadIdx.x) + 816))] = kernel[((((((((int)blockIdx.x) / 34) * 8192) + (((((int)threadIdx.x) + 816) >> 5) * 256)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) + 16) & 31)))];
    if (((int)threadIdx.x) < 72) {
      kernel_shared[((((int)threadIdx.x) + 952))] = kernel[((((((((int)blockIdx.x) / 34) * 8192) + (((((int)threadIdx.x) + 952) >> 5) * 256)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) + 24) & 31)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)))] * kernel_shared[((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 1))] * kernel_shared[((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 32))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 32))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 136))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 1))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 137))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 1))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 136))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 33))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 137))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 33))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 272))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 2))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 273))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 2))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 272))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 34))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 273))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 34))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 408))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 3))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 409))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 3))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 408))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 35))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 409))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 35))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 64))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 64))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 96))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 96))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 136))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 65))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 137))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 65))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 136))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 97))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 137))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 97))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 272))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 66))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 273))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 66))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 272))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 98))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 273))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 98))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 408))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 67))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 409))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 67))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 408))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 99))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 409))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 99))]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 128))]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 128))]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 160))]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 160))]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 136))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 129))]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 137))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 129))]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 136))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 161))]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 137))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 161))]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 272))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 130))]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 273))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 130))]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 272))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 162))]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 273))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 162))]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 408))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 131))]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 409))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 131))]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 408))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 163))]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 409))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 163))]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 192))]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 192))]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 224))]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 224))]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 136))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 193))]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 137))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 193))]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 136))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 225))]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 137))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 225))]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 272))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 194))]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 273))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 194))]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 272))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 226))]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 273))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 226))]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 408))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 195))]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 409))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 195))]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 408))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 227))]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 409))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 227))]));
      compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 256))]));
      compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 256))]));
      compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[(((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 288))]));
      compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 288))]));
      compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 136))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 257))]));
      compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 137))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 257))]));
      compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 136))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 289))]));
      compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 137))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 289))]));
      compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 272))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 258))]));
      compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 273))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 258))]));
      compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 272))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 290))]));
      compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 273))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 290))]));
      compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 408))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 259))]));
      compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 409))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 259))]));
      compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 408))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 291))]));
      compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 409))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 291))]));
      compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[(((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 320))]));
      compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 320))]));
      compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[(((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 352))]));
      compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 352))]));
      compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 136))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 321))]));
      compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 137))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 321))]));
      compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 136))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 353))]));
      compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 137))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 353))]));
      compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 272))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 322))]));
      compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 273))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 322))]));
      compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 272))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 354))]));
      compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 273))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 354))]));
      compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 408))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 323))]));
      compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 409))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 323))]));
      compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 408))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 355))]));
      compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 409))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 355))]));
      compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[(((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 384))]));
      compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 384))]));
      compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[(((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 416))]));
      compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 416))]));
      compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 136))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 385))]));
      compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 137))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 385))]));
      compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 136))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 417))]));
      compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 137))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 417))]));
      compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 272))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 386))]));
      compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 273))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 386))]));
      compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 272))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 418))]));
      compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 273))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 418))]));
      compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 408))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 387))]));
      compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 409))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 387))]));
      compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 408))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 419))]));
      compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 409))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 419))]));
      compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[(((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 448))]));
      compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 448))]));
      compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[(((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 480))]));
      compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 480))]));
      compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 136))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 449))]));
      compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 137))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 449))]));
      compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 136))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 481))]));
      compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 137))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 481))]));
      compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 272))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 450))]));
      compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 273))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 450))]));
      compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 272))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 482))]));
      compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 273))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 482))]));
      compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 408))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 451))]));
      compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 409))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 451))]));
      compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 408))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 483))]));
      compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[((((rc_outer_inner * 544) + ((((int)threadIdx.x) % 68) * 2)) + 409))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 512) + (rc_outer_inner * 4)) + 483))]));
    }
  }
  for (int ff_inner = 0; ff_inner < 16; ++ff_inner) {
    for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
      compute[((((((((((int)blockIdx.x) / 34) * 147968) + ((((int)threadIdx.x) / 68) * 73984)) + (ff_inner * 4624)) + ((((int)blockIdx.x) % 34) * 136)) + ((((int)threadIdx.x) % 68) * 2)) + xx_inner))] = compute_local[(((ff_inner * 2) + xx_inner))];
    }
  }
}



void conv_kernel_wrapper(int N_B, int N_C, int N_H, int N_W, int N_F, int N_R, int N_S, int PaddingH, int PaddingW,
                        int StrideH, int StrideW, int N_X, int N_Y, const float *Input, 
                        const float *Kernel, float *Output, int itr) {
                        hipEvent_t start;
                        CHECK(hipEventCreate(&start));
                        hipEvent_t stop;
                        CHECK(hipEventCreate(&stop));
                        float *dev_Input;
                        float *dev_Kernel;
                        float *dev_Output;
                    
                        CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                        CHECK(hipMemcpy(dev_Kernel, Kernel, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                        CHECK(hipMemcpy(dev_Input, Input, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Output, sizeof(float) * N_B * N_F * N_Y * N_X));
                        CHECK(hipMemset(dev_Output, 0, sizeof(float) * N_B * N_F * N_Y * N_X));
                    
dim3 size_grid(136,1,1);

dim3 size_block(136,1,1);
 
                       printf("warmup....\n");
                        default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                    
                        double *time_arr = (double *)malloc(itr * sizeof(double));
                        for (int j = 1; j < itr; j++) {
                            CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                            CHECK(hipMemcpy(dev_Kernel, Kernel+j*N_F * N_C * N_R * N_S, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                            CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                            CHECK(hipMemcpy(dev_Input, Input+j*N_B * N_C * N_H * N_W, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                            /* Record the start event*/
                            CHECK(hipEventRecord(start, NULL));
                            default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                            /* Record the end event */
                            CHECK(hipEventRecord(stop, NULL));
                            /* Wait for the stop event to complete*/
                            CHECK(hipEventSynchronize(stop));
                            float msec = 0.0f;
                            hipEventElapsedTime(&msec, start, stop);
                            time_arr[j] = msec;
                        }
                    
                       
                    
                        //hipError_t err = hipGetLastError();
                        //if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
                        double msecTotal = 0;
                        for (int j = 1; j < itr; j++) {   
                            printf("%0.5lf\n", time_arr[j]);
                            msecTotal += time_arr[j];
                        }
                    
                        double msecPerMatrixMul = (msecTotal) / itr;
                        double flopsPerMatrixMul = 2.0 * N_B * N_C * N_X * N_Y * N_F * N_R * N_S;
                    
                        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
                    
                        printf("\n===========================================================\n");
                        printf("Input size: %d X %d X %d X %d\n", N_B, N_C, N_H, N_W);
                        printf("Kernel size: %d X %d X %d X %d\n", N_F, N_C, N_R, N_S);
                        printf("Output size: %d X %d X %d X %d\n", N_B, N_F, N_Y, N_X);
                        printf("flops: %0.5lf\n", (flopsPerMatrixMul * 1.0e-9f));
                        printf("Time is %0.5lf \n", (msecPerMatrixMul / 1000.0f));
                        printf("Gflops %0.5lf \n", gigaFlops);
                        printf("===========================================================\n");
                    
                        /*write back to Host Output -- batch size is 1*/
                        CHECK(hipMemcpy(Output, dev_Output, sizeof(float) * N_B * N_F * N_X * N_Y, hipMemcpyDeviceToHost));
                    
                        CHECK(hipFree(dev_Input));
                        CHECK(hipFree(dev_Kernel));
                        CHECK(hipFree(dev_Output));
                    
                    }
            