#include "hip/hip_runtime.h"

            #include <cassert>
            #include <stdlib.h>
            #include <hip/hip_runtime.h>
            #include "common.h"
            

            
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[16];
  __shared__ float pad_temp_shared[1008];
  __shared__ float kernel_shared[432];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(8)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(9)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(10)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(11)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(12)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(13)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  compute_local[(14)] = 0.000000e+00f;
  compute_local[(7)] = 0.000000e+00f;
  compute_local[(15)] = 0.000000e+00f;
  if (((int)threadIdx.x) < 112) {
    pad_temp_shared[((((int)threadIdx.x) * 9))] = data[((((((((((int)threadIdx.x) * 9) / 336) * 12100) + (((((int)blockIdx.x) % 54) >> 1) * 440)) + ((((((int)threadIdx.x) * 9) % 336) / 56) * 110)) + ((((int)blockIdx.x) & 1) * 54)) + ((((int)threadIdx.x) * 9) % 56)))];
  }
  if (((int)threadIdx.x) < 112) {
    pad_temp_shared[(((((int)threadIdx.x) * 9) + 1))] = data[(((((((((((int)threadIdx.x) * 9) + 1) / 336) * 12100) + (((((int)blockIdx.x) % 54) >> 1) * 440)) + (((((((int)threadIdx.x) * 9) + 1) % 336) / 56) * 110)) + ((((int)blockIdx.x) & 1) * 54)) + (((((int)threadIdx.x) * 9) + 1) % 56)))];
  }
  if (((int)threadIdx.x) < 112) {
    pad_temp_shared[(((((int)threadIdx.x) * 9) + 2))] = data[(((((((((((int)threadIdx.x) * 9) + 2) / 336) * 12100) + (((((int)blockIdx.x) % 54) >> 1) * 440)) + (((((((int)threadIdx.x) * 9) + 2) % 336) / 56) * 110)) + ((((int)blockIdx.x) & 1) * 54)) + (((((int)threadIdx.x) * 9) + 2) % 56)))];
  }
  if (((int)threadIdx.x) < 112) {
    pad_temp_shared[(((((int)threadIdx.x) * 9) + 3))] = data[(((((((((((int)threadIdx.x) * 9) + 3) / 336) * 12100) + (((((int)blockIdx.x) % 54) >> 1) * 440)) + (((((((int)threadIdx.x) * 9) + 3) % 336) / 56) * 110)) + ((((int)blockIdx.x) & 1) * 54)) + (((((int)threadIdx.x) * 9) + 3) % 56)))];
  }
  if (((int)threadIdx.x) < 112) {
    pad_temp_shared[(((((int)threadIdx.x) * 9) + 4))] = data[(((((((((((int)threadIdx.x) * 9) + 4) / 336) * 12100) + (((((int)blockIdx.x) % 54) >> 1) * 440)) + (((((((int)threadIdx.x) * 9) + 4) % 336) / 56) * 110)) + ((((int)blockIdx.x) & 1) * 54)) + (((((int)threadIdx.x) * 9) + 4) % 56)))];
  }
  if (((int)threadIdx.x) < 112) {
    pad_temp_shared[(((((int)threadIdx.x) * 9) + 5))] = data[(((((((((((int)threadIdx.x) * 9) + 5) / 336) * 12100) + (((((int)blockIdx.x) % 54) >> 1) * 440)) + (((((((int)threadIdx.x) * 9) + 5) % 336) / 56) * 110)) + ((((int)blockIdx.x) & 1) * 54)) + (((((int)threadIdx.x) * 9) + 5) % 56)))];
  }
  if (((int)threadIdx.x) < 112) {
    pad_temp_shared[(((((int)threadIdx.x) * 9) + 6))] = data[(((((((((((int)threadIdx.x) * 9) + 6) / 336) * 12100) + (((((int)blockIdx.x) % 54) >> 1) * 440)) + (((((((int)threadIdx.x) * 9) + 6) % 336) / 56) * 110)) + ((((int)blockIdx.x) & 1) * 54)) + (((((int)threadIdx.x) * 9) + 6) % 56)))];
  }
  if (((int)threadIdx.x) < 112) {
    pad_temp_shared[(((((int)threadIdx.x) * 9) + 7))] = data[(((((((((((int)threadIdx.x) * 9) + 7) / 336) * 12100) + (((((int)blockIdx.x) % 54) >> 1) * 440)) + (((((((int)threadIdx.x) * 9) + 7) % 336) / 56) * 110)) + ((((int)blockIdx.x) & 1) * 54)) + (((((int)threadIdx.x) * 9) + 7) % 56)))];
  }
  if (((int)threadIdx.x) < 112) {
    pad_temp_shared[(((((int)threadIdx.x) * 9) + 8))] = data[(((((((((((int)threadIdx.x) * 9) + 8) / 336) * 12100) + (((((int)blockIdx.x) % 54) >> 1) * 440)) + (((((((int)threadIdx.x) * 9) + 8) % 336) / 56) * 110)) + ((((int)blockIdx.x) & 1) * 54)) + (((((int)threadIdx.x) * 9) + 8) % 56)))];
  }
  if (((int)threadIdx.x) < 72) {
    kernel_shared[((((int)threadIdx.x) * 6))] = kernel[((((((int)blockIdx.x) / 54) * 432) + (((int)threadIdx.x) * 6)))];
  }
  if (((int)threadIdx.x) < 72) {
    kernel_shared[(((((int)threadIdx.x) * 6) + 1))] = kernel[(((((((int)blockIdx.x) / 54) * 432) + (((int)threadIdx.x) * 6)) + 1))];
  }
  if (((int)threadIdx.x) < 72) {
    kernel_shared[(((((int)threadIdx.x) * 6) + 2))] = kernel[(((((((int)blockIdx.x) / 54) * 432) + (((int)threadIdx.x) * 6)) + 2))];
  }
  if (((int)threadIdx.x) < 72) {
    kernel_shared[(((((int)threadIdx.x) * 6) + 3))] = kernel[(((((((int)blockIdx.x) / 54) * 432) + (((int)threadIdx.x) * 6)) + 3))];
  }
  if (((int)threadIdx.x) < 72) {
    kernel_shared[(((((int)threadIdx.x) * 6) + 4))] = kernel[(((((((int)blockIdx.x) / 54) * 432) + (((int)threadIdx.x) * 6)) + 4))];
  }
  if (((int)threadIdx.x) < 72) {
    kernel_shared[(((((int)threadIdx.x) * 6) + 5))] = kernel[(((((((int)blockIdx.x) / 54) * 432) + (((int)threadIdx.x) * 6)) + 5))];
  }
  __syncthreads();
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((int)threadIdx.x) % 54))] * kernel_shared[(((((int)threadIdx.x) / 54) * 108))]));
  compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 112))] * kernel_shared[(((((int)threadIdx.x) / 54) * 108))]));
  compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 56))] * kernel_shared[(((((int)threadIdx.x) / 54) * 108))]));
  compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 168))] * kernel_shared[(((((int)threadIdx.x) / 54) * 108))]));
  compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((int)threadIdx.x) % 54))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 27))]));
  compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 112))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 27))]));
  compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 56))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 27))]));
  compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 168))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 27))]));
  compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((int)threadIdx.x) % 54))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 54))]));
  compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 112))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 54))]));
  compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 56))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 54))]));
  compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 168))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 54))]));
  compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((int)threadIdx.x) % 54))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 81))]));
  compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 112))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 81))]));
  compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 56))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 81))]));
  compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 168))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 81))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 56))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 3))]));
  compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 168))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 3))]));
  compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 112))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 3))]));
  compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 224))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 3))]));
  compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 56))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 30))]));
  compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 168))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 30))]));
  compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 112))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 30))]));
  compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 224))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 30))]));
  compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 56))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 57))]));
  compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 168))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 57))]));
  compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 112))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 57))]));
  compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 224))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 57))]));
  compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 56))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 84))]));
  compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 168))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 84))]));
  compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 112))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 84))]));
  compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 224))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 84))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 112))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 6))]));
  compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 224))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 6))]));
  compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 168))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 6))]));
  compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 280))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 6))]));
  compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 112))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 33))]));
  compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 224))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 33))]));
  compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 168))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 33))]));
  compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 280))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 33))]));
  compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 112))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 60))]));
  compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 224))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 60))]));
  compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 168))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 60))]));
  compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 280))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 60))]));
  compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 112))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 87))]));
  compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 224))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 87))]));
  compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 168))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 87))]));
  compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 280))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 87))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 1))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 1))]));
  compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 113))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 1))]));
  compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 57))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 1))]));
  compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 169))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 1))]));
  compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 1))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 28))]));
  compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 113))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 28))]));
  compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 57))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 28))]));
  compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 169))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 28))]));
  compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 1))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 55))]));
  compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 113))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 55))]));
  compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 57))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 55))]));
  compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 169))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 55))]));
  compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 1))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 82))]));
  compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 113))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 82))]));
  compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 57))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 82))]));
  compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 169))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 82))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 57))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 4))]));
  compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 169))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 4))]));
  compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 113))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 4))]));
  compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 225))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 4))]));
  compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 57))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 31))]));
  compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 169))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 31))]));
  compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 113))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 31))]));
  compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 225))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 31))]));
  compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 57))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 58))]));
  compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 169))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 58))]));
  compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 113))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 58))]));
  compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 225))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 58))]));
  compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 57))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 85))]));
  compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 169))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 85))]));
  compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 113))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 85))]));
  compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 225))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 85))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 113))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 7))]));
  compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 225))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 7))]));
  compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 169))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 7))]));
  compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 281))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 7))]));
  compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 113))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 34))]));
  compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 225))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 34))]));
  compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 169))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 34))]));
  compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 281))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 34))]));
  compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 113))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 61))]));
  compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 225))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 61))]));
  compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 169))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 61))]));
  compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 281))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 61))]));
  compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 113))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 88))]));
  compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 225))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 88))]));
  compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 169))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 88))]));
  compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 281))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 88))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 2))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 2))]));
  compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 114))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 2))]));
  compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 58))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 2))]));
  compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 170))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 2))]));
  compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 2))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 29))]));
  compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 114))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 29))]));
  compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 58))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 29))]));
  compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 170))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 29))]));
  compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 2))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 56))]));
  compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 114))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 56))]));
  compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 58))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 56))]));
  compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 170))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 56))]));
  compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 2))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 83))]));
  compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 114))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 83))]));
  compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 58))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 83))]));
  compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 170))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 83))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 58))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 5))]));
  compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 170))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 5))]));
  compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 114))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 5))]));
  compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 226))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 5))]));
  compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 58))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 32))]));
  compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 170))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 32))]));
  compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 114))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 32))]));
  compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 226))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 32))]));
  compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 58))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 59))]));
  compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 170))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 59))]));
  compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 114))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 59))]));
  compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 226))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 59))]));
  compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 58))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 86))]));
  compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 170))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 86))]));
  compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 114))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 86))]));
  compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 226))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 86))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 114))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 8))]));
  compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 226))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 8))]));
  compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 170))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 8))]));
  compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 282))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 8))]));
  compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 114))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 35))]));
  compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 226))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 35))]));
  compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 170))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 35))]));
  compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 282))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 35))]));
  compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 114))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 62))]));
  compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 226))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 62))]));
  compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 170))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 62))]));
  compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 282))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 62))]));
  compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 114))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 89))]));
  compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 226))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 89))]));
  compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 170))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 89))]));
  compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 282))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 89))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 336))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 9))]));
  compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 448))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 9))]));
  compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 392))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 9))]));
  compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 504))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 9))]));
  compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 336))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 36))]));
  compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 448))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 36))]));
  compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 392))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 36))]));
  compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 504))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 36))]));
  compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 336))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 63))]));
  compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 448))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 63))]));
  compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 392))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 63))]));
  compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 504))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 63))]));
  compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 336))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 90))]));
  compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 448))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 90))]));
  compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 392))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 90))]));
  compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 504))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 90))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 392))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 12))]));
  compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 504))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 12))]));
  compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 448))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 12))]));
  compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 560))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 12))]));
  compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 392))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 39))]));
  compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 504))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 39))]));
  compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 448))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 39))]));
  compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 560))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 39))]));
  compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 392))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 66))]));
  compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 504))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 66))]));
  compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 448))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 66))]));
  compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 560))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 66))]));
  compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 392))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 93))]));
  compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 504))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 93))]));
  compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 448))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 93))]));
  compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 560))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 93))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 448))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 15))]));
  compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 560))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 15))]));
  compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 504))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 15))]));
  compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 616))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 15))]));
  compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 448))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 42))]));
  compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 560))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 42))]));
  compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 504))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 42))]));
  compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 616))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 42))]));
  compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 448))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 69))]));
  compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 560))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 69))]));
  compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 504))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 69))]));
  compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 616))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 69))]));
  compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 448))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 96))]));
  compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 560))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 96))]));
  compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 504))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 96))]));
  compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 616))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 96))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 337))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 10))]));
  compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 449))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 10))]));
  compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 393))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 10))]));
  compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 505))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 10))]));
  compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 337))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 37))]));
  compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 449))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 37))]));
  compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 393))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 37))]));
  compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 505))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 37))]));
  compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 337))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 64))]));
  compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 449))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 64))]));
  compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 393))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 64))]));
  compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 505))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 64))]));
  compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 337))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 91))]));
  compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 449))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 91))]));
  compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 393))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 91))]));
  compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 505))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 91))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 393))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 13))]));
  compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 505))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 13))]));
  compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 449))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 13))]));
  compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 561))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 13))]));
  compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 393))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 40))]));
  compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 505))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 40))]));
  compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 449))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 40))]));
  compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 561))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 40))]));
  compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 393))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 67))]));
  compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 505))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 67))]));
  compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 449))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 67))]));
  compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 561))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 67))]));
  compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 393))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 94))]));
  compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 505))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 94))]));
  compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 449))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 94))]));
  compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 561))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 94))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 449))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 16))]));
  compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 561))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 16))]));
  compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 505))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 16))]));
  compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 617))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 16))]));
  compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 449))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 43))]));
  compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 561))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 43))]));
  compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 505))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 43))]));
  compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 617))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 43))]));
  compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 449))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 70))]));
  compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 561))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 70))]));
  compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 505))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 70))]));
  compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 617))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 70))]));
  compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 449))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 97))]));
  compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 561))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 97))]));
  compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 505))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 97))]));
  compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 617))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 97))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 338))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 11))]));
  compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 450))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 11))]));
  compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 394))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 11))]));
  compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 506))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 11))]));
  compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 338))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 38))]));
  compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 450))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 38))]));
  compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 394))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 38))]));
  compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 506))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 38))]));
  compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 338))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 65))]));
  compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 450))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 65))]));
  compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 394))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 65))]));
  compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 506))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 65))]));
  compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 338))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 92))]));
  compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 450))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 92))]));
  compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 394))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 92))]));
  compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 506))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 92))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 394))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 14))]));
  compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 506))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 14))]));
  compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 450))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 14))]));
  compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 562))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 14))]));
  compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 394))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 41))]));
  compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 506))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 41))]));
  compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 450))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 41))]));
  compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 562))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 41))]));
  compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 394))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 68))]));
  compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 506))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 68))]));
  compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 450))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 68))]));
  compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 562))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 68))]));
  compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 394))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 95))]));
  compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 506))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 95))]));
  compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 450))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 95))]));
  compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 562))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 95))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 450))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 17))]));
  compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 562))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 17))]));
  compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 506))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 17))]));
  compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 618))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 17))]));
  compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 450))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 44))]));
  compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 562))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 44))]));
  compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 506))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 44))]));
  compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 618))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 44))]));
  compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 450))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 71))]));
  compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 562))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 71))]));
  compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 506))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 71))]));
  compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 618))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 71))]));
  compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 450))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 98))]));
  compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 562))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 98))]));
  compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 506))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 98))]));
  compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 618))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 98))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 672))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 18))]));
  compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 784))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 18))]));
  compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 728))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 18))]));
  compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 840))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 18))]));
  compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 672))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 45))]));
  compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 784))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 45))]));
  compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 728))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 45))]));
  compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 840))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 45))]));
  compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 672))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 72))]));
  compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 784))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 72))]));
  compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 728))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 72))]));
  compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 840))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 72))]));
  compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 672))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 99))]));
  compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 784))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 99))]));
  compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 728))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 99))]));
  compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 840))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 99))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 728))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 21))]));
  compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 840))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 21))]));
  compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 784))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 21))]));
  compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 896))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 21))]));
  compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 728))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 48))]));
  compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 840))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 48))]));
  compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 784))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 48))]));
  compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 896))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 48))]));
  compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 728))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 75))]));
  compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 840))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 75))]));
  compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 784))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 75))]));
  compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 896))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 75))]));
  compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 728))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 102))]));
  compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 840))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 102))]));
  compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 784))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 102))]));
  compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 896))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 102))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 784))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 24))]));
  compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 896))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 24))]));
  compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 840))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 24))]));
  compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 952))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 24))]));
  compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 784))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 51))]));
  compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 896))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 51))]));
  compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 840))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 51))]));
  compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 952))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 51))]));
  compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 784))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 78))]));
  compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 896))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 78))]));
  compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 840))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 78))]));
  compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 952))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 78))]));
  compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 784))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 105))]));
  compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 896))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 105))]));
  compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 840))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 105))]));
  compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 952))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 105))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 673))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 19))]));
  compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 785))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 19))]));
  compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 729))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 19))]));
  compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 841))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 19))]));
  compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 673))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 46))]));
  compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 785))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 46))]));
  compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 729))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 46))]));
  compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 841))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 46))]));
  compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 673))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 73))]));
  compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 785))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 73))]));
  compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 729))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 73))]));
  compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 841))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 73))]));
  compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 673))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 100))]));
  compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 785))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 100))]));
  compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 729))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 100))]));
  compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 841))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 100))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 729))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 22))]));
  compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 841))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 22))]));
  compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 785))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 22))]));
  compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 897))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 22))]));
  compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 729))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 49))]));
  compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 841))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 49))]));
  compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 785))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 49))]));
  compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 897))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 49))]));
  compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 729))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 76))]));
  compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 841))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 76))]));
  compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 785))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 76))]));
  compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 897))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 76))]));
  compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 729))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 103))]));
  compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 841))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 103))]));
  compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 785))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 103))]));
  compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 897))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 103))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 785))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 25))]));
  compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 897))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 25))]));
  compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 841))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 25))]));
  compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 953))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 25))]));
  compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 785))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 52))]));
  compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 897))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 52))]));
  compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 841))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 52))]));
  compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 953))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 52))]));
  compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 785))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 79))]));
  compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 897))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 79))]));
  compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 841))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 79))]));
  compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 953))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 79))]));
  compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 785))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 106))]));
  compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 897))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 106))]));
  compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 841))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 106))]));
  compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 953))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 106))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 674))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 20))]));
  compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 786))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 20))]));
  compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 730))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 20))]));
  compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 842))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 20))]));
  compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 674))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 47))]));
  compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 786))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 47))]));
  compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 730))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 47))]));
  compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 842))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 47))]));
  compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 674))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 74))]));
  compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 786))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 74))]));
  compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 730))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 74))]));
  compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 842))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 74))]));
  compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 674))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 101))]));
  compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 786))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 101))]));
  compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 730))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 101))]));
  compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 842))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 101))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 730))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 23))]));
  compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 842))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 23))]));
  compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 786))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 23))]));
  compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 898))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 23))]));
  compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 730))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 50))]));
  compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 842))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 50))]));
  compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 786))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 50))]));
  compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 898))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 50))]));
  compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 730))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 77))]));
  compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 842))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 77))]));
  compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 786))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 77))]));
  compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 898))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 77))]));
  compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 730))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 104))]));
  compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 842))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 104))]));
  compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 786))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 104))]));
  compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 898))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 104))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 786))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 26))]));
  compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 898))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 26))]));
  compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 842))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 26))]));
  compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 954))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 26))]));
  compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 786))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 53))]));
  compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 898))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 53))]));
  compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 842))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 53))]));
  compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 954))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 53))]));
  compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 786))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 80))]));
  compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 898))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 80))]));
  compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 842))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 80))]));
  compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 954))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 80))]));
  compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 786))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 107))]));
  compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 898))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 107))]));
  compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 842))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 107))]));
  compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((((int)threadIdx.x) % 54) + 954))] * kernel_shared[((((((int)threadIdx.x) / 54) * 108) + 107))]));
  for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
      compute[(((((((((((int)blockIdx.x) / 54) * 186624) + ((((int)threadIdx.x) / 54) * 46656)) + (ff_inner * 11664)) + (((((int)blockIdx.x) % 54) >> 1) * 432)) + (yy_inner * 108)) + ((((int)blockIdx.x) & 1) * 54)) + (((int)threadIdx.x) % 54)))] = compute_local[(((ff_inner * 2) + yy_inner))];
      compute[((((((((((((int)blockIdx.x) / 54) * 186624) + ((((int)threadIdx.x) / 54) * 46656)) + (ff_inner * 11664)) + (((((int)blockIdx.x) % 54) >> 1) * 432)) + (yy_inner * 108)) + ((((int)blockIdx.x) & 1) * 54)) + (((int)threadIdx.x) % 54)) + 216))] = compute_local[((((ff_inner * 2) + yy_inner) + 8))];
    }
  }
}



void conv_kernel_wrapper(int N_B, int N_C, int N_H, int N_W, int N_F, int N_R, int N_S, int PaddingH, int PaddingW,
                        int StrideH, int StrideW, int N_X, int N_Y, const float *Input, 
                        const float *Kernel, float *Output, int itr) {
                        hipEvent_t start;
                        CHECK(hipEventCreate(&start));
                        hipEvent_t stop;
                        CHECK(hipEventCreate(&stop));
                        float *dev_Input;
                        float *dev_Kernel;
                        float *dev_Output;
                    
                        CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                        CHECK(hipMemcpy(dev_Kernel, Kernel, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                        CHECK(hipMemcpy(dev_Input, Input, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Output, sizeof(float) * N_B * N_F * N_Y * N_X));
                        CHECK(hipMemset(dev_Output, 0, sizeof(float) * N_B * N_F * N_Y * N_X));
                    
dim3 size_grid(108,1,1);

dim3 size_block(216,1,1);
 
                       printf("warmup....\n");
                        default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                    
                        double *time_arr = (double *)malloc(itr * sizeof(double));
                        for (int j = 1; j < itr; j++) {
                            CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                            CHECK(hipMemcpy(dev_Kernel, Kernel+j*N_F * N_C * N_R * N_S, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                            CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                            CHECK(hipMemcpy(dev_Input, Input+j*N_B * N_C * N_H * N_W, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                            /* Record the start event*/
                            CHECK(hipEventRecord(start, NULL));
                            default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                            /* Record the end event */
                            CHECK(hipEventRecord(stop, NULL));
                            /* Wait for the stop event to complete*/
                            CHECK(hipEventSynchronize(stop));
                            float msec = 0.0f;
                            hipEventElapsedTime(&msec, start, stop);
                            time_arr[j] = msec;
                        }
                    
                       
                    
                        //hipError_t err = hipGetLastError();
                        //if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
                        double msecTotal = 0;
                        for (int j = 1; j < itr; j++) {   
                            printf("%0.5lf\n", time_arr[j]);
                            msecTotal += time_arr[j];
                        }
                    
                        double msecPerMatrixMul = (msecTotal) / itr;
                        double flopsPerMatrixMul = 2.0 * N_B * N_C * N_X * N_Y * N_F * N_R * N_S;
                    
                        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
                    
                        printf("\n===========================================================\n");
                        printf("Input size: %d X %d X %d X %d\n", N_B, N_C, N_H, N_W);
                        printf("Kernel size: %d X %d X %d X %d\n", N_F, N_C, N_R, N_S);
                        printf("Output size: %d X %d X %d X %d\n", N_B, N_F, N_Y, N_X);
                        printf("flops: %0.5lf\n", (flopsPerMatrixMul * 1.0e-9f));
                        printf("Time is %0.5lf \n", (msecPerMatrixMul / 1000.0f));
                        printf("Gflops %0.5lf \n", gigaFlops);
                        printf("===========================================================\n");
                    
                        /*write back to Host Output -- batch size is 1*/
                        CHECK(hipMemcpy(Output, dev_Output, sizeof(float) * N_B * N_F * N_X * N_Y, hipMemcpyDeviceToHost));
                    
                        CHECK(hipFree(dev_Input));
                        CHECK(hipFree(dev_Kernel));
                        CHECK(hipFree(dev_Output));
                    
                    }
            