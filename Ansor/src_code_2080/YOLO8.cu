#include "hip/hip_runtime.h"

            #include <cassert>
            #include <stdlib.h>
            #include <hip/hip_runtime.h>
            #include "common.h"
            

            
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[34];
  __shared__ float pad_temp_shared[228];
  __shared__ float kernel_shared[2304];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(17)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(18)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(19)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(20)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(21)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(22)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  compute_local[(23)] = 0.000000e+00f;
  compute_local[(7)] = 0.000000e+00f;
  compute_local[(24)] = 0.000000e+00f;
  compute_local[(8)] = 0.000000e+00f;
  compute_local[(25)] = 0.000000e+00f;
  compute_local[(9)] = 0.000000e+00f;
  compute_local[(26)] = 0.000000e+00f;
  compute_local[(10)] = 0.000000e+00f;
  compute_local[(27)] = 0.000000e+00f;
  compute_local[(11)] = 0.000000e+00f;
  compute_local[(28)] = 0.000000e+00f;
  compute_local[(12)] = 0.000000e+00f;
  compute_local[(29)] = 0.000000e+00f;
  compute_local[(13)] = 0.000000e+00f;
  compute_local[(30)] = 0.000000e+00f;
  compute_local[(14)] = 0.000000e+00f;
  compute_local[(31)] = 0.000000e+00f;
  compute_local[(15)] = 0.000000e+00f;
  compute_local[(32)] = 0.000000e+00f;
  compute_local[(16)] = 0.000000e+00f;
  compute_local[(33)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 128; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[(((int)threadIdx.x))] = data[(((((rc_outer_outer * 1444) + ((((int)threadIdx.x) / 3) * 19)) + (((int)blockIdx.x) % 17)) + (((int)threadIdx.x) % 3)))];
    pad_temp_shared[((((int)threadIdx.x) + 32))] = data[(((((rc_outer_outer * 1444) + (((((int)threadIdx.x) + 32) / 3) * 19)) + (((int)blockIdx.x) % 17)) + ((((int)threadIdx.x) + 2) % 3)))];
    pad_temp_shared[((((int)threadIdx.x) + 64))] = data[(((((rc_outer_outer * 1444) + (((((int)threadIdx.x) + 64) / 3) * 19)) + (((int)blockIdx.x) % 17)) + ((((int)threadIdx.x) + 1) % 3)))];
    pad_temp_shared[((((int)threadIdx.x) + 96))] = data[((((((rc_outer_outer * 1444) + ((((int)threadIdx.x) / 3) * 19)) + (((int)blockIdx.x) % 17)) + (((int)threadIdx.x) % 3)) + 608))];
    pad_temp_shared[((((int)threadIdx.x) + 128))] = data[(((((rc_outer_outer * 1444) + (((((int)threadIdx.x) + 128) / 3) * 19)) + (((int)blockIdx.x) % 17)) + ((((int)threadIdx.x) + 2) % 3)))];
    pad_temp_shared[((((int)threadIdx.x) + 160))] = data[(((((rc_outer_outer * 1444) + (((((int)threadIdx.x) + 160) / 3) * 19)) + (((int)blockIdx.x) % 17)) + ((((int)threadIdx.x) + 1) % 3)))];
    pad_temp_shared[((((int)threadIdx.x) + 192))] = data[((((((rc_outer_outer * 1444) + ((((int)threadIdx.x) / 3) * 19)) + (((int)blockIdx.x) % 17)) + (((int)threadIdx.x) % 3)) + 1216))];
    if (((int)threadIdx.x) < 4) {
      pad_temp_shared[((((int)threadIdx.x) + 224))] = data[(((((rc_outer_outer * 1444) + (((((int)threadIdx.x) + 224) / 3) * 19)) + (((int)blockIdx.x) % 17)) + ((((int)threadIdx.x) + 2) % 3)))];
    }
      ((float4*)(kernel_shared + ((((int)threadIdx.x) * 4))))[0] = ((float4*)(kernel + ((((((((int)blockIdx.x) / 17) * 294912) + ((((int)threadIdx.x) / 9) * 4608)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) % 9) * 4)))))[0];
      ((float4*)(kernel_shared + (((((int)threadIdx.x) * 4) + 128))))[0] = ((float4*)(kernel + ((((((((int)blockIdx.x) / 17) * 294912) + ((((((int)threadIdx.x) * 4) + 128) / 36) * 4608)) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) * 4) + 20) % 36)))))[0];
      ((float4*)(kernel_shared + (((((int)threadIdx.x) * 4) + 256))))[0] = ((float4*)(kernel + ((((((((int)blockIdx.x) / 17) * 294912) + ((((((int)threadIdx.x) * 4) + 256) / 36) * 4608)) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) * 4) + 4) % 36)))))[0];
      ((float4*)(kernel_shared + (((((int)threadIdx.x) * 4) + 384))))[0] = ((float4*)(kernel + ((((((((int)blockIdx.x) / 17) * 294912) + ((((((int)threadIdx.x) * 4) + 384) / 36) * 4608)) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) * 4) + 24) % 36)))))[0];
      ((float4*)(kernel_shared + (((((int)threadIdx.x) * 4) + 512))))[0] = ((float4*)(kernel + ((((((((int)blockIdx.x) / 17) * 294912) + ((((((int)threadIdx.x) * 4) + 512) / 36) * 4608)) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) * 4) + 8) % 36)))))[0];
      ((float4*)(kernel_shared + (((((int)threadIdx.x) * 4) + 640))))[0] = ((float4*)(kernel + ((((((((int)blockIdx.x) / 17) * 294912) + ((((((int)threadIdx.x) * 4) + 640) / 36) * 4608)) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) * 4) + 28) % 36)))))[0];
      ((float4*)(kernel_shared + (((((int)threadIdx.x) * 4) + 768))))[0] = ((float4*)(kernel + ((((((((int)blockIdx.x) / 17) * 294912) + ((((((int)threadIdx.x) * 4) + 768) / 36) * 4608)) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) * 4) + 12) % 36)))))[0];
      ((float4*)(kernel_shared + (((((int)threadIdx.x) * 4) + 896))))[0] = ((float4*)(kernel + ((((((((int)blockIdx.x) / 17) * 294912) + ((((((int)threadIdx.x) * 4) + 896) / 36) * 4608)) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) * 4) + 32) % 36)))))[0];
      ((float4*)(kernel_shared + (((((int)threadIdx.x) * 4) + 1024))))[0] = ((float4*)(kernel + ((((((((int)blockIdx.x) / 17) * 294912) + ((((((int)threadIdx.x) * 4) + 1024) / 36) * 4608)) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) * 4) + 16) % 36)))))[0];
      ((float4*)(kernel_shared + (((((int)threadIdx.x) * 4) + 1152))))[0] = ((float4*)(kernel + (((((((((int)blockIdx.x) / 17) * 294912) + ((((int)threadIdx.x) / 9) * 4608)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) % 9) * 4)) + 147456))))[0];
      ((float4*)(kernel_shared + (((((int)threadIdx.x) * 4) + 1280))))[0] = ((float4*)(kernel + ((((((((int)blockIdx.x) / 17) * 294912) + ((((((int)threadIdx.x) * 4) + 1280) / 36) * 4608)) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) * 4) + 20) % 36)))))[0];
      ((float4*)(kernel_shared + (((((int)threadIdx.x) * 4) + 1408))))[0] = ((float4*)(kernel + ((((((((int)blockIdx.x) / 17) * 294912) + ((((((int)threadIdx.x) * 4) + 1408) / 36) * 4608)) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) * 4) + 4) % 36)))))[0];
      ((float4*)(kernel_shared + (((((int)threadIdx.x) * 4) + 1536))))[0] = ((float4*)(kernel + ((((((((int)blockIdx.x) / 17) * 294912) + ((((((int)threadIdx.x) * 4) + 1536) / 36) * 4608)) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) * 4) + 24) % 36)))))[0];
      ((float4*)(kernel_shared + (((((int)threadIdx.x) * 4) + 1664))))[0] = ((float4*)(kernel + ((((((((int)blockIdx.x) / 17) * 294912) + ((((((int)threadIdx.x) * 4) + 1664) / 36) * 4608)) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) * 4) + 8) % 36)))))[0];
      ((float4*)(kernel_shared + (((((int)threadIdx.x) * 4) + 1792))))[0] = ((float4*)(kernel + ((((((((int)blockIdx.x) / 17) * 294912) + ((((((int)threadIdx.x) * 4) + 1792) / 36) * 4608)) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) * 4) + 28) % 36)))))[0];
      ((float4*)(kernel_shared + (((((int)threadIdx.x) * 4) + 1920))))[0] = ((float4*)(kernel + ((((((((int)blockIdx.x) / 17) * 294912) + ((((((int)threadIdx.x) * 4) + 1920) / 36) * 4608)) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) * 4) + 12) % 36)))))[0];
      ((float4*)(kernel_shared + (((((int)threadIdx.x) * 4) + 2048))))[0] = ((float4*)(kernel + ((((((((int)blockIdx.x) / 17) * 294912) + ((((((int)threadIdx.x) * 4) + 2048) / 36) * 4608)) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) * 4) + 32) % 36)))))[0];
      ((float4*)(kernel_shared + (((((int)threadIdx.x) * 4) + 2176))))[0] = ((float4*)(kernel + ((((((((int)blockIdx.x) / 17) * 294912) + ((((((int)threadIdx.x) * 4) + 2176) / 36) * 4608)) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) * 4) + 16) % 36)))))[0];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((rc_outer_inner * 57))] * kernel_shared[(((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)))]));
      compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[((rc_outer_inner * 57))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 36))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((rc_outer_inner * 57) + 1))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 1))]));
      compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[(((rc_outer_inner * 57) + 1))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 37))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((rc_outer_inner * 57) + 2))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 2))]));
      compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[(((rc_outer_inner * 57) + 2))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 38))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((rc_outer_inner * 57) + 3))] * kernel_shared[(((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)))]));
      compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[(((rc_outer_inner * 57) + 3))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 36))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((rc_outer_inner * 57) + 4))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 1))]));
      compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[(((rc_outer_inner * 57) + 4))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 37))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((rc_outer_inner * 57) + 5))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 2))]));
      compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[(((rc_outer_inner * 57) + 5))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 38))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((rc_outer_inner * 57) + 6))] * kernel_shared[(((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)))]));
      compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[(((rc_outer_inner * 57) + 6))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 36))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((rc_outer_inner * 57) + 7))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 1))]));
      compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[(((rc_outer_inner * 57) + 7))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 37))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((rc_outer_inner * 57) + 8))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 2))]));
      compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[(((rc_outer_inner * 57) + 8))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 38))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((rc_outer_inner * 57) + 9))] * kernel_shared[(((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)))]));
      compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[(((rc_outer_inner * 57) + 9))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 36))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((rc_outer_inner * 57) + 10))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 1))]));
      compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[(((rc_outer_inner * 57) + 10))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 37))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((rc_outer_inner * 57) + 11))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 2))]));
      compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[(((rc_outer_inner * 57) + 11))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 38))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((rc_outer_inner * 57) + 12))] * kernel_shared[(((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)))]));
      compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[(((rc_outer_inner * 57) + 12))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 36))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((rc_outer_inner * 57) + 13))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 1))]));
      compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[(((rc_outer_inner * 57) + 13))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 37))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((rc_outer_inner * 57) + 14))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 2))]));
      compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[(((rc_outer_inner * 57) + 14))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 38))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((rc_outer_inner * 57) + 15))] * kernel_shared[(((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)))]));
      compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[(((rc_outer_inner * 57) + 15))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 36))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((rc_outer_inner * 57) + 16))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 1))]));
      compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[(((rc_outer_inner * 57) + 16))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 37))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((rc_outer_inner * 57) + 17))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 2))]));
      compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[(((rc_outer_inner * 57) + 17))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 38))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((rc_outer_inner * 57) + 18))] * kernel_shared[(((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)))]));
      compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[(((rc_outer_inner * 57) + 18))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 36))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((rc_outer_inner * 57) + 19))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 1))]));
      compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[(((rc_outer_inner * 57) + 19))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 37))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((rc_outer_inner * 57) + 20))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 2))]));
      compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[(((rc_outer_inner * 57) + 20))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 38))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((rc_outer_inner * 57) + 21))] * kernel_shared[(((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)))]));
      compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[(((rc_outer_inner * 57) + 21))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 36))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((rc_outer_inner * 57) + 22))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 1))]));
      compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[(((rc_outer_inner * 57) + 22))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 37))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((rc_outer_inner * 57) + 23))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 2))]));
      compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[(((rc_outer_inner * 57) + 23))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 38))]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((rc_outer_inner * 57) + 24))] * kernel_shared[(((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)))]));
      compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[(((rc_outer_inner * 57) + 24))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 36))]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((rc_outer_inner * 57) + 25))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 1))]));
      compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[(((rc_outer_inner * 57) + 25))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 37))]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((rc_outer_inner * 57) + 26))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 2))]));
      compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[(((rc_outer_inner * 57) + 26))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 38))]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((rc_outer_inner * 57) + 27))] * kernel_shared[(((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)))]));
      compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[(((rc_outer_inner * 57) + 27))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 36))]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((rc_outer_inner * 57) + 28))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 1))]));
      compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[(((rc_outer_inner * 57) + 28))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 37))]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((rc_outer_inner * 57) + 29))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 2))]));
      compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[(((rc_outer_inner * 57) + 29))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 38))]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((rc_outer_inner * 57) + 30))] * kernel_shared[(((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)))]));
      compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[(((rc_outer_inner * 57) + 30))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 36))]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((rc_outer_inner * 57) + 31))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 1))]));
      compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[(((rc_outer_inner * 57) + 31))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 37))]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((rc_outer_inner * 57) + 32))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 2))]));
      compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[(((rc_outer_inner * 57) + 32))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 38))]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((rc_outer_inner * 57) + 33))] * kernel_shared[(((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)))]));
      compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[(((rc_outer_inner * 57) + 33))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 36))]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((rc_outer_inner * 57) + 34))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 1))]));
      compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[(((rc_outer_inner * 57) + 34))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 37))]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((rc_outer_inner * 57) + 35))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 2))]));
      compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[(((rc_outer_inner * 57) + 35))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 38))]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((rc_outer_inner * 57) + 36))] * kernel_shared[(((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)))]));
      compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[(((rc_outer_inner * 57) + 36))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 36))]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((rc_outer_inner * 57) + 37))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 1))]));
      compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[(((rc_outer_inner * 57) + 37))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 37))]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((rc_outer_inner * 57) + 38))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 2))]));
      compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[(((rc_outer_inner * 57) + 38))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 38))]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((rc_outer_inner * 57) + 39))] * kernel_shared[(((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)))]));
      compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[(((rc_outer_inner * 57) + 39))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 36))]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((rc_outer_inner * 57) + 40))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 1))]));
      compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[(((rc_outer_inner * 57) + 40))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 37))]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((rc_outer_inner * 57) + 41))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 2))]));
      compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[(((rc_outer_inner * 57) + 41))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 38))]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((rc_outer_inner * 57) + 42))] * kernel_shared[(((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)))]));
      compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[(((rc_outer_inner * 57) + 42))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 36))]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((rc_outer_inner * 57) + 43))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 1))]));
      compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[(((rc_outer_inner * 57) + 43))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 37))]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((rc_outer_inner * 57) + 44))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 2))]));
      compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[(((rc_outer_inner * 57) + 44))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 38))]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((rc_outer_inner * 57) + 45))] * kernel_shared[(((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)))]));
      compute_local[(32)] = (compute_local[(32)] + (pad_temp_shared[(((rc_outer_inner * 57) + 45))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 36))]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((rc_outer_inner * 57) + 46))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 1))]));
      compute_local[(32)] = (compute_local[(32)] + (pad_temp_shared[(((rc_outer_inner * 57) + 46))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 37))]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((rc_outer_inner * 57) + 47))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 2))]));
      compute_local[(32)] = (compute_local[(32)] + (pad_temp_shared[(((rc_outer_inner * 57) + 47))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 38))]));
      compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(((rc_outer_inner * 57) + 48))] * kernel_shared[(((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)))]));
      compute_local[(33)] = (compute_local[(33)] + (pad_temp_shared[(((rc_outer_inner * 57) + 48))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 36))]));
      compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(((rc_outer_inner * 57) + 49))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 1))]));
      compute_local[(33)] = (compute_local[(33)] + (pad_temp_shared[(((rc_outer_inner * 57) + 49))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 37))]));
      compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(((rc_outer_inner * 57) + 50))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 2))]));
      compute_local[(33)] = (compute_local[(33)] + (pad_temp_shared[(((rc_outer_inner * 57) + 50))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 38))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((rc_outer_inner * 57) + 3))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 3))]));
      compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[(((rc_outer_inner * 57) + 3))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 39))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((rc_outer_inner * 57) + 4))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 4))]));
      compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[(((rc_outer_inner * 57) + 4))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 40))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((rc_outer_inner * 57) + 5))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 5))]));
      compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[(((rc_outer_inner * 57) + 5))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 41))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((rc_outer_inner * 57) + 6))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 3))]));
      compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[(((rc_outer_inner * 57) + 6))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 39))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((rc_outer_inner * 57) + 7))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 4))]));
      compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[(((rc_outer_inner * 57) + 7))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 40))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((rc_outer_inner * 57) + 8))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 5))]));
      compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[(((rc_outer_inner * 57) + 8))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 41))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((rc_outer_inner * 57) + 9))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 3))]));
      compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[(((rc_outer_inner * 57) + 9))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 39))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((rc_outer_inner * 57) + 10))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 4))]));
      compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[(((rc_outer_inner * 57) + 10))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 40))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((rc_outer_inner * 57) + 11))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 5))]));
      compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[(((rc_outer_inner * 57) + 11))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 41))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((rc_outer_inner * 57) + 12))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 3))]));
      compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[(((rc_outer_inner * 57) + 12))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 39))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((rc_outer_inner * 57) + 13))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 4))]));
      compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[(((rc_outer_inner * 57) + 13))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 40))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((rc_outer_inner * 57) + 14))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 5))]));
      compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[(((rc_outer_inner * 57) + 14))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 41))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((rc_outer_inner * 57) + 15))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 3))]));
      compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[(((rc_outer_inner * 57) + 15))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 39))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((rc_outer_inner * 57) + 16))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 4))]));
      compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[(((rc_outer_inner * 57) + 16))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 40))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((rc_outer_inner * 57) + 17))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 5))]));
      compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[(((rc_outer_inner * 57) + 17))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 41))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((rc_outer_inner * 57) + 18))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 3))]));
      compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[(((rc_outer_inner * 57) + 18))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 39))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((rc_outer_inner * 57) + 19))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 4))]));
      compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[(((rc_outer_inner * 57) + 19))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 40))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((rc_outer_inner * 57) + 20))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 5))]));
      compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[(((rc_outer_inner * 57) + 20))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 41))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((rc_outer_inner * 57) + 21))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 3))]));
      compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[(((rc_outer_inner * 57) + 21))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 39))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((rc_outer_inner * 57) + 22))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 4))]));
      compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[(((rc_outer_inner * 57) + 22))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 40))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((rc_outer_inner * 57) + 23))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 5))]));
      compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[(((rc_outer_inner * 57) + 23))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 41))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((rc_outer_inner * 57) + 24))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 3))]));
      compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[(((rc_outer_inner * 57) + 24))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 39))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((rc_outer_inner * 57) + 25))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 4))]));
      compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[(((rc_outer_inner * 57) + 25))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 40))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((rc_outer_inner * 57) + 26))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 5))]));
      compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[(((rc_outer_inner * 57) + 26))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 41))]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((rc_outer_inner * 57) + 27))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 3))]));
      compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[(((rc_outer_inner * 57) + 27))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 39))]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((rc_outer_inner * 57) + 28))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 4))]));
      compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[(((rc_outer_inner * 57) + 28))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 40))]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((rc_outer_inner * 57) + 29))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 5))]));
      compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[(((rc_outer_inner * 57) + 29))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 41))]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((rc_outer_inner * 57) + 30))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 3))]));
      compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[(((rc_outer_inner * 57) + 30))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 39))]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((rc_outer_inner * 57) + 31))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 4))]));
      compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[(((rc_outer_inner * 57) + 31))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 40))]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((rc_outer_inner * 57) + 32))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 5))]));
      compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[(((rc_outer_inner * 57) + 32))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 41))]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((rc_outer_inner * 57) + 33))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 3))]));
      compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[(((rc_outer_inner * 57) + 33))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 39))]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((rc_outer_inner * 57) + 34))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 4))]));
      compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[(((rc_outer_inner * 57) + 34))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 40))]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((rc_outer_inner * 57) + 35))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 5))]));
      compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[(((rc_outer_inner * 57) + 35))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 41))]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((rc_outer_inner * 57) + 36))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 3))]));
      compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[(((rc_outer_inner * 57) + 36))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 39))]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((rc_outer_inner * 57) + 37))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 4))]));
      compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[(((rc_outer_inner * 57) + 37))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 40))]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((rc_outer_inner * 57) + 38))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 5))]));
      compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[(((rc_outer_inner * 57) + 38))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 41))]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((rc_outer_inner * 57) + 39))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 3))]));
      compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[(((rc_outer_inner * 57) + 39))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 39))]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((rc_outer_inner * 57) + 40))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 4))]));
      compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[(((rc_outer_inner * 57) + 40))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 40))]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((rc_outer_inner * 57) + 41))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 5))]));
      compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[(((rc_outer_inner * 57) + 41))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 41))]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((rc_outer_inner * 57) + 42))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 3))]));
      compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[(((rc_outer_inner * 57) + 42))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 39))]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((rc_outer_inner * 57) + 43))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 4))]));
      compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[(((rc_outer_inner * 57) + 43))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 40))]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((rc_outer_inner * 57) + 44))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 5))]));
      compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[(((rc_outer_inner * 57) + 44))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 41))]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((rc_outer_inner * 57) + 45))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 3))]));
      compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[(((rc_outer_inner * 57) + 45))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 39))]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((rc_outer_inner * 57) + 46))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 4))]));
      compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[(((rc_outer_inner * 57) + 46))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 40))]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((rc_outer_inner * 57) + 47))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 5))]));
      compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[(((rc_outer_inner * 57) + 47))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 41))]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((rc_outer_inner * 57) + 48))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 3))]));
      compute_local[(32)] = (compute_local[(32)] + (pad_temp_shared[(((rc_outer_inner * 57) + 48))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 39))]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((rc_outer_inner * 57) + 49))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 4))]));
      compute_local[(32)] = (compute_local[(32)] + (pad_temp_shared[(((rc_outer_inner * 57) + 49))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 40))]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((rc_outer_inner * 57) + 50))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 5))]));
      compute_local[(32)] = (compute_local[(32)] + (pad_temp_shared[(((rc_outer_inner * 57) + 50))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 41))]));
      compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(((rc_outer_inner * 57) + 51))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 3))]));
      compute_local[(33)] = (compute_local[(33)] + (pad_temp_shared[(((rc_outer_inner * 57) + 51))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 39))]));
      compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(((rc_outer_inner * 57) + 52))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 4))]));
      compute_local[(33)] = (compute_local[(33)] + (pad_temp_shared[(((rc_outer_inner * 57) + 52))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 40))]));
      compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(((rc_outer_inner * 57) + 53))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 5))]));
      compute_local[(33)] = (compute_local[(33)] + (pad_temp_shared[(((rc_outer_inner * 57) + 53))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 41))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((rc_outer_inner * 57) + 6))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 6))]));
      compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[(((rc_outer_inner * 57) + 6))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 42))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((rc_outer_inner * 57) + 7))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 7))]));
      compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[(((rc_outer_inner * 57) + 7))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 43))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((rc_outer_inner * 57) + 8))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 8))]));
      compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[(((rc_outer_inner * 57) + 8))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 44))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((rc_outer_inner * 57) + 9))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 6))]));
      compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[(((rc_outer_inner * 57) + 9))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 42))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((rc_outer_inner * 57) + 10))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 7))]));
      compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[(((rc_outer_inner * 57) + 10))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 43))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((rc_outer_inner * 57) + 11))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 8))]));
      compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[(((rc_outer_inner * 57) + 11))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 44))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((rc_outer_inner * 57) + 12))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 6))]));
      compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[(((rc_outer_inner * 57) + 12))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 42))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((rc_outer_inner * 57) + 13))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 7))]));
      compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[(((rc_outer_inner * 57) + 13))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 43))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((rc_outer_inner * 57) + 14))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 8))]));
      compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[(((rc_outer_inner * 57) + 14))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 44))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((rc_outer_inner * 57) + 15))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 6))]));
      compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[(((rc_outer_inner * 57) + 15))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 42))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((rc_outer_inner * 57) + 16))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 7))]));
      compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[(((rc_outer_inner * 57) + 16))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 43))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((rc_outer_inner * 57) + 17))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 8))]));
      compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[(((rc_outer_inner * 57) + 17))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 44))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((rc_outer_inner * 57) + 18))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 6))]));
      compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[(((rc_outer_inner * 57) + 18))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 42))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((rc_outer_inner * 57) + 19))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 7))]));
      compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[(((rc_outer_inner * 57) + 19))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 43))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((rc_outer_inner * 57) + 20))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 8))]));
      compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[(((rc_outer_inner * 57) + 20))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 44))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((rc_outer_inner * 57) + 21))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 6))]));
      compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[(((rc_outer_inner * 57) + 21))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 42))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((rc_outer_inner * 57) + 22))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 7))]));
      compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[(((rc_outer_inner * 57) + 22))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 43))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((rc_outer_inner * 57) + 23))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 8))]));
      compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[(((rc_outer_inner * 57) + 23))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 44))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((rc_outer_inner * 57) + 24))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 6))]));
      compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[(((rc_outer_inner * 57) + 24))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 42))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((rc_outer_inner * 57) + 25))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 7))]));
      compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[(((rc_outer_inner * 57) + 25))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 43))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((rc_outer_inner * 57) + 26))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 8))]));
      compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[(((rc_outer_inner * 57) + 26))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 44))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((rc_outer_inner * 57) + 27))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 6))]));
      compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[(((rc_outer_inner * 57) + 27))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 42))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((rc_outer_inner * 57) + 28))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 7))]));
      compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[(((rc_outer_inner * 57) + 28))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 43))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((rc_outer_inner * 57) + 29))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 8))]));
      compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[(((rc_outer_inner * 57) + 29))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 44))]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((rc_outer_inner * 57) + 30))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 6))]));
      compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[(((rc_outer_inner * 57) + 30))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 42))]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((rc_outer_inner * 57) + 31))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 7))]));
      compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[(((rc_outer_inner * 57) + 31))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 43))]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((rc_outer_inner * 57) + 32))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 8))]));
      compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[(((rc_outer_inner * 57) + 32))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 44))]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((rc_outer_inner * 57) + 33))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 6))]));
      compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[(((rc_outer_inner * 57) + 33))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 42))]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((rc_outer_inner * 57) + 34))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 7))]));
      compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[(((rc_outer_inner * 57) + 34))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 43))]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((rc_outer_inner * 57) + 35))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 8))]));
      compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[(((rc_outer_inner * 57) + 35))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 44))]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((rc_outer_inner * 57) + 36))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 6))]));
      compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[(((rc_outer_inner * 57) + 36))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 42))]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((rc_outer_inner * 57) + 37))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 7))]));
      compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[(((rc_outer_inner * 57) + 37))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 43))]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((rc_outer_inner * 57) + 38))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 8))]));
      compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[(((rc_outer_inner * 57) + 38))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 44))]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((rc_outer_inner * 57) + 39))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 6))]));
      compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[(((rc_outer_inner * 57) + 39))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 42))]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((rc_outer_inner * 57) + 40))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 7))]));
      compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[(((rc_outer_inner * 57) + 40))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 43))]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((rc_outer_inner * 57) + 41))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 8))]));
      compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[(((rc_outer_inner * 57) + 41))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 44))]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((rc_outer_inner * 57) + 42))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 6))]));
      compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[(((rc_outer_inner * 57) + 42))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 42))]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((rc_outer_inner * 57) + 43))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 7))]));
      compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[(((rc_outer_inner * 57) + 43))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 43))]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((rc_outer_inner * 57) + 44))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 8))]));
      compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[(((rc_outer_inner * 57) + 44))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 44))]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((rc_outer_inner * 57) + 45))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 6))]));
      compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[(((rc_outer_inner * 57) + 45))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 42))]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((rc_outer_inner * 57) + 46))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 7))]));
      compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[(((rc_outer_inner * 57) + 46))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 43))]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((rc_outer_inner * 57) + 47))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 8))]));
      compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[(((rc_outer_inner * 57) + 47))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 44))]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((rc_outer_inner * 57) + 48))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 6))]));
      compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[(((rc_outer_inner * 57) + 48))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 42))]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((rc_outer_inner * 57) + 49))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 7))]));
      compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[(((rc_outer_inner * 57) + 49))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 43))]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((rc_outer_inner * 57) + 50))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 8))]));
      compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[(((rc_outer_inner * 57) + 50))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 44))]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((rc_outer_inner * 57) + 51))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 6))]));
      compute_local[(32)] = (compute_local[(32)] + (pad_temp_shared[(((rc_outer_inner * 57) + 51))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 42))]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((rc_outer_inner * 57) + 52))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 7))]));
      compute_local[(32)] = (compute_local[(32)] + (pad_temp_shared[(((rc_outer_inner * 57) + 52))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 43))]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((rc_outer_inner * 57) + 53))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 8))]));
      compute_local[(32)] = (compute_local[(32)] + (pad_temp_shared[(((rc_outer_inner * 57) + 53))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 44))]));
      compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(((rc_outer_inner * 57) + 54))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 6))]));
      compute_local[(33)] = (compute_local[(33)] + (pad_temp_shared[(((rc_outer_inner * 57) + 54))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 42))]));
      compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(((rc_outer_inner * 57) + 55))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 7))]));
      compute_local[(33)] = (compute_local[(33)] + (pad_temp_shared[(((rc_outer_inner * 57) + 55))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 43))]));
      compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(((rc_outer_inner * 57) + 56))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 8))]));
      compute_local[(33)] = (compute_local[(33)] + (pad_temp_shared[(((rc_outer_inner * 57) + 56))] * kernel_shared[((((((int)threadIdx.x) * 72) + (rc_outer_inner * 9)) + 44))]));
    }
  }
  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 17; ++yy_inner) {
      compute[(((((((((int)blockIdx.x) / 17) * 18496) + (((int)threadIdx.x) * 578)) + (ff_inner * 289)) + (yy_inner * 17)) + (((int)blockIdx.x) % 17)))] = compute_local[(((ff_inner * 17) + yy_inner))];
    }
  }
}



void conv_kernel_wrapper(int N_B, int N_C, int N_H, int N_W, int N_F, int N_R, int N_S, int PaddingH, int PaddingW,
                        int StrideH, int StrideW, int N_X, int N_Y, const float *Input, 
                        const float *Kernel, float *Output, int itr) {
                        hipEvent_t start;
                        CHECK(hipEventCreate(&start));
                        hipEvent_t stop;
                        CHECK(hipEventCreate(&stop));
                        float *dev_Input;
                        float *dev_Kernel;
                        float *dev_Output;
                    
                        CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                        CHECK(hipMemcpy(dev_Kernel, Kernel, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                        CHECK(hipMemcpy(dev_Input, Input, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Output, sizeof(float) * N_B * N_F * N_Y * N_X));
                        CHECK(hipMemset(dev_Output, 0, sizeof(float) * N_B * N_F * N_Y * N_X));
                    
dim3 size_grid(272,1,1);

dim3 size_block(32,1,1);
 
                       printf("warmup....\n");
                        default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                    
                        double *time_arr = (double *)malloc(itr * sizeof(double));
                        for (int j = 1; j < itr; j++) {
                            CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                            CHECK(hipMemcpy(dev_Kernel, Kernel+j*N_F * N_C * N_R * N_S, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                            CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                            CHECK(hipMemcpy(dev_Input, Input+j*N_B * N_C * N_H * N_W, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                            /* Record the start event*/
                            CHECK(hipEventRecord(start, NULL));
                            default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                            /* Record the end event */
                            CHECK(hipEventRecord(stop, NULL));
                            /* Wait for the stop event to complete*/
                            CHECK(hipEventSynchronize(stop));
                            float msec = 0.0f;
                            hipEventElapsedTime(&msec, start, stop);
                            time_arr[j] = msec;
                        }
                    
                       
                    
                        //hipError_t err = hipGetLastError();
                        //if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
                        double msecTotal = 0;
                        for (int j = 1; j < itr; j++) {   
                            printf("%0.5lf\n", time_arr[j]);
                            msecTotal += time_arr[j];
                        }
                    
                        double msecPerMatrixMul = (msecTotal) / itr;
                        double flopsPerMatrixMul = 2.0 * N_B * N_C * N_X * N_Y * N_F * N_R * N_S;
                    
                        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
                    
                        printf("\n===========================================================\n");
                        printf("Input size: %d X %d X %d X %d\n", N_B, N_C, N_H, N_W);
                        printf("Kernel size: %d X %d X %d X %d\n", N_F, N_C, N_R, N_S);
                        printf("Output size: %d X %d X %d X %d\n", N_B, N_F, N_Y, N_X);
                        printf("flops: %0.5lf\n", (flopsPerMatrixMul * 1.0e-9f));
                        printf("Time is %0.5lf \n", (msecPerMatrixMul / 1000.0f));
                        printf("Gflops %0.5lf \n", gigaFlops);
                        printf("===========================================================\n");
                    
                        /*write back to Host Output -- batch size is 1*/
                        CHECK(hipMemcpy(Output, dev_Output, sizeof(float) * N_B * N_F * N_X * N_Y, hipMemcpyDeviceToHost));
                    
                        CHECK(hipFree(dev_Input));
                        CHECK(hipFree(dev_Kernel));
                        CHECK(hipFree(dev_Output));
                    
                    }
            