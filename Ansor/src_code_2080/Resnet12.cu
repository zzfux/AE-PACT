#include "hip/hip_runtime.h"

            #include <cassert>
            #include <stdlib.h>
            #include <hip/hip_runtime.h>
            #include "common.h"
            

            
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[8];
  __shared__ float pad_temp_shared[5184];
  __shared__ float kernel_shared[4608];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(7)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[(((int)threadIdx.x))] = data[(((rc_outer_outer * 5184) + ((int)threadIdx.x)))];
    pad_temp_shared[((((int)threadIdx.x) + 49))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 49))];
    pad_temp_shared[((((int)threadIdx.x) + 98))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 98))];
    pad_temp_shared[((((int)threadIdx.x) + 147))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 147))];
    pad_temp_shared[((((int)threadIdx.x) + 196))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 196))];
    pad_temp_shared[((((int)threadIdx.x) + 245))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 245))];
    pad_temp_shared[((((int)threadIdx.x) + 294))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 294))];
    pad_temp_shared[((((int)threadIdx.x) + 343))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 343))];
    pad_temp_shared[((((int)threadIdx.x) + 392))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 392))];
    pad_temp_shared[((((int)threadIdx.x) + 441))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 441))];
    pad_temp_shared[((((int)threadIdx.x) + 490))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 490))];
    pad_temp_shared[((((int)threadIdx.x) + 539))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 539))];
    pad_temp_shared[((((int)threadIdx.x) + 588))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 588))];
    pad_temp_shared[((((int)threadIdx.x) + 637))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 637))];
    pad_temp_shared[((((int)threadIdx.x) + 686))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 686))];
    pad_temp_shared[((((int)threadIdx.x) + 735))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 735))];
    pad_temp_shared[((((int)threadIdx.x) + 784))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 784))];
    pad_temp_shared[((((int)threadIdx.x) + 833))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 833))];
    pad_temp_shared[((((int)threadIdx.x) + 882))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 882))];
    pad_temp_shared[((((int)threadIdx.x) + 931))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 931))];
    pad_temp_shared[((((int)threadIdx.x) + 980))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 980))];
    pad_temp_shared[((((int)threadIdx.x) + 1029))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 1029))];
    pad_temp_shared[((((int)threadIdx.x) + 1078))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 1078))];
    pad_temp_shared[((((int)threadIdx.x) + 1127))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 1127))];
    pad_temp_shared[((((int)threadIdx.x) + 1176))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 1176))];
    pad_temp_shared[((((int)threadIdx.x) + 1225))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 1225))];
    pad_temp_shared[((((int)threadIdx.x) + 1274))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 1274))];
    pad_temp_shared[((((int)threadIdx.x) + 1323))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 1323))];
    pad_temp_shared[((((int)threadIdx.x) + 1372))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 1372))];
    pad_temp_shared[((((int)threadIdx.x) + 1421))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 1421))];
    pad_temp_shared[((((int)threadIdx.x) + 1470))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 1470))];
    pad_temp_shared[((((int)threadIdx.x) + 1519))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 1519))];
    pad_temp_shared[((((int)threadIdx.x) + 1568))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 1568))];
    pad_temp_shared[((((int)threadIdx.x) + 1617))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 1617))];
    pad_temp_shared[((((int)threadIdx.x) + 1666))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 1666))];
    pad_temp_shared[((((int)threadIdx.x) + 1715))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 1715))];
    pad_temp_shared[((((int)threadIdx.x) + 1764))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 1764))];
    pad_temp_shared[((((int)threadIdx.x) + 1813))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 1813))];
    pad_temp_shared[((((int)threadIdx.x) + 1862))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 1862))];
    pad_temp_shared[((((int)threadIdx.x) + 1911))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 1911))];
    pad_temp_shared[((((int)threadIdx.x) + 1960))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 1960))];
    pad_temp_shared[((((int)threadIdx.x) + 2009))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 2009))];
    pad_temp_shared[((((int)threadIdx.x) + 2058))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 2058))];
    pad_temp_shared[((((int)threadIdx.x) + 2107))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 2107))];
    pad_temp_shared[((((int)threadIdx.x) + 2156))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 2156))];
    pad_temp_shared[((((int)threadIdx.x) + 2205))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 2205))];
    pad_temp_shared[((((int)threadIdx.x) + 2254))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 2254))];
    pad_temp_shared[((((int)threadIdx.x) + 2303))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 2303))];
    pad_temp_shared[((((int)threadIdx.x) + 2352))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 2352))];
    pad_temp_shared[((((int)threadIdx.x) + 2401))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 2401))];
    pad_temp_shared[((((int)threadIdx.x) + 2450))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 2450))];
    pad_temp_shared[((((int)threadIdx.x) + 2499))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 2499))];
    pad_temp_shared[((((int)threadIdx.x) + 2548))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 2548))];
    pad_temp_shared[((((int)threadIdx.x) + 2597))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 2597))];
    pad_temp_shared[((((int)threadIdx.x) + 2646))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 2646))];
    pad_temp_shared[((((int)threadIdx.x) + 2695))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 2695))];
    pad_temp_shared[((((int)threadIdx.x) + 2744))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 2744))];
    pad_temp_shared[((((int)threadIdx.x) + 2793))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 2793))];
    pad_temp_shared[((((int)threadIdx.x) + 2842))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 2842))];
    pad_temp_shared[((((int)threadIdx.x) + 2891))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 2891))];
    pad_temp_shared[((((int)threadIdx.x) + 2940))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 2940))];
    pad_temp_shared[((((int)threadIdx.x) + 2989))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 2989))];
    pad_temp_shared[((((int)threadIdx.x) + 3038))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 3038))];
    pad_temp_shared[((((int)threadIdx.x) + 3087))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 3087))];
    pad_temp_shared[((((int)threadIdx.x) + 3136))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 3136))];
    pad_temp_shared[((((int)threadIdx.x) + 3185))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 3185))];
    pad_temp_shared[((((int)threadIdx.x) + 3234))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 3234))];
    pad_temp_shared[((((int)threadIdx.x) + 3283))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 3283))];
    pad_temp_shared[((((int)threadIdx.x) + 3332))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 3332))];
    pad_temp_shared[((((int)threadIdx.x) + 3381))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 3381))];
    pad_temp_shared[((((int)threadIdx.x) + 3430))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 3430))];
    pad_temp_shared[((((int)threadIdx.x) + 3479))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 3479))];
    pad_temp_shared[((((int)threadIdx.x) + 3528))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 3528))];
    pad_temp_shared[((((int)threadIdx.x) + 3577))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 3577))];
    pad_temp_shared[((((int)threadIdx.x) + 3626))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 3626))];
    pad_temp_shared[((((int)threadIdx.x) + 3675))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 3675))];
    pad_temp_shared[((((int)threadIdx.x) + 3724))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 3724))];
    pad_temp_shared[((((int)threadIdx.x) + 3773))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 3773))];
    pad_temp_shared[((((int)threadIdx.x) + 3822))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 3822))];
    pad_temp_shared[((((int)threadIdx.x) + 3871))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 3871))];
    pad_temp_shared[((((int)threadIdx.x) + 3920))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 3920))];
    pad_temp_shared[((((int)threadIdx.x) + 3969))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 3969))];
    pad_temp_shared[((((int)threadIdx.x) + 4018))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 4018))];
    pad_temp_shared[((((int)threadIdx.x) + 4067))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 4067))];
    pad_temp_shared[((((int)threadIdx.x) + 4116))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 4116))];
    pad_temp_shared[((((int)threadIdx.x) + 4165))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 4165))];
    pad_temp_shared[((((int)threadIdx.x) + 4214))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 4214))];
    pad_temp_shared[((((int)threadIdx.x) + 4263))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 4263))];
    pad_temp_shared[((((int)threadIdx.x) + 4312))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 4312))];
    pad_temp_shared[((((int)threadIdx.x) + 4361))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 4361))];
    pad_temp_shared[((((int)threadIdx.x) + 4410))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 4410))];
    pad_temp_shared[((((int)threadIdx.x) + 4459))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 4459))];
    pad_temp_shared[((((int)threadIdx.x) + 4508))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 4508))];
    pad_temp_shared[((((int)threadIdx.x) + 4557))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 4557))];
    pad_temp_shared[((((int)threadIdx.x) + 4606))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 4606))];
    pad_temp_shared[((((int)threadIdx.x) + 4655))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 4655))];
    pad_temp_shared[((((int)threadIdx.x) + 4704))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 4704))];
    pad_temp_shared[((((int)threadIdx.x) + 4753))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 4753))];
    pad_temp_shared[((((int)threadIdx.x) + 4802))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 4802))];
    pad_temp_shared[((((int)threadIdx.x) + 4851))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 4851))];
    pad_temp_shared[((((int)threadIdx.x) + 4900))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 4900))];
    pad_temp_shared[((((int)threadIdx.x) + 4949))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 4949))];
    pad_temp_shared[((((int)threadIdx.x) + 4998))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 4998))];
    pad_temp_shared[((((int)threadIdx.x) + 5047))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 5047))];
    pad_temp_shared[((((int)threadIdx.x) + 5096))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 5096))];
    if (((int)threadIdx.x) < 39) {
      pad_temp_shared[((((int)threadIdx.x) + 5145))] = data[((((rc_outer_outer * 5184) + ((int)threadIdx.x)) + 5145))];
    }
    kernel_shared[((((int)threadIdx.x) * 6))] = kernel[((((((int)blockIdx.x) * 36864) + (rc_outer_outer * 576)) + (((int)threadIdx.x) * 6)))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 1))] = kernel[(((((((int)blockIdx.x) * 36864) + (rc_outer_outer * 576)) + (((int)threadIdx.x) * 6)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 2))] = kernel[(((((((int)blockIdx.x) * 36864) + (rc_outer_outer * 576)) + (((int)threadIdx.x) * 6)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 3))] = kernel[(((((((int)blockIdx.x) * 36864) + (rc_outer_outer * 576)) + (((int)threadIdx.x) * 6)) + 3))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 4))] = kernel[(((((((int)blockIdx.x) * 36864) + (rc_outer_outer * 576)) + (((int)threadIdx.x) * 6)) + 4))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 5))] = kernel[(((((((int)blockIdx.x) * 36864) + (rc_outer_outer * 576)) + (((int)threadIdx.x) * 6)) + 5))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 294))] = kernel[(((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 98) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 98) % 192) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 295))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 98) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 98) % 192) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 296))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 98) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 98) % 192) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 297))] = kernel[(((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 99) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 99) % 192) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 298))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 99) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 99) % 192) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 299))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 99) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 99) % 192) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 588))] = kernel[(((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 196) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 4) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 589))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 196) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 4) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 590))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 196) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 4) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 591))] = kernel[(((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 197) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 5) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 592))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 197) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 5) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 593))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 197) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 5) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 882))] = kernel[(((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 294) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 102) % 192) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 883))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 294) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 102) % 192) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 884))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 294) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 102) % 192) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 885))] = kernel[(((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 295) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 103) % 192) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 886))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 295) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 103) % 192) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 887))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 295) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 103) % 192) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 1176))] = kernel[(((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 392) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 8) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 1177))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 392) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 8) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 1178))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 392) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 8) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 1179))] = kernel[(((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 393) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 9) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 1180))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 393) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 9) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 1181))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 393) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 9) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 1470))] = kernel[(((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 490) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 106) % 192) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 1471))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 490) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 106) % 192) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 1472))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 490) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 106) % 192) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 1473))] = kernel[(((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 491) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 107) % 192) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 1474))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 491) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 107) % 192) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 1475))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 491) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 107) % 192) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 1764))] = kernel[(((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 588) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 12) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 1765))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 588) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 12) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 1766))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 588) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 12) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 1767))] = kernel[(((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 589) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 13) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 1768))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 589) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 13) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 1769))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 589) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 13) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 2058))] = kernel[(((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 686) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 110) % 192) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 2059))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 686) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 110) % 192) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 2060))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 686) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 110) % 192) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 2061))] = kernel[(((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 687) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 111) % 192) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 2062))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 687) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 111) % 192) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 2063))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 687) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 111) % 192) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 2352))] = kernel[(((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 784) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 16) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 2353))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 784) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 16) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 2354))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 784) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 16) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 2355))] = kernel[(((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 785) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 17) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 2356))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 785) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 17) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 2357))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 785) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 17) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 2646))] = kernel[(((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 882) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 114) % 192) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 2647))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 882) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 114) % 192) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 2648))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 882) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 114) % 192) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 2649))] = kernel[(((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 883) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 115) % 192) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 2650))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 883) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 115) % 192) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 2651))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 883) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 115) % 192) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 2940))] = kernel[(((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 980) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 20) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 2941))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 980) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 20) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 2942))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 980) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 20) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 2943))] = kernel[(((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 981) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 21) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 2944))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 981) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 21) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 2945))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 981) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 21) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 3234))] = kernel[(((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 1078) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 118) % 192) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 3235))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 1078) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 118) % 192) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 3236))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 1078) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 118) % 192) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 3237))] = kernel[(((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 1079) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 119) % 192) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 3238))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 1079) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 119) % 192) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 3239))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 1079) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 119) % 192) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 3528))] = kernel[(((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 1176) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 24) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 3529))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 1176) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 24) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 3530))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 1176) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 24) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 3531))] = kernel[(((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 1177) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 25) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 3532))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 1177) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 25) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 3533))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 1177) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 25) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 3822))] = kernel[(((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 1274) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 122) % 192) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 3823))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 1274) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 122) % 192) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 3824))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 1274) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 122) % 192) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 3825))] = kernel[(((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 1275) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 123) % 192) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 3826))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 1275) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 123) % 192) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 3827))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 1275) / 192) * 4608)) + (rc_outer_outer * 576)) + ((((((int)threadIdx.x) * 2) + 123) % 192) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 4116))] = kernel[(((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 1372) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 28) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 4117))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 1372) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 28) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 4118))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 1372) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 28) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 4119))] = kernel[(((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 1373) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 29) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 4120))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 1373) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 29) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 6) + 4121))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 1373) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 29) * 3)) + 2))];
    if (((int)threadIdx.x) < 33) {
      kernel_shared[(((((int)threadIdx.x) * 6) + 4410))] = kernel[(((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 1470) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 126) * 3)))];
    }
    if (((int)threadIdx.x) < 33) {
      kernel_shared[(((((int)threadIdx.x) * 6) + 4411))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 1470) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 126) * 3)) + 1))];
    }
    if (((int)threadIdx.x) < 33) {
      kernel_shared[(((((int)threadIdx.x) * 6) + 4412))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 1470) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 126) * 3)) + 2))];
    }
    if (((int)threadIdx.x) < 33) {
      kernel_shared[(((((int)threadIdx.x) * 6) + 4413))] = kernel[(((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 1471) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 127) * 3)))];
    }
    if (((int)threadIdx.x) < 33) {
      kernel_shared[(((((int)threadIdx.x) * 6) + 4414))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 1471) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 127) * 3)) + 1))];
    }
    if (((int)threadIdx.x) < 33) {
      kernel_shared[(((((int)threadIdx.x) * 6) + 4415))] = kernel[((((((((int)blockIdx.x) * 36864) + ((((((int)threadIdx.x) * 2) + 1471) / 192) * 4608)) + (rc_outer_outer * 576)) + (((((int)threadIdx.x) * 2) + 127) * 3)) + 2))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 16; ++rc_outer_inner) {
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)))] * kernel_shared[((rc_outer_inner * 36))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)))] * kernel_shared[(((rc_outer_inner * 36) + 2304))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)))] * kernel_shared[(((rc_outer_inner * 36) + 576))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)))] * kernel_shared[(((rc_outer_inner * 36) + 2880))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)))] * kernel_shared[(((rc_outer_inner * 36) + 1152))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)))] * kernel_shared[(((rc_outer_inner * 36) + 3456))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)))] * kernel_shared[(((rc_outer_inner * 36) + 1728))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)))] * kernel_shared[(((rc_outer_inner * 36) + 4032))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 1))] * kernel_shared[(((rc_outer_inner * 36) + 1))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 1))] * kernel_shared[(((rc_outer_inner * 36) + 2305))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 1))] * kernel_shared[(((rc_outer_inner * 36) + 577))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 1))] * kernel_shared[(((rc_outer_inner * 36) + 2881))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 1))] * kernel_shared[(((rc_outer_inner * 36) + 1153))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 1))] * kernel_shared[(((rc_outer_inner * 36) + 3457))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 1))] * kernel_shared[(((rc_outer_inner * 36) + 1729))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 1))] * kernel_shared[(((rc_outer_inner * 36) + 4033))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 2))] * kernel_shared[(((rc_outer_inner * 36) + 2))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 2))] * kernel_shared[(((rc_outer_inner * 36) + 2306))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 2))] * kernel_shared[(((rc_outer_inner * 36) + 578))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 2))] * kernel_shared[(((rc_outer_inner * 36) + 2882))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 2))] * kernel_shared[(((rc_outer_inner * 36) + 1154))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 2))] * kernel_shared[(((rc_outer_inner * 36) + 3458))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 2))] * kernel_shared[(((rc_outer_inner * 36) + 1730))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 2))] * kernel_shared[(((rc_outer_inner * 36) + 4034))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 81))] * kernel_shared[(((rc_outer_inner * 36) + 9))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 81))] * kernel_shared[(((rc_outer_inner * 36) + 2313))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 81))] * kernel_shared[(((rc_outer_inner * 36) + 585))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 81))] * kernel_shared[(((rc_outer_inner * 36) + 2889))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 81))] * kernel_shared[(((rc_outer_inner * 36) + 1161))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 81))] * kernel_shared[(((rc_outer_inner * 36) + 3465))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 81))] * kernel_shared[(((rc_outer_inner * 36) + 1737))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 81))] * kernel_shared[(((rc_outer_inner * 36) + 4041))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 82))] * kernel_shared[(((rc_outer_inner * 36) + 10))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 82))] * kernel_shared[(((rc_outer_inner * 36) + 2314))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 82))] * kernel_shared[(((rc_outer_inner * 36) + 586))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 82))] * kernel_shared[(((rc_outer_inner * 36) + 2890))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 82))] * kernel_shared[(((rc_outer_inner * 36) + 1162))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 82))] * kernel_shared[(((rc_outer_inner * 36) + 3466))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 82))] * kernel_shared[(((rc_outer_inner * 36) + 1738))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 82))] * kernel_shared[(((rc_outer_inner * 36) + 4042))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 83))] * kernel_shared[(((rc_outer_inner * 36) + 11))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 83))] * kernel_shared[(((rc_outer_inner * 36) + 2315))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 83))] * kernel_shared[(((rc_outer_inner * 36) + 587))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 83))] * kernel_shared[(((rc_outer_inner * 36) + 2891))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 83))] * kernel_shared[(((rc_outer_inner * 36) + 1163))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 83))] * kernel_shared[(((rc_outer_inner * 36) + 3467))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 83))] * kernel_shared[(((rc_outer_inner * 36) + 1739))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 83))] * kernel_shared[(((rc_outer_inner * 36) + 4043))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 162))] * kernel_shared[(((rc_outer_inner * 36) + 18))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 162))] * kernel_shared[(((rc_outer_inner * 36) + 2322))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 162))] * kernel_shared[(((rc_outer_inner * 36) + 594))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 162))] * kernel_shared[(((rc_outer_inner * 36) + 2898))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 162))] * kernel_shared[(((rc_outer_inner * 36) + 1170))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 162))] * kernel_shared[(((rc_outer_inner * 36) + 3474))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 162))] * kernel_shared[(((rc_outer_inner * 36) + 1746))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 162))] * kernel_shared[(((rc_outer_inner * 36) + 4050))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 163))] * kernel_shared[(((rc_outer_inner * 36) + 19))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 163))] * kernel_shared[(((rc_outer_inner * 36) + 2323))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 163))] * kernel_shared[(((rc_outer_inner * 36) + 595))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 163))] * kernel_shared[(((rc_outer_inner * 36) + 2899))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 163))] * kernel_shared[(((rc_outer_inner * 36) + 1171))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 163))] * kernel_shared[(((rc_outer_inner * 36) + 3475))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 163))] * kernel_shared[(((rc_outer_inner * 36) + 1747))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 163))] * kernel_shared[(((rc_outer_inner * 36) + 4051))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 164))] * kernel_shared[(((rc_outer_inner * 36) + 20))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 164))] * kernel_shared[(((rc_outer_inner * 36) + 2324))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 164))] * kernel_shared[(((rc_outer_inner * 36) + 596))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 164))] * kernel_shared[(((rc_outer_inner * 36) + 2900))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 164))] * kernel_shared[(((rc_outer_inner * 36) + 1172))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 164))] * kernel_shared[(((rc_outer_inner * 36) + 3476))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 164))] * kernel_shared[(((rc_outer_inner * 36) + 1748))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 164))] * kernel_shared[(((rc_outer_inner * 36) + 4052))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 243))] * kernel_shared[(((rc_outer_inner * 36) + 27))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 243))] * kernel_shared[(((rc_outer_inner * 36) + 2331))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 243))] * kernel_shared[(((rc_outer_inner * 36) + 603))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 243))] * kernel_shared[(((rc_outer_inner * 36) + 2907))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 243))] * kernel_shared[(((rc_outer_inner * 36) + 1179))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 243))] * kernel_shared[(((rc_outer_inner * 36) + 3483))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 243))] * kernel_shared[(((rc_outer_inner * 36) + 1755))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 243))] * kernel_shared[(((rc_outer_inner * 36) + 4059))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 244))] * kernel_shared[(((rc_outer_inner * 36) + 28))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 244))] * kernel_shared[(((rc_outer_inner * 36) + 2332))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 244))] * kernel_shared[(((rc_outer_inner * 36) + 604))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 244))] * kernel_shared[(((rc_outer_inner * 36) + 2908))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 244))] * kernel_shared[(((rc_outer_inner * 36) + 1180))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 244))] * kernel_shared[(((rc_outer_inner * 36) + 3484))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 244))] * kernel_shared[(((rc_outer_inner * 36) + 1756))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 244))] * kernel_shared[(((rc_outer_inner * 36) + 4060))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 245))] * kernel_shared[(((rc_outer_inner * 36) + 29))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 245))] * kernel_shared[(((rc_outer_inner * 36) + 2333))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 245))] * kernel_shared[(((rc_outer_inner * 36) + 605))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 245))] * kernel_shared[(((rc_outer_inner * 36) + 2909))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 245))] * kernel_shared[(((rc_outer_inner * 36) + 1181))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 245))] * kernel_shared[(((rc_outer_inner * 36) + 3485))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 245))] * kernel_shared[(((rc_outer_inner * 36) + 1757))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 245))] * kernel_shared[(((rc_outer_inner * 36) + 4061))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 9))] * kernel_shared[(((rc_outer_inner * 36) + 3))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 9))] * kernel_shared[(((rc_outer_inner * 36) + 2307))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 9))] * kernel_shared[(((rc_outer_inner * 36) + 579))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 9))] * kernel_shared[(((rc_outer_inner * 36) + 2883))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 9))] * kernel_shared[(((rc_outer_inner * 36) + 1155))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 9))] * kernel_shared[(((rc_outer_inner * 36) + 3459))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 9))] * kernel_shared[(((rc_outer_inner * 36) + 1731))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 9))] * kernel_shared[(((rc_outer_inner * 36) + 4035))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 10))] * kernel_shared[(((rc_outer_inner * 36) + 4))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 10))] * kernel_shared[(((rc_outer_inner * 36) + 2308))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 10))] * kernel_shared[(((rc_outer_inner * 36) + 580))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 10))] * kernel_shared[(((rc_outer_inner * 36) + 2884))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 10))] * kernel_shared[(((rc_outer_inner * 36) + 1156))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 10))] * kernel_shared[(((rc_outer_inner * 36) + 3460))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 10))] * kernel_shared[(((rc_outer_inner * 36) + 1732))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 10))] * kernel_shared[(((rc_outer_inner * 36) + 4036))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 11))] * kernel_shared[(((rc_outer_inner * 36) + 5))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 11))] * kernel_shared[(((rc_outer_inner * 36) + 2309))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 11))] * kernel_shared[(((rc_outer_inner * 36) + 581))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 11))] * kernel_shared[(((rc_outer_inner * 36) + 2885))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 11))] * kernel_shared[(((rc_outer_inner * 36) + 1157))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 11))] * kernel_shared[(((rc_outer_inner * 36) + 3461))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 11))] * kernel_shared[(((rc_outer_inner * 36) + 1733))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 11))] * kernel_shared[(((rc_outer_inner * 36) + 4037))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 90))] * kernel_shared[(((rc_outer_inner * 36) + 12))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 90))] * kernel_shared[(((rc_outer_inner * 36) + 2316))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 90))] * kernel_shared[(((rc_outer_inner * 36) + 588))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 90))] * kernel_shared[(((rc_outer_inner * 36) + 2892))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 90))] * kernel_shared[(((rc_outer_inner * 36) + 1164))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 90))] * kernel_shared[(((rc_outer_inner * 36) + 3468))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 90))] * kernel_shared[(((rc_outer_inner * 36) + 1740))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 90))] * kernel_shared[(((rc_outer_inner * 36) + 4044))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 91))] * kernel_shared[(((rc_outer_inner * 36) + 13))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 91))] * kernel_shared[(((rc_outer_inner * 36) + 2317))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 91))] * kernel_shared[(((rc_outer_inner * 36) + 589))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 91))] * kernel_shared[(((rc_outer_inner * 36) + 2893))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 91))] * kernel_shared[(((rc_outer_inner * 36) + 1165))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 91))] * kernel_shared[(((rc_outer_inner * 36) + 3469))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 91))] * kernel_shared[(((rc_outer_inner * 36) + 1741))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 91))] * kernel_shared[(((rc_outer_inner * 36) + 4045))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 92))] * kernel_shared[(((rc_outer_inner * 36) + 14))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 92))] * kernel_shared[(((rc_outer_inner * 36) + 2318))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 92))] * kernel_shared[(((rc_outer_inner * 36) + 590))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 92))] * kernel_shared[(((rc_outer_inner * 36) + 2894))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 92))] * kernel_shared[(((rc_outer_inner * 36) + 1166))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 92))] * kernel_shared[(((rc_outer_inner * 36) + 3470))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 92))] * kernel_shared[(((rc_outer_inner * 36) + 1742))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 92))] * kernel_shared[(((rc_outer_inner * 36) + 4046))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 171))] * kernel_shared[(((rc_outer_inner * 36) + 21))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 171))] * kernel_shared[(((rc_outer_inner * 36) + 2325))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 171))] * kernel_shared[(((rc_outer_inner * 36) + 597))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 171))] * kernel_shared[(((rc_outer_inner * 36) + 2901))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 171))] * kernel_shared[(((rc_outer_inner * 36) + 1173))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 171))] * kernel_shared[(((rc_outer_inner * 36) + 3477))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 171))] * kernel_shared[(((rc_outer_inner * 36) + 1749))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 171))] * kernel_shared[(((rc_outer_inner * 36) + 4053))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 172))] * kernel_shared[(((rc_outer_inner * 36) + 22))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 172))] * kernel_shared[(((rc_outer_inner * 36) + 2326))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 172))] * kernel_shared[(((rc_outer_inner * 36) + 598))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 172))] * kernel_shared[(((rc_outer_inner * 36) + 2902))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 172))] * kernel_shared[(((rc_outer_inner * 36) + 1174))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 172))] * kernel_shared[(((rc_outer_inner * 36) + 3478))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 172))] * kernel_shared[(((rc_outer_inner * 36) + 1750))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 172))] * kernel_shared[(((rc_outer_inner * 36) + 4054))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 173))] * kernel_shared[(((rc_outer_inner * 36) + 23))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 173))] * kernel_shared[(((rc_outer_inner * 36) + 2327))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 173))] * kernel_shared[(((rc_outer_inner * 36) + 599))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 173))] * kernel_shared[(((rc_outer_inner * 36) + 2903))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 173))] * kernel_shared[(((rc_outer_inner * 36) + 1175))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 173))] * kernel_shared[(((rc_outer_inner * 36) + 3479))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 173))] * kernel_shared[(((rc_outer_inner * 36) + 1751))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 173))] * kernel_shared[(((rc_outer_inner * 36) + 4055))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 252))] * kernel_shared[(((rc_outer_inner * 36) + 30))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 252))] * kernel_shared[(((rc_outer_inner * 36) + 2334))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 252))] * kernel_shared[(((rc_outer_inner * 36) + 606))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 252))] * kernel_shared[(((rc_outer_inner * 36) + 2910))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 252))] * kernel_shared[(((rc_outer_inner * 36) + 1182))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 252))] * kernel_shared[(((rc_outer_inner * 36) + 3486))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 252))] * kernel_shared[(((rc_outer_inner * 36) + 1758))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 252))] * kernel_shared[(((rc_outer_inner * 36) + 4062))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 253))] * kernel_shared[(((rc_outer_inner * 36) + 31))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 253))] * kernel_shared[(((rc_outer_inner * 36) + 2335))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 253))] * kernel_shared[(((rc_outer_inner * 36) + 607))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 253))] * kernel_shared[(((rc_outer_inner * 36) + 2911))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 253))] * kernel_shared[(((rc_outer_inner * 36) + 1183))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 253))] * kernel_shared[(((rc_outer_inner * 36) + 3487))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 253))] * kernel_shared[(((rc_outer_inner * 36) + 1759))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 253))] * kernel_shared[(((rc_outer_inner * 36) + 4063))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 254))] * kernel_shared[(((rc_outer_inner * 36) + 32))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 254))] * kernel_shared[(((rc_outer_inner * 36) + 2336))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 254))] * kernel_shared[(((rc_outer_inner * 36) + 608))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 254))] * kernel_shared[(((rc_outer_inner * 36) + 2912))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 254))] * kernel_shared[(((rc_outer_inner * 36) + 1184))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 254))] * kernel_shared[(((rc_outer_inner * 36) + 3488))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 254))] * kernel_shared[(((rc_outer_inner * 36) + 1760))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 254))] * kernel_shared[(((rc_outer_inner * 36) + 4064))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 18))] * kernel_shared[(((rc_outer_inner * 36) + 6))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 18))] * kernel_shared[(((rc_outer_inner * 36) + 2310))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 18))] * kernel_shared[(((rc_outer_inner * 36) + 582))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 18))] * kernel_shared[(((rc_outer_inner * 36) + 2886))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 18))] * kernel_shared[(((rc_outer_inner * 36) + 1158))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 18))] * kernel_shared[(((rc_outer_inner * 36) + 3462))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 18))] * kernel_shared[(((rc_outer_inner * 36) + 1734))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 18))] * kernel_shared[(((rc_outer_inner * 36) + 4038))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 19))] * kernel_shared[(((rc_outer_inner * 36) + 7))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 19))] * kernel_shared[(((rc_outer_inner * 36) + 2311))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 19))] * kernel_shared[(((rc_outer_inner * 36) + 583))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 19))] * kernel_shared[(((rc_outer_inner * 36) + 2887))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 19))] * kernel_shared[(((rc_outer_inner * 36) + 1159))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 19))] * kernel_shared[(((rc_outer_inner * 36) + 3463))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 19))] * kernel_shared[(((rc_outer_inner * 36) + 1735))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 19))] * kernel_shared[(((rc_outer_inner * 36) + 4039))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 20))] * kernel_shared[(((rc_outer_inner * 36) + 8))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 20))] * kernel_shared[(((rc_outer_inner * 36) + 2312))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 20))] * kernel_shared[(((rc_outer_inner * 36) + 584))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 20))] * kernel_shared[(((rc_outer_inner * 36) + 2888))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 20))] * kernel_shared[(((rc_outer_inner * 36) + 1160))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 20))] * kernel_shared[(((rc_outer_inner * 36) + 3464))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 20))] * kernel_shared[(((rc_outer_inner * 36) + 1736))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 20))] * kernel_shared[(((rc_outer_inner * 36) + 4040))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 99))] * kernel_shared[(((rc_outer_inner * 36) + 15))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 99))] * kernel_shared[(((rc_outer_inner * 36) + 2319))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 99))] * kernel_shared[(((rc_outer_inner * 36) + 591))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 99))] * kernel_shared[(((rc_outer_inner * 36) + 2895))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 99))] * kernel_shared[(((rc_outer_inner * 36) + 1167))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 99))] * kernel_shared[(((rc_outer_inner * 36) + 3471))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 99))] * kernel_shared[(((rc_outer_inner * 36) + 1743))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 99))] * kernel_shared[(((rc_outer_inner * 36) + 4047))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 100))] * kernel_shared[(((rc_outer_inner * 36) + 16))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 100))] * kernel_shared[(((rc_outer_inner * 36) + 2320))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 100))] * kernel_shared[(((rc_outer_inner * 36) + 592))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 100))] * kernel_shared[(((rc_outer_inner * 36) + 2896))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 100))] * kernel_shared[(((rc_outer_inner * 36) + 1168))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 100))] * kernel_shared[(((rc_outer_inner * 36) + 3472))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 100))] * kernel_shared[(((rc_outer_inner * 36) + 1744))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 100))] * kernel_shared[(((rc_outer_inner * 36) + 4048))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 101))] * kernel_shared[(((rc_outer_inner * 36) + 17))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 101))] * kernel_shared[(((rc_outer_inner * 36) + 2321))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 101))] * kernel_shared[(((rc_outer_inner * 36) + 593))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 101))] * kernel_shared[(((rc_outer_inner * 36) + 2897))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 101))] * kernel_shared[(((rc_outer_inner * 36) + 1169))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 101))] * kernel_shared[(((rc_outer_inner * 36) + 3473))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 101))] * kernel_shared[(((rc_outer_inner * 36) + 1745))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 101))] * kernel_shared[(((rc_outer_inner * 36) + 4049))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 180))] * kernel_shared[(((rc_outer_inner * 36) + 24))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 180))] * kernel_shared[(((rc_outer_inner * 36) + 2328))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 180))] * kernel_shared[(((rc_outer_inner * 36) + 600))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 180))] * kernel_shared[(((rc_outer_inner * 36) + 2904))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 180))] * kernel_shared[(((rc_outer_inner * 36) + 1176))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 180))] * kernel_shared[(((rc_outer_inner * 36) + 3480))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 180))] * kernel_shared[(((rc_outer_inner * 36) + 1752))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 180))] * kernel_shared[(((rc_outer_inner * 36) + 4056))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 181))] * kernel_shared[(((rc_outer_inner * 36) + 25))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 181))] * kernel_shared[(((rc_outer_inner * 36) + 2329))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 181))] * kernel_shared[(((rc_outer_inner * 36) + 601))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 181))] * kernel_shared[(((rc_outer_inner * 36) + 2905))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 181))] * kernel_shared[(((rc_outer_inner * 36) + 1177))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 181))] * kernel_shared[(((rc_outer_inner * 36) + 3481))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 181))] * kernel_shared[(((rc_outer_inner * 36) + 1753))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 181))] * kernel_shared[(((rc_outer_inner * 36) + 4057))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 182))] * kernel_shared[(((rc_outer_inner * 36) + 26))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 182))] * kernel_shared[(((rc_outer_inner * 36) + 2330))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 182))] * kernel_shared[(((rc_outer_inner * 36) + 602))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 182))] * kernel_shared[(((rc_outer_inner * 36) + 2906))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 182))] * kernel_shared[(((rc_outer_inner * 36) + 1178))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 182))] * kernel_shared[(((rc_outer_inner * 36) + 3482))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 182))] * kernel_shared[(((rc_outer_inner * 36) + 1754))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 182))] * kernel_shared[(((rc_outer_inner * 36) + 4058))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 261))] * kernel_shared[(((rc_outer_inner * 36) + 33))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 261))] * kernel_shared[(((rc_outer_inner * 36) + 2337))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 261))] * kernel_shared[(((rc_outer_inner * 36) + 609))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 261))] * kernel_shared[(((rc_outer_inner * 36) + 2913))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 261))] * kernel_shared[(((rc_outer_inner * 36) + 1185))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 261))] * kernel_shared[(((rc_outer_inner * 36) + 3489))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 261))] * kernel_shared[(((rc_outer_inner * 36) + 1761))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 261))] * kernel_shared[(((rc_outer_inner * 36) + 4065))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 262))] * kernel_shared[(((rc_outer_inner * 36) + 34))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 262))] * kernel_shared[(((rc_outer_inner * 36) + 2338))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 262))] * kernel_shared[(((rc_outer_inner * 36) + 610))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 262))] * kernel_shared[(((rc_outer_inner * 36) + 2914))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 262))] * kernel_shared[(((rc_outer_inner * 36) + 1186))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 262))] * kernel_shared[(((rc_outer_inner * 36) + 3490))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 262))] * kernel_shared[(((rc_outer_inner * 36) + 1762))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 262))] * kernel_shared[(((rc_outer_inner * 36) + 4066))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 263))] * kernel_shared[(((rc_outer_inner * 36) + 35))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 263))] * kernel_shared[(((rc_outer_inner * 36) + 2339))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 263))] * kernel_shared[(((rc_outer_inner * 36) + 611))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 263))] * kernel_shared[(((rc_outer_inner * 36) + 2915))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 263))] * kernel_shared[(((rc_outer_inner * 36) + 1187))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 263))] * kernel_shared[(((rc_outer_inner * 36) + 3491))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 263))] * kernel_shared[(((rc_outer_inner * 36) + 1763))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((rc_outer_inner * 324) + ((((int)threadIdx.x) / 7) * 9)) + (((int)threadIdx.x) % 7)) + 263))] * kernel_shared[(((rc_outer_inner * 36) + 4067))]));
    }
  }
  for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
    compute[((((((int)blockIdx.x) * 392) + (ff_inner * 49)) + ((int)threadIdx.x)))] = compute_local[(ff_inner)];
    compute[(((((((int)blockIdx.x) * 392) + (ff_inner * 49)) + ((int)threadIdx.x)) + 196))] = compute_local[((ff_inner + 4))];
  }
}



void conv_kernel_wrapper(int N_B, int N_C, int N_H, int N_W, int N_F, int N_R, int N_S, int PaddingH, int PaddingW,
                        int StrideH, int StrideW, int N_X, int N_Y, const float *Input, 
                        const float *Kernel, float *Output, int itr) {
                        hipEvent_t start;
                        CHECK(hipEventCreate(&start));
                        hipEvent_t stop;
                        CHECK(hipEventCreate(&stop));
                        float *dev_Input;
                        float *dev_Kernel;
                        float *dev_Output;
                    
                        CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                        CHECK(hipMemcpy(dev_Kernel, Kernel, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                        CHECK(hipMemcpy(dev_Input, Input, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Output, sizeof(float) * N_B * N_F * N_Y * N_X));
                        CHECK(hipMemset(dev_Output, 0, sizeof(float) * N_B * N_F * N_Y * N_X));
                    
dim3 size_grid(64,1,1);

dim3 size_block(49,1,1);
 
                       printf("warmup....\n");
                        default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                    
                        double *time_arr = (double *)malloc(itr * sizeof(double));
                        for (int j = 1; j < itr; j++) {
                            CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                            CHECK(hipMemcpy(dev_Kernel, Kernel+j*N_F * N_C * N_R * N_S, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                            CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                            CHECK(hipMemcpy(dev_Input, Input+j*N_B * N_C * N_H * N_W, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                            /* Record the start event*/
                            CHECK(hipEventRecord(start, NULL));
                            default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                            /* Record the end event */
                            CHECK(hipEventRecord(stop, NULL));
                            /* Wait for the stop event to complete*/
                            CHECK(hipEventSynchronize(stop));
                            float msec = 0.0f;
                            hipEventElapsedTime(&msec, start, stop);
                            time_arr[j] = msec;
                        }
                    
                       
                    
                        //hipError_t err = hipGetLastError();
                        //if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
                        double msecTotal = 0;
                        for (int j = 1; j < itr; j++) {   
                            printf("%0.5lf\n", time_arr[j]);
                            msecTotal += time_arr[j];
                        }
                    
                        double msecPerMatrixMul = (msecTotal) / itr;
                        double flopsPerMatrixMul = 2.0 * N_B * N_C * N_X * N_Y * N_F * N_R * N_S;
                    
                        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
                    
                        printf("\n===========================================================\n");
                        printf("Input size: %d X %d X %d X %d\n", N_B, N_C, N_H, N_W);
                        printf("Kernel size: %d X %d X %d X %d\n", N_F, N_C, N_R, N_S);
                        printf("Output size: %d X %d X %d X %d\n", N_B, N_F, N_Y, N_X);
                        printf("flops: %0.5lf\n", (flopsPerMatrixMul * 1.0e-9f));
                        printf("Time is %0.5lf \n", (msecPerMatrixMul / 1000.0f));
                        printf("Gflops %0.5lf \n", gigaFlops);
                        printf("===========================================================\n");
                    
                        /*write back to Host Output -- batch size is 1*/
                        CHECK(hipMemcpy(Output, dev_Output, sizeof(float) * N_B * N_F * N_X * N_Y, hipMemcpyDeviceToHost));
                    
                        CHECK(hipFree(dev_Input));
                        CHECK(hipFree(dev_Kernel));
                        CHECK(hipFree(dev_Output));
                    
                    }
            