#include "hip/hip_runtime.h"

            #include <cassert>
            #include <stdlib.h>
            #include <hip/hip_runtime.h>
            #include "common.h"
            

            
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[64];
  __shared__ float pad_temp_shared[1088];
  __shared__ float kernel_shared[512];
  for (int ff_c_outer_inner_init = 0; ff_c_outer_inner_init < 4; ++ff_c_outer_inner_init) {
    compute_local[((ff_c_outer_inner_init * 2))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 2) + 8))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 2) + 16))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 2) + 24))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 2) + 32))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 2) + 40))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 2) + 48))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 2) + 56))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 2) + 1))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 2) + 9))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 2) + 17))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 2) + 25))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 2) + 33))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 2) + 41))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 2) + 49))] = 0.000000e+00f;
    compute_local[(((ff_c_outer_inner_init * 2) + 57))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
      ((float2*)(pad_temp_shared + ((((int)threadIdx.x) * 2))))[0] = ((float2*)(data + (((((((rc_outer_outer * 147968) + ((((int)threadIdx.x) / 68) * 18496)) + ((((int)blockIdx.x) >> 2) * 544)) + (((((int)threadIdx.x) % 68) / 17) * 136)) + ((((int)blockIdx.x) & 3) * 34)) + ((((int)threadIdx.x) % 17) * 2)))))[0];
      ((float2*)(pad_temp_shared + (((((int)threadIdx.x) * 2) + 272))))[0] = ((float2*)(data + ((((((((rc_outer_outer * 147968) + ((((int)threadIdx.x) / 68) * 18496)) + ((((int)blockIdx.x) >> 2) * 544)) + (((((int)threadIdx.x) % 68) / 17) * 136)) + ((((int)blockIdx.x) & 3) * 34)) + ((((int)threadIdx.x) % 17) * 2)) + 36992))))[0];
      ((float2*)(pad_temp_shared + (((((int)threadIdx.x) * 2) + 544))))[0] = ((float2*)(data + ((((((((rc_outer_outer * 147968) + ((((int)threadIdx.x) / 68) * 18496)) + ((((int)blockIdx.x) >> 2) * 544)) + (((((int)threadIdx.x) % 68) / 17) * 136)) + ((((int)blockIdx.x) & 3) * 34)) + ((((int)threadIdx.x) % 17) * 2)) + 73984))))[0];
      ((float2*)(pad_temp_shared + (((((int)threadIdx.x) * 2) + 816))))[0] = ((float2*)(data + ((((((((rc_outer_outer * 147968) + ((((int)threadIdx.x) / 68) * 18496)) + ((((int)blockIdx.x) >> 2) * 544)) + (((((int)threadIdx.x) % 68) / 17) * 136)) + ((((int)blockIdx.x) & 3) * 34)) + ((((int)threadIdx.x) % 17) * 2)) + 110976))))[0];
    kernel_shared[(((int)threadIdx.x))] = kernel[(((((((int)threadIdx.x) >> 3) * 128) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    kernel_shared[((((int)threadIdx.x) + 136))] = kernel[((((((((int)threadIdx.x) >> 3) * 128) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 2176))];
    kernel_shared[((((int)threadIdx.x) + 272))] = kernel[((((((((int)threadIdx.x) >> 3) * 128) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 4352))];
    if (((int)threadIdx.x) < 104) {
      kernel_shared[((((int)threadIdx.x) + 408))] = kernel[((((((((int)threadIdx.x) >> 3) * 128) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 6528))];
    }
    __syncthreads();
    for (int ff_c_outer_inner = 0; ff_c_outer_inner < 4; ++ff_c_outer_inner) {
      for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
        compute_local[((ff_c_outer_inner * 2))] = (compute_local[((ff_c_outer_inner * 2))] + (pad_temp_shared[(((rc_inner * 136) + (((int)threadIdx.x) % 34)))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 32) + (ff_c_outer_inner * 8)) + rc_inner))]));
        compute_local[(((ff_c_outer_inner * 2) + 8))] = (compute_local[(((ff_c_outer_inner * 2) + 8))] + (pad_temp_shared[((((rc_inner * 136) + (((int)threadIdx.x) % 34)) + 68))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 32) + (ff_c_outer_inner * 8)) + rc_inner))]));
        compute_local[(((ff_c_outer_inner * 2) + 16))] = (compute_local[(((ff_c_outer_inner * 2) + 16))] + (pad_temp_shared[(((rc_inner * 136) + (((int)threadIdx.x) % 34)))] * kernel_shared[((((((((int)threadIdx.x) / 34) * 32) + (ff_c_outer_inner * 8)) + rc_inner) + 128))]));
        compute_local[(((ff_c_outer_inner * 2) + 24))] = (compute_local[(((ff_c_outer_inner * 2) + 24))] + (pad_temp_shared[((((rc_inner * 136) + (((int)threadIdx.x) % 34)) + 68))] * kernel_shared[((((((((int)threadIdx.x) / 34) * 32) + (ff_c_outer_inner * 8)) + rc_inner) + 128))]));
        compute_local[(((ff_c_outer_inner * 2) + 32))] = (compute_local[(((ff_c_outer_inner * 2) + 32))] + (pad_temp_shared[(((rc_inner * 136) + (((int)threadIdx.x) % 34)))] * kernel_shared[((((((((int)threadIdx.x) / 34) * 32) + (ff_c_outer_inner * 8)) + rc_inner) + 256))]));
        compute_local[(((ff_c_outer_inner * 2) + 40))] = (compute_local[(((ff_c_outer_inner * 2) + 40))] + (pad_temp_shared[((((rc_inner * 136) + (((int)threadIdx.x) % 34)) + 68))] * kernel_shared[((((((((int)threadIdx.x) / 34) * 32) + (ff_c_outer_inner * 8)) + rc_inner) + 256))]));
        compute_local[(((ff_c_outer_inner * 2) + 48))] = (compute_local[(((ff_c_outer_inner * 2) + 48))] + (pad_temp_shared[(((rc_inner * 136) + (((int)threadIdx.x) % 34)))] * kernel_shared[((((((((int)threadIdx.x) / 34) * 32) + (ff_c_outer_inner * 8)) + rc_inner) + 384))]));
        compute_local[(((ff_c_outer_inner * 2) + 56))] = (compute_local[(((ff_c_outer_inner * 2) + 56))] + (pad_temp_shared[((((rc_inner * 136) + (((int)threadIdx.x) % 34)) + 68))] * kernel_shared[((((((((int)threadIdx.x) / 34) * 32) + (ff_c_outer_inner * 8)) + rc_inner) + 384))]));
        compute_local[(((ff_c_outer_inner * 2) + 1))] = (compute_local[(((ff_c_outer_inner * 2) + 1))] + (pad_temp_shared[((((rc_inner * 136) + (((int)threadIdx.x) % 34)) + 34))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 32) + (ff_c_outer_inner * 8)) + rc_inner))]));
        compute_local[(((ff_c_outer_inner * 2) + 9))] = (compute_local[(((ff_c_outer_inner * 2) + 9))] + (pad_temp_shared[((((rc_inner * 136) + (((int)threadIdx.x) % 34)) + 102))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 32) + (ff_c_outer_inner * 8)) + rc_inner))]));
        compute_local[(((ff_c_outer_inner * 2) + 17))] = (compute_local[(((ff_c_outer_inner * 2) + 17))] + (pad_temp_shared[((((rc_inner * 136) + (((int)threadIdx.x) % 34)) + 34))] * kernel_shared[((((((((int)threadIdx.x) / 34) * 32) + (ff_c_outer_inner * 8)) + rc_inner) + 128))]));
        compute_local[(((ff_c_outer_inner * 2) + 25))] = (compute_local[(((ff_c_outer_inner * 2) + 25))] + (pad_temp_shared[((((rc_inner * 136) + (((int)threadIdx.x) % 34)) + 102))] * kernel_shared[((((((((int)threadIdx.x) / 34) * 32) + (ff_c_outer_inner * 8)) + rc_inner) + 128))]));
        compute_local[(((ff_c_outer_inner * 2) + 33))] = (compute_local[(((ff_c_outer_inner * 2) + 33))] + (pad_temp_shared[((((rc_inner * 136) + (((int)threadIdx.x) % 34)) + 34))] * kernel_shared[((((((((int)threadIdx.x) / 34) * 32) + (ff_c_outer_inner * 8)) + rc_inner) + 256))]));
        compute_local[(((ff_c_outer_inner * 2) + 41))] = (compute_local[(((ff_c_outer_inner * 2) + 41))] + (pad_temp_shared[((((rc_inner * 136) + (((int)threadIdx.x) % 34)) + 102))] * kernel_shared[((((((((int)threadIdx.x) / 34) * 32) + (ff_c_outer_inner * 8)) + rc_inner) + 256))]));
        compute_local[(((ff_c_outer_inner * 2) + 49))] = (compute_local[(((ff_c_outer_inner * 2) + 49))] + (pad_temp_shared[((((rc_inner * 136) + (((int)threadIdx.x) % 34)) + 34))] * kernel_shared[((((((((int)threadIdx.x) / 34) * 32) + (ff_c_outer_inner * 8)) + rc_inner) + 384))]));
        compute_local[(((ff_c_outer_inner * 2) + 57))] = (compute_local[(((ff_c_outer_inner * 2) + 57))] + (pad_temp_shared[((((rc_inner * 136) + (((int)threadIdx.x) % 34)) + 102))] * kernel_shared[((((((((int)threadIdx.x) / 34) * 32) + (ff_c_outer_inner * 8)) + rc_inner) + 384))]));
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
      compute[((((((((((int)threadIdx.x) / 34) * 73984) + (ff_inner * 18496)) + ((((int)blockIdx.x) >> 2) * 544)) + (yy_inner * 136)) + ((((int)blockIdx.x) & 3) * 34)) + (((int)threadIdx.x) % 34)))] = compute_local[(((ff_inner * 2) + yy_inner))];
      compute[(((((((((((int)threadIdx.x) / 34) * 73984) + (ff_inner * 18496)) + ((((int)blockIdx.x) >> 2) * 544)) + (yy_inner * 136)) + ((((int)blockIdx.x) & 3) * 34)) + (((int)threadIdx.x) % 34)) + 272))] = compute_local[((((ff_inner * 2) + yy_inner) + 8))];
      compute[(((((((((((int)threadIdx.x) / 34) * 73984) + (ff_inner * 18496)) + ((((int)blockIdx.x) >> 2) * 544)) + (yy_inner * 136)) + ((((int)blockIdx.x) & 3) * 34)) + (((int)threadIdx.x) % 34)) + 295936))] = compute_local[((((ff_inner * 2) + yy_inner) + 16))];
      compute[(((((((((((int)threadIdx.x) / 34) * 73984) + (ff_inner * 18496)) + ((((int)blockIdx.x) >> 2) * 544)) + (yy_inner * 136)) + ((((int)blockIdx.x) & 3) * 34)) + (((int)threadIdx.x) % 34)) + 296208))] = compute_local[((((ff_inner * 2) + yy_inner) + 24))];
      compute[(((((((((((int)threadIdx.x) / 34) * 73984) + (ff_inner * 18496)) + ((((int)blockIdx.x) >> 2) * 544)) + (yy_inner * 136)) + ((((int)blockIdx.x) & 3) * 34)) + (((int)threadIdx.x) % 34)) + 591872))] = compute_local[((((ff_inner * 2) + yy_inner) + 32))];
      compute[(((((((((((int)threadIdx.x) / 34) * 73984) + (ff_inner * 18496)) + ((((int)blockIdx.x) >> 2) * 544)) + (yy_inner * 136)) + ((((int)blockIdx.x) & 3) * 34)) + (((int)threadIdx.x) % 34)) + 592144))] = compute_local[((((ff_inner * 2) + yy_inner) + 40))];
      compute[(((((((((((int)threadIdx.x) / 34) * 73984) + (ff_inner * 18496)) + ((((int)blockIdx.x) >> 2) * 544)) + (yy_inner * 136)) + ((((int)blockIdx.x) & 3) * 34)) + (((int)threadIdx.x) % 34)) + 887808))] = compute_local[((((ff_inner * 2) + yy_inner) + 48))];
      compute[(((((((((((int)threadIdx.x) / 34) * 73984) + (ff_inner * 18496)) + ((((int)blockIdx.x) >> 2) * 544)) + (yy_inner * 136)) + ((((int)blockIdx.x) & 3) * 34)) + (((int)threadIdx.x) % 34)) + 888080))] = compute_local[((((ff_inner * 2) + yy_inner) + 56))];
    }
  }
}



void conv_kernel_wrapper(int N_B, int N_C, int N_H, int N_W, int N_F, int N_R, int N_S, int PaddingH, int PaddingW,
                        int StrideH, int StrideW, int N_X, int N_Y, const float *Input, 
                        const float *Kernel, float *Output, int itr) {
                        hipEvent_t start;
                        CHECK(hipEventCreate(&start));
                        hipEvent_t stop;
                        CHECK(hipEventCreate(&stop));
                        float *dev_Input;
                        float *dev_Kernel;
                        float *dev_Output;
                    
                        CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                        CHECK(hipMemcpy(dev_Kernel, Kernel, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                        CHECK(hipMemcpy(dev_Input, Input, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Output, sizeof(float) * N_B * N_F * N_Y * N_X));
                        CHECK(hipMemset(dev_Output, 0, sizeof(float) * N_B * N_F * N_Y * N_X));
                    
dim3 size_grid(136,1,1);

dim3 size_block(136,1,1);
 
                       printf("warmup....\n");
                        default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                    
                        double *time_arr = (double *)malloc(itr * sizeof(double));
                        for (int j = 1; j < itr; j++) {
                            CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                            CHECK(hipMemcpy(dev_Kernel, Kernel+j*N_F * N_C * N_R * N_S, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                            CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                            CHECK(hipMemcpy(dev_Input, Input+j*N_B * N_C * N_H * N_W, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                            /* Record the start event*/
                            CHECK(hipEventRecord(start, NULL));
                            default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                            /* Record the end event */
                            CHECK(hipEventRecord(stop, NULL));
                            /* Wait for the stop event to complete*/
                            CHECK(hipEventSynchronize(stop));
                            float msec = 0.0f;
                            hipEventElapsedTime(&msec, start, stop);
                            time_arr[j] = msec;
                        }
                    
                       
                    
                        //hipError_t err = hipGetLastError();
                        //if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
                        double msecTotal = 0;
                        for (int j = 1; j < itr; j++) {   
                            printf("%0.5lf\n", time_arr[j]);
                            msecTotal += time_arr[j];
                        }
                    
                        double msecPerMatrixMul = (msecTotal) / itr;
                        double flopsPerMatrixMul = 2.0 * N_B * N_C * N_X * N_Y * N_F * N_R * N_S;
                    
                        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
                    
                        printf("\n===========================================================\n");
                        printf("Input size: %d X %d X %d X %d\n", N_B, N_C, N_H, N_W);
                        printf("Kernel size: %d X %d X %d X %d\n", N_F, N_C, N_R, N_S);
                        printf("Output size: %d X %d X %d X %d\n", N_B, N_F, N_Y, N_X);
                        printf("flops: %0.5lf\n", (flopsPerMatrixMul * 1.0e-9f));
                        printf("Time is %0.5lf \n", (msecPerMatrixMul / 1000.0f));
                        printf("Gflops %0.5lf \n", gigaFlops);
                        printf("===========================================================\n");
                    
                        /*write back to Host Output -- batch size is 1*/
                        CHECK(hipMemcpy(Output, dev_Output, sizeof(float) * N_B * N_F * N_X * N_Y, hipMemcpyDeviceToHost));
                    
                        CHECK(hipFree(dev_Input));
                        CHECK(hipFree(dev_Kernel));
                        CHECK(hipFree(dev_Output));
                    
                    }
            