#include "hip/hip_runtime.h"

            #include <cassert>
            #include <stdlib.h>
            #include <hip/hip_runtime.h>
            #include "common.h"
            

            
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[16];
  __shared__ float pad_temp_shared[3520];
  __shared__ float kernel_shared[4096];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(8)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(9)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(10)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(11)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(12)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(13)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  compute_local[(14)] = 0.000000e+00f;
  compute_local[(7)] = 0.000000e+00f;
  compute_local[(15)] = 0.000000e+00f;
  pad_temp_shared[(((int)threadIdx.x))] = data[(((((((int)threadIdx.x) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + (((int)threadIdx.x) % 55)))];
  pad_temp_shared[((((int)threadIdx.x) + 112))] = data[((((((((int)threadIdx.x) + 112) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 2) % 55)))];
  pad_temp_shared[((((int)threadIdx.x) + 224))] = data[((((((((int)threadIdx.x) + 224) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 4) % 55)))];
  pad_temp_shared[((((int)threadIdx.x) + 336))] = data[((((((((int)threadIdx.x) + 336) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 6) % 55)))];
  pad_temp_shared[((((int)threadIdx.x) + 448))] = data[((((((((int)threadIdx.x) + 448) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 8) % 55)))];
  pad_temp_shared[((((int)threadIdx.x) + 560))] = data[((((((((int)threadIdx.x) + 560) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 10) % 55)))];
  pad_temp_shared[((((int)threadIdx.x) + 672))] = data[((((((((int)threadIdx.x) + 672) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 12) % 55)))];
  pad_temp_shared[((((int)threadIdx.x) + 784))] = data[((((((((int)threadIdx.x) + 784) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 14) % 55)))];
  pad_temp_shared[((((int)threadIdx.x) + 896))] = data[((((((((int)threadIdx.x) + 896) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 16) % 55)))];
  pad_temp_shared[((((int)threadIdx.x) + 1008))] = data[((((((((int)threadIdx.x) + 1008) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 18) % 55)))];
  pad_temp_shared[((((int)threadIdx.x) + 1120))] = data[((((((((int)threadIdx.x) + 1120) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 20) % 55)))];
  pad_temp_shared[((((int)threadIdx.x) + 1232))] = data[((((((((int)threadIdx.x) + 1232) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 22) % 55)))];
  pad_temp_shared[((((int)threadIdx.x) + 1344))] = data[((((((((int)threadIdx.x) + 1344) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 24) % 55)))];
  pad_temp_shared[((((int)threadIdx.x) + 1456))] = data[((((((((int)threadIdx.x) + 1456) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 26) % 55)))];
  pad_temp_shared[((((int)threadIdx.x) + 1568))] = data[((((((((int)threadIdx.x) + 1568) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 28) % 55)))];
  pad_temp_shared[((((int)threadIdx.x) + 1680))] = data[((((((((int)threadIdx.x) + 1680) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 30) % 55)))];
  pad_temp_shared[((((int)threadIdx.x) + 1792))] = data[((((((((int)threadIdx.x) + 1792) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 32) % 55)))];
  pad_temp_shared[((((int)threadIdx.x) + 1904))] = data[((((((((int)threadIdx.x) + 1904) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 34) % 55)))];
  pad_temp_shared[((((int)threadIdx.x) + 2016))] = data[((((((((int)threadIdx.x) + 2016) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 36) % 55)))];
  pad_temp_shared[((((int)threadIdx.x) + 2128))] = data[((((((((int)threadIdx.x) + 2128) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 38) % 55)))];
  pad_temp_shared[((((int)threadIdx.x) + 2240))] = data[((((((((int)threadIdx.x) + 2240) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 40) % 55)))];
  pad_temp_shared[((((int)threadIdx.x) + 2352))] = data[((((((((int)threadIdx.x) + 2352) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 42) % 55)))];
  pad_temp_shared[((((int)threadIdx.x) + 2464))] = data[((((((((int)threadIdx.x) + 2464) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 44) % 55)))];
  pad_temp_shared[((((int)threadIdx.x) + 2576))] = data[((((((((int)threadIdx.x) + 2576) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 46) % 55)))];
  pad_temp_shared[((((int)threadIdx.x) + 2688))] = data[((((((((int)threadIdx.x) + 2688) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 48) % 55)))];
  pad_temp_shared[((((int)threadIdx.x) + 2800))] = data[((((((((int)threadIdx.x) + 2800) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 50) % 55)))];
  pad_temp_shared[((((int)threadIdx.x) + 2912))] = data[((((((((int)threadIdx.x) + 2912) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 52) % 55)))];
  pad_temp_shared[((((int)threadIdx.x) + 3024))] = data[((((((((int)threadIdx.x) + 3024) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 54) % 55)))];
  pad_temp_shared[((((int)threadIdx.x) + 3136))] = data[((((((((int)threadIdx.x) + 3136) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 1) % 55)))];
  pad_temp_shared[((((int)threadIdx.x) + 3248))] = data[((((((((int)threadIdx.x) + 3248) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 3) % 55)))];
  pad_temp_shared[((((int)threadIdx.x) + 3360))] = data[((((((((int)threadIdx.x) + 3360) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 5) % 55)))];
  if (((int)threadIdx.x) < 48) {
    pad_temp_shared[((((int)threadIdx.x) + 3472))] = data[((((((((int)threadIdx.x) + 3472) / 55) * 3136) + ((((int)blockIdx.x) % 28) * 112)) + (((int)threadIdx.x) + 7)))];
  }
  kernel_shared[(((int)threadIdx.x))] = kernel[((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)))];
  kernel_shared[((((int)threadIdx.x) + 112))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 112))];
  kernel_shared[((((int)threadIdx.x) + 224))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 224))];
  kernel_shared[((((int)threadIdx.x) + 336))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 336))];
  kernel_shared[((((int)threadIdx.x) + 448))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 448))];
  kernel_shared[((((int)threadIdx.x) + 560))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 560))];
  kernel_shared[((((int)threadIdx.x) + 672))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 672))];
  kernel_shared[((((int)threadIdx.x) + 784))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 784))];
  kernel_shared[((((int)threadIdx.x) + 896))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 896))];
  kernel_shared[((((int)threadIdx.x) + 1008))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 1008))];
  kernel_shared[((((int)threadIdx.x) + 1120))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 1120))];
  kernel_shared[((((int)threadIdx.x) + 1232))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 1232))];
  kernel_shared[((((int)threadIdx.x) + 1344))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 1344))];
  kernel_shared[((((int)threadIdx.x) + 1456))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 1456))];
  kernel_shared[((((int)threadIdx.x) + 1568))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 1568))];
  kernel_shared[((((int)threadIdx.x) + 1680))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 1680))];
  kernel_shared[((((int)threadIdx.x) + 1792))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 1792))];
  kernel_shared[((((int)threadIdx.x) + 1904))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 1904))];
  kernel_shared[((((int)threadIdx.x) + 2016))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 2016))];
  kernel_shared[((((int)threadIdx.x) + 2128))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 2128))];
  kernel_shared[((((int)threadIdx.x) + 2240))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 2240))];
  kernel_shared[((((int)threadIdx.x) + 2352))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 2352))];
  kernel_shared[((((int)threadIdx.x) + 2464))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 2464))];
  kernel_shared[((((int)threadIdx.x) + 2576))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 2576))];
  kernel_shared[((((int)threadIdx.x) + 2688))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 2688))];
  kernel_shared[((((int)threadIdx.x) + 2800))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 2800))];
  kernel_shared[((((int)threadIdx.x) + 2912))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 2912))];
  kernel_shared[((((int)threadIdx.x) + 3024))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 3024))];
  kernel_shared[((((int)threadIdx.x) + 3136))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 3136))];
  kernel_shared[((((int)threadIdx.x) + 3248))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 3248))];
  kernel_shared[((((int)threadIdx.x) + 3360))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 3360))];
  kernel_shared[((((int)threadIdx.x) + 3472))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 3472))];
  kernel_shared[((((int)threadIdx.x) + 3584))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 3584))];
  kernel_shared[((((int)threadIdx.x) + 3696))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 3696))];
  kernel_shared[((((int)threadIdx.x) + 3808))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 3808))];
  kernel_shared[((((int)threadIdx.x) + 3920))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 3920))];
  if (((int)threadIdx.x) < 64) {
    kernel_shared[((((int)threadIdx.x) + 4032))] = kernel[(((((((int)blockIdx.x) / 28) * 4096) + ((int)threadIdx.x)) + 4032))];
  }
  __syncthreads();
  for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)))] * kernel_shared[((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 28))] * kernel_shared[((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 64))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 28))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 64))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 55))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 1))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 83))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 1))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 55))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 65))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 83))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 65))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 110))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 2))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 138))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 2))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 110))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 66))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 138))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 66))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 165))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 3))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 193))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 3))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 165))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 67))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 193))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 67))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 220))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 4))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 248))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 4))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 220))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 68))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 248))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 68))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 275))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 5))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 303))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 5))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 275))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 69))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 303))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 69))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 330))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 6))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 358))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 6))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 330))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 70))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 358))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 70))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 385))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 7))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 413))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 7))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 385))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 71))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 413))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 71))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 128))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 28))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 128))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 192))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 28))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 192))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 55))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 129))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 83))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 129))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 55))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 193))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 83))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 193))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 110))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 130))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 138))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 130))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 110))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 194))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 138))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 194))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 165))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 131))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 193))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 131))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 165))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 195))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 193))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 195))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 220))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 132))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 248))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 132))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 220))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 196))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 248))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 196))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 275))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 133))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 303))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 133))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 275))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 197))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 303))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 197))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 330))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 134))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 358))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 134))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 330))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 198))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 358))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 198))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 385))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 135))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 413))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 135))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 385))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 199))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 413))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 199))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 256))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 28))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 256))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 320))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 28))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 320))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 55))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 257))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 83))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 257))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 55))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 321))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 83))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 321))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 110))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 258))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 138))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 258))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 110))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 322))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 138))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 322))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 165))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 259))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 193))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 259))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 165))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 323))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 193))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 323))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 220))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 260))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 248))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 260))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 220))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 324))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 248))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 324))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 275))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 261))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 303))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 261))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 275))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 325))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 303))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 325))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 330))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 262))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 358))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 262))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 330))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 326))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 358))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 326))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 385))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 263))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 413))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 263))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 385))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 327))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 413))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 327))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 384))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 28))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 384))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 448))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 28))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 448))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 55))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 385))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 83))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 385))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 55))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 449))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 83))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 449))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 110))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 386))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 138))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 386))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 110))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 450))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 138))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 450))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 165))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 387))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 193))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 387))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 165))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 451))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 193))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 451))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 220))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 388))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 248))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 388))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 220))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 452))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 248))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 452))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 275))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 389))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 303))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 389))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 275))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 453))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 303))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 453))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 330))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 390))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 358))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 390))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 330))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 454))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 358))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 454))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 385))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 391))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 413))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 391))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 385))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 455))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((rc_outer_inner * 440) + ((((int)threadIdx.x) % 14) * 2)) + 413))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 512) + (rc_outer_inner * 8)) + 455))]));
  }
  for (int ff_inner = 0; ff_inner < 8; ++ff_inner) {
    compute[(((((((((int)blockIdx.x) / 28) * 50176) + ((((int)threadIdx.x) / 14) * 6272)) + (ff_inner * 784)) + ((((int)blockIdx.x) % 28) * 28)) + (((int)threadIdx.x) % 14)))] = compute_local[(ff_inner)];
    compute[((((((((((int)blockIdx.x) / 28) * 50176) + ((((int)threadIdx.x) / 14) * 6272)) + (ff_inner * 784)) + ((((int)blockIdx.x) % 28) * 28)) + (((int)threadIdx.x) % 14)) + 14))] = compute_local[((ff_inner + 8))];
  }
}



void conv_kernel_wrapper(int N_B, int N_C, int N_H, int N_W, int N_F, int N_R, int N_S, int PaddingH, int PaddingW,
                        int StrideH, int StrideW, int N_X, int N_Y, const float *Input, 
                        const float *Kernel, float *Output, int itr) {
                        hipEvent_t start;
                        CHECK(hipEventCreate(&start));
                        hipEvent_t stop;
                        CHECK(hipEventCreate(&stop));
                        float *dev_Input;
                        float *dev_Kernel;
                        float *dev_Output;
                    
                        CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                        CHECK(hipMemcpy(dev_Kernel, Kernel, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                        CHECK(hipMemcpy(dev_Input, Input, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Output, sizeof(float) * N_B * N_F * N_Y * N_X));
                        CHECK(hipMemset(dev_Output, 0, sizeof(float) * N_B * N_F * N_Y * N_X));
                    
dim3 size_grid(56,1,1);

dim3 size_block(112,1,1);
 
                       printf("warmup....\n");
                        default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                    
                        double *time_arr = (double *)malloc(itr * sizeof(double));
                        for (int j = 1; j < itr; j++) {
                            CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                            CHECK(hipMemcpy(dev_Kernel, Kernel+j*N_F * N_C * N_R * N_S, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                            CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                            CHECK(hipMemcpy(dev_Input, Input+j*N_B * N_C * N_H * N_W, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                            /* Record the start event*/
                            CHECK(hipEventRecord(start, NULL));
                            default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                            /* Record the end event */
                            CHECK(hipEventRecord(stop, NULL));
                            /* Wait for the stop event to complete*/
                            CHECK(hipEventSynchronize(stop));
                            float msec = 0.0f;
                            hipEventElapsedTime(&msec, start, stop);
                            time_arr[j] = msec;
                        }
                    
                       
                    
                        //hipError_t err = hipGetLastError();
                        //if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
                        double msecTotal = 0;
                        for (int j = 1; j < itr; j++) {   
                            printf("%0.5lf\n", time_arr[j]);
                            msecTotal += time_arr[j];
                        }
                    
                        double msecPerMatrixMul = (msecTotal) / itr;
                        double flopsPerMatrixMul = 2.0 * N_B * N_C * N_X * N_Y * N_F * N_R * N_S;
                    
                        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
                    
                        printf("\n===========================================================\n");
                        printf("Input size: %d X %d X %d X %d\n", N_B, N_C, N_H, N_W);
                        printf("Kernel size: %d X %d X %d X %d\n", N_F, N_C, N_R, N_S);
                        printf("Output size: %d X %d X %d X %d\n", N_B, N_F, N_Y, N_X);
                        printf("flops: %0.5lf\n", (flopsPerMatrixMul * 1.0e-9f));
                        printf("Time is %0.5lf \n", (msecPerMatrixMul / 1000.0f));
                        printf("Gflops %0.5lf \n", gigaFlops);
                        printf("===========================================================\n");
                    
                        /*write back to Host Output -- batch size is 1*/
                        CHECK(hipMemcpy(Output, dev_Output, sizeof(float) * N_B * N_F * N_X * N_Y, hipMemcpyDeviceToHost));
                    
                        CHECK(hipFree(dev_Input));
                        CHECK(hipFree(dev_Kernel));
                        CHECK(hipFree(dev_Output));
                    
                    }
            