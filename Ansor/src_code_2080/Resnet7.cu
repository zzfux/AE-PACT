#include "hip/hip_runtime.h"

            #include <cassert>
            #include <stdlib.h>
            #include <hip/hip_runtime.h>
            #include "common.h"
            

            
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[4];
  __shared__ float pad_temp_shared[1160];
  __shared__ float kernel_shared[2304];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((((int)threadIdx.x) * 2))] = data[((((((rc_outer_outer * 7200) + (((((int)threadIdx.x) * 2) / 145) * 900)) + ((((int)blockIdx.x) % 7) * 120)) + ((((((int)threadIdx.x) * 2) % 145) / 29) * 30)) + ((((int)threadIdx.x) * 2) % 29)))];
    pad_temp_shared[(((((int)threadIdx.x) * 2) + 1))] = data[((((((rc_outer_outer * 7200) + ((((((int)threadIdx.x) * 2) + 1) / 145) * 900)) + ((((int)blockIdx.x) % 7) * 120)) + (((((((int)threadIdx.x) * 2) + 1) % 145) / 29) * 30)) + (((((int)threadIdx.x) * 2) + 1) % 29)))];
    pad_temp_shared[(((((int)threadIdx.x) * 2) + 448))] = data[((((((rc_outer_outer * 7200) + ((((((int)threadIdx.x) * 2) + 448) / 145) * 900)) + ((((int)blockIdx.x) % 7) * 120)) + (((((((int)threadIdx.x) * 2) + 13) % 145) / 29) * 30)) + (((((int)threadIdx.x) * 2) + 13) % 29)))];
    pad_temp_shared[(((((int)threadIdx.x) * 2) + 449))] = data[((((((rc_outer_outer * 7200) + ((((((int)threadIdx.x) * 2) + 449) / 145) * 900)) + ((((int)blockIdx.x) % 7) * 120)) + (((((((int)threadIdx.x) * 2) + 14) % 145) / 29) * 30)) + (((((int)threadIdx.x) * 2) + 14) % 29)))];
    if (((int)threadIdx.x) < 132) {
      pad_temp_shared[(((((int)threadIdx.x) * 2) + 896))] = data[((((((rc_outer_outer * 7200) + ((((((int)threadIdx.x) * 2) + 896) / 145) * 900)) + ((((int)blockIdx.x) % 7) * 120)) + (((((((int)threadIdx.x) * 2) + 26) % 145) / 29) * 30)) + (((((int)threadIdx.x) * 2) + 26) % 29)))];
    }
    if (((int)threadIdx.x) < 132) {
      pad_temp_shared[(((((int)threadIdx.x) * 2) + 897))] = data[((((((rc_outer_outer * 7200) + ((((((int)threadIdx.x) * 2) + 897) / 145) * 900)) + ((((int)blockIdx.x) % 7) * 120)) + (((((((int)threadIdx.x) * 2) + 27) % 145) / 29) * 30)) + (((((int)threadIdx.x) * 2) + 27) % 29)))];
    }
    kernel_shared[(((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 7) * 36864) + ((((int)threadIdx.x) / 72) * 1152)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) % 72)))];
    kernel_shared[((((int)threadIdx.x) + 224))] = kernel[((((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 224) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 8) % 72)))];
    kernel_shared[((((int)threadIdx.x) + 448))] = kernel[((((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 448) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 16) % 72)))];
    kernel_shared[((((int)threadIdx.x) + 672))] = kernel[((((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 672) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 24) % 72)))];
    kernel_shared[((((int)threadIdx.x) + 896))] = kernel[((((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 896) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 32) % 72)))];
    kernel_shared[((((int)threadIdx.x) + 1120))] = kernel[((((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 1120) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 40) % 72)))];
    kernel_shared[((((int)threadIdx.x) + 1344))] = kernel[((((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 1344) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 48) % 72)))];
    kernel_shared[((((int)threadIdx.x) + 1568))] = kernel[((((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 1568) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 56) % 72)))];
    kernel_shared[((((int)threadIdx.x) + 1792))] = kernel[((((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 1792) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 64) % 72)))];
    kernel_shared[((((int)threadIdx.x) + 2016))] = kernel[(((((((((int)blockIdx.x) / 7) * 36864) + ((((int)threadIdx.x) / 72) * 1152)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) % 72)) + 32256))];
    if (((int)threadIdx.x) < 64) {
      kernel_shared[((((int)threadIdx.x) + 2240))] = kernel[((((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 2240) / 72) * 1152)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) + 8)))];
    }
    __syncthreads();
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)))] * kernel_shared[(((((int)threadIdx.x) / 14) * 144))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 2))] * kernel_shared[(((((int)threadIdx.x) / 14) * 144))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 72))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 2))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 72))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 1))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 3))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 1))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 73))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 3))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 73))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 2))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 2))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 4))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 2))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 2))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 74))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 4))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 74))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 29))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 3))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 31))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 3))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 29))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 75))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 31))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 75))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 30))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 4))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 32))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 4))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 30))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 76))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 32))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 76))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 31))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 5))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 33))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 5))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 31))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 77))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 33))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 77))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 58))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 6))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 60))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 6))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 58))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 78))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 60))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 78))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 59))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 7))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 61))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 7))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 59))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 79))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 61))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 79))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 60))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 8))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 62))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 8))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 60))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 80))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 62))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 80))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 145))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 9))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 147))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 9))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 145))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 81))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 147))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 81))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 146))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 10))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 148))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 10))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 146))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 82))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 148))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 82))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 147))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 11))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 149))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 11))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 147))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 83))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 149))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 83))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 174))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 12))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 176))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 12))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 174))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 84))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 176))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 84))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 175))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 13))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 177))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 13))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 175))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 85))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 177))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 85))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 176))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 14))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 178))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 14))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 176))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 86))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 178))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 86))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 203))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 15))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 205))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 15))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 203))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 87))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 205))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 87))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 204))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 16))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 206))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 16))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 204))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 88))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 206))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 88))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 205))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 17))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 207))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 17))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 205))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 89))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 207))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 89))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 290))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 18))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 292))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 18))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 290))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 90))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 292))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 90))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 291))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 19))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 293))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 19))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 291))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 91))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 293))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 91))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 292))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 20))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 294))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 20))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 292))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 92))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 294))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 92))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 319))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 21))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 321))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 21))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 319))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 93))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 321))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 93))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 320))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 22))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 322))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 22))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 320))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 94))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 322))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 94))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 321))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 23))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 323))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 23))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 321))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 95))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 323))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 95))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 348))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 24))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 350))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 24))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 348))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 96))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 350))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 96))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 349))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 25))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 351))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 25))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 349))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 97))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 351))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 97))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 350))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 26))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 352))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 26))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 350))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 98))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 352))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 98))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 435))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 27))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 437))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 27))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 435))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 99))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 437))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 99))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 436))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 28))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 438))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 28))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 436))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 100))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 438))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 100))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 437))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 29))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 439))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 29))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 437))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 101))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 439))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 101))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 464))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 30))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 466))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 30))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 464))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 102))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 466))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 102))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 465))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 31))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 467))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 31))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 465))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 103))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 467))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 103))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 466))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 32))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 468))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 32))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 466))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 104))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 468))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 104))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 493))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 33))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 495))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 33))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 493))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 105))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 495))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 105))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 494))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 34))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 496))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 34))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 494))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 106))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 496))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 106))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 495))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 35))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 497))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 35))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 495))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 107))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 497))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 107))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 580))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 36))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 582))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 36))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 580))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 108))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 582))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 108))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 581))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 37))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 583))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 37))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 581))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 109))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 583))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 109))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 582))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 38))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 584))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 38))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 582))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 110))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 584))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 110))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 609))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 39))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 611))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 39))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 609))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 111))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 611))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 111))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 610))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 40))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 612))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 40))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 610))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 112))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 612))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 112))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 611))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 41))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 613))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 41))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 611))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 113))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 613))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 113))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 638))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 42))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 640))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 42))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 638))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 114))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 640))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 114))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 639))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 43))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 641))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 43))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 639))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 115))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 641))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 115))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 640))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 44))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 642))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 44))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 640))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 116))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 642))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 116))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 725))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 45))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 727))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 45))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 725))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 117))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 727))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 117))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 726))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 46))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 728))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 46))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 726))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 118))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 728))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 118))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 727))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 47))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 729))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 47))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 727))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 119))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 729))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 119))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 754))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 48))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 756))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 48))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 754))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 120))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 756))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 120))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 755))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 49))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 757))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 49))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 755))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 121))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 757))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 121))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 756))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 50))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 758))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 50))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 756))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 122))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 758))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 122))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 783))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 51))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 785))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 51))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 783))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 123))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 785))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 123))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 784))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 52))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 786))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 52))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 784))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 124))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 786))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 124))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 785))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 53))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 787))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 53))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 785))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 125))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 787))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 125))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 870))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 54))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 872))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 54))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 870))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 126))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 872))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 126))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 871))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 55))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 873))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 55))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 871))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 127))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 873))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 127))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 872))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 56))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 874))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 56))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 872))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 128))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 874))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 128))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 899))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 57))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 901))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 57))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 899))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 129))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 901))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 129))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 900))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 58))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 902))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 58))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 900))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 130))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 902))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 130))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 901))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 59))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 903))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 59))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 901))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 131))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 903))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 131))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 928))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 60))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 930))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 60))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 928))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 132))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 930))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 132))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 929))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 61))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 931))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 61))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 929))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 133))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 931))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 133))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 930))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 62))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 932))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 62))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 930))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 134))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 932))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 134))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1015))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 63))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1017))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 63))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1015))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 135))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1017))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 135))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1016))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 64))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1018))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 64))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1016))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 136))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1018))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 136))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1017))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 65))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1019))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 65))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1017))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 137))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1019))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 137))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1044))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 66))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1046))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 66))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1044))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 138))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1046))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 138))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1045))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 67))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1047))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 67))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1045))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 139))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1047))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 139))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1046))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 68))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1048))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 68))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1046))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 140))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1048))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 140))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1073))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 69))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1075))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 69))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1073))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 141))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1075))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 141))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1074))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 70))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1076))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 70))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1074))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 142))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1076))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 142))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1075))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 71))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1077))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 71))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1075))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 143))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((((int)threadIdx.x) % 14) / 7) * 58) + ((((int)threadIdx.x) % 7) * 4)) + 1077))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + 143))]));
  }
  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
    for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
      compute[((((((((((int)blockIdx.x) / 7) * 6272) + ((((int)threadIdx.x) / 14) * 392)) + (ff_inner * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) % 14) * 2)) + xx_inner))] = compute_local[(((ff_inner * 2) + xx_inner))];
    }
  }
}



void conv_kernel_wrapper(int N_B, int N_C, int N_H, int N_W, int N_F, int N_R, int N_S, int PaddingH, int PaddingW,
                        int StrideH, int StrideW, int N_X, int N_Y, const float *Input, 
                        const float *Kernel, float *Output, int itr) {
                        hipEvent_t start;
                        CHECK(hipEventCreate(&start));
                        hipEvent_t stop;
                        CHECK(hipEventCreate(&stop));
                        float *dev_Input;
                        float *dev_Kernel;
                        float *dev_Output;
                    
                        CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                        CHECK(hipMemcpy(dev_Kernel, Kernel, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                        CHECK(hipMemcpy(dev_Input, Input, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Output, sizeof(float) * N_B * N_F * N_Y * N_X));
                        CHECK(hipMemset(dev_Output, 0, sizeof(float) * N_B * N_F * N_Y * N_X));
                    
dim3 size_grid(56,1,1);

dim3 size_block(224,1,1);
 
                       printf("warmup....\n");
                        default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                    
                        double *time_arr = (double *)malloc(itr * sizeof(double));
                        for (int j = 1; j < itr; j++) {
                            CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                            CHECK(hipMemcpy(dev_Kernel, Kernel+j*N_F * N_C * N_R * N_S, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                            CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                            CHECK(hipMemcpy(dev_Input, Input+j*N_B * N_C * N_H * N_W, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                            /* Record the start event*/
                            CHECK(hipEventRecord(start, NULL));
                            default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                            /* Record the end event */
                            CHECK(hipEventRecord(stop, NULL));
                            /* Wait for the stop event to complete*/
                            CHECK(hipEventSynchronize(stop));
                            float msec = 0.0f;
                            hipEventElapsedTime(&msec, start, stop);
                            time_arr[j] = msec;
                        }
                    
                       
                    
                        //hipError_t err = hipGetLastError();
                        //if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
                        double msecTotal = 0;
                        for (int j = 1; j < itr; j++) {   
                            printf("%0.5lf\n", time_arr[j]);
                            msecTotal += time_arr[j];
                        }
                    
                        double msecPerMatrixMul = (msecTotal) / itr;
                        double flopsPerMatrixMul = 2.0 * N_B * N_C * N_X * N_Y * N_F * N_R * N_S;
                    
                        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
                    
                        printf("\n===========================================================\n");
                        printf("Input size: %d X %d X %d X %d\n", N_B, N_C, N_H, N_W);
                        printf("Kernel size: %d X %d X %d X %d\n", N_F, N_C, N_R, N_S);
                        printf("Output size: %d X %d X %d X %d\n", N_B, N_F, N_Y, N_X);
                        printf("flops: %0.5lf\n", (flopsPerMatrixMul * 1.0e-9f));
                        printf("Time is %0.5lf \n", (msecPerMatrixMul / 1000.0f));
                        printf("Gflops %0.5lf \n", gigaFlops);
                        printf("===========================================================\n");
                    
                        /*write back to Host Output -- batch size is 1*/
                        CHECK(hipMemcpy(Output, dev_Output, sizeof(float) * N_B * N_F * N_X * N_Y, hipMemcpyDeviceToHost));
                    
                        CHECK(hipFree(dev_Input));
                        CHECK(hipFree(dev_Kernel));
                        CHECK(hipFree(dev_Output));
                    
                    }
            