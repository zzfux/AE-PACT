#include "hip/hip_runtime.h"

            #include <cassert>
            #include <stdlib.h>
            #include <hip/hip_runtime.h>
            #include "common.h"
            

            
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[8];
  __shared__ float pad_temp_shared[1792];
  __shared__ float kernel_shared[2304];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(7)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[(((int)threadIdx.x))] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 224) * 3136)) + ((((int)blockIdx.x) % 27) * 112)) + (((int)threadIdx.x) % 224)))];
    pad_temp_shared[((((int)threadIdx.x) + 432))] = data[(((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 432) / 224) * 3136)) + ((((int)blockIdx.x) % 27) * 112)) + ((((int)threadIdx.x) + 208) % 224)))];
    pad_temp_shared[((((int)threadIdx.x) + 864))] = data[(((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 864) / 224) * 3136)) + ((((int)blockIdx.x) % 27) * 112)) + ((((int)threadIdx.x) + 192) % 224)))];
    pad_temp_shared[((((int)threadIdx.x) + 1296))] = data[(((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 1296) / 224) * 3136)) + ((((int)blockIdx.x) % 27) * 112)) + ((((int)threadIdx.x) + 176) % 224)))];
    if (((int)threadIdx.x) < 64) {
      pad_temp_shared[((((int)threadIdx.x) + 1728))] = data[(((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 1728) / 224) * 3136)) + ((((int)blockIdx.x) % 27) * 112)) + (((int)threadIdx.x) + 160)))];
    }
    kernel_shared[(((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 27) * 18432) + ((((int)threadIdx.x) / 72) * 576)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) % 72)))];
    kernel_shared[((((int)threadIdx.x) + 432))] = kernel[(((((((((int)blockIdx.x) / 27) * 18432) + ((((int)threadIdx.x) / 72) * 576)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) % 72)) + 3456))];
    kernel_shared[((((int)threadIdx.x) + 864))] = kernel[(((((((((int)blockIdx.x) / 27) * 18432) + ((((int)threadIdx.x) / 72) * 576)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) % 72)) + 6912))];
    kernel_shared[((((int)threadIdx.x) + 1296))] = kernel[(((((((((int)blockIdx.x) / 27) * 18432) + ((((int)threadIdx.x) / 72) * 576)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) % 72)) + 10368))];
    kernel_shared[((((int)threadIdx.x) + 1728))] = kernel[(((((((((int)blockIdx.x) / 27) * 18432) + ((((int)threadIdx.x) / 72) * 576)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) % 72)) + 13824))];
    if (((int)threadIdx.x) < 144) {
      kernel_shared[((((int)threadIdx.x) + 2160))] = kernel[(((((((((int)blockIdx.x) / 27) * 18432) + ((((int)threadIdx.x) / 72) * 576)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) % 72)) + 17280))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)))] * kernel_shared[((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1152))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 72))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1224))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1153))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 73))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1225))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 2))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 2))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 2))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1154))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 2))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 74))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 2))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1226))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 56))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 3))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 56))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1155))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 56))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 75))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 56))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1227))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 57))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 4))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 57))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1156))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 57))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 76))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 57))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1228))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 58))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 5))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 58))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1157))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 58))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 77))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 58))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1229))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 112))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 6))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 112))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1158))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 112))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 78))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 112))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1230))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 113))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 7))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 113))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1159))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 113))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 79))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 113))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1231))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 114))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 8))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 114))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1160))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 114))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 80))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 114))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1232))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 224))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 9))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 224))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1161))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 224))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 81))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 224))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1233))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 225))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 10))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 225))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1162))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 225))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 82))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 225))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1234))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 226))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 11))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 226))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1163))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 226))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 83))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 226))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1235))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 280))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 12))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 280))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1164))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 280))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 84))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 280))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1236))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 281))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 13))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 281))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1165))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 281))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 85))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 281))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1237))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 282))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 14))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 282))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1166))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 282))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 86))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 282))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1238))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 336))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 15))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 336))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1167))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 336))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 87))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 336))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1239))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 337))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 16))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 337))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1168))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 337))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 88))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 337))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1240))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 338))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 17))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 338))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1169))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 338))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 89))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 338))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1241))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 448))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 18))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 448))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1170))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 448))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 90))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 448))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1242))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 449))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 19))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 449))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1171))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 449))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 91))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 449))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1243))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 450))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 20))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 450))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1172))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 450))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 92))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 450))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1244))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 504))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 21))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 504))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1173))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 504))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 93))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 504))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1245))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 505))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 22))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 505))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1174))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 505))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 94))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 505))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1246))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 506))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 23))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 506))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1175))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 506))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 95))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 506))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1247))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 560))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 24))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 560))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1176))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 560))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 96))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 560))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1248))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 561))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 25))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 561))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1177))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 561))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 97))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 561))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1249))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 562))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 26))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 562))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1178))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 562))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 98))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 562))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1250))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 672))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 27))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 672))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1179))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 672))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 99))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 672))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1251))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 673))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 28))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 673))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1180))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 673))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 100))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 673))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1252))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 674))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 29))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 674))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1181))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 674))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 101))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 674))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1253))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 728))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 30))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 728))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1182))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 728))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 102))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 728))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1254))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 729))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 31))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 729))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1183))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 729))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 103))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 729))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1255))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 730))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 32))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 730))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1184))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 730))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 104))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 730))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1256))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 784))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 33))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 784))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1185))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 784))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 105))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 784))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1257))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 785))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 34))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 785))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1186))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 785))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 106))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 785))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1258))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 786))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 35))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 786))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1187))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 786))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 107))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 786))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1259))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 56))] * kernel_shared[((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 56))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1152))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 56))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 72))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 56))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1224))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 57))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 57))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1153))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 57))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 73))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 57))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1225))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 58))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 2))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 58))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1154))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 58))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 74))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 58))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1226))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 112))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 3))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 112))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1155))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 112))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 75))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 112))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1227))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 113))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 4))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 113))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1156))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 113))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 76))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 113))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1228))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 114))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 5))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 114))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1157))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 114))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 77))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 114))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1229))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 168))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 6))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 168))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1158))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 168))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 78))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 168))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1230))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 169))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 7))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 169))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1159))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 169))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 79))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 169))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1231))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 170))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 8))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 170))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1160))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 170))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 80))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 170))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1232))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 280))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 9))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 280))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1161))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 280))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 81))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 280))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1233))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 281))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 10))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 281))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1162))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 281))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 82))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 281))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1234))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 282))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 11))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 282))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1163))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 282))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 83))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 282))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1235))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 336))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 12))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 336))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1164))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 336))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 84))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 336))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1236))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 337))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 13))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 337))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1165))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 337))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 85))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 337))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1237))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 338))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 14))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 338))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1166))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 338))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 86))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 338))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1238))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 392))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 15))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 392))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1167))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 392))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 87))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 392))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1239))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 393))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 16))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 393))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1168))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 393))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 88))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 393))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1240))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 394))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 17))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 394))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1169))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 394))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 89))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 394))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1241))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 504))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 18))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 504))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1170))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 504))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 90))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 504))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1242))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 505))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 19))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 505))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1171))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 505))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 91))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 505))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1243))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 506))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 20))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 506))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1172))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 506))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 92))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 506))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1244))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 560))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 21))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 560))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1173))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 560))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 93))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 560))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1245))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 561))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 22))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 561))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1174))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 561))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 94))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 561))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1246))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 562))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 23))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 562))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1175))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 562))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 95))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 562))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1247))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 616))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 24))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 616))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1176))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 616))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 96))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 616))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1248))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 617))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 25))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 617))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1177))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 617))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 97))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 617))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1249))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 618))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 26))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 618))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1178))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 618))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 98))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 618))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1250))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 728))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 27))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 728))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1179))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 728))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 99))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 728))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1251))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 729))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 28))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 729))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1180))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 729))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 100))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 729))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1252))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 730))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 29))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 730))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1181))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 730))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 101))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 730))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1253))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 784))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 30))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 784))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1182))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 784))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 102))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 784))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1254))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 785))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 31))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 785))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1183))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 785))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 103))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 785))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1255))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 786))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 32))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 786))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1184))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 786))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 104))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 786))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1256))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 840))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 33))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 840))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1185))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 840))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 105))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 840))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1257))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 841))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 34))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 841))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1186))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 841))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 106))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 841))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1258))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 842))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 35))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 842))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1187))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 842))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 107))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((int)threadIdx.x) % 54)) + 842))] * kernel_shared[(((((((int)threadIdx.x) / 54) * 144) + (rc_outer_inner * 36)) + 1259))]));
    }
  }
  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
      compute[((((((((((int)blockIdx.x) / 27) * 93312) + ((((int)threadIdx.x) / 54) * 5832)) + (ff_inner * 2916)) + ((((int)blockIdx.x) % 27) * 108)) + (yy_inner * 54)) + (((int)threadIdx.x) % 54)))] = compute_local[(((ff_inner * 2) + yy_inner))];
      compute[(((((((((((int)blockIdx.x) / 27) * 93312) + ((((int)threadIdx.x) / 54) * 5832)) + (ff_inner * 2916)) + ((((int)blockIdx.x) % 27) * 108)) + (yy_inner * 54)) + (((int)threadIdx.x) % 54)) + 46656))] = compute_local[((((ff_inner * 2) + yy_inner) + 4))];
    }
  }
}



void conv_kernel_wrapper(int N_B, int N_C, int N_H, int N_W, int N_F, int N_R, int N_S, int PaddingH, int PaddingW,
                        int StrideH, int StrideW, int N_X, int N_Y, const float *Input, 
                        const float *Kernel, float *Output, int itr) {
                        hipEvent_t start;
                        CHECK(hipEventCreate(&start));
                        hipEvent_t stop;
                        CHECK(hipEventCreate(&stop));
                        float *dev_Input;
                        float *dev_Kernel;
                        float *dev_Output;
                    
                        CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                        CHECK(hipMemcpy(dev_Kernel, Kernel, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                        CHECK(hipMemcpy(dev_Input, Input, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Output, sizeof(float) * N_B * N_F * N_Y * N_X));
                        CHECK(hipMemset(dev_Output, 0, sizeof(float) * N_B * N_F * N_Y * N_X));
                    
dim3 size_grid(54,1,1);

dim3 size_block(432,1,1);
 
                       printf("warmup....\n");
                        default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                    
                        double *time_arr = (double *)malloc(itr * sizeof(double));
                        for (int j = 1; j < itr; j++) {
                            CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                            CHECK(hipMemcpy(dev_Kernel, Kernel+j*N_F * N_C * N_R * N_S, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                            CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                            CHECK(hipMemcpy(dev_Input, Input+j*N_B * N_C * N_H * N_W, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                            /* Record the start event*/
                            CHECK(hipEventRecord(start, NULL));
                            default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                            /* Record the end event */
                            CHECK(hipEventRecord(stop, NULL));
                            /* Wait for the stop event to complete*/
                            CHECK(hipEventSynchronize(stop));
                            float msec = 0.0f;
                            hipEventElapsedTime(&msec, start, stop);
                            time_arr[j] = msec;
                        }
                    
                       
                    
                        //hipError_t err = hipGetLastError();
                        //if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
                        double msecTotal = 0;
                        for (int j = 1; j < itr; j++) {   
                            printf("%0.5lf\n", time_arr[j]);
                            msecTotal += time_arr[j];
                        }
                    
                        double msecPerMatrixMul = (msecTotal) / itr;
                        double flopsPerMatrixMul = 2.0 * N_B * N_C * N_X * N_Y * N_F * N_R * N_S;
                    
                        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
                    
                        printf("\n===========================================================\n");
                        printf("Input size: %d X %d X %d X %d\n", N_B, N_C, N_H, N_W);
                        printf("Kernel size: %d X %d X %d X %d\n", N_F, N_C, N_R, N_S);
                        printf("Output size: %d X %d X %d X %d\n", N_B, N_F, N_Y, N_X);
                        printf("flops: %0.5lf\n", (flopsPerMatrixMul * 1.0e-9f));
                        printf("Time is %0.5lf \n", (msecPerMatrixMul / 1000.0f));
                        printf("Gflops %0.5lf \n", gigaFlops);
                        printf("===========================================================\n");
                    
                        /*write back to Host Output -- batch size is 1*/
                        CHECK(hipMemcpy(Output, dev_Output, sizeof(float) * N_B * N_F * N_X * N_Y, hipMemcpyDeviceToHost));
                    
                        CHECK(hipFree(dev_Input));
                        CHECK(hipFree(dev_Kernel));
                        CHECK(hipFree(dev_Output));
                    
                    }
            