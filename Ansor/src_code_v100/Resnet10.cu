#include "hip/hip_runtime.h"

            #include <cassert>
            #include <stdlib.h>
            #include <hip/hip_runtime.h>
            #include "common.h"
            

            
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[2];
  __shared__ float pad_temp_shared[360];
  __shared__ float kernel_shared[2304];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((((int)threadIdx.x) * 2))] = data[((((((rc_outer_outer * 2048) + (((((int)threadIdx.x) * 2) / 45) * 256)) + ((((int)blockIdx.x) % 7) * 32)) + ((((((int)threadIdx.x) * 2) % 45) / 15) * 16)) + ((((int)threadIdx.x) * 2) % 15)))];
    pad_temp_shared[(((((int)threadIdx.x) * 2) + 1))] = data[((((((rc_outer_outer * 2048) + ((((((int)threadIdx.x) * 2) + 1) / 45) * 256)) + ((((int)blockIdx.x) % 7) * 32)) + (((((((int)threadIdx.x) * 2) + 1) % 45) / 15) * 16)) + (((((int)threadIdx.x) * 2) + 1) % 15)))];
    if (((int)threadIdx.x) < 68) {
      pad_temp_shared[(((((int)threadIdx.x) * 2) + 224))] = data[((((((rc_outer_outer * 2048) + ((((((int)threadIdx.x) * 2) + 224) / 45) * 256)) + ((((int)blockIdx.x) % 7) * 32)) + (((((((int)threadIdx.x) * 2) + 44) % 45) / 15) * 16)) + (((((int)threadIdx.x) * 2) + 14) % 15)))];
    }
    if (((int)threadIdx.x) < 68) {
      pad_temp_shared[(((((int)threadIdx.x) * 2) + 225))] = data[(((((((rc_outer_outer * 2048) + (((((int)threadIdx.x) * 2) / 45) * 256)) + ((((int)blockIdx.x) % 7) * 32)) + ((((((int)threadIdx.x) * 2) % 45) / 15) * 16)) + ((((int)threadIdx.x) * 2) % 15)) + 1280))];
    }
    kernel_shared[((((int)threadIdx.x) * 3))] = kernel[((((((((int)blockIdx.x) / 7) * 73728) + ((((int)threadIdx.x) / 24) * 2304)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 24) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 3) + 1))] = kernel[(((((((((int)blockIdx.x) / 7) * 73728) + ((((int)threadIdx.x) / 24) * 2304)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 24) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 3) + 2))] = kernel[(((((((((int)blockIdx.x) / 7) * 73728) + ((((int)threadIdx.x) / 24) * 2304)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 24) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 3) + 336))] = kernel[((((((((int)blockIdx.x) / 7) * 73728) + (((((int)threadIdx.x) + 112) / 24) * 2304)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 16) % 24) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 3) + 337))] = kernel[(((((((((int)blockIdx.x) / 7) * 73728) + (((((int)threadIdx.x) + 112) / 24) * 2304)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 16) % 24) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 3) + 338))] = kernel[(((((((((int)blockIdx.x) / 7) * 73728) + (((((int)threadIdx.x) + 112) / 24) * 2304)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 16) % 24) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 3) + 672))] = kernel[((((((((int)blockIdx.x) / 7) * 73728) + (((((int)threadIdx.x) + 224) / 24) * 2304)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 8) % 24) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 3) + 673))] = kernel[(((((((((int)blockIdx.x) / 7) * 73728) + (((((int)threadIdx.x) + 224) / 24) * 2304)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 8) % 24) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 3) + 674))] = kernel[(((((((((int)blockIdx.x) / 7) * 73728) + (((((int)threadIdx.x) + 224) / 24) * 2304)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 8) % 24) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 3) + 1008))] = kernel[(((((((((int)blockIdx.x) / 7) * 73728) + ((((int)threadIdx.x) / 24) * 2304)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 24) * 3)) + 32256))];
    kernel_shared[(((((int)threadIdx.x) * 3) + 1009))] = kernel[(((((((((int)blockIdx.x) / 7) * 73728) + ((((int)threadIdx.x) / 24) * 2304)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 24) * 3)) + 32257))];
    kernel_shared[(((((int)threadIdx.x) * 3) + 1010))] = kernel[(((((((((int)blockIdx.x) / 7) * 73728) + ((((int)threadIdx.x) / 24) * 2304)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 24) * 3)) + 32258))];
    kernel_shared[(((((int)threadIdx.x) * 3) + 1344))] = kernel[((((((((int)blockIdx.x) / 7) * 73728) + (((((int)threadIdx.x) + 448) / 24) * 2304)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 16) % 24) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 3) + 1345))] = kernel[(((((((((int)blockIdx.x) / 7) * 73728) + (((((int)threadIdx.x) + 448) / 24) * 2304)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 16) % 24) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 3) + 1346))] = kernel[(((((((((int)blockIdx.x) / 7) * 73728) + (((((int)threadIdx.x) + 448) / 24) * 2304)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 16) % 24) * 3)) + 2))];
    kernel_shared[(((((int)threadIdx.x) * 3) + 1680))] = kernel[((((((((int)blockIdx.x) / 7) * 73728) + (((((int)threadIdx.x) + 560) / 24) * 2304)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 8) % 24) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 3) + 1681))] = kernel[(((((((((int)blockIdx.x) / 7) * 73728) + (((((int)threadIdx.x) + 560) / 24) * 2304)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 8) % 24) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 3) + 1682))] = kernel[(((((((((int)blockIdx.x) / 7) * 73728) + (((((int)threadIdx.x) + 560) / 24) * 2304)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 8) % 24) * 3)) + 2))];
    if (((int)threadIdx.x) < 96) {
      kernel_shared[(((((int)threadIdx.x) * 3) + 2016))] = kernel[(((((((((int)blockIdx.x) / 7) * 73728) + ((((int)threadIdx.x) / 24) * 2304)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 24) * 3)) + 64512))];
      kernel_shared[(((((int)threadIdx.x) * 3) + 2017))] = kernel[(((((((((int)blockIdx.x) / 7) * 73728) + ((((int)threadIdx.x) / 24) * 2304)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 24) * 3)) + 64513))];
      kernel_shared[(((((int)threadIdx.x) * 3) + 2018))] = kernel[(((((((((int)blockIdx.x) / 7) * 73728) + ((((int)threadIdx.x) / 24) * 2304)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 24) * 3)) + 64514))];
    }
    __syncthreads();
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 2))] * kernel_shared[(((((int)threadIdx.x) / 7) * 144))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 15))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 3))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 30))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 6))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 45))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 9))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 60))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 12))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 75))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 15))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 2))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 72))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 15))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 75))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 30))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 78))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 45))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 81))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 60))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 84))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 75))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 87))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 1))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 1))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 16))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 4))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 31))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 7))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 46))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 10))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 61))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 13))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 76))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 16))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 1))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 73))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 16))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 76))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 31))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 79))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 46))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 82))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 61))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 85))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 76))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 88))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 2))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 2))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 17))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 5))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 32))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 8))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 47))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 11))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 62))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 14))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 77))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 17))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 2))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 74))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 17))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 77))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 32))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 80))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 47))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 83))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 62))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 86))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 77))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 89))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 90))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 18))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 105))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 21))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 120))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 24))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 135))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 27))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 150))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 30))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 165))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 33))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 90))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 90))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 105))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 93))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 120))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 96))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 135))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 99))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 150))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 102))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 165))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 105))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 91))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 19))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 106))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 22))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 121))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 25))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 136))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 28))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 151))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 31))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 166))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 34))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 91))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 91))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 106))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 94))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 121))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 97))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 136))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 100))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 151))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 103))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 166))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 106))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 92))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 20))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 107))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 23))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 122))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 26))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 137))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 29))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 152))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 32))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 167))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 35))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 92))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 92))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 107))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 95))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 122))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 98))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 137))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 101))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 152))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 104))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 167))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 107))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 180))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 36))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 195))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 39))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 210))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 42))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 225))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 45))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 240))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 48))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 255))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 51))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 180))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 108))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 195))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 111))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 210))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 114))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 225))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 117))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 240))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 120))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 255))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 123))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 181))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 37))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 196))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 40))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 211))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 43))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 226))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 46))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 241))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 49))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 256))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 52))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 181))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 109))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 196))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 112))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 211))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 115))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 226))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 118))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 241))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 121))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 256))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 124))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 182))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 38))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 197))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 41))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 212))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 44))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 227))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 47))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 242))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 50))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 257))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 53))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 182))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 110))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 197))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 113))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 212))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 116))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 227))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 119))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 242))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 122))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 257))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 125))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 270))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 54))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 285))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 57))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 300))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 60))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 315))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 63))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 330))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 66))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 345))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 69))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 270))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 126))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 285))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 129))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 300))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 132))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 315))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 135))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 330))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 138))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 345))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 141))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 271))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 55))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 286))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 58))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 301))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 61))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 316))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 64))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 331))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 67))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 346))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 70))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 271))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 127))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 286))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 130))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 301))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 133))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 316))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 136))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 331))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 139))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 346))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 142))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 272))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 56))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 287))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 59))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 302))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 62))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 317))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 65))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 332))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 68))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 347))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 71))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 272))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 128))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 287))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 131))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 302))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 134))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 317))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 137))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 332))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 140))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 347))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + 143))]));
  }
  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
    compute[(((((((((int)blockIdx.x) / 7) * 1568) + ((((int)threadIdx.x) / 7) * 98)) + (ff_inner * 49)) + ((((int)blockIdx.x) % 7) * 7)) + (((int)threadIdx.x) % 7)))] = compute_local[(ff_inner)];
  }
}



void conv_kernel_wrapper(int N_B, int N_C, int N_H, int N_W, int N_F, int N_R, int N_S, int PaddingH, int PaddingW,
                        int StrideH, int StrideW, int N_X, int N_Y, const float *Input, 
                        const float *Kernel, float *Output, int itr) {
                        hipEvent_t start;
                        CHECK(hipEventCreate(&start));
                        hipEvent_t stop;
                        CHECK(hipEventCreate(&stop));
                        float *dev_Input;
                        float *dev_Kernel;
                        float *dev_Output;
                    
                        CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                        CHECK(hipMemcpy(dev_Kernel, Kernel, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                        CHECK(hipMemcpy(dev_Input, Input, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Output, sizeof(float) * N_B * N_F * N_Y * N_X));
                        CHECK(hipMemset(dev_Output, 0, sizeof(float) * N_B * N_F * N_Y * N_X));
                    
dim3 size_grid(112,1,1);

dim3 size_block(112,1,1);
 
                       printf("warmup....\n");
                        default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                    
                        double *time_arr = (double *)malloc(itr * sizeof(double));
                        for (int j = 1; j < itr; j++) {
                            CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                            CHECK(hipMemcpy(dev_Kernel, Kernel+j*N_F * N_C * N_R * N_S, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                            CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                            CHECK(hipMemcpy(dev_Input, Input+j*N_B * N_C * N_H * N_W, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                            /* Record the start event*/
                            CHECK(hipEventRecord(start, NULL));
                            default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                            /* Record the end event */
                            CHECK(hipEventRecord(stop, NULL));
                            /* Wait for the stop event to complete*/
                            CHECK(hipEventSynchronize(stop));
                            float msec = 0.0f;
                            hipEventElapsedTime(&msec, start, stop);
                            time_arr[j] = msec;
                        }
                    
                       
                    
                        //hipError_t err = hipGetLastError();
                        //if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
                        double msecTotal = 0;
                        for (int j = 1; j < itr; j++) {   
                            printf("%0.5lf\n", time_arr[j]);
                            msecTotal += time_arr[j];
                        }
                    
                        double msecPerMatrixMul = (msecTotal) / itr;
                        double flopsPerMatrixMul = 2.0 * N_B * N_C * N_X * N_Y * N_F * N_R * N_S;
                    
                        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
                    
                        printf("\n===========================================================\n");
                        printf("Input size: %d X %d X %d X %d\n", N_B, N_C, N_H, N_W);
                        printf("Kernel size: %d X %d X %d X %d\n", N_F, N_C, N_R, N_S);
                        printf("Output size: %d X %d X %d X %d\n", N_B, N_F, N_Y, N_X);
                        printf("flops: %0.5lf\n", (flopsPerMatrixMul * 1.0e-9f));
                        printf("Time is %0.5lf \n", (msecPerMatrixMul / 1000.0f));
                        printf("Gflops %0.5lf \n", gigaFlops);
                        printf("===========================================================\n");
                    
                        /*write back to Host Output -- batch size is 1*/
                        CHECK(hipMemcpy(Output, dev_Output, sizeof(float) * N_B * N_F * N_X * N_Y, hipMemcpyDeviceToHost));
                    
                        CHECK(hipFree(dev_Input));
                        CHECK(hipFree(dev_Kernel));
                        CHECK(hipFree(dev_Output));
                    
                    }
            