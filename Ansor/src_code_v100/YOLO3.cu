#include "hip/hip_runtime.h"

            #include <cassert>
            #include <stdlib.h>
            #include <hip/hip_runtime.h>
            #include "common.h"
            

            
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[32];
  __shared__ float pad_temp_shared[4352];
  __shared__ float kernel_shared[2048];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  compute_local[(7)] = 0.000000e+00f;
  compute_local[(8)] = 0.000000e+00f;
  compute_local[(9)] = 0.000000e+00f;
  compute_local[(10)] = 0.000000e+00f;
  compute_local[(11)] = 0.000000e+00f;
  compute_local[(12)] = 0.000000e+00f;
  compute_local[(13)] = 0.000000e+00f;
  compute_local[(14)] = 0.000000e+00f;
  compute_local[(15)] = 0.000000e+00f;
  compute_local[(16)] = 0.000000e+00f;
  compute_local[(17)] = 0.000000e+00f;
  compute_local[(18)] = 0.000000e+00f;
  compute_local[(19)] = 0.000000e+00f;
  compute_local[(20)] = 0.000000e+00f;
  compute_local[(21)] = 0.000000e+00f;
  compute_local[(22)] = 0.000000e+00f;
  compute_local[(23)] = 0.000000e+00f;
  compute_local[(24)] = 0.000000e+00f;
  compute_local[(25)] = 0.000000e+00f;
  compute_local[(26)] = 0.000000e+00f;
  compute_local[(27)] = 0.000000e+00f;
  compute_local[(28)] = 0.000000e+00f;
  compute_local[(29)] = 0.000000e+00f;
  compute_local[(30)] = 0.000000e+00f;
  compute_local[(31)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[(((int)threadIdx.x))] = data[(((((((rc_outer_outer * 591872) + ((((int)threadIdx.x) / 136) * 18496)) + ((((int)blockIdx.x) >> 2) * 544)) + (((((int)threadIdx.x) % 136) / 34) * 136)) + ((((int)blockIdx.x) & 3) * 34)) + (((int)threadIdx.x) % 34)))];
    pad_temp_shared[((((int)threadIdx.x) + 272))] = data[((((((((rc_outer_outer * 591872) + ((((int)threadIdx.x) / 136) * 18496)) + ((((int)blockIdx.x) >> 2) * 544)) + (((((int)threadIdx.x) % 136) / 34) * 136)) + ((((int)blockIdx.x) & 3) * 34)) + (((int)threadIdx.x) % 34)) + 36992))];
    pad_temp_shared[((((int)threadIdx.x) + 544))] = data[((((((((rc_outer_outer * 591872) + ((((int)threadIdx.x) / 136) * 18496)) + ((((int)blockIdx.x) >> 2) * 544)) + (((((int)threadIdx.x) % 136) / 34) * 136)) + ((((int)blockIdx.x) & 3) * 34)) + (((int)threadIdx.x) % 34)) + 73984))];
    pad_temp_shared[((((int)threadIdx.x) + 816))] = data[((((((((rc_outer_outer * 591872) + ((((int)threadIdx.x) / 136) * 18496)) + ((((int)blockIdx.x) >> 2) * 544)) + (((((int)threadIdx.x) % 136) / 34) * 136)) + ((((int)blockIdx.x) & 3) * 34)) + (((int)threadIdx.x) % 34)) + 110976))];
    pad_temp_shared[((((int)threadIdx.x) + 1088))] = data[((((((((rc_outer_outer * 591872) + ((((int)threadIdx.x) / 136) * 18496)) + ((((int)blockIdx.x) >> 2) * 544)) + (((((int)threadIdx.x) % 136) / 34) * 136)) + ((((int)blockIdx.x) & 3) * 34)) + (((int)threadIdx.x) % 34)) + 147968))];
    pad_temp_shared[((((int)threadIdx.x) + 1360))] = data[((((((((rc_outer_outer * 591872) + ((((int)threadIdx.x) / 136) * 18496)) + ((((int)blockIdx.x) >> 2) * 544)) + (((((int)threadIdx.x) % 136) / 34) * 136)) + ((((int)blockIdx.x) & 3) * 34)) + (((int)threadIdx.x) % 34)) + 184960))];
    pad_temp_shared[((((int)threadIdx.x) + 1632))] = data[((((((((rc_outer_outer * 591872) + ((((int)threadIdx.x) / 136) * 18496)) + ((((int)blockIdx.x) >> 2) * 544)) + (((((int)threadIdx.x) % 136) / 34) * 136)) + ((((int)blockIdx.x) & 3) * 34)) + (((int)threadIdx.x) % 34)) + 221952))];
    pad_temp_shared[((((int)threadIdx.x) + 1904))] = data[((((((((rc_outer_outer * 591872) + ((((int)threadIdx.x) / 136) * 18496)) + ((((int)blockIdx.x) >> 2) * 544)) + (((((int)threadIdx.x) % 136) / 34) * 136)) + ((((int)blockIdx.x) & 3) * 34)) + (((int)threadIdx.x) % 34)) + 258944))];
    pad_temp_shared[((((int)threadIdx.x) + 2176))] = data[((((((((rc_outer_outer * 591872) + ((((int)threadIdx.x) / 136) * 18496)) + ((((int)blockIdx.x) >> 2) * 544)) + (((((int)threadIdx.x) % 136) / 34) * 136)) + ((((int)blockIdx.x) & 3) * 34)) + (((int)threadIdx.x) % 34)) + 295936))];
    pad_temp_shared[((((int)threadIdx.x) + 2448))] = data[((((((((rc_outer_outer * 591872) + ((((int)threadIdx.x) / 136) * 18496)) + ((((int)blockIdx.x) >> 2) * 544)) + (((((int)threadIdx.x) % 136) / 34) * 136)) + ((((int)blockIdx.x) & 3) * 34)) + (((int)threadIdx.x) % 34)) + 332928))];
    pad_temp_shared[((((int)threadIdx.x) + 2720))] = data[((((((((rc_outer_outer * 591872) + ((((int)threadIdx.x) / 136) * 18496)) + ((((int)blockIdx.x) >> 2) * 544)) + (((((int)threadIdx.x) % 136) / 34) * 136)) + ((((int)blockIdx.x) & 3) * 34)) + (((int)threadIdx.x) % 34)) + 369920))];
    pad_temp_shared[((((int)threadIdx.x) + 2992))] = data[((((((((rc_outer_outer * 591872) + ((((int)threadIdx.x) / 136) * 18496)) + ((((int)blockIdx.x) >> 2) * 544)) + (((((int)threadIdx.x) % 136) / 34) * 136)) + ((((int)blockIdx.x) & 3) * 34)) + (((int)threadIdx.x) % 34)) + 406912))];
    pad_temp_shared[((((int)threadIdx.x) + 3264))] = data[((((((((rc_outer_outer * 591872) + ((((int)threadIdx.x) / 136) * 18496)) + ((((int)blockIdx.x) >> 2) * 544)) + (((((int)threadIdx.x) % 136) / 34) * 136)) + ((((int)blockIdx.x) & 3) * 34)) + (((int)threadIdx.x) % 34)) + 443904))];
    pad_temp_shared[((((int)threadIdx.x) + 3536))] = data[((((((((rc_outer_outer * 591872) + ((((int)threadIdx.x) / 136) * 18496)) + ((((int)blockIdx.x) >> 2) * 544)) + (((((int)threadIdx.x) % 136) / 34) * 136)) + ((((int)blockIdx.x) & 3) * 34)) + (((int)threadIdx.x) % 34)) + 480896))];
    pad_temp_shared[((((int)threadIdx.x) + 3808))] = data[((((((((rc_outer_outer * 591872) + ((((int)threadIdx.x) / 136) * 18496)) + ((((int)blockIdx.x) >> 2) * 544)) + (((((int)threadIdx.x) % 136) / 34) * 136)) + ((((int)blockIdx.x) & 3) * 34)) + (((int)threadIdx.x) % 34)) + 517888))];
    pad_temp_shared[((((int)threadIdx.x) + 4080))] = data[((((((((rc_outer_outer * 591872) + ((((int)threadIdx.x) / 136) * 18496)) + ((((int)blockIdx.x) >> 2) * 544)) + (((((int)threadIdx.x) % 136) / 34) * 136)) + ((((int)blockIdx.x) & 3) * 34)) + (((int)threadIdx.x) % 34)) + 554880))];
    kernel_shared[((((int)threadIdx.x) * 2))] = kernel[(((((((int)threadIdx.x) >> 4) * 128) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)))];
    kernel_shared[(((((int)threadIdx.x) * 2) + 1))] = kernel[(((((((((int)threadIdx.x) * 2) + 1) >> 5) * 128) + (rc_outer_outer * 32)) + (((((int)threadIdx.x) * 2) + 1) & 31)))];
    kernel_shared[(((((int)threadIdx.x) * 2) + 544))] = kernel[((((((((int)threadIdx.x) >> 4) * 128) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 2176))];
    kernel_shared[(((((int)threadIdx.x) * 2) + 545))] = kernel[((((((((((int)threadIdx.x) * 2) + 1) >> 5) * 128) + (rc_outer_outer * 32)) + (((((int)threadIdx.x) * 2) + 1) & 31)) + 2176))];
    kernel_shared[(((((int)threadIdx.x) * 2) + 1088))] = kernel[((((((((int)threadIdx.x) >> 4) * 128) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 4352))];
    kernel_shared[(((((int)threadIdx.x) * 2) + 1089))] = kernel[((((((((((int)threadIdx.x) * 2) + 1) >> 5) * 128) + (rc_outer_outer * 32)) + (((((int)threadIdx.x) * 2) + 1) & 31)) + 4352))];
    if (((int)threadIdx.x) < 208) {
      kernel_shared[(((((int)threadIdx.x) * 2) + 1632))] = kernel[((((((((int)threadIdx.x) >> 4) * 128) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 6528))];
    }
    if (((int)threadIdx.x) < 208) {
      kernel_shared[(((((int)threadIdx.x) * 2) + 1633))] = kernel[((((((((((int)threadIdx.x) * 2) + 1) >> 5) * 128) + (rc_outer_outer * 32)) + (((((int)threadIdx.x) * 2) + 1) & 31)) + 6528))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)))] * kernel_shared[((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 34))] * kernel_shared[((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 68))] * kernel_shared[((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 102))] * kernel_shared[((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 32))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 34))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 32))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 68))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 32))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 102))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 32))]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 64))]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 34))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 64))]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 68))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 64))]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 102))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 64))]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 96))]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 34))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 96))]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 68))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 96))]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 102))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 96))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 136))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 1))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 170))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 1))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 204))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 1))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 238))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 1))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 136))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 33))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 170))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 33))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 204))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 33))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 238))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 33))]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 136))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 65))]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 170))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 65))]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 204))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 65))]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 238))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 65))]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 136))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 97))]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 170))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 97))]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 204))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 97))]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 238))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 97))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 272))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 2))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 306))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 2))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 340))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 2))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 374))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 2))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 272))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 34))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 306))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 34))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 340))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 34))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 374))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 34))]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 272))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 66))]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 306))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 66))]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 340))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 66))]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 374))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 66))]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 272))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 98))]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 306))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 98))]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 340))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 98))]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 374))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 98))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 408))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 3))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 442))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 3))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 476))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 3))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 510))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 3))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 408))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 35))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 442))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 35))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 476))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 35))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 510))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 35))]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 408))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 67))]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 442))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 67))]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 476))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 67))]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 510))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 67))]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 408))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 99))]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 442))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 99))]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 476))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 99))]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 510))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 99))]));
      compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 128))]));
      compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 34))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 128))]));
      compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 68))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 128))]));
      compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 102))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 128))]));
      compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[(((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 160))]));
      compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 34))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 160))]));
      compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 68))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 160))]));
      compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 102))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 160))]));
      compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[(((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 192))]));
      compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 34))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 192))]));
      compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 68))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 192))]));
      compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 102))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 192))]));
      compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[(((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 224))]));
      compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 34))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 224))]));
      compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 68))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 224))]));
      compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 102))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 224))]));
      compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 136))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 129))]));
      compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 170))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 129))]));
      compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 204))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 129))]));
      compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 238))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 129))]));
      compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 136))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 161))]));
      compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 170))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 161))]));
      compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 204))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 161))]));
      compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 238))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 161))]));
      compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 136))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 193))]));
      compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 170))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 193))]));
      compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 204))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 193))]));
      compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 238))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 193))]));
      compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 136))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 225))]));
      compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 170))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 225))]));
      compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 204))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 225))]));
      compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 238))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 225))]));
      compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 272))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 130))]));
      compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 306))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 130))]));
      compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 340))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 130))]));
      compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 374))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 130))]));
      compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 272))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 162))]));
      compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 306))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 162))]));
      compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 340))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 162))]));
      compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 374))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 162))]));
      compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 272))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 194))]));
      compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 306))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 194))]));
      compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 340))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 194))]));
      compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 374))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 194))]));
      compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 272))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 226))]));
      compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 306))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 226))]));
      compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 340))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 226))]));
      compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 374))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 226))]));
      compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 408))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 131))]));
      compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 442))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 131))]));
      compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 476))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 131))]));
      compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 510))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 131))]));
      compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 408))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 163))]));
      compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 442))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 163))]));
      compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 476))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 163))]));
      compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 510))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 163))]));
      compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 408))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 195))]));
      compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 442))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 195))]));
      compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 476))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 195))]));
      compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 510))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 195))]));
      compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 408))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 227))]));
      compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 442))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 227))]));
      compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 476))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 227))]));
      compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[((((rc_outer_inner * 544) + (((int)threadIdx.x) % 34)) + 510))] * kernel_shared[(((((((int)threadIdx.x) / 34) * 256) + (rc_outer_inner * 4)) + 227))]));
    }
  }
  for (int ff_inner = 0; ff_inner < 8; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 4; ++yy_inner) {
      compute[((((((((((int)threadIdx.x) / 34) * 147968) + (ff_inner * 18496)) + ((((int)blockIdx.x) >> 2) * 544)) + (yy_inner * 136)) + ((((int)blockIdx.x) & 3) * 34)) + (((int)threadIdx.x) % 34)))] = compute_local[(((ff_inner * 4) + yy_inner))];
    }
  }
}



void conv_kernel_wrapper(int N_B, int N_C, int N_H, int N_W, int N_F, int N_R, int N_S, int PaddingH, int PaddingW,
                        int StrideH, int StrideW, int N_X, int N_Y, const float *Input, 
                        const float *Kernel, float *Output, int itr) {
                        hipEvent_t start;
                        CHECK(hipEventCreate(&start));
                        hipEvent_t stop;
                        CHECK(hipEventCreate(&stop));
                        float *dev_Input;
                        float *dev_Kernel;
                        float *dev_Output;
                    
                        CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                        CHECK(hipMemcpy(dev_Kernel, Kernel, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                        CHECK(hipMemcpy(dev_Input, Input, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Output, sizeof(float) * N_B * N_F * N_Y * N_X));
                        CHECK(hipMemset(dev_Output, 0, sizeof(float) * N_B * N_F * N_Y * N_X));
                    
dim3 size_grid(136,1,1);

dim3 size_block(272,1,1);
 
                       printf("warmup....\n");
                        default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                    
                        double *time_arr = (double *)malloc(itr * sizeof(double));
                        for (int j = 1; j < itr; j++) {
                            CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                            CHECK(hipMemcpy(dev_Kernel, Kernel+j*N_F * N_C * N_R * N_S, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                            CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                            CHECK(hipMemcpy(dev_Input, Input+j*N_B * N_C * N_H * N_W, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                            /* Record the start event*/
                            CHECK(hipEventRecord(start, NULL));
                            default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                            /* Record the end event */
                            CHECK(hipEventRecord(stop, NULL));
                            /* Wait for the stop event to complete*/
                            CHECK(hipEventSynchronize(stop));
                            float msec = 0.0f;
                            hipEventElapsedTime(&msec, start, stop);
                            time_arr[j] = msec;
                        }
                    
                       
                    
                        //hipError_t err = hipGetLastError();
                        //if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
                        double msecTotal = 0;
                        for (int j = 1; j < itr; j++) {   
                            printf("%0.5lf\n", time_arr[j]);
                            msecTotal += time_arr[j];
                        }
                    
                        double msecPerMatrixMul = (msecTotal) / itr;
                        double flopsPerMatrixMul = 2.0 * N_B * N_C * N_X * N_Y * N_F * N_R * N_S;
                    
                        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
                    
                        printf("\n===========================================================\n");
                        printf("Input size: %d X %d X %d X %d\n", N_B, N_C, N_H, N_W);
                        printf("Kernel size: %d X %d X %d X %d\n", N_F, N_C, N_R, N_S);
                        printf("Output size: %d X %d X %d X %d\n", N_B, N_F, N_Y, N_X);
                        printf("flops: %0.5lf\n", (flopsPerMatrixMul * 1.0e-9f));
                        printf("Time is %0.5lf \n", (msecPerMatrixMul / 1000.0f));
                        printf("Gflops %0.5lf \n", gigaFlops);
                        printf("===========================================================\n");
                    
                        /*write back to Host Output -- batch size is 1*/
                        CHECK(hipMemcpy(Output, dev_Output, sizeof(float) * N_B * N_F * N_X * N_Y, hipMemcpyDeviceToHost));
                    
                        CHECK(hipFree(dev_Input));
                        CHECK(hipFree(dev_Kernel));
                        CHECK(hipFree(dev_Output));
                    
                    }
            