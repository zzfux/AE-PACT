#include "hip/hip_runtime.h"

            #include <cassert>
            #include <stdlib.h>
            #include <hip/hip_runtime.h>
            #include "common.h"
            

            
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[8];
  __shared__ float pad_temp_shared[3584];
  __shared__ float kernel_shared[512];
  for (int ff_c_outer_inner_init = 0; ff_c_outer_inner_init < 2; ++ff_c_outer_inner_init) {
    for (int ff_c_inner_init = 0; ff_c_inner_init < 2; ++ff_c_inner_init) {
      compute_local[(((ff_c_outer_inner_init * 2) + ff_c_inner_init))] = 0.000000e+00f;
      compute_local[((((ff_c_outer_inner_init * 2) + ff_c_inner_init) + 4))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 2; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 8; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
        ((float2*)(pad_temp_shared + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 2)))))[0] = ((float2*)(data + ((((((rc_outer_outer * 100352) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12544)) + ((((int)threadIdx.x) / 56) * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) % 56) * 2)))))[0];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 7) + (((int)threadIdx.x) >> 5)) < 16) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 224) + ((int)threadIdx.x)) < 512) {
          if (((((((int)blockIdx.x) / 28) * 16) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 7)) + (((int)threadIdx.x) >> 5)) < 64) {
            kernel_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 224) + ((int)threadIdx.x)))] = kernel[(((((((((int)blockIdx.x) / 28) * 1024) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 448)) + ((((int)threadIdx.x) >> 5) * 64)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)))];
          }
        }
      }
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int ff_c_outer_inner = 0; ff_c_outer_inner < 2; ++ff_c_outer_inner) {
        for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
          for (int ff_c_inner = 0; ff_c_inner < 2; ++ff_c_inner) {
            compute_local[(((ff_c_outer_inner * 2) + ff_c_inner))] = (compute_local[(((ff_c_outer_inner * 2) + ff_c_inner))] + (pad_temp_shared[((((rc_outer_inner * 1792) + (rc_inner * 112)) + (((int)threadIdx.x) % 112)))] * kernel_shared[(((((((((int)threadIdx.x) / 112) * 128) + (ff_c_outer_inner * 64)) + (ff_c_inner * 32)) + (rc_outer_inner * 16)) + rc_inner))]));
            compute_local[((((ff_c_outer_inner * 2) + ff_c_inner) + 4))] = (compute_local[((((ff_c_outer_inner * 2) + ff_c_inner) + 4))] + (pad_temp_shared[((((rc_outer_inner * 1792) + (rc_inner * 112)) + (((int)threadIdx.x) % 112)))] * kernel_shared[((((((((((int)threadIdx.x) / 112) * 128) + (ff_c_outer_inner * 64)) + (ff_c_inner * 32)) + (rc_outer_inner * 16)) + rc_inner) + 256))]));
          }
        }
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
    compute[(((((((((int)blockIdx.x) / 28) * 50176) + ((((int)threadIdx.x) / 112) * 12544)) + (ff_inner * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + (((int)threadIdx.x) % 112)))] = compute_local[(ff_inner)];
    compute[((((((((((int)blockIdx.x) / 28) * 50176) + ((((int)threadIdx.x) / 112) * 12544)) + (ff_inner * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + (((int)threadIdx.x) % 112)) + 25088))] = compute_local[((ff_inner + 4))];
  }
}



void conv_kernel_wrapper(int N_B, int N_C, int N_H, int N_W, int N_F, int N_R, int N_S, int PaddingH, int PaddingW,
                        int StrideH, int StrideW, int N_X, int N_Y, const float *Input, 
                        const float *Kernel, float *Output, int itr) {
                        hipEvent_t start;
                        CHECK(hipEventCreate(&start));
                        hipEvent_t stop;
                        CHECK(hipEventCreate(&stop));
                        float *dev_Input;
                        float *dev_Kernel;
                        float *dev_Output;
                    
                        CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                        CHECK(hipMemcpy(dev_Kernel, Kernel, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                        CHECK(hipMemcpy(dev_Input, Input, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Output, sizeof(float) * N_B * N_F * N_Y * N_X));
                        CHECK(hipMemset(dev_Output, 0, sizeof(float) * N_B * N_F * N_Y * N_X));
                    
dim3 size_grid(112,1,1);

dim3 size_block(224,1,1);
 
                       printf("warmup....\n");
                        default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                    
                        double *time_arr = (double *)malloc(itr * sizeof(double));
                        for (int j = 1; j < itr; j++) {
                            CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                            CHECK(hipMemcpy(dev_Kernel, Kernel+j*N_F * N_C * N_R * N_S, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                            CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                            CHECK(hipMemcpy(dev_Input, Input+j*N_B * N_C * N_H * N_W, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                            /* Record the start event*/
                            CHECK(hipEventRecord(start, NULL));
                            default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                            /* Record the end event */
                            CHECK(hipEventRecord(stop, NULL));
                            /* Wait for the stop event to complete*/
                            CHECK(hipEventSynchronize(stop));
                            float msec = 0.0f;
                            hipEventElapsedTime(&msec, start, stop);
                            time_arr[j] = msec;
                        }
                    
                       
                    
                        //hipError_t err = hipGetLastError();
                        //if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
                        double msecTotal = 0;
                        for (int j = 1; j < itr; j++) {   
                            printf("%0.5lf\n", time_arr[j]);
                            msecTotal += time_arr[j];
                        }
                    
                        double msecPerMatrixMul = (msecTotal) / itr;
                        double flopsPerMatrixMul = 2.0 * N_B * N_C * N_X * N_Y * N_F * N_R * N_S;
                    
                        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
                    
                        printf("\n===========================================================\n");
                        printf("Input size: %d X %d X %d X %d\n", N_B, N_C, N_H, N_W);
                        printf("Kernel size: %d X %d X %d X %d\n", N_F, N_C, N_R, N_S);
                        printf("Output size: %d X %d X %d X %d\n", N_B, N_F, N_Y, N_X);
                        printf("flops: %0.5lf\n", (flopsPerMatrixMul * 1.0e-9f));
                        printf("Time is %0.5lf \n", (msecPerMatrixMul / 1000.0f));
                        printf("Gflops %0.5lf \n", gigaFlops);
                        printf("===========================================================\n");
                    
                        /*write back to Host Output -- batch size is 1*/
                        CHECK(hipMemcpy(Output, dev_Output, sizeof(float) * N_B * N_F * N_X * N_Y, hipMemcpyDeviceToHost));
                    
                        CHECK(hipFree(dev_Input));
                        CHECK(hipFree(dev_Kernel));
                        CHECK(hipFree(dev_Output));
                    
                    }
            