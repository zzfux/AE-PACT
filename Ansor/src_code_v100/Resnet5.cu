#include "hip/hip_runtime.h"

            #include <cassert>
            #include <stdlib.h>
            #include <hip/hip_runtime.h>
            #include "common.h"
            

            
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[4];
  __shared__ float pad_temp_shared[112];
  __shared__ float kernel_shared[1024];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[(((int)threadIdx.x))] = data[((((((rc_outer_outer * 50176) + ((((int)threadIdx.x) / 7) * 3136)) + (((((int)blockIdx.x) % 196) / 7) * 112)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) % 7)))];
    if (((int)threadIdx.x) < 48) {
      pad_temp_shared[((((int)threadIdx.x) + 64))] = data[((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 64) / 7) * 3136)) + (((((int)blockIdx.x) % 196) / 7) * 112)) + ((((int)blockIdx.x) % 7) * 8)) + ((((int)threadIdx.x) + 1) % 7)))];
    }
      ((float4*)(kernel_shared + ((((int)threadIdx.x) * 4))))[0] = ((float4*)(kernel + ((((((((int)blockIdx.x) / 196) * 4096) + ((((int)threadIdx.x) >> 2) * 64)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 3) * 4)))))[0];
      ((float4*)(kernel_shared + (((((int)threadIdx.x) * 4) + 256))))[0] = ((float4*)(kernel + (((((((((int)blockIdx.x) / 196) * 4096) + ((((int)threadIdx.x) >> 2) * 64)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 1024))))[0];
      ((float4*)(kernel_shared + (((((int)threadIdx.x) * 4) + 512))))[0] = ((float4*)(kernel + (((((((((int)blockIdx.x) / 196) * 4096) + ((((int)threadIdx.x) >> 2) * 64)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 2048))))[0];
      ((float4*)(kernel_shared + (((((int)threadIdx.x) * 4) + 768))))[0] = ((float4*)(kernel + (((((((((int)blockIdx.x) / 196) * 4096) + ((((int)threadIdx.x) >> 2) * 64)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 3072))))[0];
    __syncthreads();
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) & 3) * 2))] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) & 3) * 2))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 256))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) & 3) * 2))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 512))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) & 3) * 2))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 768))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 7))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 1))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 7))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 257))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 7))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 513))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 7))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 769))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 14))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 2))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 14))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 258))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 14))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 514))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 14))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 770))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 21))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 3))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 21))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 259))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 21))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 515))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 21))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 771))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 28))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 4))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 28))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 260))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 28))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 516))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 28))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 772))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 35))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 5))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 35))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 261))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 35))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 517))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 35))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 773))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 42))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 6))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 42))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 262))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 42))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 518))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 42))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 774))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 49))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 7))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 49))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 263))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 49))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 519))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 49))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 775))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 56))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 8))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 56))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 264))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 56))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 520))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 56))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 776))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 63))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 9))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 63))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 265))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 63))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 521))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 63))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 777))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 70))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 10))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 70))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 266))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 70))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 522))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 70))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 778))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 77))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 11))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 77))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 267))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 77))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 523))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 77))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 779))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 84))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 12))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 84))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 268))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 84))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 524))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 84))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 780))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 91))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 13))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 91))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 269))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 91))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 525))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 91))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 781))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 98))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 14))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 98))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 270))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 98))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 526))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 98))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 782))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 105))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 15))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 105))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 271))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 105))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 527))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((((int)threadIdx.x) & 3) * 2) + 105))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 16) + 783))]));
  }
  compute[((((((((int)blockIdx.x) / 196) * 50176) + ((((int)threadIdx.x) >> 2) * 784)) + ((((int)blockIdx.x) % 196) * 4)) + (((int)threadIdx.x) & 3)))] = compute_local[(0)];
  compute[(((((((((int)blockIdx.x) / 196) * 50176) + ((((int)threadIdx.x) >> 2) * 784)) + ((((int)blockIdx.x) % 196) * 4)) + (((int)threadIdx.x) & 3)) + 12544))] = compute_local[(1)];
  compute[(((((((((int)blockIdx.x) / 196) * 50176) + ((((int)threadIdx.x) >> 2) * 784)) + ((((int)blockIdx.x) % 196) * 4)) + (((int)threadIdx.x) & 3)) + 25088))] = compute_local[(2)];
  compute[(((((((((int)blockIdx.x) / 196) * 50176) + ((((int)threadIdx.x) >> 2) * 784)) + ((((int)blockIdx.x) % 196) * 4)) + (((int)threadIdx.x) & 3)) + 37632))] = compute_local[(3)];
}



void conv_kernel_wrapper(int N_B, int N_C, int N_H, int N_W, int N_F, int N_R, int N_S, int PaddingH, int PaddingW,
                        int StrideH, int StrideW, int N_X, int N_Y, const float *Input, 
                        const float *Kernel, float *Output, int itr) {
                        hipEvent_t start;
                        CHECK(hipEventCreate(&start));
                        hipEvent_t stop;
                        CHECK(hipEventCreate(&stop));
                        float *dev_Input;
                        float *dev_Kernel;
                        float *dev_Output;
                    
                        CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                        CHECK(hipMemcpy(dev_Kernel, Kernel, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                        CHECK(hipMemcpy(dev_Input, Input, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Output, sizeof(float) * N_B * N_F * N_Y * N_X));
                        CHECK(hipMemset(dev_Output, 0, sizeof(float) * N_B * N_F * N_Y * N_X));
                    
dim3 size_grid(392,1,1);

dim3 size_block(64,1,1);
 
                       printf("warmup....\n");
                        default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                    
                        double *time_arr = (double *)malloc(itr * sizeof(double));
                        for (int j = 1; j < itr; j++) {
                            CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                            CHECK(hipMemcpy(dev_Kernel, Kernel+j*N_F * N_C * N_R * N_S, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                            CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                            CHECK(hipMemcpy(dev_Input, Input+j*N_B * N_C * N_H * N_W, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                            /* Record the start event*/
                            CHECK(hipEventRecord(start, NULL));
                            default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                            /* Record the end event */
                            CHECK(hipEventRecord(stop, NULL));
                            /* Wait for the stop event to complete*/
                            CHECK(hipEventSynchronize(stop));
                            float msec = 0.0f;
                            hipEventElapsedTime(&msec, start, stop);
                            time_arr[j] = msec;
                        }
                    
                       
                    
                        //hipError_t err = hipGetLastError();
                        //if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
                        double msecTotal = 0;
                        for (int j = 1; j < itr; j++) {   
                            printf("%0.5lf\n", time_arr[j]);
                            msecTotal += time_arr[j];
                        }
                    
                        double msecPerMatrixMul = (msecTotal) / itr;
                        double flopsPerMatrixMul = 2.0 * N_B * N_C * N_X * N_Y * N_F * N_R * N_S;
                    
                        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
                    
                        printf("\n===========================================================\n");
                        printf("Input size: %d X %d X %d X %d\n", N_B, N_C, N_H, N_W);
                        printf("Kernel size: %d X %d X %d X %d\n", N_F, N_C, N_R, N_S);
                        printf("Output size: %d X %d X %d X %d\n", N_B, N_F, N_Y, N_X);
                        printf("flops: %0.5lf\n", (flopsPerMatrixMul * 1.0e-9f));
                        printf("Time is %0.5lf \n", (msecPerMatrixMul / 1000.0f));
                        printf("Gflops %0.5lf \n", gigaFlops);
                        printf("===========================================================\n");
                    
                        /*write back to Host Output -- batch size is 1*/
                        CHECK(hipMemcpy(Output, dev_Output, sizeof(float) * N_B * N_F * N_X * N_Y, hipMemcpyDeviceToHost));
                    
                        CHECK(hipFree(dev_Input));
                        CHECK(hipFree(dev_Kernel));
                        CHECK(hipFree(dev_Output));
                    
                    }
            