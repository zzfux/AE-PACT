#include "hip/hip_runtime.h"

            #include <cassert>
            #include <stdlib.h>
            #include <hip/hip_runtime.h>
            #include "common.h"
            

            
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[16];
  __shared__ float pad_temp_shared[2176];
  __shared__ float kernel_shared[128];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  compute_local[(7)] = 0.000000e+00f;
  compute_local[(8)] = 0.000000e+00f;
  compute_local[(9)] = 0.000000e+00f;
  compute_local[(10)] = 0.000000e+00f;
  compute_local[(11)] = 0.000000e+00f;
  compute_local[(12)] = 0.000000e+00f;
  compute_local[(13)] = 0.000000e+00f;
  compute_local[(14)] = 0.000000e+00f;
  compute_local[(15)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
      ((float4*)(pad_temp_shared + ((((int)threadIdx.x) * 4))))[0] = ((float4*)(data + (((((rc_outer_outer * 36992) + ((((int)threadIdx.x) / 68) * 4624)) + ((((int)blockIdx.x) % 17) * 272)) + ((((int)threadIdx.x) % 68) * 4)))))[0];
      ((float4*)(pad_temp_shared + (((((int)threadIdx.x) * 4) + 1088))))[0] = ((float4*)(data + ((((((rc_outer_outer * 36992) + ((((int)threadIdx.x) / 68) * 4624)) + ((((int)blockIdx.x) % 17) * 272)) + ((((int)threadIdx.x) % 68) * 4)) + 18496))))[0];
    if (((int)threadIdx.x) < 128) {
      kernel_shared[(((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 17) * 4096) + ((((int)threadIdx.x) >> 3) * 256)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    }
    __syncthreads();
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * kernel_shared[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 272))] * kernel_shared[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((int)threadIdx.x))] * kernel_shared[(8)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 272))] * kernel_shared[(9)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((int)threadIdx.x))] * kernel_shared[(16)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 272))] * kernel_shared[(17)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((int)threadIdx.x))] * kernel_shared[(24)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 272))] * kernel_shared[(25)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((int)threadIdx.x))] * kernel_shared[(32)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 272))] * kernel_shared[(33)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((int)threadIdx.x))] * kernel_shared[(40)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 272))] * kernel_shared[(41)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((int)threadIdx.x))] * kernel_shared[(48)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 272))] * kernel_shared[(49)]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((int)threadIdx.x))] * kernel_shared[(56)]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 272))] * kernel_shared[(57)]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((int)threadIdx.x))] * kernel_shared[(64)]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 272))] * kernel_shared[(65)]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((int)threadIdx.x))] * kernel_shared[(72)]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 272))] * kernel_shared[(73)]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((int)threadIdx.x))] * kernel_shared[(80)]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 272))] * kernel_shared[(81)]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((int)threadIdx.x))] * kernel_shared[(88)]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 272))] * kernel_shared[(89)]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((int)threadIdx.x))] * kernel_shared[(96)]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 272))] * kernel_shared[(97)]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((int)threadIdx.x))] * kernel_shared[(104)]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 272))] * kernel_shared[(105)]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((int)threadIdx.x))] * kernel_shared[(112)]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 272))] * kernel_shared[(113)]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((int)threadIdx.x))] * kernel_shared[(120)]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 272))] * kernel_shared[(121)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 544))] * kernel_shared[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 816))] * kernel_shared[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 544))] * kernel_shared[(10)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 816))] * kernel_shared[(11)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 544))] * kernel_shared[(18)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 816))] * kernel_shared[(19)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 544))] * kernel_shared[(26)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 816))] * kernel_shared[(27)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 544))] * kernel_shared[(34)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 816))] * kernel_shared[(35)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 544))] * kernel_shared[(42)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 816))] * kernel_shared[(43)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 544))] * kernel_shared[(50)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 816))] * kernel_shared[(51)]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 544))] * kernel_shared[(58)]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 816))] * kernel_shared[(59)]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 544))] * kernel_shared[(66)]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 816))] * kernel_shared[(67)]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 544))] * kernel_shared[(74)]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 816))] * kernel_shared[(75)]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 544))] * kernel_shared[(82)]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 816))] * kernel_shared[(83)]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 544))] * kernel_shared[(90)]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 816))] * kernel_shared[(91)]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 544))] * kernel_shared[(98)]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 816))] * kernel_shared[(99)]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 544))] * kernel_shared[(106)]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 816))] * kernel_shared[(107)]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 544))] * kernel_shared[(114)]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 816))] * kernel_shared[(115)]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 544))] * kernel_shared[(122)]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 816))] * kernel_shared[(123)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 1088))] * kernel_shared[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 1360))] * kernel_shared[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 1088))] * kernel_shared[(12)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 1360))] * kernel_shared[(13)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 1088))] * kernel_shared[(20)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 1360))] * kernel_shared[(21)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 1088))] * kernel_shared[(28)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 1360))] * kernel_shared[(29)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 1088))] * kernel_shared[(36)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 1360))] * kernel_shared[(37)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 1088))] * kernel_shared[(44)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 1360))] * kernel_shared[(45)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 1088))] * kernel_shared[(52)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 1360))] * kernel_shared[(53)]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 1088))] * kernel_shared[(60)]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 1360))] * kernel_shared[(61)]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 1088))] * kernel_shared[(68)]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 1360))] * kernel_shared[(69)]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 1088))] * kernel_shared[(76)]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 1360))] * kernel_shared[(77)]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 1088))] * kernel_shared[(84)]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 1360))] * kernel_shared[(85)]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 1088))] * kernel_shared[(92)]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 1360))] * kernel_shared[(93)]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 1088))] * kernel_shared[(100)]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 1360))] * kernel_shared[(101)]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 1088))] * kernel_shared[(108)]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 1360))] * kernel_shared[(109)]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 1088))] * kernel_shared[(116)]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 1360))] * kernel_shared[(117)]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 1088))] * kernel_shared[(124)]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 1360))] * kernel_shared[(125)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 1632))] * kernel_shared[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 1904))] * kernel_shared[(7)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 1632))] * kernel_shared[(14)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 1904))] * kernel_shared[(15)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 1632))] * kernel_shared[(22)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 1904))] * kernel_shared[(23)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 1632))] * kernel_shared[(30)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 1904))] * kernel_shared[(31)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 1632))] * kernel_shared[(38)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 1904))] * kernel_shared[(39)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 1632))] * kernel_shared[(46)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 1904))] * kernel_shared[(47)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 1632))] * kernel_shared[(54)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 1904))] * kernel_shared[(55)]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 1632))] * kernel_shared[(62)]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 1904))] * kernel_shared[(63)]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 1632))] * kernel_shared[(70)]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 1904))] * kernel_shared[(71)]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 1632))] * kernel_shared[(78)]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 1904))] * kernel_shared[(79)]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 1632))] * kernel_shared[(86)]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 1904))] * kernel_shared[(87)]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 1632))] * kernel_shared[(94)]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 1904))] * kernel_shared[(95)]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 1632))] * kernel_shared[(102)]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 1904))] * kernel_shared[(103)]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 1632))] * kernel_shared[(110)]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 1904))] * kernel_shared[(111)]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 1632))] * kernel_shared[(118)]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 1904))] * kernel_shared[(119)]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 1632))] * kernel_shared[(126)]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 1904))] * kernel_shared[(127)]));
  }
  for (int ff_inner = 0; ff_inner < 16; ++ff_inner) {
    compute[((((((((int)blockIdx.x) / 17) * 73984) + (ff_inner * 4624)) + ((((int)blockIdx.x) % 17) * 272)) + ((int)threadIdx.x)))] = compute_local[(ff_inner)];
  }
}



void conv_kernel_wrapper(int N_B, int N_C, int N_H, int N_W, int N_F, int N_R, int N_S, int PaddingH, int PaddingW,
                        int StrideH, int StrideW, int N_X, int N_Y, const float *Input, 
                        const float *Kernel, float *Output, int itr) {
                        hipEvent_t start;
                        CHECK(hipEventCreate(&start));
                        hipEvent_t stop;
                        CHECK(hipEventCreate(&stop));
                        float *dev_Input;
                        float *dev_Kernel;
                        float *dev_Output;
                    
                        CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                        CHECK(hipMemcpy(dev_Kernel, Kernel, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                        CHECK(hipMemcpy(dev_Input, Input, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Output, sizeof(float) * N_B * N_F * N_Y * N_X));
                        CHECK(hipMemset(dev_Output, 0, sizeof(float) * N_B * N_F * N_Y * N_X));
                    
dim3 size_grid(136,1,1);

dim3 size_block(272,1,1);
 
                       printf("warmup....\n");
                        default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                    
                        double *time_arr = (double *)malloc(itr * sizeof(double));
                        for (int j = 1; j < itr; j++) {
                            CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                            CHECK(hipMemcpy(dev_Kernel, Kernel+j*N_F * N_C * N_R * N_S, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                            CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                            CHECK(hipMemcpy(dev_Input, Input+j*N_B * N_C * N_H * N_W, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                            /* Record the start event*/
                            CHECK(hipEventRecord(start, NULL));
                            default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                            /* Record the end event */
                            CHECK(hipEventRecord(stop, NULL));
                            /* Wait for the stop event to complete*/
                            CHECK(hipEventSynchronize(stop));
                            float msec = 0.0f;
                            hipEventElapsedTime(&msec, start, stop);
                            time_arr[j] = msec;
                        }
                    
                       
                    
                        //hipError_t err = hipGetLastError();
                        //if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
                        double msecTotal = 0;
                        for (int j = 1; j < itr; j++) {   
                            printf("%0.5lf\n", time_arr[j]);
                            msecTotal += time_arr[j];
                        }
                    
                        double msecPerMatrixMul = (msecTotal) / itr;
                        double flopsPerMatrixMul = 2.0 * N_B * N_C * N_X * N_Y * N_F * N_R * N_S;
                    
                        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
                    
                        printf("\n===========================================================\n");
                        printf("Input size: %d X %d X %d X %d\n", N_B, N_C, N_H, N_W);
                        printf("Kernel size: %d X %d X %d X %d\n", N_F, N_C, N_R, N_S);
                        printf("Output size: %d X %d X %d X %d\n", N_B, N_F, N_Y, N_X);
                        printf("flops: %0.5lf\n", (flopsPerMatrixMul * 1.0e-9f));
                        printf("Time is %0.5lf \n", (msecPerMatrixMul / 1000.0f));
                        printf("Gflops %0.5lf \n", gigaFlops);
                        printf("===========================================================\n");
                    
                        /*write back to Host Output -- batch size is 1*/
                        CHECK(hipMemcpy(Output, dev_Output, sizeof(float) * N_B * N_F * N_X * N_Y, hipMemcpyDeviceToHost));
                    
                        CHECK(hipFree(dev_Input));
                        CHECK(hipFree(dev_Kernel));
                        CHECK(hipFree(dev_Output));
                    
                    }
            