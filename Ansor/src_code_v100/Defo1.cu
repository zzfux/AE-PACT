#include "hip/hip_runtime.h"

            #include <cassert>
            #include <stdlib.h>
            #include <hip/hip_runtime.h>
            #include "common.h"
            

            
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[4];
  __shared__ float pad_temp_shared[640];
  __shared__ float kernel_shared[576];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[(((int)threadIdx.x))] = data[((((((rc_outer_outer * 25088) + (((((int)blockIdx.x) % 81) / 3) * 112)) + ((((int)threadIdx.x) / 20) * 56)) + ((((int)blockIdx.x) % 3) * 18)) + (((int)threadIdx.x) % 20)))];
    pad_temp_shared[((((int)threadIdx.x) + 72))] = data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 72) / 80) * 3136)) + (((((int)blockIdx.x) % 81) / 3) * 112)) + ((((((int)threadIdx.x) + 72) % 80) / 20) * 56)) + ((((int)blockIdx.x) % 3) * 18)) + ((((int)threadIdx.x) + 12) % 20)))];
    pad_temp_shared[((((int)threadIdx.x) + 144))] = data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 144) / 80) * 3136)) + (((((int)blockIdx.x) % 81) / 3) * 112)) + ((((((int)threadIdx.x) + 64) % 80) / 20) * 56)) + ((((int)blockIdx.x) % 3) * 18)) + ((((int)threadIdx.x) + 4) % 20)))];
    pad_temp_shared[((((int)threadIdx.x) + 216))] = data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 216) / 80) * 3136)) + (((((int)blockIdx.x) % 81) / 3) * 112)) + ((((((int)threadIdx.x) + 56) % 80) / 20) * 56)) + ((((int)blockIdx.x) % 3) * 18)) + ((((int)threadIdx.x) + 16) % 20)))];
    pad_temp_shared[((((int)threadIdx.x) + 288))] = data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 288) / 80) * 3136)) + (((((int)blockIdx.x) % 81) / 3) * 112)) + ((((((int)threadIdx.x) + 48) % 80) / 20) * 56)) + ((((int)blockIdx.x) % 3) * 18)) + ((((int)threadIdx.x) + 8) % 20)))];
    pad_temp_shared[((((int)threadIdx.x) + 360))] = data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 360) / 80) * 3136)) + (((((int)blockIdx.x) % 81) / 3) * 112)) + ((((((int)threadIdx.x) + 40) % 80) / 20) * 56)) + ((((int)blockIdx.x) % 3) * 18)) + (((int)threadIdx.x) % 20)))];
    pad_temp_shared[((((int)threadIdx.x) + 432))] = data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 432) / 80) * 3136)) + (((((int)blockIdx.x) % 81) / 3) * 112)) + ((((((int)threadIdx.x) + 32) % 80) / 20) * 56)) + ((((int)blockIdx.x) % 3) * 18)) + ((((int)threadIdx.x) + 12) % 20)))];
    pad_temp_shared[((((int)threadIdx.x) + 504))] = data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 504) / 80) * 3136)) + (((((int)blockIdx.x) % 81) / 3) * 112)) + ((((((int)threadIdx.x) + 24) % 80) / 20) * 56)) + ((((int)blockIdx.x) % 3) * 18)) + ((((int)threadIdx.x) + 4) % 20)))];
    if (((int)threadIdx.x) < 64) {
      pad_temp_shared[((((int)threadIdx.x) + 576))] = data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 576) / 80) * 3136)) + (((((int)blockIdx.x) % 81) / 3) * 112)) + (((((int)threadIdx.x) + 16) / 20) * 56)) + ((((int)blockIdx.x) % 3) * 18)) + ((((int)threadIdx.x) + 16) % 20)))];
    }
      ((float2*)(kernel_shared + ((((int)threadIdx.x) * 2))))[0] = ((float2*)(kernel + ((((((((int)blockIdx.x) / 81) * 2304) + ((((int)threadIdx.x) / 36) * 288)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 36) * 2)))))[0];
      ((float2*)(kernel_shared + (((((int)threadIdx.x) * 2) + 144))))[0] = ((float2*)(kernel + (((((((((int)blockIdx.x) / 81) * 2304) + ((((int)threadIdx.x) / 36) * 288)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 36) * 2)) + 576))))[0];
      ((float2*)(kernel_shared + (((((int)threadIdx.x) * 2) + 288))))[0] = ((float2*)(kernel + (((((((((int)blockIdx.x) / 81) * 2304) + ((((int)threadIdx.x) / 36) * 288)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 36) * 2)) + 1152))))[0];
      ((float2*)(kernel_shared + (((((int)threadIdx.x) * 2) + 432))))[0] = ((float2*)(kernel + (((((((((int)blockIdx.x) / 81) * 2304) + ((((int)threadIdx.x) / 36) * 288)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 36) * 2)) + 1728))))[0];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int xx_c_outer_inner = 0; xx_c_outer_inner < 2; ++xx_c_outer_inner) {
        compute_local[(xx_c_outer_inner)] = (compute_local[(xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner))] * kernel_shared[((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)))]));
        compute_local[((xx_c_outer_inner + 2))] = (compute_local[((xx_c_outer_inner + 2))] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 20))] * kernel_shared[((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)))]));
        compute_local[(xx_c_outer_inner)] = (compute_local[(xx_c_outer_inner)] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 1))]));
        compute_local[((xx_c_outer_inner + 2))] = (compute_local[((xx_c_outer_inner + 2))] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 21))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 1))]));
        compute_local[(xx_c_outer_inner)] = (compute_local[(xx_c_outer_inner)] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 2))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 2))]));
        compute_local[((xx_c_outer_inner + 2))] = (compute_local[((xx_c_outer_inner + 2))] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 22))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 2))]));
        compute_local[(xx_c_outer_inner)] = (compute_local[(xx_c_outer_inner)] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 20))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 3))]));
        compute_local[((xx_c_outer_inner + 2))] = (compute_local[((xx_c_outer_inner + 2))] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 40))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 3))]));
        compute_local[(xx_c_outer_inner)] = (compute_local[(xx_c_outer_inner)] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 21))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 4))]));
        compute_local[((xx_c_outer_inner + 2))] = (compute_local[((xx_c_outer_inner + 2))] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 41))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 4))]));
        compute_local[(xx_c_outer_inner)] = (compute_local[(xx_c_outer_inner)] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 22))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 5))]));
        compute_local[((xx_c_outer_inner + 2))] = (compute_local[((xx_c_outer_inner + 2))] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 42))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 5))]));
        compute_local[(xx_c_outer_inner)] = (compute_local[(xx_c_outer_inner)] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 40))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 6))]));
        compute_local[((xx_c_outer_inner + 2))] = (compute_local[((xx_c_outer_inner + 2))] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 60))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 6))]));
        compute_local[(xx_c_outer_inner)] = (compute_local[(xx_c_outer_inner)] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 41))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 7))]));
        compute_local[((xx_c_outer_inner + 2))] = (compute_local[((xx_c_outer_inner + 2))] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 61))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 7))]));
        compute_local[(xx_c_outer_inner)] = (compute_local[(xx_c_outer_inner)] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 42))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 8))]));
        compute_local[((xx_c_outer_inner + 2))] = (compute_local[((xx_c_outer_inner + 2))] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 62))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 8))]));
        compute_local[(xx_c_outer_inner)] = (compute_local[(xx_c_outer_inner)] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 80))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 9))]));
        compute_local[((xx_c_outer_inner + 2))] = (compute_local[((xx_c_outer_inner + 2))] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 100))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 9))]));
        compute_local[(xx_c_outer_inner)] = (compute_local[(xx_c_outer_inner)] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 81))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 10))]));
        compute_local[((xx_c_outer_inner + 2))] = (compute_local[((xx_c_outer_inner + 2))] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 101))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 10))]));
        compute_local[(xx_c_outer_inner)] = (compute_local[(xx_c_outer_inner)] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 82))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 11))]));
        compute_local[((xx_c_outer_inner + 2))] = (compute_local[((xx_c_outer_inner + 2))] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 102))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 11))]));
        compute_local[(xx_c_outer_inner)] = (compute_local[(xx_c_outer_inner)] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 100))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 12))]));
        compute_local[((xx_c_outer_inner + 2))] = (compute_local[((xx_c_outer_inner + 2))] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 120))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 12))]));
        compute_local[(xx_c_outer_inner)] = (compute_local[(xx_c_outer_inner)] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 101))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 13))]));
        compute_local[((xx_c_outer_inner + 2))] = (compute_local[((xx_c_outer_inner + 2))] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 121))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 13))]));
        compute_local[(xx_c_outer_inner)] = (compute_local[(xx_c_outer_inner)] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 102))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 14))]));
        compute_local[((xx_c_outer_inner + 2))] = (compute_local[((xx_c_outer_inner + 2))] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 122))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 14))]));
        compute_local[(xx_c_outer_inner)] = (compute_local[(xx_c_outer_inner)] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 120))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 15))]));
        compute_local[((xx_c_outer_inner + 2))] = (compute_local[((xx_c_outer_inner + 2))] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 140))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 15))]));
        compute_local[(xx_c_outer_inner)] = (compute_local[(xx_c_outer_inner)] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 121))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 16))]));
        compute_local[((xx_c_outer_inner + 2))] = (compute_local[((xx_c_outer_inner + 2))] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 141))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 16))]));
        compute_local[(xx_c_outer_inner)] = (compute_local[(xx_c_outer_inner)] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 122))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 17))]));
        compute_local[((xx_c_outer_inner + 2))] = (compute_local[((xx_c_outer_inner + 2))] + (pad_temp_shared[(((((rc_outer_inner * 160) + ((((int)threadIdx.x) % 9) * 2)) + xx_c_outer_inner) + 142))] * kernel_shared[(((((((int)threadIdx.x) / 9) * 72) + (rc_outer_inner * 18)) + 17))]));
      }
    }
  }
  for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
    compute[((((((((((int)blockIdx.x) / 81) * 23328) + ((((int)threadIdx.x) / 9) * 2916)) + (((((int)blockIdx.x) % 81) / 3) * 108)) + ((((int)blockIdx.x) % 3) * 18)) + ((((int)threadIdx.x) % 9) * 2)) + xx_inner))] = compute_local[(xx_inner)];
    compute[(((((((((((int)blockIdx.x) / 81) * 23328) + ((((int)threadIdx.x) / 9) * 2916)) + (((((int)blockIdx.x) % 81) / 3) * 108)) + ((((int)blockIdx.x) % 3) * 18)) + ((((int)threadIdx.x) % 9) * 2)) + xx_inner) + 54))] = compute_local[((xx_inner + 2))];
  }
}



void conv_kernel_wrapper(int N_B, int N_C, int N_H, int N_W, int N_F, int N_R, int N_S, int PaddingH, int PaddingW,
                        int StrideH, int StrideW, int N_X, int N_Y, const float *Input, 
                        const float *Kernel, float *Output, int itr) {
                        hipEvent_t start;
                        CHECK(hipEventCreate(&start));
                        hipEvent_t stop;
                        CHECK(hipEventCreate(&stop));
                        float *dev_Input;
                        float *dev_Kernel;
                        float *dev_Output;
                    
                        CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                        CHECK(hipMemcpy(dev_Kernel, Kernel, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                        CHECK(hipMemcpy(dev_Input, Input, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Output, sizeof(float) * N_B * N_F * N_Y * N_X));
                        CHECK(hipMemset(dev_Output, 0, sizeof(float) * N_B * N_F * N_Y * N_X));
                    
dim3 size_grid(648,1,1);

dim3 size_block(72,1,1);
 
                       printf("warmup....\n");
                        default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                    
                        double *time_arr = (double *)malloc(itr * sizeof(double));
                        for (int j = 1; j < itr; j++) {
                            CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                            CHECK(hipMemcpy(dev_Kernel, Kernel+j*N_F * N_C * N_R * N_S, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                            CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                            CHECK(hipMemcpy(dev_Input, Input+j*N_B * N_C * N_H * N_W, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                            /* Record the start event*/
                            CHECK(hipEventRecord(start, NULL));
                            default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                            /* Record the end event */
                            CHECK(hipEventRecord(stop, NULL));
                            /* Wait for the stop event to complete*/
                            CHECK(hipEventSynchronize(stop));
                            float msec = 0.0f;
                            hipEventElapsedTime(&msec, start, stop);
                            time_arr[j] = msec;
                        }
                    
                       
                    
                        //hipError_t err = hipGetLastError();
                        //if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
                        double msecTotal = 0;
                        for (int j = 1; j < itr; j++) {   
                            printf("%0.5lf\n", time_arr[j]);
                            msecTotal += time_arr[j];
                        }
                    
                        double msecPerMatrixMul = (msecTotal) / itr;
                        double flopsPerMatrixMul = 2.0 * N_B * N_C * N_X * N_Y * N_F * N_R * N_S;
                    
                        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
                    
                        printf("\n===========================================================\n");
                        printf("Input size: %d X %d X %d X %d\n", N_B, N_C, N_H, N_W);
                        printf("Kernel size: %d X %d X %d X %d\n", N_F, N_C, N_R, N_S);
                        printf("Output size: %d X %d X %d X %d\n", N_B, N_F, N_Y, N_X);
                        printf("flops: %0.5lf\n", (flopsPerMatrixMul * 1.0e-9f));
                        printf("Time is %0.5lf \n", (msecPerMatrixMul / 1000.0f));
                        printf("Gflops %0.5lf \n", gigaFlops);
                        printf("===========================================================\n");
                    
                        /*write back to Host Output -- batch size is 1*/
                        CHECK(hipMemcpy(Output, dev_Output, sizeof(float) * N_B * N_F * N_X * N_Y, hipMemcpyDeviceToHost));
                    
                        CHECK(hipFree(dev_Input));
                        CHECK(hipFree(dev_Kernel));
                        CHECK(hipFree(dev_Output));
                    
                    }
            