#include "hip/hip_runtime.h"

            #include <cassert>
            #include <stdlib.h>
            #include <hip/hip_runtime.h>
            #include "common.h"
            

            
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[6];
  __shared__ float pad_temp_shared[580];
  __shared__ float kernel_shared[576];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[(((int)threadIdx.x))] = data[(((((rc_outer_outer * 3364) + ((((int)threadIdx.x) / 145) * 841)) + ((((int)blockIdx.x) % 9) * 87)) + (((int)threadIdx.x) % 145)))];
    pad_temp_shared[((((int)threadIdx.x) + 216))] = data[(((((rc_outer_outer * 3364) + (((((int)threadIdx.x) + 216) / 145) * 841)) + ((((int)blockIdx.x) % 9) * 87)) + ((((int)threadIdx.x) + 71) % 145)))];
    if (((int)threadIdx.x) < 148) {
      pad_temp_shared[((((int)threadIdx.x) + 432))] = data[(((((rc_outer_outer * 3364) + (((((int)threadIdx.x) + 432) / 145) * 841)) + ((((int)blockIdx.x) % 9) * 87)) + ((((int)threadIdx.x) + 142) % 145)))];
    }
    kernel_shared[(((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 9) * 9216) + ((((int)threadIdx.x) / 36) * 576)) + (rc_outer_outer * 36)) + (((int)threadIdx.x) % 36)))];
    kernel_shared[((((int)threadIdx.x) + 216))] = kernel[(((((((((int)blockIdx.x) / 9) * 9216) + ((((int)threadIdx.x) / 36) * 576)) + (rc_outer_outer * 36)) + (((int)threadIdx.x) % 36)) + 3456))];
    if (((int)threadIdx.x) < 144) {
      kernel_shared[((((int)threadIdx.x) + 432))] = kernel[(((((((((int)blockIdx.x) / 9) * 9216) + ((((int)threadIdx.x) / 36) * 576)) + (rc_outer_outer * 36)) + (((int)threadIdx.x) % 36)) + 6912))];
    }
    __syncthreads();
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((int)threadIdx.x) % 27))] * kernel_shared[(((((int)threadIdx.x) / 27) * 72))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 29))] * kernel_shared[(((((int)threadIdx.x) / 27) * 72))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 58))] * kernel_shared[(((((int)threadIdx.x) / 27) * 72))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((int)threadIdx.x) % 27))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 36))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 29))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 36))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 58))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 36))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 1))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 1))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 30))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 1))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 59))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 1))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 1))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 37))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 30))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 37))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 59))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 37))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 2))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 2))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 31))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 2))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 60))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 2))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 2))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 38))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 31))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 38))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 60))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 38))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 29))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 3))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 58))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 3))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 87))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 3))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 29))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 39))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 58))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 39))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 87))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 39))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 30))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 4))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 59))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 4))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 88))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 4))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 30))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 40))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 59))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 40))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 88))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 40))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 31))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 5))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 60))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 5))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 89))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 5))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 31))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 41))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 60))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 41))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 89))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 41))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 58))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 6))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 87))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 6))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 116))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 6))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 58))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 42))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 87))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 42))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 116))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 42))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 59))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 7))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 88))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 7))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 117))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 7))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 59))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 43))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 88))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 43))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 117))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 43))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 60))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 8))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 89))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 8))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 118))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 8))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 60))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 44))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 89))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 44))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 118))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 44))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 145))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 9))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 174))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 9))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 203))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 9))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 145))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 45))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 174))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 45))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 203))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 45))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 146))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 10))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 175))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 10))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 204))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 10))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 146))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 46))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 175))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 46))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 204))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 46))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 147))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 11))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 176))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 11))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 205))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 11))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 147))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 47))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 176))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 47))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 205))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 47))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 174))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 12))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 203))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 12))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 232))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 12))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 174))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 48))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 203))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 48))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 232))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 48))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 175))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 13))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 204))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 13))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 233))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 13))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 175))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 49))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 204))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 49))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 233))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 49))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 176))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 14))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 205))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 14))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 234))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 14))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 176))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 50))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 205))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 50))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 234))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 50))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 203))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 15))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 232))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 15))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 261))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 15))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 203))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 51))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 232))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 51))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 261))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 51))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 204))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 16))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 233))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 16))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 262))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 16))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 204))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 52))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 233))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 52))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 262))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 52))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 205))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 17))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 234))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 17))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 263))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 17))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 205))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 53))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 234))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 53))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 263))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 53))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 290))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 18))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 319))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 18))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 348))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 18))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 290))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 54))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 319))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 54))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 348))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 54))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 291))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 19))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 320))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 19))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 349))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 19))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 291))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 55))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 320))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 55))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 349))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 55))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 292))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 20))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 321))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 20))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 350))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 20))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 292))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 56))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 321))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 56))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 350))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 56))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 319))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 21))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 348))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 21))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 377))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 21))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 319))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 57))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 348))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 57))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 377))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 57))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 320))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 22))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 349))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 22))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 378))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 22))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 320))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 58))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 349))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 58))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 378))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 58))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 321))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 23))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 350))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 23))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 379))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 23))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 321))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 59))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 350))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 59))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 379))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 59))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 348))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 24))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 377))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 24))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 406))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 24))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 348))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 60))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 377))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 60))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 406))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 60))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 349))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 25))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 378))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 25))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 407))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 25))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 349))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 61))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 378))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 61))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 407))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 61))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 350))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 26))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 379))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 26))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 408))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 26))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 350))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 62))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 379))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 62))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 408))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 62))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 435))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 27))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 464))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 27))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 493))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 27))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 435))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 63))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 464))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 63))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 493))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 63))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 436))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 28))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 465))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 28))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 494))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 28))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 436))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 64))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 465))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 64))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 494))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 64))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 437))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 29))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 466))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 29))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 495))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 29))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 437))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 65))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 466))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 65))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 495))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 65))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 464))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 30))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 493))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 30))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 522))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 30))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 464))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 66))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 493))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 66))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 522))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 66))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 465))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 31))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 494))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 31))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 523))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 31))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 465))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 67))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 494))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 67))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 523))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 67))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 466))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 32))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 495))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 32))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 524))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 32))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 466))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 68))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 495))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 68))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 524))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 68))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 493))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 33))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 522))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 33))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 551))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 33))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 493))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 69))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 522))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 69))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 551))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 69))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 494))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 34))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 523))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 34))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 552))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 34))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 494))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 70))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 523))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 70))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 552))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 70))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 495))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 35))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 524))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 35))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 553))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 35))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 495))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 71))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 524))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 71))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((int)threadIdx.x) % 27) + 553))] * kernel_shared[((((((int)threadIdx.x) / 27) * 72) + 71))]));
  }
  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
    compute[(((((((((int)blockIdx.x) / 9) * 11664) + ((((int)threadIdx.x) / 27) * 1458)) + (ff_inner * 729)) + ((((int)blockIdx.x) % 9) * 81)) + (((int)threadIdx.x) % 27)))] = compute_local[(ff_inner)];
    compute[((((((((((int)blockIdx.x) / 9) * 11664) + ((((int)threadIdx.x) / 27) * 1458)) + (ff_inner * 729)) + ((((int)blockIdx.x) % 9) * 81)) + (((int)threadIdx.x) % 27)) + 27))] = compute_local[((ff_inner + 2))];
    compute[((((((((((int)blockIdx.x) / 9) * 11664) + ((((int)threadIdx.x) / 27) * 1458)) + (ff_inner * 729)) + ((((int)blockIdx.x) % 9) * 81)) + (((int)threadIdx.x) % 27)) + 54))] = compute_local[((ff_inner + 4))];
  }
}



void conv_kernel_wrapper(int N_B, int N_C, int N_H, int N_W, int N_F, int N_R, int N_S, int PaddingH, int PaddingW,
                        int StrideH, int StrideW, int N_X, int N_Y, const float *Input, 
                        const float *Kernel, float *Output, int itr) {
                        hipEvent_t start;
                        CHECK(hipEventCreate(&start));
                        hipEvent_t stop;
                        CHECK(hipEventCreate(&stop));
                        float *dev_Input;
                        float *dev_Kernel;
                        float *dev_Output;
                    
                        CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                        CHECK(hipMemcpy(dev_Kernel, Kernel, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                        CHECK(hipMemcpy(dev_Input, Input, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Output, sizeof(float) * N_B * N_F * N_Y * N_X));
                        CHECK(hipMemset(dev_Output, 0, sizeof(float) * N_B * N_F * N_Y * N_X));
                    
dim3 size_grid(72,1,1);

dim3 size_block(216,1,1);
 
                       printf("warmup....\n");
                        default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                    
                        double *time_arr = (double *)malloc(itr * sizeof(double));
                        for (int j = 1; j < itr; j++) {
                            CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                            CHECK(hipMemcpy(dev_Kernel, Kernel+j*N_F * N_C * N_R * N_S, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                            CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                            CHECK(hipMemcpy(dev_Input, Input+j*N_B * N_C * N_H * N_W, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                            /* Record the start event*/
                            CHECK(hipEventRecord(start, NULL));
                            default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                            /* Record the end event */
                            CHECK(hipEventRecord(stop, NULL));
                            /* Wait for the stop event to complete*/
                            CHECK(hipEventSynchronize(stop));
                            float msec = 0.0f;
                            hipEventElapsedTime(&msec, start, stop);
                            time_arr[j] = msec;
                        }
                    
                       
                    
                        //hipError_t err = hipGetLastError();
                        //if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
                        double msecTotal = 0;
                        for (int j = 1; j < itr; j++) {   
                            printf("%0.5lf\n", time_arr[j]);
                            msecTotal += time_arr[j];
                        }
                    
                        double msecPerMatrixMul = (msecTotal) / itr;
                        double flopsPerMatrixMul = 2.0 * N_B * N_C * N_X * N_Y * N_F * N_R * N_S;
                    
                        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
                    
                        printf("\n===========================================================\n");
                        printf("Input size: %d X %d X %d X %d\n", N_B, N_C, N_H, N_W);
                        printf("Kernel size: %d X %d X %d X %d\n", N_F, N_C, N_R, N_S);
                        printf("Output size: %d X %d X %d X %d\n", N_B, N_F, N_Y, N_X);
                        printf("flops: %0.5lf\n", (flopsPerMatrixMul * 1.0e-9f));
                        printf("Time is %0.5lf \n", (msecPerMatrixMul / 1000.0f));
                        printf("Gflops %0.5lf \n", gigaFlops);
                        printf("===========================================================\n");
                    
                        /*write back to Host Output -- batch size is 1*/
                        CHECK(hipMemcpy(Output, dev_Output, sizeof(float) * N_B * N_F * N_X * N_Y, hipMemcpyDeviceToHost));
                    
                        CHECK(hipFree(dev_Input));
                        CHECK(hipFree(dev_Kernel));
                        CHECK(hipFree(dev_Output));
                    
                    }
            