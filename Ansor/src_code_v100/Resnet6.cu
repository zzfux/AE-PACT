#include "hip/hip_runtime.h"

            #include <cassert>
            #include <stdlib.h>
            #include <hip/hip_runtime.h>
            #include "common.h"
            

            
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[8];
  __shared__ float pad_temp_shared[1536];
  __shared__ float kernel_shared[2304];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(7)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((((int)threadIdx.x) * 3))] = data[(((((((rc_outer_outer * 14400) + ((((int)threadIdx.x) >> 5) * 900)) + (((((int)blockIdx.x) % 14) / 7) * 420)) + (((((int)threadIdx.x) & 31) >> 1) * 30)) + ((((int)blockIdx.x) % 7) * 4)) + ((((int)threadIdx.x) & 1) * 3)))];
    pad_temp_shared[(((((int)threadIdx.x) * 3) + 1))] = data[(((((((rc_outer_outer * 14400) + ((((((int)threadIdx.x) * 3) + 1) / 96) * 900)) + (((((int)blockIdx.x) % 14) / 7) * 420)) + (((((((int)threadIdx.x) * 3) + 1) % 96) / 6) * 30)) + ((((int)blockIdx.x) % 7) * 4)) + (((((int)threadIdx.x) * 3) + 1) % 6)))];
    pad_temp_shared[(((((int)threadIdx.x) * 3) + 2))] = data[(((((((rc_outer_outer * 14400) + ((((((int)threadIdx.x) * 3) + 2) / 96) * 900)) + (((((int)blockIdx.x) % 14) / 7) * 420)) + (((((((int)threadIdx.x) * 3) + 2) % 96) / 6) * 30)) + ((((int)blockIdx.x) % 7) * 4)) + (((((int)threadIdx.x) * 3) + 2) % 6)))];
    pad_temp_shared[(((((((((int)threadIdx.x) * 3) + 336) / 96) * 96) + ((((((int)threadIdx.x) >> 1) + 8) & 15) * 6)) + ((((int)threadIdx.x) & 1) * 3)))] = data[(((((((rc_outer_outer * 14400) + ((((((int)threadIdx.x) * 3) + 336) / 96) * 900)) + (((((int)blockIdx.x) % 14) / 7) * 420)) + ((((((int)threadIdx.x) >> 1) + 8) & 15) * 30)) + ((((int)blockIdx.x) % 7) * 4)) + ((((int)threadIdx.x) & 1) * 3)))];
    pad_temp_shared[(((((((((int)threadIdx.x) * 3) + 337) / 96) * 96) + ((((((((int)threadIdx.x) * 3) + 1) / 6) + 8) & 15) * 6)) + (((((int)threadIdx.x) * 3) + 1) % 6)))] = data[(((((((rc_outer_outer * 14400) + ((((((int)threadIdx.x) * 3) + 337) / 96) * 900)) + (((((int)blockIdx.x) % 14) / 7) * 420)) + ((((((((int)threadIdx.x) * 3) + 1) / 6) + 8) & 15) * 30)) + ((((int)blockIdx.x) % 7) * 4)) + (((((int)threadIdx.x) * 3) + 1) % 6)))];
    pad_temp_shared[(((((((((int)threadIdx.x) * 3) + 338) / 96) * 96) + ((((((((int)threadIdx.x) * 3) + 2) / 6) + 8) & 15) * 6)) + (((((int)threadIdx.x) * 3) + 2) % 6)))] = data[(((((((rc_outer_outer * 14400) + ((((((int)threadIdx.x) * 3) + 338) / 96) * 900)) + (((((int)blockIdx.x) % 14) / 7) * 420)) + ((((((((int)threadIdx.x) * 3) + 2) / 6) + 8) & 15) * 30)) + ((((int)blockIdx.x) % 7) * 4)) + (((((int)threadIdx.x) * 3) + 2) % 6)))];
    pad_temp_shared[(((((int)threadIdx.x) * 3) + 672))] = data[((((((((rc_outer_outer * 14400) + ((((int)threadIdx.x) >> 5) * 900)) + (((((int)blockIdx.x) % 14) / 7) * 420)) + (((((int)threadIdx.x) & 31) >> 1) * 30)) + ((((int)blockIdx.x) % 7) * 4)) + ((((int)threadIdx.x) & 1) * 3)) + 6300))];
    pad_temp_shared[((((((((int)threadIdx.x) * 3) + 673) / 96) * 96) + (((((int)threadIdx.x) * 3) + 1) % 96)))] = data[(((((((rc_outer_outer * 14400) + ((((((int)threadIdx.x) * 3) + 673) / 96) * 900)) + (((((int)blockIdx.x) % 14) / 7) * 420)) + (((((((int)threadIdx.x) * 3) + 1) % 96) / 6) * 30)) + ((((int)blockIdx.x) % 7) * 4)) + (((((int)threadIdx.x) * 3) + 1) % 6)))];
    pad_temp_shared[((((((((int)threadIdx.x) * 3) + 674) / 96) * 96) + (((((int)threadIdx.x) * 3) + 2) % 96)))] = data[(((((((rc_outer_outer * 14400) + ((((((int)threadIdx.x) * 3) + 674) / 96) * 900)) + (((((int)blockIdx.x) % 14) / 7) * 420)) + (((((((int)threadIdx.x) * 3) + 2) % 96) / 6) * 30)) + ((((int)blockIdx.x) % 7) * 4)) + (((((int)threadIdx.x) * 3) + 2) % 6)))];
    pad_temp_shared[(((((((((int)threadIdx.x) * 3) + 1008) / 96) * 96) + ((((((int)threadIdx.x) >> 1) + 8) & 15) * 6)) + ((((int)threadIdx.x) & 1) * 3)))] = data[(((((((rc_outer_outer * 14400) + ((((((int)threadIdx.x) * 3) + 1008) / 96) * 900)) + (((((int)blockIdx.x) % 14) / 7) * 420)) + ((((((int)threadIdx.x) >> 1) + 8) & 15) * 30)) + ((((int)blockIdx.x) % 7) * 4)) + ((((int)threadIdx.x) & 1) * 3)))];
    pad_temp_shared[(((((((((int)threadIdx.x) * 3) + 1009) / 96) * 96) + ((((((((int)threadIdx.x) * 3) + 1) / 6) + 8) & 15) * 6)) + (((((int)threadIdx.x) * 3) + 1) % 6)))] = data[(((((((rc_outer_outer * 14400) + ((((((int)threadIdx.x) * 3) + 1009) / 96) * 900)) + (((((int)blockIdx.x) % 14) / 7) * 420)) + ((((((((int)threadIdx.x) * 3) + 1) / 6) + 8) & 15) * 30)) + ((((int)blockIdx.x) % 7) * 4)) + (((((int)threadIdx.x) * 3) + 1) % 6)))];
    pad_temp_shared[(((((((((int)threadIdx.x) * 3) + 1010) / 96) * 96) + ((((((((int)threadIdx.x) * 3) + 2) / 6) + 8) & 15) * 6)) + (((((int)threadIdx.x) * 3) + 2) % 6)))] = data[(((((((rc_outer_outer * 14400) + ((((((int)threadIdx.x) * 3) + 1010) / 96) * 900)) + (((((int)blockIdx.x) % 14) / 7) * 420)) + ((((((((int)threadIdx.x) * 3) + 2) / 6) + 8) & 15) * 30)) + ((((int)blockIdx.x) % 7) * 4)) + (((((int)threadIdx.x) * 3) + 2) % 6)))];
    if (((int)threadIdx.x) < 64) {
      pad_temp_shared[(((((int)threadIdx.x) * 3) + 1344))] = data[((((((((rc_outer_outer * 14400) + ((((int)threadIdx.x) >> 5) * 900)) + (((((int)blockIdx.x) % 14) / 7) * 420)) + (((((int)threadIdx.x) & 31) >> 1) * 30)) + ((((int)blockIdx.x) % 7) * 4)) + ((((int)threadIdx.x) & 1) * 3)) + 12600))];
    }
    if (((int)threadIdx.x) < 64) {
      pad_temp_shared[((((((((int)threadIdx.x) * 3) + 1345) / 96) * 96) + (((((int)threadIdx.x) * 3) + 1) % 96)))] = data[(((((((rc_outer_outer * 14400) + ((((((int)threadIdx.x) * 3) + 1345) / 96) * 900)) + (((((int)blockIdx.x) % 14) / 7) * 420)) + (((((((int)threadIdx.x) * 3) + 1) % 96) / 6) * 30)) + ((((int)blockIdx.x) % 7) * 4)) + (((((int)threadIdx.x) * 3) + 1) % 6)))];
    }
    if (((int)threadIdx.x) < 64) {
      pad_temp_shared[((((((((int)threadIdx.x) * 3) + 1346) / 96) * 96) + (((((int)threadIdx.x) * 3) + 2) % 96)))] = data[(((((((rc_outer_outer * 14400) + ((((((int)threadIdx.x) * 3) + 1346) / 96) * 900)) + (((((int)blockIdx.x) % 14) / 7) * 420)) + (((((((int)threadIdx.x) * 3) + 2) % 96) / 6) * 30)) + ((((int)blockIdx.x) % 7) * 4)) + (((((int)threadIdx.x) * 3) + 2) % 6)))];
    }
    kernel_shared[((((int)threadIdx.x) * 2))] = kernel[((((((((int)blockIdx.x) / 14) * 18432) + ((((int)threadIdx.x) / 72) * 1152)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) % 72) * 2)))];
    kernel_shared[(((((int)threadIdx.x) * 2) + 1))] = kernel[((((((((int)blockIdx.x) / 14) * 18432) + ((((((int)threadIdx.x) * 2) + 1) / 144) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 1) % 144)))];
    kernel_shared[(((((int)threadIdx.x) * 2) + 224))] = kernel[((((((((int)blockIdx.x) / 14) * 18432) + ((((((int)threadIdx.x) * 2) + 224) / 144) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 80) % 144)))];
    kernel_shared[(((((int)threadIdx.x) * 2) + 225))] = kernel[((((((((int)blockIdx.x) / 14) * 18432) + ((((((int)threadIdx.x) * 2) + 225) / 144) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 81) % 144)))];
    kernel_shared[(((((int)threadIdx.x) * 2) + 448))] = kernel[((((((((int)blockIdx.x) / 14) * 18432) + ((((((int)threadIdx.x) * 2) + 448) / 144) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 16) % 144)))];
    kernel_shared[(((((int)threadIdx.x) * 2) + 449))] = kernel[((((((((int)blockIdx.x) / 14) * 18432) + ((((((int)threadIdx.x) * 2) + 449) / 144) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 17) % 144)))];
    kernel_shared[(((((int)threadIdx.x) * 2) + 672))] = kernel[((((((((int)blockIdx.x) / 14) * 18432) + ((((((int)threadIdx.x) * 2) + 672) / 144) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 96) % 144)))];
    kernel_shared[(((((int)threadIdx.x) * 2) + 673))] = kernel[((((((((int)blockIdx.x) / 14) * 18432) + ((((((int)threadIdx.x) * 2) + 673) / 144) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 97) % 144)))];
    kernel_shared[(((((int)threadIdx.x) * 2) + 896))] = kernel[((((((((int)blockIdx.x) / 14) * 18432) + ((((((int)threadIdx.x) * 2) + 896) / 144) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 32) % 144)))];
    kernel_shared[(((((int)threadIdx.x) * 2) + 897))] = kernel[((((((((int)blockIdx.x) / 14) * 18432) + ((((((int)threadIdx.x) * 2) + 897) / 144) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 33) % 144)))];
    kernel_shared[(((((int)threadIdx.x) * 2) + 1120))] = kernel[((((((((int)blockIdx.x) / 14) * 18432) + ((((((int)threadIdx.x) * 2) + 1120) / 144) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 112) % 144)))];
    kernel_shared[(((((int)threadIdx.x) * 2) + 1121))] = kernel[((((((((int)blockIdx.x) / 14) * 18432) + ((((((int)threadIdx.x) * 2) + 1121) / 144) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 113) % 144)))];
    kernel_shared[(((((int)threadIdx.x) * 2) + 1344))] = kernel[((((((((int)blockIdx.x) / 14) * 18432) + ((((((int)threadIdx.x) * 2) + 1344) / 144) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 48) % 144)))];
    kernel_shared[(((((int)threadIdx.x) * 2) + 1345))] = kernel[((((((((int)blockIdx.x) / 14) * 18432) + ((((((int)threadIdx.x) * 2) + 1345) / 144) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 49) % 144)))];
    kernel_shared[(((((int)threadIdx.x) * 2) + 1568))] = kernel[((((((((int)blockIdx.x) / 14) * 18432) + ((((((int)threadIdx.x) * 2) + 1568) / 144) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 128) % 144)))];
    kernel_shared[(((((int)threadIdx.x) * 2) + 1569))] = kernel[((((((((int)blockIdx.x) / 14) * 18432) + ((((((int)threadIdx.x) * 2) + 1569) / 144) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 129) % 144)))];
    kernel_shared[(((((int)threadIdx.x) * 2) + 1792))] = kernel[((((((((int)blockIdx.x) / 14) * 18432) + ((((((int)threadIdx.x) * 2) + 1792) / 144) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 64) % 144)))];
    kernel_shared[(((((int)threadIdx.x) * 2) + 1793))] = kernel[((((((((int)blockIdx.x) / 14) * 18432) + ((((((int)threadIdx.x) * 2) + 1793) / 144) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 65) % 144)))];
    kernel_shared[(((((int)threadIdx.x) * 2) + 2016))] = kernel[(((((((((int)blockIdx.x) / 14) * 18432) + ((((int)threadIdx.x) / 72) * 1152)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) % 72) * 2)) + 16128))];
    kernel_shared[(((((int)threadIdx.x) * 2) + 2017))] = kernel[((((((((int)blockIdx.x) / 14) * 18432) + ((((((int)threadIdx.x) * 2) + 2017) / 144) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 1) % 144)))];
    if (((int)threadIdx.x) < 32) {
      kernel_shared[(((((int)threadIdx.x) * 2) + 2240))] = kernel[((((((((int)blockIdx.x) / 14) * 18432) + ((((((int)threadIdx.x) * 2) + 2240) / 144) * 1152)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) * 2) + 80)))];
    }
    if (((int)threadIdx.x) < 32) {
      kernel_shared[(((((int)threadIdx.x) * 2) + 2241))] = kernel[((((((((int)blockIdx.x) / 14) * 18432) + ((((((int)threadIdx.x) * 2) + 2241) / 144) * 1152)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) * 2) + 81)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 2))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1152))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 2))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1152))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 1))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 3))] * kernel_shared[((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1152))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 3))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1152))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 6))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 3))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 8))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 3))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 6))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1155))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 8))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1155))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 7))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 3))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 9))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 3))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 7))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1155))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 9))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1155))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 12))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 6))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 14))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 6))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 12))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1158))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 14))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1158))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 13))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 6))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 15))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 6))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 13))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1158))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 15))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1158))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 96))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 9))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 98))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 9))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 96))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1161))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 98))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1161))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 97))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 9))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 99))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 9))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 97))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1161))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 99))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1161))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 102))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 12))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 104))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 12))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 102))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1164))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 104))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1164))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 103))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 12))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 105))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 12))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 103))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1164))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 105))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1164))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 108))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 15))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 110))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 15))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 108))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1167))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 110))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1167))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 109))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 15))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 111))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 15))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 109))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1167))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 111))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1167))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 3))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1153))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 3))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1153))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 2))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 4))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 2))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1153))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 4))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1153))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 7))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 4))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 9))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 4))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 7))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1156))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 9))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1156))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 8))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 4))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 10))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 4))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 8))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1156))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 10))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1156))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 13))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 7))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 15))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 7))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 13))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1159))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 15))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1159))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 14))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 7))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 16))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 7))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 14))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1159))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 16))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1159))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 97))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 10))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 99))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 10))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 97))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1162))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 99))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1162))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 98))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 10))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 100))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 10))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 98))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1162))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 100))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1162))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 103))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 13))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 105))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 13))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 103))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1165))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 105))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1165))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 104))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 13))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 106))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 13))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 104))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1165))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 106))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1165))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 109))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 16))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 111))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 16))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 109))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1168))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 111))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1168))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 110))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 16))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 112))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 16))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 110))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1168))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 112))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1168))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 2))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 2))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 4))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 2))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 2))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1154))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 4))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1154))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 3))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 2))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 5))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 2))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 3))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1154))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 5))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1154))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 8))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 5))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 10))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 5))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 8))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1157))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 10))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1157))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 9))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 5))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 11))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 5))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 9))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1157))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 11))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1157))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 14))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 8))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 16))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 8))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 14))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1160))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 16))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1160))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 15))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 8))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 17))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 8))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 15))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1160))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 17))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1160))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 98))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 11))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 100))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 11))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 98))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1163))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 100))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1163))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 99))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 11))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 101))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 11))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 99))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1163))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 101))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1163))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 104))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 14))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 106))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 14))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 104))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1166))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 106))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1166))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 105))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 14))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 107))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 14))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 105))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1166))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 107))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1166))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 110))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 17))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 112))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 17))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 110))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1169))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 112))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1169))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 111))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 17))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 113))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 17))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 111))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1169))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 192) + ((((int)threadIdx.x) % 14) * 6)) + 113))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 18)) + 1169))]));
    }
  }
  for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
    compute[((((((((((int)blockIdx.x) / 14) * 12544) + ((((int)threadIdx.x) / 14) * 784)) + (((((int)blockIdx.x) % 14) / 7) * 392)) + ((((int)threadIdx.x) % 14) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + xx_inner))] = compute_local[(xx_inner)];
    compute[(((((((((((int)blockIdx.x) / 14) * 12544) + ((((int)threadIdx.x) / 14) * 784)) + (((((int)blockIdx.x) % 14) / 7) * 392)) + ((((int)threadIdx.x) % 14) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + xx_inner) + 2))] = compute_local[((xx_inner + 2))];
    compute[(((((((((((int)blockIdx.x) / 14) * 12544) + ((((int)threadIdx.x) / 14) * 784)) + (((((int)blockIdx.x) % 14) / 7) * 392)) + ((((int)threadIdx.x) % 14) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + xx_inner) + 6272))] = compute_local[((xx_inner + 4))];
    compute[(((((((((((int)blockIdx.x) / 14) * 12544) + ((((int)threadIdx.x) / 14) * 784)) + (((((int)blockIdx.x) % 14) / 7) * 392)) + ((((int)threadIdx.x) % 14) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + xx_inner) + 6274))] = compute_local[((xx_inner + 6))];
  }
}



void conv_kernel_wrapper(int N_B, int N_C, int N_H, int N_W, int N_F, int N_R, int N_S, int PaddingH, int PaddingW,
                        int StrideH, int StrideW, int N_X, int N_Y, const float *Input, 
                        const float *Kernel, float *Output, int itr) {
                        hipEvent_t start;
                        CHECK(hipEventCreate(&start));
                        hipEvent_t stop;
                        CHECK(hipEventCreate(&stop));
                        float *dev_Input;
                        float *dev_Kernel;
                        float *dev_Output;
                    
                        CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                        CHECK(hipMemcpy(dev_Kernel, Kernel, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                        CHECK(hipMemcpy(dev_Input, Input, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Output, sizeof(float) * N_B * N_F * N_Y * N_X));
                        CHECK(hipMemset(dev_Output, 0, sizeof(float) * N_B * N_F * N_Y * N_X));
                    
dim3 size_grid(112,1,1);

dim3 size_block(112,1,1);
 
                       printf("warmup....\n");
                        default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                    
                        double *time_arr = (double *)malloc(itr * sizeof(double));
                        for (int j = 1; j < itr; j++) {
                            CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                            CHECK(hipMemcpy(dev_Kernel, Kernel+j*N_F * N_C * N_R * N_S, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                            CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                            CHECK(hipMemcpy(dev_Input, Input+j*N_B * N_C * N_H * N_W, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                            /* Record the start event*/
                            CHECK(hipEventRecord(start, NULL));
                            default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                            /* Record the end event */
                            CHECK(hipEventRecord(stop, NULL));
                            /* Wait for the stop event to complete*/
                            CHECK(hipEventSynchronize(stop));
                            float msec = 0.0f;
                            hipEventElapsedTime(&msec, start, stop);
                            time_arr[j] = msec;
                        }
                    
                       
                    
                        //hipError_t err = hipGetLastError();
                        //if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
                        double msecTotal = 0;
                        for (int j = 1; j < itr; j++) {   
                            printf("%0.5lf\n", time_arr[j]);
                            msecTotal += time_arr[j];
                        }
                    
                        double msecPerMatrixMul = (msecTotal) / itr;
                        double flopsPerMatrixMul = 2.0 * N_B * N_C * N_X * N_Y * N_F * N_R * N_S;
                    
                        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
                    
                        printf("\n===========================================================\n");
                        printf("Input size: %d X %d X %d X %d\n", N_B, N_C, N_H, N_W);
                        printf("Kernel size: %d X %d X %d X %d\n", N_F, N_C, N_R, N_S);
                        printf("Output size: %d X %d X %d X %d\n", N_B, N_F, N_Y, N_X);
                        printf("flops: %0.5lf\n", (flopsPerMatrixMul * 1.0e-9f));
                        printf("Time is %0.5lf \n", (msecPerMatrixMul / 1000.0f));
                        printf("Gflops %0.5lf \n", gigaFlops);
                        printf("===========================================================\n");
                    
                        /*write back to Host Output -- batch size is 1*/
                        CHECK(hipMemcpy(Output, dev_Output, sizeof(float) * N_B * N_F * N_X * N_Y, hipMemcpyDeviceToHost));
                    
                        CHECK(hipFree(dev_Input));
                        CHECK(hipFree(dev_Kernel));
                        CHECK(hipFree(dev_Output));
                    
                    }
            