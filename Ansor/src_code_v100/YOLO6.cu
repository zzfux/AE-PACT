#include "hip/hip_runtime.h"

            #include <cassert>
            #include <stdlib.h>
            #include <hip/hip_runtime.h>
            #include "common.h"
            

            
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[16];
  __shared__ float pad_temp_shared[544];
  __shared__ float kernel_shared[384];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  compute_local[(8)] = 0.000000e+00f;
  compute_local[(10)] = 0.000000e+00f;
  compute_local[(12)] = 0.000000e+00f;
  compute_local[(14)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(7)] = 0.000000e+00f;
  compute_local[(9)] = 0.000000e+00f;
  compute_local[(11)] = 0.000000e+00f;
  compute_local[(13)] = 0.000000e+00f;
  compute_local[(15)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    for (int rx_outer_outer = 0; rx_outer_outer < 3; ++rx_outer_outer) {
      __syncthreads();
      pad_temp_shared[(((int)threadIdx.x))] = data[((((((rc_outer_outer * 5184) + ((((int)blockIdx.x) % 17) * 72)) + ((((int)threadIdx.x) / 34) * 36)) + rx_outer_outer) + (((int)threadIdx.x) % 34)))];
      pad_temp_shared[((((int)threadIdx.x) + 136))] = data[(((((((rc_outer_outer * 5184) + ((((int)blockIdx.x) % 17) * 72)) + ((((int)threadIdx.x) / 34) * 36)) + rx_outer_outer) + (((int)threadIdx.x) % 34)) + 1296))];
      pad_temp_shared[((((int)threadIdx.x) + 272))] = data[(((((((rc_outer_outer * 5184) + ((((int)blockIdx.x) % 17) * 72)) + ((((int)threadIdx.x) / 34) * 36)) + rx_outer_outer) + (((int)threadIdx.x) % 34)) + 2592))];
      pad_temp_shared[((((int)threadIdx.x) + 408))] = data[(((((((rc_outer_outer * 5184) + ((((int)blockIdx.x) % 17) * 72)) + ((((int)threadIdx.x) / 34) * 36)) + rx_outer_outer) + (((int)threadIdx.x) % 34)) + 3888))];
      kernel_shared[(((int)threadIdx.x))] = kernel[(((((((((int)blockIdx.x) / 17) * 73728) + ((((int)threadIdx.x) / 12) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) % 12) * 3)) + rx_outer_outer))];
      kernel_shared[((((int)threadIdx.x) + 136))] = kernel[(((((((((int)blockIdx.x) / 17) * 73728) + (((((int)threadIdx.x) + 136) / 12) * 2304)) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) + 4) % 12) * 3)) + rx_outer_outer))];
      if (((int)threadIdx.x) < 112) {
        kernel_shared[((((int)threadIdx.x) + 272))] = kernel[(((((((((int)blockIdx.x) / 17) * 73728) + (((((int)threadIdx.x) + 272) / 12) * 2304)) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) + 8) % 12) * 3)) + rx_outer_outer))];
      }
      __syncthreads();
      for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)))] * kernel_shared[((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)))]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 34))] * kernel_shared[((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)))]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 12))]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 34))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 12))]));
        compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 24))]));
        compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 34))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 24))]));
        compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 36))]));
        compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 34))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 36))]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 34))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 1))]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 68))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 1))]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 34))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 13))]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 68))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 13))]));
        compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 34))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 25))]));
        compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 68))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 25))]));
        compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 34))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 37))]));
        compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 68))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 37))]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 68))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 2))]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 102))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 2))]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 68))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 14))]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 102))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 14))]));
        compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 68))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 26))]));
        compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 102))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 26))]));
        compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 68))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 38))]));
        compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 102))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 38))]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 1))] * kernel_shared[((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)))]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 35))] * kernel_shared[((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)))]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 12))]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 35))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 12))]));
        compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 24))]));
        compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 35))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 24))]));
        compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 36))]));
        compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 35))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 36))]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 35))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 1))]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 69))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 1))]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 35))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 13))]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 69))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 13))]));
        compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 35))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 25))]));
        compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 69))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 25))]));
        compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 35))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 37))]));
        compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 69))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 37))]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 69))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 2))]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 103))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 2))]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 69))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 14))]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 103))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 14))]));
        compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 69))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 26))]));
        compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 103))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 26))]));
        compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 69))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 38))]));
        compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((rc_outer_inner * 136) + ((((int)threadIdx.x) % 17) * 2)) + 103))] * kernel_shared[(((((((int)threadIdx.x) / 17) * 48) + (rc_outer_inner * 3)) + 38))]));
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
      for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
        compute[(((((((((((int)blockIdx.x) / 17) * 36992) + ((((int)threadIdx.x) / 17) * 4624)) + (ff_inner * 1156)) + ((((int)blockIdx.x) % 17) * 68)) + (yy_inner * 34)) + ((((int)threadIdx.x) % 17) * 2)) + xx_inner))] = compute_local[((((ff_inner * 4) + (yy_inner * 2)) + xx_inner))];
      }
    }
  }
}



void conv_kernel_wrapper(int N_B, int N_C, int N_H, int N_W, int N_F, int N_R, int N_S, int PaddingH, int PaddingW,
                        int StrideH, int StrideW, int N_X, int N_Y, const float *Input, 
                        const float *Kernel, float *Output, int itr) {
                        hipEvent_t start;
                        CHECK(hipEventCreate(&start));
                        hipEvent_t stop;
                        CHECK(hipEventCreate(&stop));
                        float *dev_Input;
                        float *dev_Kernel;
                        float *dev_Output;
                    
                        CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                        CHECK(hipMemcpy(dev_Kernel, Kernel, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                        CHECK(hipMemcpy(dev_Input, Input, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Output, sizeof(float) * N_B * N_F * N_Y * N_X));
                        CHECK(hipMemset(dev_Output, 0, sizeof(float) * N_B * N_F * N_Y * N_X));
                    
dim3 size_grid(272,1,1);

dim3 size_block(136,1,1);
 
                       printf("warmup....\n");
                        default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                    
                        double *time_arr = (double *)malloc(itr * sizeof(double));
                        for (int j = 1; j < itr; j++) {
                            CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                            CHECK(hipMemcpy(dev_Kernel, Kernel+j*N_F * N_C * N_R * N_S, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                            CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                            CHECK(hipMemcpy(dev_Input, Input+j*N_B * N_C * N_H * N_W, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                            /* Record the start event*/
                            CHECK(hipEventRecord(start, NULL));
                            default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                            /* Record the end event */
                            CHECK(hipEventRecord(stop, NULL));
                            /* Wait for the stop event to complete*/
                            CHECK(hipEventSynchronize(stop));
                            float msec = 0.0f;
                            hipEventElapsedTime(&msec, start, stop);
                            time_arr[j] = msec;
                        }
                    
                       
                    
                        //hipError_t err = hipGetLastError();
                        //if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
                        double msecTotal = 0;
                        for (int j = 1; j < itr; j++) {   
                            printf("%0.5lf\n", time_arr[j]);
                            msecTotal += time_arr[j];
                        }
                    
                        double msecPerMatrixMul = (msecTotal) / itr;
                        double flopsPerMatrixMul = 2.0 * N_B * N_C * N_X * N_Y * N_F * N_R * N_S;
                    
                        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
                    
                        printf("\n===========================================================\n");
                        printf("Input size: %d X %d X %d X %d\n", N_B, N_C, N_H, N_W);
                        printf("Kernel size: %d X %d X %d X %d\n", N_F, N_C, N_R, N_S);
                        printf("Output size: %d X %d X %d X %d\n", N_B, N_F, N_Y, N_X);
                        printf("flops: %0.5lf\n", (flopsPerMatrixMul * 1.0e-9f));
                        printf("Time is %0.5lf \n", (msecPerMatrixMul / 1000.0f));
                        printf("Gflops %0.5lf \n", gigaFlops);
                        printf("===========================================================\n");
                    
                        /*write back to Host Output -- batch size is 1*/
                        CHECK(hipMemcpy(Output, dev_Output, sizeof(float) * N_B * N_F * N_X * N_Y, hipMemcpyDeviceToHost));
                    
                        CHECK(hipFree(dev_Input));
                        CHECK(hipFree(dev_Kernel));
                        CHECK(hipFree(dev_Output));
                    
                    }
            