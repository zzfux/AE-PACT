#include "hip/hip_runtime.h"

            #include <cassert>
            #include <stdlib.h>
            #include <hip/hip_runtime.h>
            #include "common.h"
            

            
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[32];
  __shared__ float pad_temp_shared[1680];
  __shared__ float kernel_shared[1152];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(8)] = 0.000000e+00f;
  compute_local[(12)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(9)] = 0.000000e+00f;
  compute_local[(13)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  compute_local[(10)] = 0.000000e+00f;
  compute_local[(14)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(7)] = 0.000000e+00f;
  compute_local[(11)] = 0.000000e+00f;
  compute_local[(15)] = 0.000000e+00f;
  compute_local[(16)] = 0.000000e+00f;
  compute_local[(20)] = 0.000000e+00f;
  compute_local[(24)] = 0.000000e+00f;
  compute_local[(28)] = 0.000000e+00f;
  compute_local[(17)] = 0.000000e+00f;
  compute_local[(21)] = 0.000000e+00f;
  compute_local[(25)] = 0.000000e+00f;
  compute_local[(29)] = 0.000000e+00f;
  compute_local[(18)] = 0.000000e+00f;
  compute_local[(22)] = 0.000000e+00f;
  compute_local[(26)] = 0.000000e+00f;
  compute_local[(30)] = 0.000000e+00f;
  compute_local[(19)] = 0.000000e+00f;
  compute_local[(23)] = 0.000000e+00f;
  compute_local[(27)] = 0.000000e+00f;
  compute_local[(31)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 240) {
      pad_temp_shared[((((int)threadIdx.x) * 7))] = data[(((((rc_outer_outer * 19600) + ((((int)threadIdx.x) / 60) * 4900)) + ((((int)blockIdx.x) % 17) * 280)) + ((((int)threadIdx.x) % 60) * 7)))];
    }
    if (((int)threadIdx.x) < 240) {
      pad_temp_shared[(((((int)threadIdx.x) * 7) + 1))] = data[(((((rc_outer_outer * 19600) + ((((((int)threadIdx.x) * 7) + 1) / 420) * 4900)) + ((((int)blockIdx.x) % 17) * 280)) + (((((int)threadIdx.x) * 7) + 1) % 420)))];
    }
    if (((int)threadIdx.x) < 240) {
      pad_temp_shared[(((((int)threadIdx.x) * 7) + 2))] = data[(((((rc_outer_outer * 19600) + ((((((int)threadIdx.x) * 7) + 2) / 420) * 4900)) + ((((int)blockIdx.x) % 17) * 280)) + (((((int)threadIdx.x) * 7) + 2) % 420)))];
    }
    if (((int)threadIdx.x) < 240) {
      pad_temp_shared[(((((int)threadIdx.x) * 7) + 3))] = data[(((((rc_outer_outer * 19600) + ((((((int)threadIdx.x) * 7) + 3) / 420) * 4900)) + ((((int)blockIdx.x) % 17) * 280)) + (((((int)threadIdx.x) * 7) + 3) % 420)))];
    }
    if (((int)threadIdx.x) < 240) {
      pad_temp_shared[(((((int)threadIdx.x) * 7) + 4))] = data[(((((rc_outer_outer * 19600) + ((((((int)threadIdx.x) * 7) + 4) / 420) * 4900)) + ((((int)blockIdx.x) % 17) * 280)) + (((((int)threadIdx.x) * 7) + 4) % 420)))];
    }
    if (((int)threadIdx.x) < 240) {
      pad_temp_shared[(((((int)threadIdx.x) * 7) + 5))] = data[(((((rc_outer_outer * 19600) + ((((((int)threadIdx.x) * 7) + 5) / 420) * 4900)) + ((((int)blockIdx.x) % 17) * 280)) + (((((int)threadIdx.x) * 7) + 5) % 420)))];
    }
    if (((int)threadIdx.x) < 240) {
      pad_temp_shared[(((((int)threadIdx.x) * 7) + 6))] = data[(((((rc_outer_outer * 19600) + ((((((int)threadIdx.x) * 7) + 6) / 420) * 4900)) + ((((int)blockIdx.x) % 17) * 280)) + (((((int)threadIdx.x) * 7) + 6) % 420)))];
    }
    kernel_shared[((((int)threadIdx.x) * 3))] = kernel[((((((((int)blockIdx.x) / 17) * 36864) + ((((int)threadIdx.x) / 12) * 1152)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) % 12) * 3)))];
    kernel_shared[(((((int)threadIdx.x) * 3) + 1))] = kernel[(((((((((int)blockIdx.x) / 17) * 36864) + ((((int)threadIdx.x) / 12) * 1152)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) % 12) * 3)) + 1))];
    kernel_shared[(((((int)threadIdx.x) * 3) + 2))] = kernel[(((((((((int)blockIdx.x) / 17) * 36864) + ((((int)threadIdx.x) / 12) * 1152)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) % 12) * 3)) + 2))];
    if (((int)threadIdx.x) < 112) {
      kernel_shared[(((((int)threadIdx.x) * 3) + 816))] = kernel[((((((((int)blockIdx.x) / 17) * 36864) + (((((int)threadIdx.x) + 272) / 12) * 1152)) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) + 8) % 12) * 3)))];
      kernel_shared[(((((int)threadIdx.x) * 3) + 817))] = kernel[(((((((((int)blockIdx.x) / 17) * 36864) + (((((int)threadIdx.x) + 272) / 12) * 1152)) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) + 8) % 12) * 3)) + 1))];
      kernel_shared[(((((int)threadIdx.x) * 3) + 818))] = kernel[(((((((((int)blockIdx.x) / 17) * 36864) + (((((int)threadIdx.x) + 272) / 12) * 1152)) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) + 8) % 12) * 3)) + 2))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)))] * kernel_shared[((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 36))]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 72))]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 108))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 70))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 3))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 70))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 39))]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 70))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 75))]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 70))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 111))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 140))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 6))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 140))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 42))]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 140))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 78))]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 140))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 114))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 70))] * kernel_shared[((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 70))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 36))]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 70))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 72))]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 70))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 108))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 140))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 3))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 140))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 39))]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 140))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 75))]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 140))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 111))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 210))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 6))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 210))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 42))]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 210))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 78))]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 210))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 114))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 140))] * kernel_shared[((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 140))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 36))]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 140))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 72))]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 140))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 108))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 210))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 3))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 210))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 39))]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 210))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 75))]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 210))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 111))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 280))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 6))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 280))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 42))]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 280))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 78))]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 280))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 114))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 210))] * kernel_shared[((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 210))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 36))]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 210))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 72))]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 210))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 108))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 280))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 3))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 280))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 39))]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 280))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 75))]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 280))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 111))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 350))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 6))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 350))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 42))]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 350))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 78))]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 350))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 114))]));
      compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 144))]));
      compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[(((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 180))]));
      compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[(((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 216))]));
      compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[(((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 252))]));
      compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 70))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 147))]));
      compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 70))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 183))]));
      compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 70))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 219))]));
      compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 70))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 255))]));
      compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 140))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 150))]));
      compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 140))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 186))]));
      compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 140))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 222))]));
      compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 140))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 258))]));
      compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 70))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 144))]));
      compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 70))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 180))]));
      compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 70))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 216))]));
      compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 70))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 252))]));
      compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 140))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 147))]));
      compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 140))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 183))]));
      compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 140))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 219))]));
      compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 140))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 255))]));
      compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 210))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 150))]));
      compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 210))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 186))]));
      compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 210))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 222))]));
      compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 210))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 258))]));
      compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 140))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 144))]));
      compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 140))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 180))]));
      compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 140))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 216))]));
      compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 140))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 252))]));
      compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 210))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 147))]));
      compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 210))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 183))]));
      compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 210))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 219))]));
      compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 210))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 255))]));
      compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 280))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 150))]));
      compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 280))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 186))]));
      compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 280))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 222))]));
      compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 280))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 258))]));
      compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 210))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 144))]));
      compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 210))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 180))]));
      compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 210))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 216))]));
      compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 210))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 252))]));
      compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 280))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 147))]));
      compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 280))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 183))]));
      compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 280))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 219))]));
      compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 280))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 255))]));
      compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 350))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 150))]));
      compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 350))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 186))]));
      compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 350))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 222))]));
      compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 350))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 258))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 1))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 37))]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 73))]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 109))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 71))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 4))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 71))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 40))]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 71))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 76))]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 71))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 112))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 141))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 7))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 141))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 43))]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 141))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 79))]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 141))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 115))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 71))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 1))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 71))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 37))]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 71))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 73))]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 71))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 109))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 141))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 4))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 141))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 40))]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 141))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 76))]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 141))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 112))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 211))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 7))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 211))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 43))]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 211))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 79))]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 211))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 115))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 141))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 1))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 141))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 37))]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 141))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 73))]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 141))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 109))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 211))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 4))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 211))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 40))]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 211))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 76))]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 211))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 112))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 281))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 7))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 281))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 43))]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 281))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 79))]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 281))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 115))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 211))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 1))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 211))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 37))]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 211))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 73))]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 211))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 109))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 281))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 4))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 281))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 40))]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 281))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 76))]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 281))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 112))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 351))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 7))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 351))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 43))]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 351))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 79))]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 351))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 115))]));
      compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 145))]));
      compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 181))]));
      compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 217))]));
      compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 1))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 253))]));
      compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 71))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 148))]));
      compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 71))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 184))]));
      compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 71))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 220))]));
      compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 71))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 256))]));
      compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 141))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 151))]));
      compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 141))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 187))]));
      compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 141))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 223))]));
      compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 141))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 259))]));
      compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 71))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 145))]));
      compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 71))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 181))]));
      compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 71))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 217))]));
      compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 71))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 253))]));
      compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 141))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 148))]));
      compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 141))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 184))]));
      compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 141))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 220))]));
      compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 141))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 256))]));
      compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 211))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 151))]));
      compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 211))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 187))]));
      compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 211))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 223))]));
      compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 211))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 259))]));
      compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 141))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 145))]));
      compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 141))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 181))]));
      compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 141))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 217))]));
      compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 141))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 253))]));
      compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 211))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 148))]));
      compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 211))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 184))]));
      compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 211))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 220))]));
      compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 211))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 256))]));
      compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 281))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 151))]));
      compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 281))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 187))]));
      compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 281))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 223))]));
      compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 281))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 259))]));
      compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 211))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 145))]));
      compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 211))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 181))]));
      compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 211))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 217))]));
      compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 211))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 253))]));
      compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 281))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 148))]));
      compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 281))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 184))]));
      compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 281))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 220))]));
      compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 281))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 256))]));
      compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 351))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 151))]));
      compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 351))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 187))]));
      compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 351))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 223))]));
      compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 351))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 259))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 2))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 2))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 2))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 38))]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 2))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 74))]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 2))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 110))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 72))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 5))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 72))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 41))]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 72))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 77))]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 72))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 113))]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 142))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 8))]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 142))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 44))]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 142))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 80))]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 142))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 116))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 72))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 2))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 72))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 38))]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 72))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 74))]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 72))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 110))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 142))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 5))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 142))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 41))]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 142))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 77))]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 142))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 113))]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 212))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 8))]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 212))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 44))]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 212))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 80))]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 212))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 116))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 142))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 2))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 142))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 38))]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 142))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 74))]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 142))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 110))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 212))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 5))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 212))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 41))]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 212))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 77))]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 212))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 113))]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 282))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 8))]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 282))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 44))]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 282))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 80))]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 282))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 116))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 212))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 2))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 212))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 38))]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 212))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 74))]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 212))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 110))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 282))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 5))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 282))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 41))]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 282))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 77))]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 282))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 113))]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 352))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 8))]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 352))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 44))]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 352))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 80))]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 352))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 116))]));
      compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 2))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 146))]));
      compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 2))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 182))]));
      compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 2))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 218))]));
      compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 2))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 254))]));
      compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 72))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 149))]));
      compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 72))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 185))]));
      compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 72))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 221))]));
      compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 72))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 257))]));
      compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 142))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 152))]));
      compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 142))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 188))]));
      compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 142))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 224))]));
      compute_local[(28)] = (compute_local[(28)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 142))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 260))]));
      compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 72))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 146))]));
      compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 72))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 182))]));
      compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 72))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 218))]));
      compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 72))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 254))]));
      compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 142))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 149))]));
      compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 142))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 185))]));
      compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 142))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 221))]));
      compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 142))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 257))]));
      compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 212))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 152))]));
      compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 212))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 188))]));
      compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 212))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 224))]));
      compute_local[(29)] = (compute_local[(29)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 212))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 260))]));
      compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 142))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 146))]));
      compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 142))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 182))]));
      compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 142))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 218))]));
      compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 142))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 254))]));
      compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 212))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 149))]));
      compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 212))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 185))]));
      compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 212))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 221))]));
      compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 212))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 257))]));
      compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 282))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 152))]));
      compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 282))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 188))]));
      compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 282))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 224))]));
      compute_local[(30)] = (compute_local[(30)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 282))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 260))]));
      compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 212))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 146))]));
      compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 212))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 182))]));
      compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 212))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 218))]));
      compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 212))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 254))]));
      compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 282))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 149))]));
      compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 282))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 185))]));
      compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 282))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 221))]));
      compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 282))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 257))]));
      compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 352))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 152))]));
      compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 352))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 188))]));
      compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 352))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 224))]));
      compute_local[(31)] = (compute_local[(31)] + (pad_temp_shared[((((rc_outer_inner * 420) + (((int)threadIdx.x) % 68)) + 352))] * kernel_shared[(((((((int)threadIdx.x) / 68) * 288) + (rc_outer_inner * 9)) + 260))]));
    }
  }
  for (int ff_inner = 0; ff_inner < 8; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 4; ++yy_inner) {
      compute[((((((((((int)blockIdx.x) / 17) * 147968) + ((((int)threadIdx.x) / 68) * 36992)) + (ff_inner * 4624)) + ((((int)blockIdx.x) % 17) * 272)) + (yy_inner * 68)) + (((int)threadIdx.x) % 68)))] = compute_local[(((ff_inner * 4) + yy_inner))];
    }
  }
}



void conv_kernel_wrapper(int N_B, int N_C, int N_H, int N_W, int N_F, int N_R, int N_S, int PaddingH, int PaddingW,
                        int StrideH, int StrideW, int N_X, int N_Y, const float *Input, 
                        const float *Kernel, float *Output, int itr) {
                        hipEvent_t start;
                        CHECK(hipEventCreate(&start));
                        hipEvent_t stop;
                        CHECK(hipEventCreate(&stop));
                        float *dev_Input;
                        float *dev_Kernel;
                        float *dev_Output;
                    
                        CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                        CHECK(hipMemcpy(dev_Kernel, Kernel, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                        CHECK(hipMemcpy(dev_Input, Input, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Output, sizeof(float) * N_B * N_F * N_Y * N_X));
                        CHECK(hipMemset(dev_Output, 0, sizeof(float) * N_B * N_F * N_Y * N_X));
                    
dim3 size_grid(136,1,1);

dim3 size_block(272,1,1);
 
                       printf("warmup....\n");
                        default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                    
                        double *time_arr = (double *)malloc(itr * sizeof(double));
                        for (int j = 1; j < itr; j++) {
                            CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                            CHECK(hipMemcpy(dev_Kernel, Kernel+j*N_F * N_C * N_R * N_S, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                            CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                            CHECK(hipMemcpy(dev_Input, Input+j*N_B * N_C * N_H * N_W, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                            /* Record the start event*/
                            CHECK(hipEventRecord(start, NULL));
                            default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                            /* Record the end event */
                            CHECK(hipEventRecord(stop, NULL));
                            /* Wait for the stop event to complete*/
                            CHECK(hipEventSynchronize(stop));
                            float msec = 0.0f;
                            hipEventElapsedTime(&msec, start, stop);
                            time_arr[j] = msec;
                        }
                    
                       
                    
                        //hipError_t err = hipGetLastError();
                        //if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
                        double msecTotal = 0;
                        for (int j = 1; j < itr; j++) {   
                            printf("%0.5lf\n", time_arr[j]);
                            msecTotal += time_arr[j];
                        }
                    
                        double msecPerMatrixMul = (msecTotal) / itr;
                        double flopsPerMatrixMul = 2.0 * N_B * N_C * N_X * N_Y * N_F * N_R * N_S;
                    
                        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
                    
                        printf("\n===========================================================\n");
                        printf("Input size: %d X %d X %d X %d\n", N_B, N_C, N_H, N_W);
                        printf("Kernel size: %d X %d X %d X %d\n", N_F, N_C, N_R, N_S);
                        printf("Output size: %d X %d X %d X %d\n", N_B, N_F, N_Y, N_X);
                        printf("flops: %0.5lf\n", (flopsPerMatrixMul * 1.0e-9f));
                        printf("Time is %0.5lf \n", (msecPerMatrixMul / 1000.0f));
                        printf("Gflops %0.5lf \n", gigaFlops);
                        printf("===========================================================\n");
                    
                        /*write back to Host Output -- batch size is 1*/
                        CHECK(hipMemcpy(Output, dev_Output, sizeof(float) * N_B * N_F * N_X * N_Y, hipMemcpyDeviceToHost));
                    
                        CHECK(hipFree(dev_Input));
                        CHECK(hipFree(dev_Kernel));
                        CHECK(hipFree(dev_Output));
                    
                    }
            