#include "hip/hip_runtime.h"

            #include <cassert>
            #include <stdlib.h>
            #include <hip/hip_runtime.h>
            #include "common.h"
            

            
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[1];
  __shared__ float pad_temp_shared[208];
  __shared__ float kernel_shared[256];
  compute_local[(0)] = 0.000000e+00f;
  pad_temp_shared[(((int)threadIdx.x))] = data[(((((((int)threadIdx.x) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 13)))];
  if (((int)threadIdx.x) < 96) {
    pad_temp_shared[((((int)threadIdx.x) + 112))] = data[((((((((int)threadIdx.x) + 112) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 8) % 13)))];
  }
  kernel_shared[(((int)threadIdx.x))] = kernel[(((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)))];
  kernel_shared[((((int)threadIdx.x) + 112))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 1792))];
  if (((int)threadIdx.x) < 32) {
    kernel_shared[((((int)threadIdx.x) + 224))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 3584))];
  }
  __syncthreads();
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 2))] * kernel_shared[(((((int)threadIdx.x) / 7) * 16))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 13))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 1))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 26))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 2))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 39))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 3))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 52))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 4))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 65))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 5))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 78))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 6))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 91))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 7))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 104))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 8))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 117))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 9))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 130))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 10))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 143))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 11))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 156))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 12))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 169))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 13))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 182))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 14))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 195))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 15))]));
  __syncthreads();
  pad_temp_shared[(((int)threadIdx.x))] = data[((((((((int)threadIdx.x) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 13)) + 3136))];
  if (((int)threadIdx.x) < 96) {
    pad_temp_shared[((((int)threadIdx.x) + 112))] = data[(((((((((int)threadIdx.x) + 112) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 8) % 13)) + 3136))];
  }
  kernel_shared[(((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 16))];
  kernel_shared[((((int)threadIdx.x) + 112))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 1808))];
  if (((int)threadIdx.x) < 32) {
    kernel_shared[((((int)threadIdx.x) + 224))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 3600))];
  }
  __syncthreads();
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 2))] * kernel_shared[(((((int)threadIdx.x) / 7) * 16))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 13))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 1))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 26))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 2))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 39))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 3))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 52))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 4))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 65))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 5))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 78))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 6))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 91))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 7))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 104))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 8))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 117))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 9))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 130))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 10))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 143))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 11))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 156))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 12))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 169))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 13))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 182))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 14))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 195))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 15))]));
  __syncthreads();
  pad_temp_shared[(((int)threadIdx.x))] = data[((((((((int)threadIdx.x) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 13)) + 6272))];
  if (((int)threadIdx.x) < 96) {
    pad_temp_shared[((((int)threadIdx.x) + 112))] = data[(((((((((int)threadIdx.x) + 112) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 8) % 13)) + 6272))];
  }
  kernel_shared[(((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 32))];
  kernel_shared[((((int)threadIdx.x) + 112))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 1824))];
  if (((int)threadIdx.x) < 32) {
    kernel_shared[((((int)threadIdx.x) + 224))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 3616))];
  }
  __syncthreads();
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 2))] * kernel_shared[(((((int)threadIdx.x) / 7) * 16))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 13))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 1))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 26))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 2))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 39))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 3))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 52))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 4))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 65))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 5))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 78))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 6))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 91))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 7))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 104))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 8))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 117))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 9))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 130))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 10))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 143))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 11))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 156))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 12))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 169))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 13))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 182))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 14))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 195))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 15))]));
  __syncthreads();
  pad_temp_shared[(((int)threadIdx.x))] = data[((((((((int)threadIdx.x) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 13)) + 9408))];
  if (((int)threadIdx.x) < 96) {
    pad_temp_shared[((((int)threadIdx.x) + 112))] = data[(((((((((int)threadIdx.x) + 112) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 8) % 13)) + 9408))];
  }
  kernel_shared[(((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 48))];
  kernel_shared[((((int)threadIdx.x) + 112))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 1840))];
  if (((int)threadIdx.x) < 32) {
    kernel_shared[((((int)threadIdx.x) + 224))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 3632))];
  }
  __syncthreads();
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 2))] * kernel_shared[(((((int)threadIdx.x) / 7) * 16))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 13))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 1))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 26))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 2))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 39))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 3))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 52))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 4))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 65))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 5))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 78))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 6))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 91))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 7))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 104))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 8))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 117))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 9))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 130))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 10))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 143))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 11))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 156))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 12))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 169))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 13))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 182))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 14))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 195))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 15))]));
  __syncthreads();
  pad_temp_shared[(((int)threadIdx.x))] = data[((((((((int)threadIdx.x) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 13)) + 12544))];
  if (((int)threadIdx.x) < 96) {
    pad_temp_shared[((((int)threadIdx.x) + 112))] = data[(((((((((int)threadIdx.x) + 112) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 8) % 13)) + 12544))];
  }
  kernel_shared[(((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 64))];
  kernel_shared[((((int)threadIdx.x) + 112))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 1856))];
  if (((int)threadIdx.x) < 32) {
    kernel_shared[((((int)threadIdx.x) + 224))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 3648))];
  }
  __syncthreads();
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 2))] * kernel_shared[(((((int)threadIdx.x) / 7) * 16))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 13))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 1))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 26))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 2))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 39))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 3))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 52))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 4))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 65))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 5))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 78))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 6))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 91))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 7))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 104))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 8))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 117))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 9))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 130))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 10))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 143))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 11))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 156))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 12))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 169))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 13))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 182))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 14))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 195))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 15))]));
  __syncthreads();
  pad_temp_shared[(((int)threadIdx.x))] = data[((((((((int)threadIdx.x) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 13)) + 15680))];
  if (((int)threadIdx.x) < 96) {
    pad_temp_shared[((((int)threadIdx.x) + 112))] = data[(((((((((int)threadIdx.x) + 112) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 8) % 13)) + 15680))];
  }
  kernel_shared[(((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 80))];
  kernel_shared[((((int)threadIdx.x) + 112))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 1872))];
  if (((int)threadIdx.x) < 32) {
    kernel_shared[((((int)threadIdx.x) + 224))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 3664))];
  }
  __syncthreads();
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 2))] * kernel_shared[(((((int)threadIdx.x) / 7) * 16))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 13))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 1))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 26))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 2))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 39))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 3))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 52))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 4))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 65))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 5))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 78))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 6))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 91))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 7))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 104))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 8))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 117))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 9))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 130))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 10))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 143))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 11))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 156))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 12))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 169))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 13))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 182))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 14))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 195))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 15))]));
  __syncthreads();
  pad_temp_shared[(((int)threadIdx.x))] = data[((((((((int)threadIdx.x) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 13)) + 18816))];
  if (((int)threadIdx.x) < 96) {
    pad_temp_shared[((((int)threadIdx.x) + 112))] = data[(((((((((int)threadIdx.x) + 112) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 8) % 13)) + 18816))];
  }
  kernel_shared[(((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 96))];
  kernel_shared[((((int)threadIdx.x) + 112))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 1888))];
  if (((int)threadIdx.x) < 32) {
    kernel_shared[((((int)threadIdx.x) + 224))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 3680))];
  }
  __syncthreads();
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 2))] * kernel_shared[(((((int)threadIdx.x) / 7) * 16))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 13))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 1))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 26))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 2))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 39))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 3))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 52))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 4))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 65))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 5))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 78))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 6))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 91))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 7))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 104))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 8))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 117))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 9))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 130))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 10))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 143))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 11))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 156))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 12))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 169))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 13))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 182))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 14))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 195))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 15))]));
  __syncthreads();
  pad_temp_shared[(((int)threadIdx.x))] = data[((((((((int)threadIdx.x) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 13)) + 21952))];
  if (((int)threadIdx.x) < 96) {
    pad_temp_shared[((((int)threadIdx.x) + 112))] = data[(((((((((int)threadIdx.x) + 112) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 8) % 13)) + 21952))];
  }
  kernel_shared[(((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 112))];
  kernel_shared[((((int)threadIdx.x) + 112))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 1904))];
  if (((int)threadIdx.x) < 32) {
    kernel_shared[((((int)threadIdx.x) + 224))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 3696))];
  }
  __syncthreads();
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 2))] * kernel_shared[(((((int)threadIdx.x) / 7) * 16))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 13))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 1))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 26))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 2))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 39))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 3))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 52))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 4))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 65))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 5))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 78))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 6))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 91))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 7))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 104))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 8))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 117))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 9))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 130))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 10))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 143))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 11))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 156))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 12))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 169))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 13))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 182))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 14))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 195))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 15))]));
  __syncthreads();
  pad_temp_shared[(((int)threadIdx.x))] = data[((((((((int)threadIdx.x) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 13)) + 25088))];
  if (((int)threadIdx.x) < 96) {
    pad_temp_shared[((((int)threadIdx.x) + 112))] = data[(((((((((int)threadIdx.x) + 112) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 8) % 13)) + 25088))];
  }
  kernel_shared[(((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 128))];
  kernel_shared[((((int)threadIdx.x) + 112))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 1920))];
  if (((int)threadIdx.x) < 32) {
    kernel_shared[((((int)threadIdx.x) + 224))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 3712))];
  }
  __syncthreads();
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 2))] * kernel_shared[(((((int)threadIdx.x) / 7) * 16))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 13))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 1))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 26))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 2))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 39))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 3))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 52))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 4))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 65))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 5))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 78))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 6))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 91))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 7))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 104))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 8))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 117))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 9))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 130))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 10))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 143))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 11))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 156))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 12))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 169))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 13))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 182))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 14))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 195))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 15))]));
  __syncthreads();
  pad_temp_shared[(((int)threadIdx.x))] = data[((((((((int)threadIdx.x) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 13)) + 28224))];
  if (((int)threadIdx.x) < 96) {
    pad_temp_shared[((((int)threadIdx.x) + 112))] = data[(((((((((int)threadIdx.x) + 112) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 8) % 13)) + 28224))];
  }
  kernel_shared[(((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 144))];
  kernel_shared[((((int)threadIdx.x) + 112))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 1936))];
  if (((int)threadIdx.x) < 32) {
    kernel_shared[((((int)threadIdx.x) + 224))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 3728))];
  }
  __syncthreads();
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 2))] * kernel_shared[(((((int)threadIdx.x) / 7) * 16))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 13))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 1))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 26))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 2))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 39))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 3))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 52))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 4))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 65))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 5))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 78))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 6))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 91))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 7))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 104))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 8))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 117))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 9))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 130))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 10))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 143))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 11))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 156))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 12))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 169))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 13))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 182))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 14))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 195))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 15))]));
  __syncthreads();
  pad_temp_shared[(((int)threadIdx.x))] = data[((((((((int)threadIdx.x) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 13)) + 31360))];
  if (((int)threadIdx.x) < 96) {
    pad_temp_shared[((((int)threadIdx.x) + 112))] = data[(((((((((int)threadIdx.x) + 112) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 8) % 13)) + 31360))];
  }
  kernel_shared[(((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 160))];
  kernel_shared[((((int)threadIdx.x) + 112))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 1952))];
  if (((int)threadIdx.x) < 32) {
    kernel_shared[((((int)threadIdx.x) + 224))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 3744))];
  }
  __syncthreads();
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 2))] * kernel_shared[(((((int)threadIdx.x) / 7) * 16))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 13))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 1))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 26))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 2))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 39))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 3))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 52))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 4))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 65))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 5))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 78))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 6))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 91))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 7))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 104))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 8))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 117))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 9))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 130))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 10))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 143))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 11))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 156))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 12))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 169))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 13))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 182))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 14))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 195))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 15))]));
  __syncthreads();
  pad_temp_shared[(((int)threadIdx.x))] = data[((((((((int)threadIdx.x) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 13)) + 34496))];
  if (((int)threadIdx.x) < 96) {
    pad_temp_shared[((((int)threadIdx.x) + 112))] = data[(((((((((int)threadIdx.x) + 112) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 8) % 13)) + 34496))];
  }
  kernel_shared[(((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 176))];
  kernel_shared[((((int)threadIdx.x) + 112))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 1968))];
  if (((int)threadIdx.x) < 32) {
    kernel_shared[((((int)threadIdx.x) + 224))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 3760))];
  }
  __syncthreads();
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 2))] * kernel_shared[(((((int)threadIdx.x) / 7) * 16))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 13))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 1))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 26))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 2))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 39))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 3))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 52))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 4))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 65))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 5))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 78))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 6))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 91))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 7))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 104))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 8))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 117))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 9))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 130))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 10))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 143))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 11))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 156))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 12))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 169))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 13))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 182))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 14))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 195))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 15))]));
  __syncthreads();
  pad_temp_shared[(((int)threadIdx.x))] = data[((((((((int)threadIdx.x) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 13)) + 37632))];
  if (((int)threadIdx.x) < 96) {
    pad_temp_shared[((((int)threadIdx.x) + 112))] = data[(((((((((int)threadIdx.x) + 112) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 8) % 13)) + 37632))];
  }
  kernel_shared[(((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 192))];
  kernel_shared[((((int)threadIdx.x) + 112))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 1984))];
  if (((int)threadIdx.x) < 32) {
    kernel_shared[((((int)threadIdx.x) + 224))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 3776))];
  }
  __syncthreads();
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 2))] * kernel_shared[(((((int)threadIdx.x) / 7) * 16))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 13))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 1))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 26))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 2))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 39))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 3))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 52))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 4))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 65))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 5))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 78))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 6))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 91))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 7))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 104))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 8))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 117))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 9))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 130))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 10))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 143))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 11))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 156))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 12))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 169))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 13))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 182))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 14))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 195))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 15))]));
  __syncthreads();
  pad_temp_shared[(((int)threadIdx.x))] = data[((((((((int)threadIdx.x) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 13)) + 40768))];
  if (((int)threadIdx.x) < 96) {
    pad_temp_shared[((((int)threadIdx.x) + 112))] = data[(((((((((int)threadIdx.x) + 112) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 8) % 13)) + 40768))];
  }
  kernel_shared[(((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 208))];
  kernel_shared[((((int)threadIdx.x) + 112))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 2000))];
  if (((int)threadIdx.x) < 32) {
    kernel_shared[((((int)threadIdx.x) + 224))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 3792))];
  }
  __syncthreads();
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 2))] * kernel_shared[(((((int)threadIdx.x) / 7) * 16))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 13))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 1))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 26))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 2))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 39))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 3))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 52))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 4))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 65))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 5))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 78))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 6))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 91))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 7))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 104))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 8))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 117))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 9))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 130))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 10))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 143))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 11))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 156))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 12))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 169))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 13))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 182))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 14))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 195))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 15))]));
  __syncthreads();
  pad_temp_shared[(((int)threadIdx.x))] = data[((((((((int)threadIdx.x) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 13)) + 43904))];
  if (((int)threadIdx.x) < 96) {
    pad_temp_shared[((((int)threadIdx.x) + 112))] = data[(((((((((int)threadIdx.x) + 112) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 8) % 13)) + 43904))];
  }
  kernel_shared[(((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 224))];
  kernel_shared[((((int)threadIdx.x) + 112))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 2016))];
  if (((int)threadIdx.x) < 32) {
    kernel_shared[((((int)threadIdx.x) + 224))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 3808))];
  }
  __syncthreads();
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 2))] * kernel_shared[(((((int)threadIdx.x) / 7) * 16))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 13))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 1))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 26))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 2))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 39))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 3))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 52))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 4))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 65))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 5))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 78))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 6))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 91))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 7))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 104))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 8))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 117))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 9))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 130))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 10))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 143))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 11))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 156))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 12))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 169))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 13))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 182))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 14))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 195))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 15))]));
  __syncthreads();
  pad_temp_shared[(((int)threadIdx.x))] = data[((((((((int)threadIdx.x) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 13)) + 47040))];
  if (((int)threadIdx.x) < 96) {
    pad_temp_shared[((((int)threadIdx.x) + 112))] = data[(((((((((int)threadIdx.x) + 112) / 13) * 196) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 8) % 13)) + 47040))];
  }
  kernel_shared[(((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 240))];
  kernel_shared[((((int)threadIdx.x) + 112))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 2032))];
  if (((int)threadIdx.x) < 32) {
    kernel_shared[((((int)threadIdx.x) + 224))] = kernel[((((((((int)blockIdx.x) / 7) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 3824))];
  }
  __syncthreads();
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 2))] * kernel_shared[(((((int)threadIdx.x) / 7) * 16))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 13))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 1))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 26))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 2))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 39))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 3))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 52))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 4))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 65))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 5))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 78))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 6))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 91))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 7))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 104))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 8))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 117))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 9))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 130))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 10))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 143))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 11))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 156))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 12))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 169))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 13))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 182))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 14))]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) % 7) * 2) + 195))] * kernel_shared[((((((int)threadIdx.x) / 7) * 16) + 15))]));
  compute[((((((((int)blockIdx.x) / 7) * 784) + ((((int)threadIdx.x) / 7) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + (((int)threadIdx.x) % 7)))] = compute_local[(0)];
}



void conv_kernel_wrapper(int N_B, int N_C, int N_H, int N_W, int N_F, int N_R, int N_S, int PaddingH, int PaddingW,
                        int StrideH, int StrideW, int N_X, int N_Y, const float *Input, 
                        const float *Kernel, float *Output, int itr) {
                        hipEvent_t start;
                        CHECK(hipEventCreate(&start));
                        hipEvent_t stop;
                        CHECK(hipEventCreate(&stop));
                        float *dev_Input;
                        float *dev_Kernel;
                        float *dev_Output;
                    
                        CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                        CHECK(hipMemcpy(dev_Kernel, Kernel, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                        CHECK(hipMemcpy(dev_Input, Input, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Output, sizeof(float) * N_B * N_F * N_Y * N_X));
                        CHECK(hipMemset(dev_Output, 0, sizeof(float) * N_B * N_F * N_Y * N_X));
                    
dim3 size_grid(224,1,1);

dim3 size_block(112,1,1);
 
                       printf("warmup....\n");
                        default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                    
                        double *time_arr = (double *)malloc(itr * sizeof(double));
                        for (int j = 1; j < itr; j++) {
                            CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                            CHECK(hipMemcpy(dev_Kernel, Kernel+j*N_F * N_C * N_R * N_S, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                            CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                            CHECK(hipMemcpy(dev_Input, Input+j*N_B * N_C * N_H * N_W, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                            /* Record the start event*/
                            CHECK(hipEventRecord(start, NULL));
                            default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                            /* Record the end event */
                            CHECK(hipEventRecord(stop, NULL));
                            /* Wait for the stop event to complete*/
                            CHECK(hipEventSynchronize(stop));
                            float msec = 0.0f;
                            hipEventElapsedTime(&msec, start, stop);
                            time_arr[j] = msec;
                        }
                    
                       
                    
                        //hipError_t err = hipGetLastError();
                        //if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
                        double msecTotal = 0;
                        for (int j = 1; j < itr; j++) {   
                            printf("%0.5lf\n", time_arr[j]);
                            msecTotal += time_arr[j];
                        }
                    
                        double msecPerMatrixMul = (msecTotal) / itr;
                        double flopsPerMatrixMul = 2.0 * N_B * N_C * N_X * N_Y * N_F * N_R * N_S;
                    
                        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
                    
                        printf("\n===========================================================\n");
                        printf("Input size: %d X %d X %d X %d\n", N_B, N_C, N_H, N_W);
                        printf("Kernel size: %d X %d X %d X %d\n", N_F, N_C, N_R, N_S);
                        printf("Output size: %d X %d X %d X %d\n", N_B, N_F, N_Y, N_X);
                        printf("flops: %0.5lf\n", (flopsPerMatrixMul * 1.0e-9f));
                        printf("Time is %0.5lf \n", (msecPerMatrixMul / 1000.0f));
                        printf("Gflops %0.5lf \n", gigaFlops);
                        printf("===========================================================\n");
                    
                        /*write back to Host Output -- batch size is 1*/
                        CHECK(hipMemcpy(Output, dev_Output, sizeof(float) * N_B * N_F * N_X * N_Y, hipMemcpyDeviceToHost));
                    
                        CHECK(hipFree(dev_Input));
                        CHECK(hipFree(dev_Kernel));
                        CHECK(hipFree(dev_Output));
                    
                    }
            