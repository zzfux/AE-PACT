#include "hip/hip_runtime.h"

            #include <cassert>
            #include <stdlib.h>
            #include <hip/hip_runtime.h>
            #include "common.h"
            

            
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[2];
  __shared__ float pad_temp_shared[1296];
  __shared__ float kernel_shared[288];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[(((int)threadIdx.x))] = data[(((rc_outer_outer * 1296) + ((int)threadIdx.x)))];
    pad_temp_shared[((((int)threadIdx.x) + 49))] = data[((((rc_outer_outer * 1296) + ((int)threadIdx.x)) + 49))];
    pad_temp_shared[((((int)threadIdx.x) + 98))] = data[((((rc_outer_outer * 1296) + ((int)threadIdx.x)) + 98))];
    pad_temp_shared[((((int)threadIdx.x) + 147))] = data[((((rc_outer_outer * 1296) + ((int)threadIdx.x)) + 147))];
    pad_temp_shared[((((int)threadIdx.x) + 196))] = data[((((rc_outer_outer * 1296) + ((int)threadIdx.x)) + 196))];
    pad_temp_shared[((((int)threadIdx.x) + 245))] = data[((((rc_outer_outer * 1296) + ((int)threadIdx.x)) + 245))];
    pad_temp_shared[((((int)threadIdx.x) + 294))] = data[((((rc_outer_outer * 1296) + ((int)threadIdx.x)) + 294))];
    pad_temp_shared[((((int)threadIdx.x) + 343))] = data[((((rc_outer_outer * 1296) + ((int)threadIdx.x)) + 343))];
    pad_temp_shared[((((int)threadIdx.x) + 392))] = data[((((rc_outer_outer * 1296) + ((int)threadIdx.x)) + 392))];
    pad_temp_shared[((((int)threadIdx.x) + 441))] = data[((((rc_outer_outer * 1296) + ((int)threadIdx.x)) + 441))];
    pad_temp_shared[((((int)threadIdx.x) + 490))] = data[((((rc_outer_outer * 1296) + ((int)threadIdx.x)) + 490))];
    pad_temp_shared[((((int)threadIdx.x) + 539))] = data[((((rc_outer_outer * 1296) + ((int)threadIdx.x)) + 539))];
    pad_temp_shared[((((int)threadIdx.x) + 588))] = data[((((rc_outer_outer * 1296) + ((int)threadIdx.x)) + 588))];
    pad_temp_shared[((((int)threadIdx.x) + 637))] = data[((((rc_outer_outer * 1296) + ((int)threadIdx.x)) + 637))];
    pad_temp_shared[((((int)threadIdx.x) + 686))] = data[((((rc_outer_outer * 1296) + ((int)threadIdx.x)) + 686))];
    pad_temp_shared[((((int)threadIdx.x) + 735))] = data[((((rc_outer_outer * 1296) + ((int)threadIdx.x)) + 735))];
    pad_temp_shared[((((int)threadIdx.x) + 784))] = data[((((rc_outer_outer * 1296) + ((int)threadIdx.x)) + 784))];
    pad_temp_shared[((((int)threadIdx.x) + 833))] = data[((((rc_outer_outer * 1296) + ((int)threadIdx.x)) + 833))];
    pad_temp_shared[((((int)threadIdx.x) + 882))] = data[((((rc_outer_outer * 1296) + ((int)threadIdx.x)) + 882))];
    pad_temp_shared[((((int)threadIdx.x) + 931))] = data[((((rc_outer_outer * 1296) + ((int)threadIdx.x)) + 931))];
    pad_temp_shared[((((int)threadIdx.x) + 980))] = data[((((rc_outer_outer * 1296) + ((int)threadIdx.x)) + 980))];
    pad_temp_shared[((((int)threadIdx.x) + 1029))] = data[((((rc_outer_outer * 1296) + ((int)threadIdx.x)) + 1029))];
    pad_temp_shared[((((int)threadIdx.x) + 1078))] = data[((((rc_outer_outer * 1296) + ((int)threadIdx.x)) + 1078))];
    pad_temp_shared[((((int)threadIdx.x) + 1127))] = data[((((rc_outer_outer * 1296) + ((int)threadIdx.x)) + 1127))];
    pad_temp_shared[((((int)threadIdx.x) + 1176))] = data[((((rc_outer_outer * 1296) + ((int)threadIdx.x)) + 1176))];
    pad_temp_shared[((((int)threadIdx.x) + 1225))] = data[((((rc_outer_outer * 1296) + ((int)threadIdx.x)) + 1225))];
    if (((int)threadIdx.x) < 22) {
      pad_temp_shared[((((int)threadIdx.x) + 1274))] = data[((((rc_outer_outer * 1296) + ((int)threadIdx.x)) + 1274))];
    }
    kernel_shared[(((int)threadIdx.x))] = kernel[((((((int)blockIdx.x) * 9216) + (rc_outer_outer * 144)) + ((int)threadIdx.x)))];
    kernel_shared[((((int)threadIdx.x) + 49))] = kernel[(((((((int)blockIdx.x) * 9216) + (rc_outer_outer * 144)) + ((int)threadIdx.x)) + 49))];
    kernel_shared[((((int)threadIdx.x) + 98))] = kernel[(((((((int)blockIdx.x) * 9216) + (((((int)threadIdx.x) + 98) / 144) * 4608)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 98) % 144)))];
    kernel_shared[((((int)threadIdx.x) + 147))] = kernel[(((((((int)blockIdx.x) * 9216) + (((((int)threadIdx.x) + 147) / 144) * 4608)) + (rc_outer_outer * 144)) + (((int)threadIdx.x) + 3)))];
    kernel_shared[((((int)threadIdx.x) + 196))] = kernel[(((((((int)blockIdx.x) * 9216) + (((((int)threadIdx.x) + 196) / 144) * 4608)) + (rc_outer_outer * 144)) + (((int)threadIdx.x) + 52)))];
    if (((int)threadIdx.x) < 43) {
      kernel_shared[((((int)threadIdx.x) + 245))] = kernel[(((((((int)blockIdx.x) * 9216) + (((((int)threadIdx.x) + 245) / 144) * 4608)) + (rc_outer_outer * 144)) + (((int)threadIdx.x) + 101)))];
    }
    __syncthreads();
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)))] * kernel_shared[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)))] * kernel_shared[(144)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 9))] * kernel_shared[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 9))] * kernel_shared[(147)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 18))] * kernel_shared[(6)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 18))] * kernel_shared[(150)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 81))] * kernel_shared[(9)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 81))] * kernel_shared[(153)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 90))] * kernel_shared[(12)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 90))] * kernel_shared[(156)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 99))] * kernel_shared[(15)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 99))] * kernel_shared[(159)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 162))] * kernel_shared[(18)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 162))] * kernel_shared[(162)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 171))] * kernel_shared[(21)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 171))] * kernel_shared[(165)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 180))] * kernel_shared[(24)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 180))] * kernel_shared[(168)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 243))] * kernel_shared[(27)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 243))] * kernel_shared[(171)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 252))] * kernel_shared[(30)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 252))] * kernel_shared[(174)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 261))] * kernel_shared[(33)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 261))] * kernel_shared[(177)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 324))] * kernel_shared[(36)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 324))] * kernel_shared[(180)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 333))] * kernel_shared[(39)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 333))] * kernel_shared[(183)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 342))] * kernel_shared[(42)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 342))] * kernel_shared[(186)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 405))] * kernel_shared[(45)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 405))] * kernel_shared[(189)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 414))] * kernel_shared[(48)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 414))] * kernel_shared[(192)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 423))] * kernel_shared[(51)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 423))] * kernel_shared[(195)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 486))] * kernel_shared[(54)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 486))] * kernel_shared[(198)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 495))] * kernel_shared[(57)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 495))] * kernel_shared[(201)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 504))] * kernel_shared[(60)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 504))] * kernel_shared[(204)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 567))] * kernel_shared[(63)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 567))] * kernel_shared[(207)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 576))] * kernel_shared[(66)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 576))] * kernel_shared[(210)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 585))] * kernel_shared[(69)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 585))] * kernel_shared[(213)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1))] * kernel_shared[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1))] * kernel_shared[(145)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 10))] * kernel_shared[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 10))] * kernel_shared[(148)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 19))] * kernel_shared[(7)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 19))] * kernel_shared[(151)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 82))] * kernel_shared[(10)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 82))] * kernel_shared[(154)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 91))] * kernel_shared[(13)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 91))] * kernel_shared[(157)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 100))] * kernel_shared[(16)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 100))] * kernel_shared[(160)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 163))] * kernel_shared[(19)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 163))] * kernel_shared[(163)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 172))] * kernel_shared[(22)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 172))] * kernel_shared[(166)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 181))] * kernel_shared[(25)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 181))] * kernel_shared[(169)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 244))] * kernel_shared[(28)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 244))] * kernel_shared[(172)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 253))] * kernel_shared[(31)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 253))] * kernel_shared[(175)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 262))] * kernel_shared[(34)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 262))] * kernel_shared[(178)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 325))] * kernel_shared[(37)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 325))] * kernel_shared[(181)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 334))] * kernel_shared[(40)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 334))] * kernel_shared[(184)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 343))] * kernel_shared[(43)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 343))] * kernel_shared[(187)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 406))] * kernel_shared[(46)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 406))] * kernel_shared[(190)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 415))] * kernel_shared[(49)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 415))] * kernel_shared[(193)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 424))] * kernel_shared[(52)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 424))] * kernel_shared[(196)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 487))] * kernel_shared[(55)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 487))] * kernel_shared[(199)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 496))] * kernel_shared[(58)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 496))] * kernel_shared[(202)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 505))] * kernel_shared[(61)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 505))] * kernel_shared[(205)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 568))] * kernel_shared[(64)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 568))] * kernel_shared[(208)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 577))] * kernel_shared[(67)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 577))] * kernel_shared[(211)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 586))] * kernel_shared[(70)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 586))] * kernel_shared[(214)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 2))] * kernel_shared[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 2))] * kernel_shared[(146)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 11))] * kernel_shared[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 11))] * kernel_shared[(149)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 20))] * kernel_shared[(8)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 20))] * kernel_shared[(152)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 83))] * kernel_shared[(11)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 83))] * kernel_shared[(155)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 92))] * kernel_shared[(14)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 92))] * kernel_shared[(158)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 101))] * kernel_shared[(17)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 101))] * kernel_shared[(161)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 164))] * kernel_shared[(20)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 164))] * kernel_shared[(164)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 173))] * kernel_shared[(23)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 173))] * kernel_shared[(167)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 182))] * kernel_shared[(26)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 182))] * kernel_shared[(170)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 245))] * kernel_shared[(29)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 245))] * kernel_shared[(173)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 254))] * kernel_shared[(32)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 254))] * kernel_shared[(176)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 263))] * kernel_shared[(35)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 263))] * kernel_shared[(179)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 326))] * kernel_shared[(38)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 326))] * kernel_shared[(182)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 335))] * kernel_shared[(41)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 335))] * kernel_shared[(185)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 344))] * kernel_shared[(44)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 344))] * kernel_shared[(188)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 407))] * kernel_shared[(47)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 407))] * kernel_shared[(191)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 416))] * kernel_shared[(50)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 416))] * kernel_shared[(194)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 425))] * kernel_shared[(53)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 425))] * kernel_shared[(197)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 488))] * kernel_shared[(56)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 488))] * kernel_shared[(200)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 497))] * kernel_shared[(59)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 497))] * kernel_shared[(203)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 506))] * kernel_shared[(62)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 506))] * kernel_shared[(206)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 569))] * kernel_shared[(65)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 569))] * kernel_shared[(209)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 578))] * kernel_shared[(68)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 578))] * kernel_shared[(212)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 587))] * kernel_shared[(71)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 587))] * kernel_shared[(215)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 648))] * kernel_shared[(72)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 648))] * kernel_shared[(216)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 657))] * kernel_shared[(75)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 657))] * kernel_shared[(219)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 666))] * kernel_shared[(78)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 666))] * kernel_shared[(222)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 729))] * kernel_shared[(81)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 729))] * kernel_shared[(225)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 738))] * kernel_shared[(84)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 738))] * kernel_shared[(228)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 747))] * kernel_shared[(87)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 747))] * kernel_shared[(231)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 810))] * kernel_shared[(90)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 810))] * kernel_shared[(234)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 819))] * kernel_shared[(93)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 819))] * kernel_shared[(237)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 828))] * kernel_shared[(96)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 828))] * kernel_shared[(240)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 891))] * kernel_shared[(99)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 891))] * kernel_shared[(243)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 900))] * kernel_shared[(102)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 900))] * kernel_shared[(246)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 909))] * kernel_shared[(105)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 909))] * kernel_shared[(249)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 972))] * kernel_shared[(108)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 972))] * kernel_shared[(252)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 981))] * kernel_shared[(111)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 981))] * kernel_shared[(255)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 990))] * kernel_shared[(114)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 990))] * kernel_shared[(258)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1053))] * kernel_shared[(117)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1053))] * kernel_shared[(261)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1062))] * kernel_shared[(120)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1062))] * kernel_shared[(264)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1071))] * kernel_shared[(123)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1071))] * kernel_shared[(267)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1134))] * kernel_shared[(126)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1134))] * kernel_shared[(270)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1143))] * kernel_shared[(129)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1143))] * kernel_shared[(273)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1152))] * kernel_shared[(132)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1152))] * kernel_shared[(276)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1215))] * kernel_shared[(135)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1215))] * kernel_shared[(279)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1224))] * kernel_shared[(138)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1224))] * kernel_shared[(282)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1233))] * kernel_shared[(141)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1233))] * kernel_shared[(285)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 649))] * kernel_shared[(73)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 649))] * kernel_shared[(217)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 658))] * kernel_shared[(76)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 658))] * kernel_shared[(220)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 667))] * kernel_shared[(79)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 667))] * kernel_shared[(223)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 730))] * kernel_shared[(82)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 730))] * kernel_shared[(226)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 739))] * kernel_shared[(85)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 739))] * kernel_shared[(229)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 748))] * kernel_shared[(88)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 748))] * kernel_shared[(232)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 811))] * kernel_shared[(91)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 811))] * kernel_shared[(235)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 820))] * kernel_shared[(94)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 820))] * kernel_shared[(238)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 829))] * kernel_shared[(97)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 829))] * kernel_shared[(241)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 892))] * kernel_shared[(100)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 892))] * kernel_shared[(244)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 901))] * kernel_shared[(103)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 901))] * kernel_shared[(247)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 910))] * kernel_shared[(106)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 910))] * kernel_shared[(250)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 973))] * kernel_shared[(109)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 973))] * kernel_shared[(253)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 982))] * kernel_shared[(112)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 982))] * kernel_shared[(256)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 991))] * kernel_shared[(115)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 991))] * kernel_shared[(259)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1054))] * kernel_shared[(118)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1054))] * kernel_shared[(262)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1063))] * kernel_shared[(121)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1063))] * kernel_shared[(265)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1072))] * kernel_shared[(124)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1072))] * kernel_shared[(268)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1135))] * kernel_shared[(127)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1135))] * kernel_shared[(271)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1144))] * kernel_shared[(130)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1144))] * kernel_shared[(274)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1153))] * kernel_shared[(133)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1153))] * kernel_shared[(277)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1216))] * kernel_shared[(136)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1216))] * kernel_shared[(280)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1225))] * kernel_shared[(139)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1225))] * kernel_shared[(283)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1234))] * kernel_shared[(142)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1234))] * kernel_shared[(286)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 650))] * kernel_shared[(74)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 650))] * kernel_shared[(218)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 659))] * kernel_shared[(77)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 659))] * kernel_shared[(221)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 668))] * kernel_shared[(80)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 668))] * kernel_shared[(224)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 731))] * kernel_shared[(83)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 731))] * kernel_shared[(227)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 740))] * kernel_shared[(86)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 740))] * kernel_shared[(230)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 749))] * kernel_shared[(89)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 749))] * kernel_shared[(233)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 812))] * kernel_shared[(92)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 812))] * kernel_shared[(236)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 821))] * kernel_shared[(95)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 821))] * kernel_shared[(239)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 830))] * kernel_shared[(98)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 830))] * kernel_shared[(242)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 893))] * kernel_shared[(101)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 893))] * kernel_shared[(245)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 902))] * kernel_shared[(104)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 902))] * kernel_shared[(248)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 911))] * kernel_shared[(107)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 911))] * kernel_shared[(251)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 974))] * kernel_shared[(110)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 974))] * kernel_shared[(254)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 983))] * kernel_shared[(113)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 983))] * kernel_shared[(257)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 992))] * kernel_shared[(116)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 992))] * kernel_shared[(260)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1055))] * kernel_shared[(119)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1055))] * kernel_shared[(263)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1064))] * kernel_shared[(122)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1064))] * kernel_shared[(266)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1073))] * kernel_shared[(125)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1073))] * kernel_shared[(269)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1136))] * kernel_shared[(128)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1136))] * kernel_shared[(272)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1145))] * kernel_shared[(131)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1145))] * kernel_shared[(275)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1154))] * kernel_shared[(134)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1154))] * kernel_shared[(278)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1217))] * kernel_shared[(137)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1217))] * kernel_shared[(281)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1226))] * kernel_shared[(140)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1226))] * kernel_shared[(284)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1235))] * kernel_shared[(143)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((int)threadIdx.x) / 7) * 9) + (((int)threadIdx.x) % 7)) + 1235))] * kernel_shared[(287)]));
  }
  compute[(((((int)blockIdx.x) * 98) + ((int)threadIdx.x)))] = compute_local[(0)];
  compute[((((((int)blockIdx.x) * 98) + ((int)threadIdx.x)) + 49))] = compute_local[(1)];
}



void conv_kernel_wrapper(int N_B, int N_C, int N_H, int N_W, int N_F, int N_R, int N_S, int PaddingH, int PaddingW,
                        int StrideH, int StrideW, int N_X, int N_Y, const float *Input, 
                        const float *Kernel, float *Output, int itr) {
                        hipEvent_t start;
                        CHECK(hipEventCreate(&start));
                        hipEvent_t stop;
                        CHECK(hipEventCreate(&stop));
                        float *dev_Input;
                        float *dev_Kernel;
                        float *dev_Output;
                    
                        CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                        CHECK(hipMemcpy(dev_Kernel, Kernel, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                        CHECK(hipMemcpy(dev_Input, Input, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Output, sizeof(float) * N_B * N_F * N_Y * N_X));
                        CHECK(hipMemset(dev_Output, 0, sizeof(float) * N_B * N_F * N_Y * N_X));
                    
dim3 size_grid(256,1,1);

dim3 size_block(49,1,1);
 
                       printf("warmup....\n");
                        default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                    
                        double *time_arr = (double *)malloc(itr * sizeof(double));
                        for (int j = 1; j < itr; j++) {
                            CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                            CHECK(hipMemcpy(dev_Kernel, Kernel+j*N_F * N_C * N_R * N_S, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                            CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                            CHECK(hipMemcpy(dev_Input, Input+j*N_B * N_C * N_H * N_W, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                            /* Record the start event*/
                            CHECK(hipEventRecord(start, NULL));
                            default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                            /* Record the end event */
                            CHECK(hipEventRecord(stop, NULL));
                            /* Wait for the stop event to complete*/
                            CHECK(hipEventSynchronize(stop));
                            float msec = 0.0f;
                            hipEventElapsedTime(&msec, start, stop);
                            time_arr[j] = msec;
                        }
                    
                       
                    
                        //hipError_t err = hipGetLastError();
                        //if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
                        double msecTotal = 0;
                        for (int j = 1; j < itr; j++) {   
                            printf("%0.5lf\n", time_arr[j]);
                            msecTotal += time_arr[j];
                        }
                    
                        double msecPerMatrixMul = (msecTotal) / itr;
                        double flopsPerMatrixMul = 2.0 * N_B * N_C * N_X * N_Y * N_F * N_R * N_S;
                    
                        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
                    
                        printf("\n===========================================================\n");
                        printf("Input size: %d X %d X %d X %d\n", N_B, N_C, N_H, N_W);
                        printf("Kernel size: %d X %d X %d X %d\n", N_F, N_C, N_R, N_S);
                        printf("Output size: %d X %d X %d X %d\n", N_B, N_F, N_Y, N_X);
                        printf("flops: %0.5lf\n", (flopsPerMatrixMul * 1.0e-9f));
                        printf("Time is %0.5lf \n", (msecPerMatrixMul / 1000.0f));
                        printf("Gflops %0.5lf \n", gigaFlops);
                        printf("===========================================================\n");
                    
                        /*write back to Host Output -- batch size is 1*/
                        CHECK(hipMemcpy(Output, dev_Output, sizeof(float) * N_B * N_F * N_X * N_Y, hipMemcpyDeviceToHost));
                    
                        CHECK(hipFree(dev_Input));
                        CHECK(hipFree(dev_Kernel));
                        CHECK(hipFree(dev_Output));
                    
                    }
            