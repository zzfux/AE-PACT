#include "hip/hip_runtime.h"

            #include <cassert>
            #include <stdlib.h>
            #include <hip/hip_runtime.h>
            #include "common.h"
            

            
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[17];
  __shared__ float pad_temp_shared[272];
  __shared__ float kernel_shared[2048];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  compute_local[(7)] = 0.000000e+00f;
  compute_local[(8)] = 0.000000e+00f;
  compute_local[(9)] = 0.000000e+00f;
  compute_local[(10)] = 0.000000e+00f;
  compute_local[(11)] = 0.000000e+00f;
  compute_local[(12)] = 0.000000e+00f;
  compute_local[(13)] = 0.000000e+00f;
  compute_local[(14)] = 0.000000e+00f;
  compute_local[(15)] = 0.000000e+00f;
  compute_local[(16)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((((int)threadIdx.x) * 2))] = data[(((((rc_outer_outer * 4624) + (((((int)threadIdx.x) * 2) / 17) * 289)) + ((((int)blockIdx.x) % 17) * 17)) + ((((int)threadIdx.x) * 2) % 17)))];
    pad_temp_shared[(((((int)threadIdx.x) * 2) + 1))] = data[(((((rc_outer_outer * 4624) + ((((((int)threadIdx.x) * 2) + 1) / 17) * 289)) + ((((int)blockIdx.x) % 17) * 17)) + (((((int)threadIdx.x) * 2) + 1) % 17)))];
    if (((int)threadIdx.x) < 8) {
      pad_temp_shared[(((((int)threadIdx.x) * 2) + 256))] = data[(((((rc_outer_outer * 4624) + ((((((int)threadIdx.x) * 2) + 256) / 17) * 289)) + ((((int)blockIdx.x) % 17) * 17)) + ((((int)threadIdx.x) * 2) + 1)))];
    }
    if (((int)threadIdx.x) < 8) {
      pad_temp_shared[(((((int)threadIdx.x) * 2) + 257))] = data[(((((rc_outer_outer * 4624) + ((((((int)threadIdx.x) * 2) + 257) / 17) * 289)) + ((((int)blockIdx.x) % 17) * 17)) + ((((int)threadIdx.x) * 2) + 2)))];
    }
      ((float2*)(kernel_shared + ((((int)threadIdx.x) * 2))))[0] = ((float2*)(kernel + ((((((((int)blockIdx.x) / 17) * 131072) + ((((int)threadIdx.x) >> 3) * 1024)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 7) * 2)))))[0];
      ((float2*)(kernel_shared + (((((int)threadIdx.x) * 2) + 256))))[0] = ((float2*)(kernel + (((((((((int)blockIdx.x) / 17) * 131072) + ((((int)threadIdx.x) >> 3) * 1024)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 7) * 2)) + 16384))))[0];
      ((float2*)(kernel_shared + (((((int)threadIdx.x) * 2) + 512))))[0] = ((float2*)(kernel + (((((((((int)blockIdx.x) / 17) * 131072) + ((((int)threadIdx.x) >> 3) * 1024)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 7) * 2)) + 32768))))[0];
      ((float2*)(kernel_shared + (((((int)threadIdx.x) * 2) + 768))))[0] = ((float2*)(kernel + (((((((((int)blockIdx.x) / 17) * 131072) + ((((int)threadIdx.x) >> 3) * 1024)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 7) * 2)) + 49152))))[0];
      ((float2*)(kernel_shared + (((((int)threadIdx.x) * 2) + 1024))))[0] = ((float2*)(kernel + (((((((((int)blockIdx.x) / 17) * 131072) + ((((int)threadIdx.x) >> 3) * 1024)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 7) * 2)) + 65536))))[0];
      ((float2*)(kernel_shared + (((((int)threadIdx.x) * 2) + 1280))))[0] = ((float2*)(kernel + (((((((((int)blockIdx.x) / 17) * 131072) + ((((int)threadIdx.x) >> 3) * 1024)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 7) * 2)) + 81920))))[0];
      ((float2*)(kernel_shared + (((((int)threadIdx.x) * 2) + 1536))))[0] = ((float2*)(kernel + (((((((((int)blockIdx.x) / 17) * 131072) + ((((int)threadIdx.x) >> 3) * 1024)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 7) * 2)) + 98304))))[0];
      ((float2*)(kernel_shared + (((((int)threadIdx.x) * 2) + 1792))))[0] = ((float2*)(kernel + (((((((((int)blockIdx.x) / 17) * 131072) + ((((int)threadIdx.x) >> 3) * 1024)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 7) * 2)) + 114688))))[0];
    __syncthreads();
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(0)] * kernel_shared[((((int)threadIdx.x) * 16))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(1)] * kernel_shared[((((int)threadIdx.x) * 16))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(2)] * kernel_shared[((((int)threadIdx.x) * 16))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(3)] * kernel_shared[((((int)threadIdx.x) * 16))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(4)] * kernel_shared[((((int)threadIdx.x) * 16))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(5)] * kernel_shared[((((int)threadIdx.x) * 16))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(6)] * kernel_shared[((((int)threadIdx.x) * 16))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(7)] * kernel_shared[((((int)threadIdx.x) * 16))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(8)] * kernel_shared[((((int)threadIdx.x) * 16))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(9)] * kernel_shared[((((int)threadIdx.x) * 16))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(10)] * kernel_shared[((((int)threadIdx.x) * 16))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(11)] * kernel_shared[((((int)threadIdx.x) * 16))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(12)] * kernel_shared[((((int)threadIdx.x) * 16))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(13)] * kernel_shared[((((int)threadIdx.x) * 16))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(14)] * kernel_shared[((((int)threadIdx.x) * 16))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(15)] * kernel_shared[((((int)threadIdx.x) * 16))]));
    compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(16)] * kernel_shared[((((int)threadIdx.x) * 16))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(17)] * kernel_shared[(((((int)threadIdx.x) * 16) + 1))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(18)] * kernel_shared[(((((int)threadIdx.x) * 16) + 1))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(19)] * kernel_shared[(((((int)threadIdx.x) * 16) + 1))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(20)] * kernel_shared[(((((int)threadIdx.x) * 16) + 1))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(21)] * kernel_shared[(((((int)threadIdx.x) * 16) + 1))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(22)] * kernel_shared[(((((int)threadIdx.x) * 16) + 1))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(23)] * kernel_shared[(((((int)threadIdx.x) * 16) + 1))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(24)] * kernel_shared[(((((int)threadIdx.x) * 16) + 1))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(25)] * kernel_shared[(((((int)threadIdx.x) * 16) + 1))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(26)] * kernel_shared[(((((int)threadIdx.x) * 16) + 1))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(27)] * kernel_shared[(((((int)threadIdx.x) * 16) + 1))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(28)] * kernel_shared[(((((int)threadIdx.x) * 16) + 1))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(29)] * kernel_shared[(((((int)threadIdx.x) * 16) + 1))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(30)] * kernel_shared[(((((int)threadIdx.x) * 16) + 1))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(31)] * kernel_shared[(((((int)threadIdx.x) * 16) + 1))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(32)] * kernel_shared[(((((int)threadIdx.x) * 16) + 1))]));
    compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(33)] * kernel_shared[(((((int)threadIdx.x) * 16) + 1))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(34)] * kernel_shared[(((((int)threadIdx.x) * 16) + 2))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(35)] * kernel_shared[(((((int)threadIdx.x) * 16) + 2))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(36)] * kernel_shared[(((((int)threadIdx.x) * 16) + 2))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(37)] * kernel_shared[(((((int)threadIdx.x) * 16) + 2))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(38)] * kernel_shared[(((((int)threadIdx.x) * 16) + 2))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(39)] * kernel_shared[(((((int)threadIdx.x) * 16) + 2))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(40)] * kernel_shared[(((((int)threadIdx.x) * 16) + 2))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(41)] * kernel_shared[(((((int)threadIdx.x) * 16) + 2))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(42)] * kernel_shared[(((((int)threadIdx.x) * 16) + 2))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(43)] * kernel_shared[(((((int)threadIdx.x) * 16) + 2))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(44)] * kernel_shared[(((((int)threadIdx.x) * 16) + 2))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(45)] * kernel_shared[(((((int)threadIdx.x) * 16) + 2))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(46)] * kernel_shared[(((((int)threadIdx.x) * 16) + 2))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(47)] * kernel_shared[(((((int)threadIdx.x) * 16) + 2))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(48)] * kernel_shared[(((((int)threadIdx.x) * 16) + 2))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(49)] * kernel_shared[(((((int)threadIdx.x) * 16) + 2))]));
    compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(50)] * kernel_shared[(((((int)threadIdx.x) * 16) + 2))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(51)] * kernel_shared[(((((int)threadIdx.x) * 16) + 3))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(52)] * kernel_shared[(((((int)threadIdx.x) * 16) + 3))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(53)] * kernel_shared[(((((int)threadIdx.x) * 16) + 3))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(54)] * kernel_shared[(((((int)threadIdx.x) * 16) + 3))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(55)] * kernel_shared[(((((int)threadIdx.x) * 16) + 3))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(56)] * kernel_shared[(((((int)threadIdx.x) * 16) + 3))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(57)] * kernel_shared[(((((int)threadIdx.x) * 16) + 3))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(58)] * kernel_shared[(((((int)threadIdx.x) * 16) + 3))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(59)] * kernel_shared[(((((int)threadIdx.x) * 16) + 3))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(60)] * kernel_shared[(((((int)threadIdx.x) * 16) + 3))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(61)] * kernel_shared[(((((int)threadIdx.x) * 16) + 3))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(62)] * kernel_shared[(((((int)threadIdx.x) * 16) + 3))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(63)] * kernel_shared[(((((int)threadIdx.x) * 16) + 3))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(64)] * kernel_shared[(((((int)threadIdx.x) * 16) + 3))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(65)] * kernel_shared[(((((int)threadIdx.x) * 16) + 3))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(66)] * kernel_shared[(((((int)threadIdx.x) * 16) + 3))]));
    compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(67)] * kernel_shared[(((((int)threadIdx.x) * 16) + 3))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(68)] * kernel_shared[(((((int)threadIdx.x) * 16) + 4))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(69)] * kernel_shared[(((((int)threadIdx.x) * 16) + 4))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(70)] * kernel_shared[(((((int)threadIdx.x) * 16) + 4))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(71)] * kernel_shared[(((((int)threadIdx.x) * 16) + 4))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(72)] * kernel_shared[(((((int)threadIdx.x) * 16) + 4))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(73)] * kernel_shared[(((((int)threadIdx.x) * 16) + 4))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(74)] * kernel_shared[(((((int)threadIdx.x) * 16) + 4))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(75)] * kernel_shared[(((((int)threadIdx.x) * 16) + 4))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(76)] * kernel_shared[(((((int)threadIdx.x) * 16) + 4))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(77)] * kernel_shared[(((((int)threadIdx.x) * 16) + 4))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(78)] * kernel_shared[(((((int)threadIdx.x) * 16) + 4))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(79)] * kernel_shared[(((((int)threadIdx.x) * 16) + 4))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(80)] * kernel_shared[(((((int)threadIdx.x) * 16) + 4))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(81)] * kernel_shared[(((((int)threadIdx.x) * 16) + 4))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(82)] * kernel_shared[(((((int)threadIdx.x) * 16) + 4))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(83)] * kernel_shared[(((((int)threadIdx.x) * 16) + 4))]));
    compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(84)] * kernel_shared[(((((int)threadIdx.x) * 16) + 4))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(85)] * kernel_shared[(((((int)threadIdx.x) * 16) + 5))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(86)] * kernel_shared[(((((int)threadIdx.x) * 16) + 5))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(87)] * kernel_shared[(((((int)threadIdx.x) * 16) + 5))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(88)] * kernel_shared[(((((int)threadIdx.x) * 16) + 5))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(89)] * kernel_shared[(((((int)threadIdx.x) * 16) + 5))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(90)] * kernel_shared[(((((int)threadIdx.x) * 16) + 5))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(91)] * kernel_shared[(((((int)threadIdx.x) * 16) + 5))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(92)] * kernel_shared[(((((int)threadIdx.x) * 16) + 5))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(93)] * kernel_shared[(((((int)threadIdx.x) * 16) + 5))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(94)] * kernel_shared[(((((int)threadIdx.x) * 16) + 5))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(95)] * kernel_shared[(((((int)threadIdx.x) * 16) + 5))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(96)] * kernel_shared[(((((int)threadIdx.x) * 16) + 5))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(97)] * kernel_shared[(((((int)threadIdx.x) * 16) + 5))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(98)] * kernel_shared[(((((int)threadIdx.x) * 16) + 5))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(99)] * kernel_shared[(((((int)threadIdx.x) * 16) + 5))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(100)] * kernel_shared[(((((int)threadIdx.x) * 16) + 5))]));
    compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(101)] * kernel_shared[(((((int)threadIdx.x) * 16) + 5))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(102)] * kernel_shared[(((((int)threadIdx.x) * 16) + 6))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(103)] * kernel_shared[(((((int)threadIdx.x) * 16) + 6))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(104)] * kernel_shared[(((((int)threadIdx.x) * 16) + 6))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(105)] * kernel_shared[(((((int)threadIdx.x) * 16) + 6))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(106)] * kernel_shared[(((((int)threadIdx.x) * 16) + 6))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(107)] * kernel_shared[(((((int)threadIdx.x) * 16) + 6))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(108)] * kernel_shared[(((((int)threadIdx.x) * 16) + 6))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(109)] * kernel_shared[(((((int)threadIdx.x) * 16) + 6))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(110)] * kernel_shared[(((((int)threadIdx.x) * 16) + 6))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(111)] * kernel_shared[(((((int)threadIdx.x) * 16) + 6))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(112)] * kernel_shared[(((((int)threadIdx.x) * 16) + 6))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(113)] * kernel_shared[(((((int)threadIdx.x) * 16) + 6))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(114)] * kernel_shared[(((((int)threadIdx.x) * 16) + 6))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(115)] * kernel_shared[(((((int)threadIdx.x) * 16) + 6))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(116)] * kernel_shared[(((((int)threadIdx.x) * 16) + 6))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(117)] * kernel_shared[(((((int)threadIdx.x) * 16) + 6))]));
    compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(118)] * kernel_shared[(((((int)threadIdx.x) * 16) + 6))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(119)] * kernel_shared[(((((int)threadIdx.x) * 16) + 7))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(120)] * kernel_shared[(((((int)threadIdx.x) * 16) + 7))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(121)] * kernel_shared[(((((int)threadIdx.x) * 16) + 7))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(122)] * kernel_shared[(((((int)threadIdx.x) * 16) + 7))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(123)] * kernel_shared[(((((int)threadIdx.x) * 16) + 7))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(124)] * kernel_shared[(((((int)threadIdx.x) * 16) + 7))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(125)] * kernel_shared[(((((int)threadIdx.x) * 16) + 7))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(126)] * kernel_shared[(((((int)threadIdx.x) * 16) + 7))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(127)] * kernel_shared[(((((int)threadIdx.x) * 16) + 7))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(128)] * kernel_shared[(((((int)threadIdx.x) * 16) + 7))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(129)] * kernel_shared[(((((int)threadIdx.x) * 16) + 7))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(130)] * kernel_shared[(((((int)threadIdx.x) * 16) + 7))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(131)] * kernel_shared[(((((int)threadIdx.x) * 16) + 7))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(132)] * kernel_shared[(((((int)threadIdx.x) * 16) + 7))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(133)] * kernel_shared[(((((int)threadIdx.x) * 16) + 7))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(134)] * kernel_shared[(((((int)threadIdx.x) * 16) + 7))]));
    compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(135)] * kernel_shared[(((((int)threadIdx.x) * 16) + 7))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(136)] * kernel_shared[(((((int)threadIdx.x) * 16) + 8))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(137)] * kernel_shared[(((((int)threadIdx.x) * 16) + 8))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(138)] * kernel_shared[(((((int)threadIdx.x) * 16) + 8))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(139)] * kernel_shared[(((((int)threadIdx.x) * 16) + 8))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(140)] * kernel_shared[(((((int)threadIdx.x) * 16) + 8))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(141)] * kernel_shared[(((((int)threadIdx.x) * 16) + 8))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(142)] * kernel_shared[(((((int)threadIdx.x) * 16) + 8))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(143)] * kernel_shared[(((((int)threadIdx.x) * 16) + 8))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(144)] * kernel_shared[(((((int)threadIdx.x) * 16) + 8))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(145)] * kernel_shared[(((((int)threadIdx.x) * 16) + 8))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(146)] * kernel_shared[(((((int)threadIdx.x) * 16) + 8))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(147)] * kernel_shared[(((((int)threadIdx.x) * 16) + 8))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(148)] * kernel_shared[(((((int)threadIdx.x) * 16) + 8))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(149)] * kernel_shared[(((((int)threadIdx.x) * 16) + 8))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(150)] * kernel_shared[(((((int)threadIdx.x) * 16) + 8))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(151)] * kernel_shared[(((((int)threadIdx.x) * 16) + 8))]));
    compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(152)] * kernel_shared[(((((int)threadIdx.x) * 16) + 8))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(153)] * kernel_shared[(((((int)threadIdx.x) * 16) + 9))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(154)] * kernel_shared[(((((int)threadIdx.x) * 16) + 9))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(155)] * kernel_shared[(((((int)threadIdx.x) * 16) + 9))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(156)] * kernel_shared[(((((int)threadIdx.x) * 16) + 9))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(157)] * kernel_shared[(((((int)threadIdx.x) * 16) + 9))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(158)] * kernel_shared[(((((int)threadIdx.x) * 16) + 9))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(159)] * kernel_shared[(((((int)threadIdx.x) * 16) + 9))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(160)] * kernel_shared[(((((int)threadIdx.x) * 16) + 9))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(161)] * kernel_shared[(((((int)threadIdx.x) * 16) + 9))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(162)] * kernel_shared[(((((int)threadIdx.x) * 16) + 9))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(163)] * kernel_shared[(((((int)threadIdx.x) * 16) + 9))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(164)] * kernel_shared[(((((int)threadIdx.x) * 16) + 9))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(165)] * kernel_shared[(((((int)threadIdx.x) * 16) + 9))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(166)] * kernel_shared[(((((int)threadIdx.x) * 16) + 9))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(167)] * kernel_shared[(((((int)threadIdx.x) * 16) + 9))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(168)] * kernel_shared[(((((int)threadIdx.x) * 16) + 9))]));
    compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(169)] * kernel_shared[(((((int)threadIdx.x) * 16) + 9))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(170)] * kernel_shared[(((((int)threadIdx.x) * 16) + 10))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(171)] * kernel_shared[(((((int)threadIdx.x) * 16) + 10))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(172)] * kernel_shared[(((((int)threadIdx.x) * 16) + 10))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(173)] * kernel_shared[(((((int)threadIdx.x) * 16) + 10))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(174)] * kernel_shared[(((((int)threadIdx.x) * 16) + 10))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(175)] * kernel_shared[(((((int)threadIdx.x) * 16) + 10))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(176)] * kernel_shared[(((((int)threadIdx.x) * 16) + 10))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(177)] * kernel_shared[(((((int)threadIdx.x) * 16) + 10))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(178)] * kernel_shared[(((((int)threadIdx.x) * 16) + 10))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(179)] * kernel_shared[(((((int)threadIdx.x) * 16) + 10))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(180)] * kernel_shared[(((((int)threadIdx.x) * 16) + 10))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(181)] * kernel_shared[(((((int)threadIdx.x) * 16) + 10))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(182)] * kernel_shared[(((((int)threadIdx.x) * 16) + 10))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(183)] * kernel_shared[(((((int)threadIdx.x) * 16) + 10))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(184)] * kernel_shared[(((((int)threadIdx.x) * 16) + 10))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(185)] * kernel_shared[(((((int)threadIdx.x) * 16) + 10))]));
    compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(186)] * kernel_shared[(((((int)threadIdx.x) * 16) + 10))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(187)] * kernel_shared[(((((int)threadIdx.x) * 16) + 11))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(188)] * kernel_shared[(((((int)threadIdx.x) * 16) + 11))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(189)] * kernel_shared[(((((int)threadIdx.x) * 16) + 11))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(190)] * kernel_shared[(((((int)threadIdx.x) * 16) + 11))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(191)] * kernel_shared[(((((int)threadIdx.x) * 16) + 11))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(192)] * kernel_shared[(((((int)threadIdx.x) * 16) + 11))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(193)] * kernel_shared[(((((int)threadIdx.x) * 16) + 11))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(194)] * kernel_shared[(((((int)threadIdx.x) * 16) + 11))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(195)] * kernel_shared[(((((int)threadIdx.x) * 16) + 11))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(196)] * kernel_shared[(((((int)threadIdx.x) * 16) + 11))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(197)] * kernel_shared[(((((int)threadIdx.x) * 16) + 11))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(198)] * kernel_shared[(((((int)threadIdx.x) * 16) + 11))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(199)] * kernel_shared[(((((int)threadIdx.x) * 16) + 11))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(200)] * kernel_shared[(((((int)threadIdx.x) * 16) + 11))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(201)] * kernel_shared[(((((int)threadIdx.x) * 16) + 11))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(202)] * kernel_shared[(((((int)threadIdx.x) * 16) + 11))]));
    compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(203)] * kernel_shared[(((((int)threadIdx.x) * 16) + 11))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(204)] * kernel_shared[(((((int)threadIdx.x) * 16) + 12))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(205)] * kernel_shared[(((((int)threadIdx.x) * 16) + 12))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(206)] * kernel_shared[(((((int)threadIdx.x) * 16) + 12))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(207)] * kernel_shared[(((((int)threadIdx.x) * 16) + 12))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(208)] * kernel_shared[(((((int)threadIdx.x) * 16) + 12))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(209)] * kernel_shared[(((((int)threadIdx.x) * 16) + 12))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(210)] * kernel_shared[(((((int)threadIdx.x) * 16) + 12))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(211)] * kernel_shared[(((((int)threadIdx.x) * 16) + 12))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(212)] * kernel_shared[(((((int)threadIdx.x) * 16) + 12))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(213)] * kernel_shared[(((((int)threadIdx.x) * 16) + 12))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(214)] * kernel_shared[(((((int)threadIdx.x) * 16) + 12))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(215)] * kernel_shared[(((((int)threadIdx.x) * 16) + 12))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(216)] * kernel_shared[(((((int)threadIdx.x) * 16) + 12))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(217)] * kernel_shared[(((((int)threadIdx.x) * 16) + 12))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(218)] * kernel_shared[(((((int)threadIdx.x) * 16) + 12))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(219)] * kernel_shared[(((((int)threadIdx.x) * 16) + 12))]));
    compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(220)] * kernel_shared[(((((int)threadIdx.x) * 16) + 12))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(221)] * kernel_shared[(((((int)threadIdx.x) * 16) + 13))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(222)] * kernel_shared[(((((int)threadIdx.x) * 16) + 13))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(223)] * kernel_shared[(((((int)threadIdx.x) * 16) + 13))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(224)] * kernel_shared[(((((int)threadIdx.x) * 16) + 13))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(225)] * kernel_shared[(((((int)threadIdx.x) * 16) + 13))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(226)] * kernel_shared[(((((int)threadIdx.x) * 16) + 13))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(227)] * kernel_shared[(((((int)threadIdx.x) * 16) + 13))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(228)] * kernel_shared[(((((int)threadIdx.x) * 16) + 13))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(229)] * kernel_shared[(((((int)threadIdx.x) * 16) + 13))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(230)] * kernel_shared[(((((int)threadIdx.x) * 16) + 13))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(231)] * kernel_shared[(((((int)threadIdx.x) * 16) + 13))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(232)] * kernel_shared[(((((int)threadIdx.x) * 16) + 13))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(233)] * kernel_shared[(((((int)threadIdx.x) * 16) + 13))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(234)] * kernel_shared[(((((int)threadIdx.x) * 16) + 13))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(235)] * kernel_shared[(((((int)threadIdx.x) * 16) + 13))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(236)] * kernel_shared[(((((int)threadIdx.x) * 16) + 13))]));
    compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(237)] * kernel_shared[(((((int)threadIdx.x) * 16) + 13))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(238)] * kernel_shared[(((((int)threadIdx.x) * 16) + 14))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(239)] * kernel_shared[(((((int)threadIdx.x) * 16) + 14))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(240)] * kernel_shared[(((((int)threadIdx.x) * 16) + 14))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(241)] * kernel_shared[(((((int)threadIdx.x) * 16) + 14))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(242)] * kernel_shared[(((((int)threadIdx.x) * 16) + 14))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(243)] * kernel_shared[(((((int)threadIdx.x) * 16) + 14))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(244)] * kernel_shared[(((((int)threadIdx.x) * 16) + 14))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(245)] * kernel_shared[(((((int)threadIdx.x) * 16) + 14))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(246)] * kernel_shared[(((((int)threadIdx.x) * 16) + 14))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(247)] * kernel_shared[(((((int)threadIdx.x) * 16) + 14))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(248)] * kernel_shared[(((((int)threadIdx.x) * 16) + 14))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(249)] * kernel_shared[(((((int)threadIdx.x) * 16) + 14))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(250)] * kernel_shared[(((((int)threadIdx.x) * 16) + 14))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(251)] * kernel_shared[(((((int)threadIdx.x) * 16) + 14))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(252)] * kernel_shared[(((((int)threadIdx.x) * 16) + 14))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(253)] * kernel_shared[(((((int)threadIdx.x) * 16) + 14))]));
    compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(254)] * kernel_shared[(((((int)threadIdx.x) * 16) + 14))]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(255)] * kernel_shared[(((((int)threadIdx.x) * 16) + 15))]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(256)] * kernel_shared[(((((int)threadIdx.x) * 16) + 15))]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(257)] * kernel_shared[(((((int)threadIdx.x) * 16) + 15))]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(258)] * kernel_shared[(((((int)threadIdx.x) * 16) + 15))]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(259)] * kernel_shared[(((((int)threadIdx.x) * 16) + 15))]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(260)] * kernel_shared[(((((int)threadIdx.x) * 16) + 15))]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(261)] * kernel_shared[(((((int)threadIdx.x) * 16) + 15))]));
    compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(262)] * kernel_shared[(((((int)threadIdx.x) * 16) + 15))]));
    compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(263)] * kernel_shared[(((((int)threadIdx.x) * 16) + 15))]));
    compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(264)] * kernel_shared[(((((int)threadIdx.x) * 16) + 15))]));
    compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(265)] * kernel_shared[(((((int)threadIdx.x) * 16) + 15))]));
    compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(266)] * kernel_shared[(((((int)threadIdx.x) * 16) + 15))]));
    compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(267)] * kernel_shared[(((((int)threadIdx.x) * 16) + 15))]));
    compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(268)] * kernel_shared[(((((int)threadIdx.x) * 16) + 15))]));
    compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(269)] * kernel_shared[(((((int)threadIdx.x) * 16) + 15))]));
    compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(270)] * kernel_shared[(((((int)threadIdx.x) * 16) + 15))]));
    compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(271)] * kernel_shared[(((((int)threadIdx.x) * 16) + 15))]));
  }
  for (int xx_inner = 0; xx_inner < 17; ++xx_inner) {
    compute[((((((((int)blockIdx.x) / 17) * 36992) + (((int)threadIdx.x) * 289)) + ((((int)blockIdx.x) % 17) * 17)) + xx_inner))] = compute_local[(xx_inner)];
  }
}



void conv_kernel_wrapper(int N_B, int N_C, int N_H, int N_W, int N_F, int N_R, int N_S, int PaddingH, int PaddingW,
                        int StrideH, int StrideW, int N_X, int N_Y, const float *Input, 
                        const float *Kernel, float *Output, int itr) {
                        hipEvent_t start;
                        CHECK(hipEventCreate(&start));
                        hipEvent_t stop;
                        CHECK(hipEventCreate(&stop));
                        float *dev_Input;
                        float *dev_Kernel;
                        float *dev_Output;
                    
                        CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                        CHECK(hipMemcpy(dev_Kernel, Kernel, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                        CHECK(hipMemcpy(dev_Input, Input, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Output, sizeof(float) * N_B * N_F * N_Y * N_X));
                        CHECK(hipMemset(dev_Output, 0, sizeof(float) * N_B * N_F * N_Y * N_X));
                    
dim3 size_grid(68,1,1);

dim3 size_block(128,1,1);
 
                       printf("warmup....\n");
                        default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                    
                        double *time_arr = (double *)malloc(itr * sizeof(double));
                        for (int j = 1; j < itr; j++) {
                            CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                            CHECK(hipMemcpy(dev_Kernel, Kernel+j*N_F * N_C * N_R * N_S, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                            CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                            CHECK(hipMemcpy(dev_Input, Input+j*N_B * N_C * N_H * N_W, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                            /* Record the start event*/
                            CHECK(hipEventRecord(start, NULL));
                            default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                            /* Record the end event */
                            CHECK(hipEventRecord(stop, NULL));
                            /* Wait for the stop event to complete*/
                            CHECK(hipEventSynchronize(stop));
                            float msec = 0.0f;
                            hipEventElapsedTime(&msec, start, stop);
                            time_arr[j] = msec;
                        }
                    
                       
                    
                        //hipError_t err = hipGetLastError();
                        //if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
                        double msecTotal = 0;
                        for (int j = 1; j < itr; j++) {   
                            printf("%0.5lf\n", time_arr[j]);
                            msecTotal += time_arr[j];
                        }
                    
                        double msecPerMatrixMul = (msecTotal) / itr;
                        double flopsPerMatrixMul = 2.0 * N_B * N_C * N_X * N_Y * N_F * N_R * N_S;
                    
                        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
                    
                        printf("\n===========================================================\n");
                        printf("Input size: %d X %d X %d X %d\n", N_B, N_C, N_H, N_W);
                        printf("Kernel size: %d X %d X %d X %d\n", N_F, N_C, N_R, N_S);
                        printf("Output size: %d X %d X %d X %d\n", N_B, N_F, N_Y, N_X);
                        printf("flops: %0.5lf\n", (flopsPerMatrixMul * 1.0e-9f));
                        printf("Time is %0.5lf \n", (msecPerMatrixMul / 1000.0f));
                        printf("Gflops %0.5lf \n", gigaFlops);
                        printf("===========================================================\n");
                    
                        /*write back to Host Output -- batch size is 1*/
                        CHECK(hipMemcpy(Output, dev_Output, sizeof(float) * N_B * N_F * N_X * N_Y, hipMemcpyDeviceToHost));
                    
                        CHECK(hipFree(dev_Input));
                        CHECK(hipFree(dev_Kernel));
                        CHECK(hipFree(dev_Output));
                    
                    }
            