#include "hip/hip_runtime.h"

            #include <cassert>
            #include <stdlib.h>
            #include <hip/hip_runtime.h>
            #include "common.h"
            

            
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[18];
  __shared__ float pad_temp_shared[1280];
  __shared__ float kernel_shared[1152];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(9)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(10)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(11)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(12)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(13)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(14)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  compute_local[(15)] = 0.000000e+00f;
  compute_local[(7)] = 0.000000e+00f;
  compute_local[(16)] = 0.000000e+00f;
  compute_local[(8)] = 0.000000e+00f;
  compute_local[(17)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[(((int)threadIdx.x))] = data[((((((rc_outer_outer * 96800) + (((((int)blockIdx.x) % 108) / 6) * 660)) + ((((int)threadIdx.x) / 20) * 110)) + ((((int)blockIdx.x) % 6) * 18)) + (((int)threadIdx.x) % 20)))];
    pad_temp_shared[((((int)threadIdx.x) + 96))] = data[(((((((rc_outer_outer * 96800) + (((((int)threadIdx.x) + 96) / 160) * 12100)) + (((((int)blockIdx.x) % 108) / 6) * 660)) + ((((((int)threadIdx.x) + 96) % 160) / 20) * 110)) + ((((int)blockIdx.x) % 6) * 18)) + ((((int)threadIdx.x) + 16) % 20)))];
    pad_temp_shared[((((int)threadIdx.x) + 192))] = data[(((((((rc_outer_outer * 96800) + (((((int)threadIdx.x) + 192) / 160) * 12100)) + (((((int)blockIdx.x) % 108) / 6) * 660)) + (((((int)threadIdx.x) + 32) / 20) * 110)) + ((((int)blockIdx.x) % 6) * 18)) + ((((int)threadIdx.x) + 12) % 20)))];
    pad_temp_shared[((((int)threadIdx.x) + 288))] = data[(((((((rc_outer_outer * 96800) + (((((int)threadIdx.x) + 288) / 160) * 12100)) + (((((int)blockIdx.x) % 108) / 6) * 660)) + ((((((int)threadIdx.x) + 128) % 160) / 20) * 110)) + ((((int)blockIdx.x) % 6) * 18)) + ((((int)threadIdx.x) + 8) % 20)))];
    pad_temp_shared[((((int)threadIdx.x) + 384))] = data[(((((((rc_outer_outer * 96800) + (((((int)threadIdx.x) + 384) / 160) * 12100)) + (((((int)blockIdx.x) % 108) / 6) * 660)) + (((((int)threadIdx.x) + 64) / 20) * 110)) + ((((int)blockIdx.x) % 6) * 18)) + ((((int)threadIdx.x) + 4) % 20)))];
    pad_temp_shared[((((int)threadIdx.x) + 480))] = data[(((((((rc_outer_outer * 96800) + (((((int)blockIdx.x) % 108) / 6) * 660)) + ((((int)threadIdx.x) / 20) * 110)) + ((((int)blockIdx.x) % 6) * 18)) + (((int)threadIdx.x) % 20)) + 36300))];
    pad_temp_shared[((((int)threadIdx.x) + 576))] = data[(((((((rc_outer_outer * 96800) + (((((int)threadIdx.x) + 576) / 160) * 12100)) + (((((int)blockIdx.x) % 108) / 6) * 660)) + ((((((int)threadIdx.x) + 96) % 160) / 20) * 110)) + ((((int)blockIdx.x) % 6) * 18)) + ((((int)threadIdx.x) + 16) % 20)))];
    pad_temp_shared[((((int)threadIdx.x) + 672))] = data[(((((((rc_outer_outer * 96800) + (((((int)threadIdx.x) + 672) / 160) * 12100)) + (((((int)blockIdx.x) % 108) / 6) * 660)) + (((((int)threadIdx.x) + 32) / 20) * 110)) + ((((int)blockIdx.x) % 6) * 18)) + ((((int)threadIdx.x) + 12) % 20)))];
    pad_temp_shared[((((int)threadIdx.x) + 768))] = data[(((((((rc_outer_outer * 96800) + (((((int)threadIdx.x) + 768) / 160) * 12100)) + (((((int)blockIdx.x) % 108) / 6) * 660)) + ((((((int)threadIdx.x) + 128) % 160) / 20) * 110)) + ((((int)blockIdx.x) % 6) * 18)) + ((((int)threadIdx.x) + 8) % 20)))];
    pad_temp_shared[((((int)threadIdx.x) + 864))] = data[(((((((rc_outer_outer * 96800) + (((((int)threadIdx.x) + 864) / 160) * 12100)) + (((((int)blockIdx.x) % 108) / 6) * 660)) + (((((int)threadIdx.x) + 64) / 20) * 110)) + ((((int)blockIdx.x) % 6) * 18)) + ((((int)threadIdx.x) + 4) % 20)))];
    pad_temp_shared[((((int)threadIdx.x) + 960))] = data[(((((((rc_outer_outer * 96800) + (((((int)blockIdx.x) % 108) / 6) * 660)) + ((((int)threadIdx.x) / 20) * 110)) + ((((int)blockIdx.x) % 6) * 18)) + (((int)threadIdx.x) % 20)) + 72600))];
    pad_temp_shared[((((int)threadIdx.x) + 1056))] = data[(((((((rc_outer_outer * 96800) + (((((int)threadIdx.x) + 1056) / 160) * 12100)) + (((((int)blockIdx.x) % 108) / 6) * 660)) + ((((((int)threadIdx.x) + 96) % 160) / 20) * 110)) + ((((int)blockIdx.x) % 6) * 18)) + ((((int)threadIdx.x) + 16) % 20)))];
    pad_temp_shared[((((int)threadIdx.x) + 1152))] = data[(((((((rc_outer_outer * 96800) + (((((int)threadIdx.x) + 1152) / 160) * 12100)) + (((((int)blockIdx.x) % 108) / 6) * 660)) + (((((int)threadIdx.x) + 32) / 20) * 110)) + ((((int)blockIdx.x) % 6) * 18)) + ((((int)threadIdx.x) + 12) % 20)))];
    if (((int)threadIdx.x) < 32) {
      pad_temp_shared[((((int)threadIdx.x) + 1248))] = data[(((((((rc_outer_outer * 96800) + (((((int)threadIdx.x) + 1248) / 160) * 12100)) + (((((int)blockIdx.x) % 108) / 6) * 660)) + (((((int)threadIdx.x) + 128) / 20) * 110)) + ((((int)blockIdx.x) % 6) * 18)) + ((((int)threadIdx.x) + 8) % 20)))];
    }
      ((float3*)(kernel_shared + ((((int)threadIdx.x) * 3))))[0] = ((float3*)(kernel + ((((((((int)blockIdx.x) / 108) * 4608) + ((((int)threadIdx.x) / 24) * 288)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 24) * 3)))))[0];
      ((float3*)(kernel_shared + (((((int)threadIdx.x) * 3) + 288))))[0] = ((float3*)(kernel + (((((((((int)blockIdx.x) / 108) * 4608) + ((((int)threadIdx.x) / 24) * 288)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 24) * 3)) + 1152))))[0];
      ((float3*)(kernel_shared + (((((int)threadIdx.x) * 3) + 576))))[0] = ((float3*)(kernel + (((((((((int)blockIdx.x) / 108) * 4608) + ((((int)threadIdx.x) / 24) * 288)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 24) * 3)) + 2304))))[0];
      ((float3*)(kernel_shared + (((((int)threadIdx.x) * 3) + 864))))[0] = ((float3*)(kernel + (((((((((int)blockIdx.x) / 108) * 4608) + ((((int)threadIdx.x) / 24) * 288)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 24) * 3)) + 3456))))[0];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int ry_outer_inner = 0; ry_outer_inner < 3; ++ry_outer_inner) {
        for (int rx_outer_inner = 0; rx_outer_inner < 3; ++rx_outer_inner) {
          compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner))] * kernel_shared[((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner))]));
          compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner))] * kernel_shared[(((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 576))]));
          compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 160))] * kernel_shared[(((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 9))]));
          compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 160))] * kernel_shared[(((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 585))]));
          compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 1))] * kernel_shared[((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner))]));
          compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 1))] * kernel_shared[(((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 576))]));
          compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 161))] * kernel_shared[(((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 9))]));
          compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 161))] * kernel_shared[(((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 585))]));
          compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 2))] * kernel_shared[((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner))]));
          compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 2))] * kernel_shared[(((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 576))]));
          compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 162))] * kernel_shared[(((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 9))]));
          compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 162))] * kernel_shared[(((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 585))]));
          compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 20))] * kernel_shared[((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner))]));
          compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 20))] * kernel_shared[(((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 576))]));
          compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 180))] * kernel_shared[(((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 9))]));
          compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 180))] * kernel_shared[(((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 585))]));
          compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 21))] * kernel_shared[((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner))]));
          compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 21))] * kernel_shared[(((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 576))]));
          compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 181))] * kernel_shared[(((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 9))]));
          compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 181))] * kernel_shared[(((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 585))]));
          compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 22))] * kernel_shared[((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner))]));
          compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 22))] * kernel_shared[(((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 576))]));
          compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 182))] * kernel_shared[(((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 9))]));
          compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 182))] * kernel_shared[(((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 585))]));
          compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 40))] * kernel_shared[((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner))]));
          compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 40))] * kernel_shared[(((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 576))]));
          compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 200))] * kernel_shared[(((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 9))]));
          compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 200))] * kernel_shared[(((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 585))]));
          compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 41))] * kernel_shared[((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner))]));
          compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 41))] * kernel_shared[(((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 576))]));
          compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 201))] * kernel_shared[(((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 9))]));
          compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 201))] * kernel_shared[(((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 585))]));
          compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 42))] * kernel_shared[((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner))]));
          compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 42))] * kernel_shared[(((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 576))]));
          compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 202))] * kernel_shared[(((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 9))]));
          compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared[(((((((rc_outer_inner * 320) + (((((int)threadIdx.x) % 12) / 6) * 60)) + (ry_outer_inner * 20)) + ((((int)threadIdx.x) % 6) * 3)) + rx_outer_inner) + 202))] * kernel_shared[(((((((((int)threadIdx.x) / 12) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 585))]));
        }
      }
    }
  }
  for (int yy_inner = 0; yy_inner < 3; ++yy_inner) {
    for (int xx_inner = 0; xx_inner < 3; ++xx_inner) {
      compute[((((((((((((int)blockIdx.x) / 108) * 186624) + ((((int)threadIdx.x) / 12) * 11664)) + (((((int)blockIdx.x) % 108) / 6) * 648)) + (((((int)threadIdx.x) % 12) / 6) * 324)) + (yy_inner * 108)) + ((((int)blockIdx.x) % 6) * 18)) + ((((int)threadIdx.x) % 6) * 3)) + xx_inner))] = compute_local[(((yy_inner * 3) + xx_inner))];
      compute[(((((((((((((int)blockIdx.x) / 108) * 186624) + ((((int)threadIdx.x) / 12) * 11664)) + (((((int)blockIdx.x) % 108) / 6) * 648)) + (((((int)threadIdx.x) % 12) / 6) * 324)) + (yy_inner * 108)) + ((((int)blockIdx.x) % 6) * 18)) + ((((int)threadIdx.x) % 6) * 3)) + xx_inner) + 93312))] = compute_local[((((yy_inner * 3) + xx_inner) + 9))];
    }
  }
}



void conv_kernel_wrapper(int N_B, int N_C, int N_H, int N_W, int N_F, int N_R, int N_S, int PaddingH, int PaddingW,
                        int StrideH, int StrideW, int N_X, int N_Y, const float *Input, 
                        const float *Kernel, float *Output, int itr) {
                        hipEvent_t start;
                        CHECK(hipEventCreate(&start));
                        hipEvent_t stop;
                        CHECK(hipEventCreate(&stop));
                        float *dev_Input;
                        float *dev_Kernel;
                        float *dev_Output;
                    
                        CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                        CHECK(hipMemcpy(dev_Kernel, Kernel, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                        CHECK(hipMemcpy(dev_Input, Input, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                        CHECK(hipMalloc(&dev_Output, sizeof(float) * N_B * N_F * N_Y * N_X));
                        CHECK(hipMemset(dev_Output, 0, sizeof(float) * N_B * N_F * N_Y * N_X));
                    
dim3 size_grid(216,1,1);

dim3 size_block(96,1,1);
 
                       printf("warmup....\n");
                        default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                    
                        double *time_arr = (double *)malloc(itr * sizeof(double));
                        for (int j = 1; j < itr; j++) {
                            CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
                            CHECK(hipMemcpy(dev_Kernel, Kernel+j*N_F * N_C * N_R * N_S, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
                    
                            CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
                            CHECK(hipMemcpy(dev_Input, Input+j*N_B * N_C * N_H * N_W, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
                    
                            /* Record the start event*/
                            CHECK(hipEventRecord(start, NULL));
                            default_function_kernel0 <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
                            /* Record the end event */
                            CHECK(hipEventRecord(stop, NULL));
                            /* Wait for the stop event to complete*/
                            CHECK(hipEventSynchronize(stop));
                            float msec = 0.0f;
                            hipEventElapsedTime(&msec, start, stop);
                            time_arr[j] = msec;
                        }
                    
                       
                    
                        //hipError_t err = hipGetLastError();
                        //if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
                        double msecTotal = 0;
                        for (int j = 1; j < itr; j++) {   
                            printf("%0.5lf\n", time_arr[j]);
                            msecTotal += time_arr[j];
                        }
                    
                        double msecPerMatrixMul = (msecTotal) / itr;
                        double flopsPerMatrixMul = 2.0 * N_B * N_C * N_X * N_Y * N_F * N_R * N_S;
                    
                        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
                    
                        printf("\n===========================================================\n");
                        printf("Input size: %d X %d X %d X %d\n", N_B, N_C, N_H, N_W);
                        printf("Kernel size: %d X %d X %d X %d\n", N_F, N_C, N_R, N_S);
                        printf("Output size: %d X %d X %d X %d\n", N_B, N_F, N_Y, N_X);
                        printf("flops: %0.5lf\n", (flopsPerMatrixMul * 1.0e-9f));
                        printf("Time is %0.5lf \n", (msecPerMatrixMul / 1000.0f));
                        printf("Gflops %0.5lf \n", gigaFlops);
                        printf("===========================================================\n");
                    
                        /*write back to Host Output -- batch size is 1*/
                        CHECK(hipMemcpy(Output, dev_Output, sizeof(float) * N_B * N_F * N_X * N_Y, hipMemcpyDeviceToHost));
                    
                        CHECK(hipFree(dev_Input));
                        CHECK(hipFree(dev_Kernel));
                        CHECK(hipFree(dev_Output));
                    
                    }
            