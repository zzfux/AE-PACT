#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "common.h"

///Only support crsf format kernel
// TODO: ASSUME Input should be padded already
// x=w y=h z=k
__global__ void
conv_kernel(const TYPE *__restrict__ Input /*(C*H*W)*/,
            const TYPE *__restrict__ Kernel /*(C*R*S*F)*/,
            TYPE *__restrict__ Output /*(F*C*Y*X)*/) {

    /* This is an attempt to build on register optimization with shared memory with an alternate view of block orientation with a split c on global 
    that also performs the operation with stride 2. */

    // Calc'ed values and vars
    int k, h, w, c, i, j, r, s, row, kReg, hReg, wReg, kGlobal;
   // Total sizes of input for register operations
    int inTotalH = 2*H+R-2; // TODO: This works but does not seem right
    int inTotalW = 2*W+S-2;


    // Compinsate for c block increase
    int blockIdxXAdjusted = blockIdx.x % BlockXCoverageNumber;

    // Find the upper corner in 3d space
    int kIndex = blockIdxXAdjusted * blockDim.x * T_k + threadIdx.x;
    //int wIndex = (blockIdx.y * blockDim.y + threadIdx.y) * T_w;
    //int hIndex = (blockIdx.z * blockDim.z + threadIdx.z) * T_h;
    int kWarpReach = 32 * T_k; // TODO: Should this be 32 exactly? - Yes?

    //int cRange = C / T_c; // TODO: Assume this will work since T_c is currently 1, but may not always be so
    //int inAdjustedTileH = T_h + R - 1; // Made into constants
    //int inAdjustedTileW = T_w + S - 1;

    //C offset calculation
    int cStart = (blockIdx.x / BlockXCoverageNumber) * CDivisionSize;
    int cBound = cStart + CDivisionSize;

    // Registers
    TYPE regOut[T_k][T_h][T_w]; //4x4x5
    TYPE regIn[inAdjustedTileH][inAdjustedTileW]; //6x7
    TYPE oneKern;

    // Zero out the registers
    for(k = 0; k < T_k; k++) {
        for (h = 0; h < T_h; h++) {
            for (w = 0; w < T_w; w++) {
                regOut[k][h][w] = 0.0f;
            }
        }
    }

    // Shared memory
    //int memTileH = T_h * blockDim.y + R - 1; // Made into constants T_h * B_h + R - 1
    //int memTileW = T_w * blockDim.x + S - 1; // T_w * B_w + S - 1
    int adjustedRowIndexMemTile = threadIdx.z * blockDim.y + threadIdx.y;
    int jumpDist = blockDim.z * blockDim.y;
    int hIndexBlock = blockIdx.z * blockDim.z * T_h;
    int wIndexBlock = blockIdx.y * blockDim.y * T_w;

    __shared__ TYPE shareBuffer_In[memTileH][memTileW]; // TODO: make non-square to avoid conflicts - if needed
    __shared__ TYPE shareBuffer_Out[OutBlockH][OutBlockW][OutBlockK+1];

    // Collapse all c into output
    for (c = cStart; c < cBound; c++) { // Assuming T_c is 1
        // Fill in shared memory - together
        for(row = adjustedRowIndexMemTile; row < memTileH; row += jumpDist) {
            if(threadIdx.x < memTileW) {
                shareBuffer_In[row][threadIdx.x] = Input[ (c)*inTotalH*inTotalW + (2 * hIndexBlock + row)*inTotalW + (2 * wIndexBlock + threadIdx.x) ];
            }
        }

        __syncthreads();

        // Fill regesters In - just ours
        for(i = 0; i < inAdjustedTileH; i++) {
            for(j = 0; j < inAdjustedTileW; j++) {
                regIn[i][j] = shareBuffer_In[threadIdx.z*2*T_h + i][threadIdx.y*2*T_w + j];
            }
        }
        
        // Use the input in regesters for all k in the tile
        kReg = 0;
#pragma unroll(1) // ReplaceLine 000001
        for (kGlobal = 0; kGlobal < kWarpReach; kGlobal+=32) { // 4
            // Do actual calculations
#pragma unroll(3) // ReplaceLine 000002
            for (r = 0; r < R; r++) {
#pragma unroll(3) // ReplaceLine 000003
                for (s = 0; s < S; s++) {
                    oneKern = Kernel[ (c)*R*S*K + (r)*S*K + (s)*K + (kIndex + kGlobal) ];
                    hReg = 0;
#pragma unroll(2) // ReplaceLine 000004
                    for (h = r; h < (T_h)*2 + r - 1; h+=2) {
                        wReg = 0;
#pragma unroll(14) // ReplaceLine 000005
                        for (w = s; w < (T_w)*2 + s - 1; w+=2) {
                            regOut[kReg][hReg][wReg] += regIn[h][w] * oneKern;
                            wReg += 1;
                        }
                        hReg += 1;
                    }
                }
            }
            kReg++; // RegisterK
        } // End k

        __syncthreads(); // Sync for next loop

    } // End c

    int kIndexTBlock = threadIdx.x;
    int hIndexTBlock = threadIdx.z * T_h;
    int wIndexTBlock = threadIdx.y * T_w;
    kWarpReach = 32;
    int kIndexBlock;

    for(k = 0; k < T_k; k++) {

        // Fill Out from Reg to shared mem
        for (h = 0; h < T_h; h++) {
            for (w = 0; w < T_w; w++) {
                shareBuffer_Out[hIndexTBlock + h][wIndexTBlock + w][kIndexTBlock] = regOut[k][h][w];
            }
        }

        __syncthreads();

        kIndexBlock = blockIdxXAdjusted * blockDim.x * T_k + k * 32;

        // Fill Out from shared memory
        for(kGlobal = 0; kGlobal < OutBlockK; kGlobal++) {
            for(row = adjustedRowIndexMemTile; row < OutBlockH; row += jumpDist) {
                if(threadIdx.x < OutBlockW) {
                    atomicAdd(&Output[ (kIndexBlock + kGlobal)*H*W + (hIndexBlock + row)*W + (wIndexBlock + threadIdx.x) ], shareBuffer_Out[row][threadIdx.x][kGlobal]);
                }
            }
        }

        __syncthreads();
    }

} // END



void conv_kernel_wrapper(const float *Input,
    const float *Kernel, float *Output, int itr) {
    hipEvent_t start;
    CHECK(hipEventCreate(&start));
    hipEvent_t stop;
    CHECK(hipEventCreate(&stop));
    float msec = 0.0f;
    float *dev_Input;
    float *dev_Kernel;
    float *dev_Output;

    CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
    CHECK(hipMemcpy(dev_Kernel, Kernel, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));

    CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
    CHECK(hipMemcpy(dev_Input, Input, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));

    CHECK(hipMalloc(&dev_Output, sizeof(float) * N_B * N_F * N_Y * N_X));
    CHECK(hipMemset(dev_Output, 0, sizeof(float) * N_B * N_F * N_Y * N_X));
   /*call conv_kernel() */
    dim3 size_block(32, B_w, B_h); 
    dim3 size_grid((N_F / (T_k*size_block.x)) * CBlockMult, W / (T_w * B_w), H / (T_h * B_h) );
    printf("warmup....\n");
    printf("block shape <%d, %d, %d>\n", 32, B_w, B_h);
    printf("grid shape <%d, %d, %d>\n", (N_F / (T_k*size_block.x)) * CBlockMult, W / (T_w * B_w), H / (T_h * B_h)  );
    printf("%d %d %d %d %d | %d %d %d %d %d %d %d  \n", K, H, W, C, R, T_k, T_h, T_w, 32, B_h, B_w, CBlockMult);
    conv_kernel <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
    
    CHECK(hipMemcpy(Output, dev_Output, sizeof(float) * N_B * N_F * N_X * N_Y, hipMemcpyDeviceToHost));
    CHECK(hipFree(dev_Input));
    CHECK(hipFree(dev_Kernel));
    CHECK(hipFree(dev_Output));


    double *time_arr = (double *)malloc(itr * sizeof(double));
    for (int j = 1; j < itr; j++) {
        CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
        CHECK(hipMemcpy(dev_Kernel, Kernel+j*N_F * N_C * N_R * N_S, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
        CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
        CHECK(hipMemcpy(dev_Input, Input+j*N_B * N_C * N_H * N_W, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
        CHECK(hipMalloc(&dev_Output, sizeof(float) * N_B * N_F * N_Y * N_X));
        CHECK(hipMemset(dev_Output, 0, sizeof(float) * N_B * N_F * N_Y * N_X));
        /* Record the start event*/
        CHECK(hipEventRecord(start, NULL));
        conv_kernel <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
        /* Record the end event */
        CHECK(hipEventRecord(stop, NULL));
        /* Wait for the stop event to complete*/
        CHECK(hipEventSynchronize(stop));
        float msec = 0.0f;
        hipEventElapsedTime(&msec, start, stop);
        time_arr[j] = msec;
        CHECK(hipFree(dev_Input));
        CHECK(hipFree(dev_Kernel));
        CHECK(hipFree(dev_Output));
    }

    for (int j = 1; j < itr; j++) {   
        printf("time %0.5lf \n", time_arr[j]);
    }

    return;
}
