#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "common.h"

__global__ void
conv_kernel(const TYPE *__restrict__ Input /*(N*C*H*W)*/,
            const TYPE *__restrict__ Kernel /*(C*R*S*F)*/,
            TYPE *__restrict__ Output /*(N*F*Y*X)*/) {

    int k, h, w, c, i, j, r, s, row, kReg, kGlobal;
    // Total sizes of input for register operations
    int inTotalH = H+R-1;
    int inTotalW = W+S-1;

    // Compinsate for c block increase
    int blockIdxXAdjusted = blockIdx.x % BlockXCoverageNumber;

    // Find the upper corner in 3d space
    int kIndex = blockIdxXAdjusted * blockDim.x * T_k + threadIdx.x;
    int kWarpReach = W_k * T_k;

    //C offset calculation
    int cStart = (blockIdx.x / BlockXCoverageNumber) * CDivisionSize;
    int cBound = cStart + CDivisionSize;

    // Registers
    TYPE regOut[T_k][T_h][T_w];
    TYPE regIn[inAdjustedTileW];
    TYPE regK[2];

    // Shared memory
    //int adjustedRowIndexMemTile = threadIdx.z * blockDim.y + threadIdx.y;
    int hIndexBlock = blockIdx.z * blockDim.z * T_h;
    int wIndexBlock = blockIdx.y * blockDim.y * T_w;

    __shared__ TYPE shareBuffer_In[2][memTileH][memTileW];
    __shared__ TYPE shareBuffer_Out[W_k*B_k][OutBlockH*OutBlockW];
    __shared__ TYPE shareBuffer_Ker[2][R*S][T_k*W_k*B_k];


    // Zero out the registers
    for(k = 0; k < T_k; k++) {
        for (h = 0; h < T_h; h++) {
            for (w = 0; w < T_w; w++) {
                regOut[k][h][w] = 0.0f;
            }
        }
    }

    int linear_id_in_TB = threadIdx.z * blockDim.x * blockDim.y+ threadIdx.y * blockDim.x + threadIdx.x;
    int num_of_warp = (blockDim.x * blockDim.y * blockDim.z) >> 5;
    int lane_id = linear_id_in_TB & 31;
    int warp_id = linear_id_in_TB >> 5;
    int jumpDist = num_of_warp;


    float *__restrict__ ingmem_ptr = &((float *) Input)[hIndexBlock * inTotalW];
    float *__restrict__ insbuf_ptr = &((float *) shareBuffer_In)[0];
    for (int i = 0; i < (memTileH*memTileW)>>1; i += (B_h*B_w*B_k*32)){
        if (i + linear_id_in_TB < (memTileH*memTileW)>>1){
            *((float2 *)insbuf_ptr+i+linear_id_in_TB) =  *((float2 *)ingmem_ptr+i+linear_id_in_TB);
        }
    }

    for (row = warp_id; row < R*S; row += jumpDist) {
        for (int rp = 0; rp < CEIL(OutBlockK, 32); rp++) {
            if (lane_id + rp*32 < OutBlockK){
                shareBuffer_Ker[0][row][lane_id+ rp*32] = Kernel[(row)*K + blockIdxXAdjusted * blockDim.x * T_k + lane_id + rp*32];
            }    
        }
    }

    int flip = 1;

    // Collapse all c into output
    for (c = cStart; c < cBound; c++) { // Assuming T_c is 1
       __syncthreads();
        if ( c+1 < cBound){
            float *__restrict__ ingmem_ptr = &((float *) Input)[(c+1)*inTotalH*inTotalW + hIndexBlock * inTotalW];
            float *__restrict__ insbuf_ptr = &((float *) shareBuffer_In)[flip*memTileW*memTileH];
            for (int i = 0; i < memTileH*memTileW/2; i += (B_h*B_w*B_k*32)){
                if (i + linear_id_in_TB < memTileH*memTileW/2){
                    *((float2 *)insbuf_ptr+i+linear_id_in_TB) =  *((float2 *)ingmem_ptr+i+linear_id_in_TB);
                }
            }

            // dbf kernSM
            for (row = warp_id; row < R*S; row += jumpDist) {
                for (int rp = 0; rp < CEIL(OutBlockK, 32); rp++) {
                    if (lane_id + rp*32 < OutBlockK){
                        shareBuffer_Ker[flip][row][lane_id + rp*32] = Kernel[ (c+1)*R*S*K + (row)*K + blockIdxXAdjusted * blockDim.x * T_k + lane_id + rp*32];
                    }
                }
            }
        }
        flip = flip ^ 1;

#pragma unroll(3) // ReplaceLine 000001
        for (r = 0; r < R; r++) {
            for(j = 0; j < inAdjustedTileW; j++) {
                regIn[j] = shareBuffer_In[flip][threadIdx.z*T_h + r][threadIdx.y*T_w + j];
            }
#pragma unroll(3) // ReplaceLine 000002
            for (s = 0; s < S; s++) {
                //Kern SM 2
                for (kReg = 0;kReg < T_k; kReg += 2) {      // loop over T_k
                    float *__restrict__ ksbuf_ptr = &((float *) shareBuffer_Ker)[flip*R*S*OutBlockK+(r*S + s)*OutBlockK];
                    *((float2 *)regK) = *((float2 *)ksbuf_ptr + threadIdx.x*(T_k/2) + kReg/2);
#pragma unroll(3) // ReplaceLine 000003
                    for (w = s; w < T_w + s; w++) {
                        regOut[kReg][0][w-s] += regIn[w] * regK[0];
                        regOut[kReg+1][0][w-s] += regIn[w] * regK[1];
                    } // Tw 17
                } //TK 4
            } //S  3
        } //R  3
    } // End c

     int kIndexBlock = blockIdxXAdjusted * blockDim.x * T_k;

    //new
    for(k = 0; k < T_k; k++) {
        for (w = 0; w < T_w; w++) {
            shareBuffer_Out[threadIdx.x][threadIdx.z*OutBlockW + threadIdx.y*T_w+w] = regOut[k][0][w];
        }
        __syncthreads();
        for(int kk = 0; kk < W_k*B_k; kk++) {
            for (int i = 0; i < OutBlockW*OutBlockH; i += (B_h*B_w*B_k*32)){
                if (i + linear_id_in_TB < OutBlockW*OutBlockH && (kIndexBlock + kk*T_k+ k) < K){
                    Output[(kIndexBlock + kk*T_k+ k) *H*W + hIndexBlock*W + (wIndexBlock + i + linear_id_in_TB)] = shareBuffer_Out[kk][i + linear_id_in_TB];
                }

            }
        }
        __syncthreads();
    } //end write out
} // END

void conv_kernel_wrapper(const float *Input,
    const float *Kernel, float *Output, int itr) {
    hipEvent_t start;
    CHECK(hipEventCreate(&start));
    hipEvent_t stop;
    CHECK(hipEventCreate(&stop));
    float msec = 0.0f;
    float *dev_Input;
    float *dev_Kernel;
    float *dev_Output;

    CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
    CHECK(hipMemcpy(dev_Kernel, Kernel, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));

    CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
    CHECK(hipMemcpy(dev_Input, Input, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));

    CHECK(hipMalloc(&dev_Output, sizeof(float) * N_B * N_F * N_Y * N_X));
    CHECK(hipMemset(dev_Output, 0, sizeof(float) * N_B * N_F * N_Y * N_X));
    /*call conv_kernel() */
    dim3 size_block(B_k*W_k, B_w*W_w, B_h*W_h);
    dim3 size_grid((N_F / (T_k*size_block.x)) * CBlockMult, CEIL(W , (T_w * size_block.y)), CEIL(H, (T_h * size_block.z)) );
    printf("warmup....\n");
    printf("block shape <%d, %d, %d>\n", B_k*W_k, B_w*W_w, B_h*W_h);
    printf("grid shape <%d, %d, %d>\n",  (N_F / (T_k*size_block.x)) * CBlockMult, CEIL(W , (T_w * size_block.y)), CEIL(H, (T_h * size_block.z)) );
    printf("%d %d %d %d %d | %d %d %d %d %d %d %d %d %d \n", K, H, W, C, R, T_h, T_w, T_k, W_h, W_w, W_k, B_h, B_w, B_k);
    conv_kernel <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
    CHECK(hipFree(dev_Input));
    CHECK(hipFree(dev_Kernel));
    CHECK(hipFree(dev_Output));
    printf("Input size: %d X %d X %d X %d\n", N_B, N_C, N_H, N_W);
    printf("Kernel size: %d X %d X %d X %d\n", N_F, N_C, N_R, N_S);
    printf("Output size: %d X %d X %d X %d\n", N_B, N_F, N_Y, N_X);

    double *time_arr = (double *)malloc(itr * sizeof(double));
    for (int j = 1; j < itr; j++) {
        CHECK(hipMalloc(&dev_Kernel, sizeof(float) * N_F * N_C * N_R * N_S));
        CHECK(hipMemcpy(dev_Kernel, Kernel+j*N_F * N_C * N_R * N_S, sizeof(float) * N_F * N_C * N_R * N_S, hipMemcpyHostToDevice));
        CHECK(hipMalloc(&dev_Input, sizeof(float) * N_B * N_C * N_H * N_W));
        CHECK(hipMemcpy(dev_Input, Input+j*N_B * N_C * N_H * N_W, sizeof(float) * N_B * N_C * N_H * N_W, hipMemcpyHostToDevice));
        CHECK(hipMalloc(&dev_Output, sizeof(float) * N_B * N_F * N_Y * N_X));
        CHECK(hipMemset(dev_Output, 0, sizeof(float) * N_B * N_F * N_Y * N_X));
        /* Record the start event*/
        CHECK(hipEventRecord(start, NULL));
        conv_kernel <<< size_grid, size_block >>>(dev_Input, dev_Kernel, dev_Output);
        /* Record the end event */
        CHECK(hipEventRecord(stop, NULL));
        /* Wait for the stop event to complete*/
        CHECK(hipEventSynchronize(stop));
        float msec = 0.0f;
        hipEventElapsedTime(&msec, start, stop);
        time_arr[j] = msec;
        CHECK(hipFree(dev_Input));
        CHECK(hipFree(dev_Kernel));
        CHECK(hipFree(dev_Output));
    }

    for (int j = 1; j < itr; j++) {   
        printf("%0.5lf \n", time_arr[j]);
    }

    return;
}
